#include "hip/hip_runtime.h"
//
// Copyright (c) Microsoft. All rights reserved.
// Licensed under the MIT license. See LICENSE.md file in the project root for full license information.
//

#include "stdafx.h"
#include "Basics.h"
#include "BestGpu.h"

#ifndef CPUONLY

#include "GPUTensor.h"
#include "GPUMatrix.h"
#include "GPUMatrixCUDAKernels.cuh"
#include "CommonMatrix.h"
#define TENSOR_OPS_DECL __device__ __host__
#include "TensorOps.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include <assert.h>
#include<limits.h>

#ifndef let
#define let const auto
#endif

#pragma comment(lib, "cudart.lib") // instruct linker to reference these libs
#pragma comment(lib, "cublas.lib")

#pragma warning(disable : 4267) // conversion from 'size_t' to 'unsigned int'; happens in CUDA <<<a,b>>> syntax if a and b are size_t
#pragma warning(disable : 4127) // conditional expression is constant; "if (sizeof(ElemType)==sizeof(float))" triggers this
#pragma warning(disable : 4702) // unreachable code; triggered for unknown reasons

#ifdef _WIN32
// thread local storage to access the current stream, initalize to default stream
__declspec(thread)
#endif
extern hipStream_t t_stream;

namespace Microsoft { namespace MSR { namespace CNTK {

// =======================================================================
// TensorView support
// =======================================================================

// TensorView computes element-wise tensor operations.
//  - supports general strides
//  - input broadcasting is supported by stride=0
//  - the operation is denoted by an opCode
//  - reduction is supported, including summation, min, max (dual to broadcasting when computing gradients)
//  - reduction operation is given by an opCode: opSum, opMin, opMax and opLogSum.
//
// This library makes extensive use of templates and macros.
// Specifically, templates are used recursively to recurse over tensor dimensions.
// For example, a tensor op of rank K is computed by looping over the last dimension
// and then calling the same function template recursively with K-1.
// Template specializations exist in order to:
//  - terminate recursion
//  - optimize for thread-parallel reduction where elements are consecutive in memory
//
// The general algorithm is very straight forward:
//
//     for all output dimensions [###]:                                 // TensorOp()
//         output[###] *= beta
//         for all reduction dimensions [***]:                          // TensorOpWithReduction()
//             output[###] += op(input1[###,***], input1[###,***], ...) * alpha
//
// Indices and dimensions used throughout this code:
//  - N = ariness; number of arguments *including output* (binary op: N=3)
//  - K = rank of output elements, regularOpDims.size(). K=0 means scalar.
//  - k = -1..K-1 = recursion index
//  - M = reduction rank, reducingOpDims.size(). M=0 means no reduction.
//  - m = -1..M-1 = recursion index
//
// Other frequently used variable names:
//  - alpha, beta: BLAS-style weights: outVal = beta * outVal + alpha * f(inVals)
//                 where beta=0 is an assignment (0 * outVal := 0, even e.g. if outVal = NaN)
//  - pointers[N]:          pointer to first element, for each argument
//  - regularOpDims[K]:     tensor dimensions of output elements to produce
//  - regularStrides[N,K]:  strides; multiply index[k] with strides[n,k] to get element offset for this dimension
//                          Broadcasting of inputs is implemented by a stride being 0.
//  - reducingOpDims[M]:    tensor dimensions of input elements to reduce over
//  - reducingStrides[N,M]: strides for input reduction. Always 0 for output argument.
//
// This code uses two custom structs, FixedArray<> and FixedMatrix<>, which
// are templated equivalents to vector<> and vector<vector<>> for CUDA code.

// -----------------------------------------------------------------------
// simple fixed-size arrays for passing dimension information by value
// since CUDA can't just take our std::array and std::vector
// -----------------------------------------------------------------------

template <typename T, size_t N>
struct FixedArray
{
    T m_data[N];
    __device__ __host__ size_t size() const
    {
        return N;
    }
    __device__ __host__ T& operator[](size_t n)
    {
        return m_data[n];
    }
    __device__ __host__ T operator[](size_t n) const
    {
        return m_data[n];
    }
    template <class VEC>
    FixedArray(const VEC& data) // construct from CPU-side STL array or vector
    {
        assert(data.size() == N);
        for (size_t n = 0; n < N; n++)
        {
            m_data[n] = (T) data[n];
            if (m_data[n] != data[n]) // overflow check
                InvalidArgument("FixedArray: Dimensions out of range, too few bits.");
        }
    }
};
template <typename T> // specialized version for 0 elements
struct FixedArray<T, 0>
{
    __device__ __host__ size_t size() const
    {
        return 0;
    }
    template <class VEC>
    FixedArray(const VEC& data)
    {
        assert(data.size() == 0);
        UNUSED(data);
    }
    FixedArray()
    {
    }
};

template <typename T, size_t N, size_t K> // N = which input/output; K = index depth
struct FixedMatrix
{
    T m_data[N][K];
    __device__ __host__ size_t getNumRows() const
    {
        return N;
    }
    __device__ __host__ size_t getNumCols() const
    {
        return K;
    }
    __device__ __host__ T& operator()(size_t n, size_t k)
    {
        return m_data[n][k];
    }
    __device__ __host__ T operator()(size_t n, size_t k) const
    {
        return m_data[n][k];
    }
    template <typename U>
    FixedMatrix(const array<SmallVector<U>, N>& data) // construct from CPU-side array of vectors
    {
        assert(data.size() == N);
        for (size_t n = 0; n < N; n++)
        {
            assert(data[n].size() == K);
            for (size_t k = 0; k < K; k++)
            {
                m_data[n][k] = (T) data[n][k];
                if (m_data[n][k] != data[n][k]) // overflow check
                    InvalidArgument("FixedArray: Dimensions out of range, too few bits.");
            }
        }
    }
};
template <typename T, size_t N> // specialized version for 0 elements
struct FixedMatrix<T, N, 0>
{
    __device__ __host__ size_t getNumRows() const
    {
        return N;
    }
    __device__ __host__ size_t getNumCols() const
    {
        return 0;
    }
    template <typename U>
    FixedMatrix(const array<SmallVector<U>, N>& data)
    {
        assert(data.size() == N);
        for (size_t n = 0; n < N; n++)
            assert(data[n].size() == 0);
        UNUSED(data);
    }
    FixedMatrix()
    {
    }
};

// -----------------------------------------------------------------------
// function to actually compute a function of (N-1) inputs based on the opcode
// -----------------------------------------------------------------------

template <class ElemType>
struct TensorOps
{
    static __device__ ElemType Compute(const FixedArray<ElemType*, 1>& pointers, ElementWiseOperator op)
    {
#define CaseNullaryTensorOp(oper)       \
    case ElementWiseOperator::op##oper: \
        return Op##oper<ElemType>()
        switch (op)
        {
            ForAllNullaryOps(CaseNullaryTensorOp);
        default:
            return OpConstOne<ElemType>(); // (failure--we only have one nullary op, so use the same, maybe it will eliminate the switch altogether)
        }
    }
    static __device__ ElemType Compute(const FixedArray<ElemType*, 2>& pointers, ElementWiseOperator op)
    {
        ElemType a = *(pointers[0]);
#define CaseUnaryTensorOp(oper)         \
    case ElementWiseOperator::op##oper: \
        return Op##oper(a)
        switch (op)
        {
            ForAllUnaryOps(CaseUnaryTensorOp);
        default:
            return 0; // (failure)
        }
    }
    static __device__ ElemType Compute(const FixedArray<ElemType*, 3>& pointers, ElementWiseOperator op)
    {
        // const ElemType & a = *(pointers[0]);    // const & for opIndex--costs quite some code bloat
        ElemType a = *(pointers[0]);
        ElemType b = *(pointers[1]);
#define CaseBinaryTensorOp(oper)        \
    case ElementWiseOperator::op##oper: \
        return Op##oper(a, b)
        switch (op)
        {
            ForAllBinaryOps(CaseBinaryTensorOp); // note: this costs about 6% compared to having only a single case
        default:
            return 0; // (failure)
        }
    }
    static __device__ ElemType Compute(const FixedArray<ElemType*, 4>& pointers, ElementWiseOperator op)
    {
#define CaseTernaryTensorOp(oper)       \
    case ElementWiseOperator::op##oper: \
        return Op##oper(*(pointers[0]), *(pointers[1]), *(pointers[2])) // reading each time, which saves mem accesses for OpCond
        switch (op)
        {
            ForAllTernaryOps(CaseTernaryTensorOp);
        default:
            return 0; // (failure)
        }
    }
};

//----------------------------------------------------------------------------
// For reductions we need the neutral elements of the corresponding binary ops
//----------------------------------------------------------------------------
template <typename ElemType> __device__ ElemType NeutralValue(ElementWiseOperator op)
{
    return 0; // error, only the explicit instantiations below should be used.
};

template<> __device__ float NeutralValue<float>(ElementWiseOperator op)
{
    switch (op)
    {
    case ElementWiseOperator::opSum:    return 0;
    case ElementWiseOperator::opLogSum: return -INFINITY;
    case ElementWiseOperator::opMin:    return FLT_MAX;
    case ElementWiseOperator::opMax:    return FLT_MIN;
    default:                            return 0; // error
    }
};

template<> __device__ double NeutralValue<double>(ElementWiseOperator op)
{
    switch (op)
    {
    case ElementWiseOperator::opSum:    return 0;
    case ElementWiseOperator::opLogSum: return -INFINITY;
    case ElementWiseOperator::opMin:    return DBL_MAX;
    case ElementWiseOperator::opMax:    return DBL_MIN;
    default:                            return 0; // error
    }
};


// ----------------------------------------------------------------------------
// Function to update an aggregate value for the specifed reduction operation
// ----------------------------------------------------------------------------

template<typename ReductionType, class ElemType> __device__ void UpdateAggregate(ReductionType& aggregate, ElemType val, ElementWiseOperator reductionOp)
{
    switch (reductionOp)
    {
    case ElementWiseOperator::opSum:
        aggregate += val;
        break;
    case ElementWiseOperator::opLogSum:
        aggregate = OpLogSum(aggregate, val);
        break;
    case ElementWiseOperator::opMin:
        if (val < aggregate)
            aggregate = val;
        break;
    case ElementWiseOperator::opMax:
        if (val > aggregate)
            aggregate = val;
        break;
    }
};


// -----------------------------------------------------------------------
// function to compute the value for a given output location (including reduction)
// -----------------------------------------------------------------------

//#define ReduceElemType double
#define ReduceElemType ElemType // (note: we could use 'double' here, but that would cause problems with CUDA cards that don't support double)

template <class ElemType, C_size_t N, C_int M, C_int m>
struct TensorOpReduce
{
    // this version for m >= 0
    static __device__ ElemType Compute(FixedArray<ElemType*, N> pointers, ElementWiseOperator op, ElementWiseOperator reductionOp,
                                       const FixedArray<C_unsigned_int, M>& reducingOpDims, const FixedMatrix<C_int, N, M>& reducingStrides)
    {
        // start with index 0
        // We may use 'double' since we are memory-bound anyway.
        ReduceElemType aggregate = TensorOpReduce<ElemType, N, M, m - 1>::Compute(pointers, op, reductionOp, reducingOpDims, reducingStrides);
        // apply this index to the pointers
        C_size_t dim = reducingOpDims[m];
        for (C_size_t k = 1 /*done with k=0 already*/; k < dim; k++)
        {
            // bump the pointers
            for (C_size_t i = 0; i < N - 1; i++) // N-1 because output is not used here
                pointers[i] += reducingStrides(i, (C_size_t) m);
            ElemType val = TensorOpReduce<ElemType, N, M, m - 1>::Compute(pointers, op, reductionOp, reducingOpDims, reducingStrides);
            UpdateAggregate<ReduceElemType, ElemType>(aggregate, val, reductionOp);
        }
        return (ElemType) aggregate;
    }
};

// this one terminates the template recursion over reduction dimensions
// The pointers are pointing to the input element.
template <class ElemType, C_size_t N, C_int M>
struct TensorOpReduce<ElemType, N, M, /*m=*/-1>
{
    // this version for m = -1
    // the pointers are pointing to the right location(s) to take the operation over
    static __device__ ElemType Compute(FixedArray<ElemType*, N> pointers, ElementWiseOperator op, ElementWiseOperator reductionOp,
                                       const FixedArray<C_unsigned_int, M>& /*reducingOpDims*/, const FixedMatrix<C_int, N, M>& /*reducingStrides*/)
    {
        return TensorOps<ElemType>::Compute(pointers, op); // finally computing something!
    }
};

// -----------------------------------------------------------------------
// function to compute one constituent of the value for a given output location
// (reduction is not done here, but by calling into here multiple times)
// -----------------------------------------------------------------------

template <class ElemType, C_size_t N, C_int M, C_int m>
struct TensorOpParallelReduce
{
    // this version for m >= 0
    static __device__ ElemType Compute(CUDA_LONG id, FixedArray<ElemType*, N> pointers, ElementWiseOperator op,
                                       const FixedArray<C_unsigned_int, M>& reducingOpDims, const FixedMatrix<C_int, N, M>& reducingStrides)
    {
        // map id (location on grid) to index[k]
        C_size_t stride = 1; // compute the stride. This seems expensive, but since we we only currently support M <= 2, this is just compile-time selection between 1 and reducingOpDims[0].
        for (int i = 0; i < m; i++)
            stride *= reducingOpDims[(C_size_t) i];
        C_size_t index = id / stride; // this dimension. For m=0, the stride is 1 and hence the division will be removed at compile time.
        id = id % stride;             // remaining dimensions inside this. For m=0 this value is ignored and hence not even computed.
        // apply this index to the pointers
        for (C_size_t i = 0; i < N - 1; i++)
            pointers[i] += index * reducingStrides(i, (C_size_t) m); // now this dimension is taken care of
        return TensorOpParallelReduce<ElemType, N, M, m - 1>::Compute(id, pointers, op, reducingOpDims, reducingStrides);
    }
};

// this one terminates the template recursion over reduction dimensions
// The pointers are pointing to the input element.
template <class ElemType, C_size_t N, C_int M>
struct TensorOpParallelReduce<ElemType, N, M, /*m=*/-1>
{
    // this version for m = -1
    // the pointers are pointing to the right location(s) to take the operation over
    static __device__ ElemType Compute(CUDA_LONG /*id*/, FixedArray<ElemType*, N> pointers, ElementWiseOperator op,
                                       const FixedArray<C_unsigned_int, M>& /*reducingOpDims*/, const FixedMatrix<C_int, N, M>& /*reducingStrides*/)
    {
        return TensorOps<ElemType>::Compute(pointers, op); // finally computing something!
    }
};

// -----------------------------------------------------------------------
// perform loop over regular index k for N-nary operations (N counting the output)
// -----------------------------------------------------------------------

// The 'pointers' only refer to a single element, so we will bump them in-place to perform indexing.
template <class ElemType, C_size_t N, C_int M, C_int K, bool parallelReduce, C_int k>
struct TensorOpElement
{
    // template-recursive version loops over indices
    static __device__ void Compute(CUDA_LONG id, ElemType beta, FixedArray<ElemType*, N>& pointers, ElemType alpha, ElementWiseOperator op, ElementWiseOperator reductionOp,
                                   const FixedArray<C_unsigned_int, K>& regularOpStrides, const FixedMatrix<C_int, N, K>& regularStrides,
                                   const FixedArray<C_unsigned_int, M>& reducingOpDims, const FixedMatrix<C_int, N, M>& reducingStrides,
                                   CUDA_LONG reductionBegin, CUDA_LONG reductionChunkSize)
    {
        // map id (location on grid) to index[k]
        C_size_t stride = regularOpStrides[(C_size_t) k];
        C_size_t index = id / stride; // this dimension
        id = id % stride;             // remaining dimensions inside this
        // apply this index to the pointers
        for (C_size_t i = 0; i < N; i++)
            pointers[i] += index * regularStrides(i, (C_size_t) k); // now this dimension is taken care of
        // process the previous index
        TensorOpElement<ElemType, N, M, K, parallelReduce, k - 1>::Compute(id, beta, pointers, alpha, op, reductionOp, regularOpStrides, regularStrides, reducingOpDims, reducingStrides, reductionBegin, reductionChunkSize);
    }
};

// specialization for k=0 where op stride is guaranteed to be 1
template <class ElemType, C_size_t N, C_int M, C_int K, bool parallelReduce>
struct TensorOpElement<ElemType, N, M, K, parallelReduce, /*k=*/0>
{
    // template-recursive version loops over indices
    static __device__ void Compute(CUDA_LONG id, ElemType beta, FixedArray<ElemType*, N>& pointers, ElemType alpha, ElementWiseOperator op, ElementWiseOperator reductionOp,
                                   const FixedArray<C_unsigned_int, K>& regularOpStrides, const FixedMatrix<C_int, N, K>& regularStrides,
                                   const FixedArray<C_unsigned_int, M>& reducingOpDims, const FixedMatrix<C_int, N, M>& reducingStrides,
                                   CUDA_LONG reductionBegin, CUDA_LONG reductionChunkSize)
    {
        // map id (location on grid) to index[k]
        C_size_t index = id; // this dimension
        // apply this index to the pointers
        for (C_size_t i = 0; i < N; i++)
            pointers[i] += index * regularStrides(i, 0); // now this dimension is taken care of
        // process the previous index
        TensorOpElement<ElemType, N, M, K, parallelReduce, -1>::Compute(/*id*/ 0, beta, pointers, alpha, op, reductionOp, regularOpStrides, regularStrides, reducingOpDims, reducingStrides, reductionBegin, reductionChunkSize);
    }
};

// specialization for k = -1 terminates the template recursion, and computes reductions in a for loop
template <class ElemType, C_size_t N, C_int M, C_int K>
struct TensorOpElement<ElemType, N, M, K, /*parallelReduce=*/false, /*k=*/-1>
{
    // template-recursion-teminating version computes the actual value for this output location
    // now the output pointers point to the right element (input pointers may still iterate for reduction)
    static __device__ void Compute(CUDA_LONG /*id*/, ElemType beta, FixedArray<ElemType*, N>& pointers, ElemType alpha, ElementWiseOperator op, ElementWiseOperator reductionOp,
                                   const FixedArray<C_unsigned_int, K>& /*regularOpStrides*/, const FixedMatrix<C_int, N, K>& /*regularStrides*/,
                                   const FixedArray<C_unsigned_int, M>& reducingOpDims, const FixedMatrix<C_int, N, M>& reducingStrides, CUDA_LONG /*reductionBegin*/, CUDA_LONG /*reductionChunkSize*/)
    {
        // compute the operation for this output coordinate
        // This may still involve a reduction over inverse-broadcasting dimensions.
        ElemType val = TensorOpReduce<ElemType, N, M, M - 1>::Compute(pointers, op, reductionOp, reducingOpDims, reducingStrides);
        // scale
        val *= alpha;
        // combine with previous value in target matrix, then write it out
        if (N < 4 || val != 0 || beta != 1) // (skip memory access if not needed) (N<4: skip this test)
        {
            auto* pout = pointers[pointers.size() - 1];
            if (beta != 0) // (skip memory access if not needed, and allow for ignoring NaNs)
                val += beta * *pout;
            // save
            *pout = val;
        }
    }
};

#undef ALLOW_ATOMIC_REDUCTION // undefine to disable use of atomicAdd() below, for testing it

// specialization for k = -1 terminates the template recursion, and computes reductions in parallel
template <class ElemType, C_size_t N, C_int M, C_int K>
struct TensorOpElement<ElemType, N, M, K, /*parallelReduce=*/true, /*k=*/-1>
{
    // template-recursion-teminating version computes the actual value for this output location
    // now the output pointers point to the right element (input pointers may still iterate for reduction)
    static __device__ void Compute(CUDA_LONG /*id*/, ElemType beta, FixedArray<ElemType*, N>& pointers, ElemType alpha, ElementWiseOperator op, ElementWiseOperator reductionOp,
                                   const FixedArray<C_unsigned_int, K>& /*regularOpStrides*/, const FixedMatrix<C_int, N, K>& /*regularStrides*/,
                                   const FixedArray<C_unsigned_int, M>& reducingOpDims, const FixedMatrix<C_int, N, M>& reducingStrides, CUDA_LONG reductionBegin, CUDA_LONG reductionChunkSize)
    {
        CUDA_LONG reductionBlock = blockIdx.z; // reduction-block index  --larger reductions are split into blocks
        CUDA_LONG tid = threadIdx.x;           // thread index
        CUDA_LONG tids = blockDim.x;           // out of how many threads  --note: last block is partial

        // determine our range  --this is a single int mul, we can stomach it (we could alternatively pass in yet another parameter)
        CUDA_LONG reductionDim = (CUDA_LONG) reducingOpDims[0];
        for (C_size_t i = 1; i < reducingOpDims.size(); i++)
            reductionDim *= reducingOpDims[i];

        // determine the redId range that we operate on
        // Each thread takes a stride tid + (multiples of tids) within this range.
        reductionBegin += reductionChunkSize * reductionBlock;
        CUDA_LONG reductionEnd = min(reductionBegin + reductionChunkSize, reductionDim);

        // compute the operation for this input coordinate
        ReduceElemType aggregate = NeutralValue<ReduceElemType>(reductionOp);

        for (CUDA_LONG redId = reductionBegin + tid; redId < reductionEnd; redId += tids)
        {
            auto val = TensorOpParallelReduce<ElemType, N, M, M - 1>::Compute(redId, pointers, op, reducingOpDims, reducingStrides);
            UpdateAggregate<ReduceElemType, ElemType>(aggregate, val, reductionOp);
        }

        // reduce    --cf https://docs.nvidia.com/cuda/samples/6_Advanced/reduction/doc/reduction.pdf
        __shared__ ReduceElemType volatile accumulators[GridDim::maxThreadsPerBlock /*tids*/];
        accumulators[tid] = aggregate;
        __syncthreads();
        static_assert(GridDim::maxThreadsPerBlock <= 1024, "GridDim::maxThreadsPerBlock too large, need to add manually unrolled steps");
        for (CUDA_LONG i = 512; i; i >>= 1)
        {
            if (tid < i && tid + i < tids)
                UpdateAggregate<volatile ReduceElemType, volatile ReduceElemType>(accumulators[tid], accumulators[tid + i], reductionOp);

            if (0 + i < tids)
                __syncthreads(); // sync if condition true for at least one thread
            // TODO: use volatile* and then we can skip the __syncthreads() for the last 32 values. See Amit's allreduce() function implementation in MatrixQuantizer_kernel.cu.
        }

        // now set final value to output coordinate
        if (tid == 0)
        {
            ElemType val = (ElemType) accumulators[0];
            // scale
            val *= alpha;
            // combine with previous value in target matrix, then write it out
            if (N < 4 || val != 0 || beta != 1) // (skip memory access if not needed) (N<4: skip this test)
            {
                auto* pout = pointers[pointers.size() - 1];
#ifdef ALLOW_ATOMIC_REDUCTION
                CUDA_LONG reductionBlocks = gridDim.z; // number of reduction blocks. If >1 we need atomicAdd
                if (reductionBlocks > 1) // multiple blocks: need to use atomicAdd()
                {
                    // in this case, outer calling code must pass beta = 1
                    atomicAdd(pout, val);
                }
                else
#endif
                {
                    if (beta != 0)
                        val += beta * *pout;
                    // save
                    *pout = val;
                }
            }
        }
    }
};

// -----------------------------------------------------------------------
// kernel and launch  --no reduction
// -----------------------------------------------------------------------

// launch tensor op with CUDA
template <class ElemType, C_size_t N, C_int M, C_int K>
__global__ void _launchTensorOp(ElemType beta, FixedArray<ElemType*, N> pointers, ElemType alpha, ElementWiseOperator op, ElementWiseOperator reductionOp,
                                FixedArray<C_unsigned_int, K> regularOpStrides, FixedMatrix<C_int, N, K> regularStrides, CUDA_LONG numElements,
                                FixedArray<C_unsigned_int, M> reducingOpDims, FixedMatrix<C_int, N, M> reducingStrides)
{
    CUDA_LONG id = GridDim::GetLinearThreadId();
    if (id < numElements) // note: there are no __syncthread() calls inside
        TensorOpElement<ElemType, N, M, K, false, K - 1>::Compute(id, beta, pointers, alpha, op, reductionOp, regularOpStrides, regularStrides, reducingOpDims, reducingStrides, 0, 0);
}

template <class ElemType, C_size_t N, C_int K>
static void LaunchTensorOp(ElemType beta, array<ElemType*, N> pointerVector, ElemType alpha, ElementWiseOperator op,
                           const SmallVector<size_t>& regularOpDims, const array<SmallVector<ptrdiff_t>, N>& regularStrideVectors)
{
    // copy all parameters to CUDA-compatible data structures
    FixedArray<ElemType*, N> pointers(pointerVector);
    SmallVector<C_size_t> regularOpStrideVector; // kernel needs the strides for converting thread index back to multi-dimensional tensor index
    C_size_t numElements = 1;
    for (C_size_t k = 0; k < regularOpDims.size(); k++)
    {
        regularOpStrideVector.push_back(numElements);
        numElements *= (C_size_t) regularOpDims[k];
    }
    FixedArray<C_unsigned_int, K> regularOpStrides(regularOpStrideVector);
    FixedMatrix<C_int, N, K> regularStrides(regularStrideVectors);
    FixedArray<C_unsigned_int, /*M=*/0> reducingOpDims; // empty reduction dimensions
    FixedMatrix<C_int, N, /*M=*/0> reducingStrides;

    // launch the kernel
    CUDA_LONG NN = (CUDA_LONG) numElements; // linear space identifying each individual input element
    SyncGuard syncGuard;
    GridDim grid(NN);
    _launchTensorOp<ElemType, N, /*M=*/0, K> <<<grid.m_blocksPerGrid, grid.m_threadsPerBlock, 0, t_stream >>>(beta, pointers, alpha, op, (ElementWiseOperator)(-1) /* dummy reductionOp */, regularOpStrides, regularStrides, grid.m_N, reducingOpDims, reducingStrides);
}

// -----------------------------------------------------------------------
// kernel and launch  --with reduction
// -----------------------------------------------------------------------

template <class ElemType, C_size_t N, C_int M, C_int K>
__global__ void _launchTensorOpWithReduction(ElemType beta, FixedArray<ElemType*, N> pointers, ElemType alpha, ElementWiseOperator op, ElementWiseOperator reductionOp,
                                             FixedArray<C_unsigned_int, K> regularOpStrides, FixedMatrix<C_int, N, K> regularStrides, CUDA_LONG numElements,
                                             FixedArray<C_unsigned_int, M> reducingOpDims, FixedMatrix<C_int, N, M> reducingStrides,
                                             CUDA_LONG reductionBegin, CUDA_LONG reductionChunkSize)
{
    CUDA_LONG id = gridDim.x * blockIdx.y + blockIdx.x; // input dimensions are Y dimension of blocks in this case, so we can use thread dim for shared-memory/parallelization
#ifndef ALLOW_ATOMIC_REDUCTION
    CUDA_LONG reductionBlock = blockIdx.z;                         // reduction-block index  --larger reductions are split into blocks
    pointers[pointers.size() - 1] += numElements * reductionBlock; // the output tensor is dense (no gaps); and there is one copy for each reduction block (those get further reduced into one later)
#endif
    if (id < numElements)                               // note: we have __syncthread() calls but only entire blocks in sync, so this is OK
        TensorOpElement<ElemType, N, M, K, true, K - 1>::Compute(id, beta, pointers, alpha, op, reductionOp, regularOpStrides, regularStrides, reducingOpDims, reducingStrides, reductionBegin, reductionChunkSize);
}

// helper function to provide a reduction buffer
template <class ElemType>
static shared_ptr<ElemType> AllocateReductionBuffer(size_t N)
{
    ElemType* deviceBufferPtr;
    CUDA_CALL(hipMalloc((void**)&deviceBufferPtr, sizeof(ElemType) * N));
    return shared_ptr<ElemType>(deviceBufferPtr, [](ElemType* deviceBufferPtr){ hipFree((void*)deviceBufferPtr); });
}

template <class ElemType>
static shared_ptr<ElemType> GetReductionBuffer(size_t N)
{
    bool dontCache = false;         // (for debugging only)
    if (t_stream != 0 || dontCache) // we cache for the NULL stream but don't bother for others, since we only ever use the NULL stream currently
        return AllocateReductionBuffer<ElemType>(N);

    static shared_ptr<ElemType> reductionBuffersCache[32]; // cache of objects    --TODO: Do we have a #define the the max somewhere? Then also use it in CPUMatrix.cu GetOnesTensor()
    static size_t reductionBuffersCacheSize[_countof(reductionBuffersCache)] = { 0 };
    let deviceId = GridDim::GetCurrentDeviceId();
    if (deviceId >= _countof(reductionBuffersCache)) // index check w.r.t. our hard-coded dimensions
        return AllocateReductionBuffer<ElemType>(N); // out of bounds: don't cache
    if (!reductionBuffersCache[deviceId])
    {
        reductionBuffersCache[deviceId] = AllocateReductionBuffer<ElemType>(N);
        reductionBuffersCacheSize[deviceId] = N;
    }
    if (N > reductionBuffersCacheSize[deviceId]) // buffer size check
        LogicError("GetReductionBuffer: Must be called with the number of multiprocs, which may not change.");
    return reductionBuffersCache[deviceId];
}

// All dimensions (N-ariness, number of input dimensions K and number of reduction dimensions M) are bound to template parameters now.
template <class ElemType, C_size_t N, C_int M, C_int K>
static void LaunchTensorOpWithReduction(ElemType beta, array<ElemType*, N> pointerVector, ElemType alpha, ElementWiseOperator op, ElementWiseOperator reductionOp,
                                        const SmallVector<size_t>& regularOpDims, const array<SmallVector<ptrdiff_t>, N>& regularStrideVectors,
                                        const SmallVector<size_t>& reducingOpDimVector, const array<SmallVector<ptrdiff_t>, N>& reducingStrideVectors)
{
    // copy all parameters to CUDA-compatible data structures
    FixedArray<ElemType*, N> pointers(pointerVector);
    SmallVector<C_size_t> regularOpStrideVector; // kernel needs the strides for converting thread index back to multi-dimensional tensor index
    C_size_t numElements = 1;
    for (C_size_t k = 0; k < regularOpDims.size(); k++)
    {
        regularOpStrideVector.push_back(numElements); // stride for dense representation of our output elements (if they were flattened)
        numElements *= (C_size_t) regularOpDims[k];
    }
    FixedArray<C_unsigned_int,    K> regularOpStrides(regularOpStrideVector);
    FixedMatrix<C_int,         N, K> regularStrides(regularStrideVectors);
    FixedArray<C_unsigned_int,    M> reducingOpDims(reducingOpDimVector);
    FixedMatrix<C_int,         N, M> reducingStrides(reducingStrideVectors);

    // launch the kernel
    CUDA_LONG NN = (CUDA_LONG) numElements; // linear space identifying each individual output element
    SyncGuard syncGuard;

    // do some optimization for reductions
    //  - example: 30 GPU procs, warp size 32 --> 960 GPU cores
    //  - NN elements must be computed, each involving a reduction over reductionDim elements
    // Cases:
    //  - #output elements NN >= GPU cores  -->  use one proc per element, do reduction in inner loop
    //    E.g. if >=960 elements are computed, each gets its own GPU thread.
    //  - reduction dimension would benefit from multiple blocks  -->  multiple blocks work on a single output element
    //    E.g.
    //     - gradient of adding a bias: reducing to a bias, e.g. 512-dim
    //     - gradient of scalar multiplication: big elementwise product reduced to a scalar (big dot product, e.g. [1024 x 1024] = 1M elements)
    //     - softmax in seq-2-seq attention model: reduce over length of attention window (e.g. 20)
    //     - summation of criterion value: scalar reduction over a few hundred or thousand samples in the minibatch
    C_size_t reductionDim = 1; // number of elements to reduce over
    for (C_size_t k = 0; k < reducingOpDimVector.size(); k++)
        reductionDim *= (C_size_t) reducingOpDimVector[k];
    GridDim grid(NN);
    let& props = GridDim::GetDeviceProps();
    // === simple case: NN large, one thread per output element
    bool disableParallelReduction = false;                       // (for debugging)
    if (reductionDim == 1 ||                                     // no reduction
        grid.m_blocksPerGrid >= props.multiProcessorCount ||     // enough output elements to fill all multiprocs
        reductionDim * numElements <= 2 * props.warpSize ||      // trivial operation not worth the trouble (2* because the more complex one also needs 2 kernel launches)
        disableParallelReduction ||                              // (for debugging)
        reductionDim * numElements <= props.multiProcessorCount) // recursive call from reduction below
    {
        // we got enough elements to generate: do one element per thread, and reduction inside
        _launchTensorOp<ElemType, N, M, K><<<grid.m_blocksPerGrid, grid.m_threadsPerBlock, 0, t_stream>>>(
            beta, pointers, alpha, op, reductionOp,
            regularOpStrides, regularStrides, grid.m_N,
            reducingOpDims, reducingStrides);
    }
    // === optimization: simple case would not use all multiprocs
    else
    {
        // m_blocksPerGrid can be thought of NN / 512, with appropriate rounding

        // we are reducing and are underutilizing the multiprocs we have: get more parallelism by doing reduction in parallel
        // If we get here, then
        //  - the total number of outputs to produce is < #multiprocs * warpSize, e.g. < 960
        //  - each output has at least two inputs, but possibly millions
        // Examples:
        //  (a1) NN=900
        //        - each multiproc processes multiple elements concurrently, each reducing over its inputs inside
        //        - use one block per output element
        //  (a2) NN=30
        //        - same as (a1) except 30 multiprocs run only a single block each
        //  (a3) NN=16
        //        - same as (a1) except only 16 multiproc run one block
        //  (b1) NN=15
        //        - 2 blocks work together on a single output element
        //  (b2) NN=1    (NN < #multiprocs, e.g. NN < 30)
        //        - multiple blocks work together on a single output element
        //        - only this case requires memory, and only K * NN
        //          where K = blocks that work together,
        //          both K and NN < #multiprocs,
        //          and K * NN = on the order of NN, but generally a bit larger due to rounding.

        // By how much do we underutilize?
        // We increase #blocks by that factor by breaking reduction into that many chunks.
        let numReductionChunks = max(props.multiProcessorCount / NN, 1); // only >1 for NN < multiProcessorCount

        // distribute NN over block X and Y
        let blockXOverBy = CeilDiv(NN, props.maxGridSize[0]);
        let numBlocksX = CeilDiv(NN, blockXOverBy);
        let numBlocksY = CeilDiv(NN, numBlocksX);
        // while block Z is for multiple blocks working together on a single output element
        let numBlocksZ = numReductionChunks;
        // Block dim is now:
        //  - X, Y: such that X*Y covers NN
        //  - Z: reduction chunks

        // reduction goes into thread dim X
        let reductionChunkSize = CeilDiv(reductionDim, numReductionChunks);
        let numThreadsX = min(reductionChunkSize, GridDim::maxThreadsPerBlock); // any that's over will be done by looping inside the kernel

        // --- cases (a1) and (a2)
        // This involves no reduction across blocks.
        if (numReductionChunks == 1)
        {
            _launchTensorOpWithReduction<ElemType, N, M, K><<<dim3(numBlocksX, numBlocksY, numBlocksZ), numThreadsX, numThreadsX * sizeof(ReduceElemType), t_stream>>>(
                beta, pointers, alpha, op, reductionOp,
                regularOpStrides, regularStrides, NN,
                reducingOpDims, reducingStrides, /*reductionBegin*/ 0, reductionChunkSize);
        }
        // --- case (b)
        // Reduction across blocks. This is the difficult one.
#ifndef ALLOW_ATOMIC_REDUCTION // temporarily disabled to ensure it is not causing the non-reproducability
        else
        {
            // we get here if NN <= #multiprocs
            assert(NN <= props.multiProcessorCount && numBlocksX == NN && numBlocksY == 1);
            // dims are:
            //  - numBlocksZ = numReductionChunks = how many multiprocs work together to produce one output element
            //  - numBlocksX = NN = number of output elements
            //  - numThreadsX = reductionChunkSize clipped to 512; reductionChunkSize > 512 is handled by an inner for loop inside of the kernel

            // we need memory for block outputs of dimension [numBlocksX x numBlocksZ]
            //  - total elements = NN * Floor(#multiprocs / NN) = <= #multiprocs
            let reductionBufferSize = props.multiProcessorCount;
            assert(reductionBufferSize >= NN * numBlocksZ);
            shared_ptr<ElemType> reductionBuffer = GetReductionBuffer<ElemType>(reductionBufferSize);

            // 'pointers', 'regularOpStrides', and 'regularStrides' are set up to point to the target memory.
            // We need to reroute them to point to our reductionBuffer.
            //  - pointer[N-1] -> replace by reductionBuffer
            //  - regularStrides -> replace [N-1] by regularOpStrides which already represent the NN elements for a dense memory layout
            //  - beta -> 0 since we write into temp memory
            //  - kernel must use block.z as second index into the output buffer; add (block.z * NN) to the pointer
            FixedArray<ElemType*, N> pointers1 = pointers;
            pointers1[N - 1] = reductionBuffer.get();
            auto regularStrideVectors1 = regularStrideVectors;
            for (size_t k = 0; k < regularOpStrides.size(); k++)
                regularStrideVectors1[N - 1][k] = (ptrdiff_t)regularOpStrideVector[k];
            FixedMatrix<C_int, N, K> regularStrides1(regularStrideVectors1);
            ElemType beta1  = 0;
            ElemType alpha1 = 1;
            _launchTensorOpWithReduction<ElemType, N, M, K> << <dim3(numBlocksX, numBlocksY, numBlocksZ), numThreadsX, numThreadsX * sizeof(ReduceElemType), t_stream >> >(
                beta1, pointers1, alpha1, op, reductionOp,
                regularOpStrides, regularStrides1, NN,
                reducingOpDims, reducingStrides, /*reductionBegin*/0, reductionChunkSize);

#if 1
            // now reduce and redistribute
            // Create a new tensor task, and execute it recursively:
            //  - input  = reductionBuffer
            //  - output = true output
            //  - op dims/strides     = output elements
            //  - reduce dims/strides = numBlocksZ
            //  - op = opCopy
            array<ElemType*, 2>                    pointerVector2{         reductionBuffer.get(),        pointerVector[N - 1] };
            const array<SmallVector<ptrdiff_t>, 2> regularStrideVectors2{  regularStrideVectors1[N - 1], regularStrideVectors[N - 1] };
            const array<SmallVector<ptrdiff_t>, 2> reducingStrideVectors2{ SmallVector<ptrdiff_t>{ NN }, SmallVector<ptrdiff_t>{ 0 } };
            const SmallVector<size_t>              reducingOpDimVector2{ (size_t)numReductionChunks };
            LaunchTensorOpWithReduction<ElemType, /*N=*/2, /*M=*/1, K>(
                beta, pointerVector2, alpha, ElementWiseOperator::opCopy, reductionOp,
                regularOpDims, regularStrideVectors2,
                reducingOpDimVector2, reducingStrideVectors2);
            // (note: ^^this will have a nested syncGuard, which is fine)

#else
            _launchTensorOp<ElemType, N, M, K><<<grid.m_blocksPerGrid, grid.m_threadsPerBlock, 0, t_stream>>>(
                beta, pointers, alpha, op, reductionOp,
                regularOpStrides, regularStrides, grid.m_N,
                reducingOpDims, reducingStrides);
            //for (size_t z = 0; z < numBlocksZ; z++)
            //    _launchTensorOpWithReduction<ElemType, N, M, K><<<dim3(numBlocksX, numBlocksY, 1), numThreadsX, numThreadsX * sizeof(ReduceElemType), t_stream>>>(z == 0 ? beta : 1, pointers, alpha, op,
            //    regularOpStrides, regularStrides, NN,
            //    reducingOpDims, reducingStrides, reductionChunkSize * z, reductionChunkSize);
            vector<ElemType> peekPartial(NN * numBlocksZ, -42);
            vector<ElemType> peekFinal(NN, -42);
            CUDA_CALL(hipMemcpy(peekPartial.data(), reductionBuffer,             sizeof(ElemType) * peekPartial.size(), hipMemcpyDeviceToHost));
            CUDA_CALL(hipMemcpy(peekFinal.data(),   pointers[pointers.size()-1], sizeof(ElemType) * peekFinal.size(),   hipMemcpyDeviceToHost));
            double s1 = 0, s2 = 0;
            for (auto v : peekPartial)
                s1 += v;
            for (auto v : peekFinal)
                s2 += v;
            sin(1.0);
#endif
        }
#else
        else if (beta == 1)
        {
            // no need to pre-scale; just add (common for gradients)
            _launchTensorOpWithReduction<ElemType, N, M, K><<<dim3(numBlocksX, numBlocksY, numBlocksZ), numThreadsX, numThreadsX * sizeof(ReduceElemType), t_stream>>>(beta, pointers, alpha, op, reductionOp, regularOpStrides, regularStrides, NN, reducingOpDims, reducingStrides, 0, reductionChunkSize);
            return;
        }
        else
        {
            // We need more than one chunk, we will use atomicAdd().
            // First reset/pre-multiply input; then do the remaining chunks using atomicAdd().
            _launchTensorOpWithReduction<ElemType, N, M, K><<<dim3(numBlocksX, numBlocksY, 1), numThreadsX, numThreadsX * sizeof(ReduceElemType), t_stream>>>(beta, pointers, alpha, op, reductionOp, regularOpStrides, regularStrides, NN, reducingOpDims, reducingStrides, 0, reductionChunkSize);
            // We will leave it like this for a while, but eventually need to revisit using temporary memory.
            _launchTensorOpWithReduction<ElemType, N, M, K><<<dim3(numBlocksX, numBlocksY, numBlocksZ - 1), numThreadsX, numThreadsX * sizeof(ReduceElemType), t_stream>>>(/*beta=*/1, pointers, alpha, op, reductionOp, regularOpStrides, regularStrides, NN, reducingOpDims, reducingStrides, reductionChunkSize, reductionChunkSize);
        }
#endif
    }
}

// -----------------------------------------------------------------------
// kernel and launch  --linear unary
// -----------------------------------------------------------------------

// for linear unary ops, we need to define a functor for every function for use as a template parameter (lambda syntax doesn't work in CUDA 7)
#define DefineUnaryTensorFunctor(oper)           \
    struct Functor##oper                         \
    {                                            \
        template <class ElemType>                \
        static __device__ ElemType f(ElemType a) \
        {                                        \
            return Op##oper(a);                  \
        }                                        \
    };
ForAllUnaryOps(DefineUnaryTensorFunctor);

// the top-level kernel for linear unary ops
// Note: If we have a beta, we have 2 memory accesses, so this optimization may no longer be needed as we are memory-bound.
template <class ElemType, class FN>
__global__ void _launchUnaryTensorOp(ElemType beta, const ElemType* pa, ElemType* pb, ElemType alpha, CUDA_LONG numElements)
{
    CUDA_LONG id = GridDim::GetLinearThreadId();
    if (id >= numElements)
        return;
    ElemType a = pa[id];
    ElemType val = FN::f(a);
    val *= alpha;
    if (beta != 0)
        val += beta * pb[id];
    pb[id] = val;
}
// version without beta and alpha
template <class ElemType, class FN>
__global__ void _launchUnaryTensorOp(const ElemType* pa, ElemType* pb, CUDA_LONG numElements)
{
    CUDA_LONG id = GridDim::GetLinearThreadId();
    if (id >= numElements)
        return;
    ElemType a = pa[id];
    ElemType val = FN::f(a);
    pb[id] = val;
}

// special case of linear unary operation
template <class ElemType>
void LaunchUnaryTensorOp(ElemType beta, const ElemType* pa, ElemType* pb, ElemType alpha, ElementWiseOperator op, size_t regularOpDim)
{
    CUDA_LONG NN = (CUDA_LONG) regularOpDim;

#define CaseLaunchUnaryTensorOp(oper)                                                                                                        \
    case ElementWiseOperator::op##oper:                                                                                                      \
        if (beta == 0 && alpha == 1)                                                                                                         \
            _launchUnaryTensorOp<ElemType, Functor##oper><<<grid.m_blocksPerGrid, grid.m_threadsPerBlock, 0, t_stream>>>(pa, pb, NN); \
        else                                                                                                                                 \
            _launchUnaryTensorOp<ElemType, Functor##oper><<<grid.m_blocksPerGrid, grid.m_threadsPerBlock, 0, t_stream>>>(beta, pa, pb, alpha, NN);\
        break;

    SyncGuard syncGuard;
    GridDim grid(NN);
    switch (op)
    {
        ForAllUnaryOps(CaseLaunchUnaryTensorOp);
    default:
        LogicError("LaunchTensorOp1: Unknown op code %d.", (int) op);
    }
}

// -----------------------------------------------------------------------
// map runtime parameters N to template parameters
// -----------------------------------------------------------------------

// tensor operation with k+1 dimensions (-1 means scalar)
template <class ElemType, C_size_t N, C_int K>
static void TensorOpWithRegularLoop(ElemType beta, const array<ElemType*, N>& pointers, ElemType alpha, ElementWiseOperator op, ElementWiseOperator reductionOp,
                                    const SmallVector<size_t>& regularOpDims, const array<SmallVector<ptrdiff_t>, N>& regularStrides,
                                    const SmallVector<size_t>& reducingOpDims, const array<SmallVector<ptrdiff_t>, N>& reducingStrides)
{
    size_t dims = reducingOpDims.size();
    switch (dims)
    {
    case 2:
        return LaunchTensorOpWithReduction<ElemType, N, 2, K>(beta, pointers, alpha, op, reductionOp, regularOpDims, regularStrides, reducingOpDims, reducingStrides);
    case 1:
        return LaunchTensorOpWithReduction<ElemType, N, 1, K>(beta, pointers, alpha, op, reductionOp, regularOpDims, regularStrides, reducingOpDims, reducingStrides);
    case 0:
        return LaunchTensorOp<ElemType, N, K>(beta, pointers, alpha, op, regularOpDims, regularStrides);
    default:
        LogicError("TensorOp: %d non-flattened reduction dimensions are not supported.", (C_int) dims);
    }
}

// tensor operation, generalized in number of arguments
// This function now expands into different k. It also eliminates the offsets by adding them to the pointers.
template <class ElemType, C_size_t N>
void TensorOpN(ElemType beta, array<ElemType*, N> pointers, ElemType alpha, ElementWiseOperator op, ElementWiseOperator reductionOp,
               const array<size_t, N>& offsets,
               const SmallVector<size_t>& regularOpDims, const array<SmallVector<ptrdiff_t>, N>& regularStrides,
               const SmallVector<size_t>& reducingOpDims, const array<SmallVector<ptrdiff_t>, N>& reducingStrides)
{
    for (C_size_t i = 0; i < N; i++) // N = a small constant, this will be unrolled
        pointers[i] += offsets[i];
    size_t dims = regularOpDims.size();
    switch (dims)
    {
    case 4:
        return TensorOpWithRegularLoop<ElemType, N, 4>(beta, pointers, alpha, op, reductionOp, regularOpDims, regularStrides, reducingOpDims, reducingStrides);
    case 3:
        return TensorOpWithRegularLoop<ElemType, N, 3>(beta, pointers, alpha, op, reductionOp, regularOpDims, regularStrides, reducingOpDims, reducingStrides);
    case 2:
        return TensorOpWithRegularLoop<ElemType, N, 2>(beta, pointers, alpha, op, reductionOp, regularOpDims, regularStrides, reducingOpDims, reducingStrides);
    case 1:
        return TensorOpWithRegularLoop<ElemType, N, 1>(beta, pointers, alpha, op, reductionOp, regularOpDims, regularStrides, reducingOpDims, reducingStrides);
    case 0:
        return TensorOpWithRegularLoop<ElemType, N, 0>(beta, pointers, alpha, op, reductionOp, regularOpDims, regularStrides, reducingOpDims, reducingStrides);
    default:
        LogicError("TensorOp: %d non-flattened input dimensions are not supported.", (C_int) dims);
    }
}

//------------------------------------------------------------------------
// explicit instantiations--these are being called from GPUMatrix.cu
//------------------------------------------------------------------------

template void TensorOpN<float, 2>(float beta, array<float*, 2> pointers, float alpha, ElementWiseOperator op, ElementWiseOperator reductionOp,
                                  const array<size_t, 2>& offsets,
                                  const SmallVector<size_t>& regularOpDims, const array<SmallVector<ptrdiff_t>, 2>& regularStrides,
                                  const SmallVector<size_t>& reducingOpDims, const array<SmallVector<ptrdiff_t>, 2>& reducingStrides);
template void TensorOpN<float, 3>(float beta, array<float*, 3> pointers, float alpha, ElementWiseOperator op, ElementWiseOperator reductionOp,
                                  const array<size_t, 3>& offsets,
                                  const SmallVector<size_t>& regularOpDims, const array<SmallVector<ptrdiff_t>, 3>& regularStrides,
                                  const SmallVector<size_t>& reducingOpDims, const array<SmallVector<ptrdiff_t>, 3>& reducingStrides);
template void TensorOpN<float, 4>(float beta, array<float*, 4> pointers, float alpha, ElementWiseOperator op, ElementWiseOperator reductionOp,
                                  const array<size_t, 4>& offsets,
                                  const SmallVector<size_t>& regularOpDims, const array<SmallVector<ptrdiff_t>, 4>& regularStrides,
                                  const SmallVector<size_t>& reducingOpDims, const array<SmallVector<ptrdiff_t>, 4>& reducingStrides);
template void TensorOpN<double, 2>(double beta, array<double*, 2> pointers, double alpha, ElementWiseOperator op, ElementWiseOperator reductionOp,
                                   const array<size_t, 2>& offsets,
                                   const SmallVector<size_t>& regularOpDims, const array<SmallVector<ptrdiff_t>, 2>& regularStrides,
                                   const SmallVector<size_t>& reducingOpDims, const array<SmallVector<ptrdiff_t>, 2>& reducingStrides);
template void TensorOpN<double, 3>(double beta, array<double*, 3> pointers, double alpha, ElementWiseOperator op, ElementWiseOperator reductionOp,
                                   const array<size_t, 3>& offsets,
                                   const SmallVector<size_t>& regularOpDims, const array<SmallVector<ptrdiff_t>, 3>& regularStrides,
                                   const SmallVector<size_t>& reducingOpDims, const array<SmallVector<ptrdiff_t>, 3>& reducingStrides);
template void TensorOpN<double, 4>(double beta, array<double*, 4> pointers, double alpha, ElementWiseOperator op, ElementWiseOperator reductionOp,
                                   const array<size_t, 4>& offsets,
                                   const SmallVector<size_t>& regularOpDims, const array<SmallVector<ptrdiff_t>, 4>& regularStrides,
                                   const SmallVector<size_t>& reducingOpDims, const array<SmallVector<ptrdiff_t>, 4>& reducingStrides);

template void LaunchUnaryTensorOp(float beta, const float* pa, float* pb, float alpha, ElementWiseOperator op, size_t regularOpDim);
template void LaunchUnaryTensorOp(double beta, const double* pa, double* pb, double alpha, ElementWiseOperator op, size_t regularOpDim);

}}}

#endif // CPUONLY
