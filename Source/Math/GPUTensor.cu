#include "hip/hip_runtime.h"
//
// Copyright (c) Microsoft. All rights reserved.
// Licensed under the MIT license. See LICENSE.md file in the project root for full license information.
//

#include "stdafx.h"
#include "Basics.h"
#include "BestGpu.h"

#ifndef CPUONLY

#include "GPUTensor.h"
#include "GPUMatrix.h"
#include "GPUMatrixCUDAKernels.cuh"
#include "CommonMatrix.h"
#define TENSOR_OPS_DECL __device__ __host__
#include "TensorOps.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include <assert.h>

#ifndef let
#define let const auto
#endif

#pragma comment(lib, "cudart.lib") // instruct linker to reference these libs
#pragma comment(lib, "cublas.lib")

#pragma warning(disable : 4267) // conversion from 'size_t' to 'unsigned int'; happens in CUDA <<<a,b>>> syntax if a and b are size_t
#pragma warning(disable : 4127) // conditional expression is constant; "if (sizeof(ElemType)==sizeof(float))" triggers this
#pragma warning(disable : 4702) // unreachable code; triggered for unknown reasons

#ifdef _WIN32
// thread local storage to access the current stream, initalize to default stream
__declspec(thread)
#endif
    extern hipStream_t t_stream;

namespace Microsoft { namespace MSR { namespace CNTK {

// =======================================================================
// TensorView support
// =======================================================================

// To save time, this makes extensive use of templates and macros.

// -----------------------------------------------------------------------
// simple fixed-size arrays for passing dimension information by value
// since CUDA can't just take our std::array and std::vector
// -----------------------------------------------------------------------

template <typename T, size_t N>
struct FixedArray
{
    T m_data[N];
    __device__ __host__ size_t size() const
    {
        return N;
    }
    __device__ __host__ T& operator[](size_t n)
    {
        return m_data[n];
    }
    __device__ __host__ T operator[](size_t n) const
    {
        return m_data[n];
    }
    template <class VEC>
    FixedArray(const VEC& data) // construct from CPU-side STL array or vector
    {
        assert(data.size() == N);
        for (size_t n = 0; n < N; n++)
        {
            m_data[n] = (T) data[n];
            if (m_data[n] != data[n]) // overflow check
                InvalidArgument("FixedArray: Dimensions out of range, too few bits.");
        }
    }
};
template <typename T> // specialized version for 0 elements
struct FixedArray<T, 0>
{
    __device__ __host__ size_t size() const
    {
        return 0;
    }
    template <class VEC>
    FixedArray(const VEC& data)
    {
        assert(data.size() == 0);
        UNUSED(data);
    }
    FixedArray()
    {
    }
};

template <typename T, size_t N, size_t K> // N = which input/output; K = index depth
struct FixedMatrix
{
    T m_data[N][K];
    __device__ __host__ size_t getNumRows() const
    {
        return N;
    }
    __device__ __host__ size_t getNumCols() const
    {
        return K;
    }
    __device__ __host__ T& operator()(size_t n, size_t k)
    {
        return m_data[n][k];
    }
    __device__ __host__ T operator()(size_t n, size_t k) const
    {
        return m_data[n][k];
    }
    template <typename U>
    FixedMatrix(const array<SmallVector<U>, N>& data) // construct from CPU-side array of vectors
    {
        assert(data.size() == N);
        for (size_t n = 0; n < N; n++)
        {
            assert(data[n].size() == K);
            for (size_t k = 0; k < K; k++)
            {
                m_data[n][k] = (T) data[n][k];
                if (m_data[n][k] != data[n][k]) // overflow check
                    InvalidArgument("FixedArray: Dimensions out of range, too few bits.");
            }
        }
    }
};
template <typename T, size_t N> // specialized version for 0 elements
struct FixedMatrix<T, N, 0>
{
    __device__ __host__ size_t getNumRows() const
    {
        return N;
    }
    __device__ __host__ size_t getNumCols() const
    {
        return 0;
    }
    template <typename U>
    FixedMatrix(const array<SmallVector<U>, N>& data)
    {
        assert(data.size() == N);
        for (size_t n = 0; n < N; n++)
            assert(data[n].size() == 0);
        UNUSED(data);
    }
    FixedMatrix()
    {
    }
};

// -----------------------------------------------------------------------
// function to actually compute a function of (N-1) inputs based on the opcode
// -----------------------------------------------------------------------

template <class ElemType>
struct TensorOps
{
    static __device__ ElemType Compute(const FixedArray<ElemType*, 1>& pointers, ElementWiseOperator op)
    {
#define CaseNullaryTensorOp(oper)       \
    case ElementWiseOperator::op##oper: \
        return Op##oper<ElemType>()
        switch (op)
        {
            ForAllNullaryOps(CaseNullaryTensorOp);
        default:
            return OpConstOne<ElemType>(); // (failure--we only have one nullary op, so use the same, maybe it will eliminate the switch altogether)
        }
    }
    static __device__ ElemType Compute(const FixedArray<ElemType*, 2>& pointers, ElementWiseOperator op)
    {
        ElemType a = *(pointers[0]);
#define CaseUnaryTensorOp(oper)         \
    case ElementWiseOperator::op##oper: \
        return Op##oper(a)
        switch (op)
        {
            ForAllUnaryOps(CaseUnaryTensorOp);
        default:
            return 0; // (failure)
        }
    }
    static __device__ ElemType Compute(const FixedArray<ElemType*, 3>& pointers, ElementWiseOperator op)
    {
        // const ElemType & a = *(pointers[0]);    // const & for opIndex--costs quite some code bloat
        ElemType a = *(pointers[0]);
        ElemType b = *(pointers[1]);
#define CaseBinaryTensorOp(oper)        \
    case ElementWiseOperator::op##oper: \
        return Op##oper(a, b)
        switch (op)
        {
            ForAllBinaryOps(CaseBinaryTensorOp); // note: this costs about 6% compared to having only a single case
        default:
            return 0; // (failure)
        }
    }
    static __device__ ElemType Compute(const FixedArray<ElemType*, 4>& pointers, ElementWiseOperator op)
    {
        ElemType a = *(pointers[0]);
        ElemType b = *(pointers[1]);
        ElemType c = *(pointers[2]);
#define CaseTernaryTensorOp(oper)       \
    case ElementWiseOperator::op##oper: \
        return Op##oper(a, b, c)
        switch (op)
        {
            ForAllTernaryOps(CaseTernaryTensorOp);
        default:
            return 0; // (failure)
        }
    }
};

// -----------------------------------------------------------------------
// function to compute the value for a given output location (this version performs reduction if needed)
// -----------------------------------------------------------------------

//#define ReduceElemType double
#define ReduceElemType ElemType

template <class ElemType, C_size_t N, C_int M, C_int m>
struct TensorOpReduce
{
    // this version for m >= 0
    static __device__ ElemType Compute(FixedArray<ElemType*, N> pointers, ElementWiseOperator op,
                                       const FixedArray<C_unsigned_int, M>& reducingOpDims, const FixedMatrix<C_int, N, M>& reducingStrides)
    {
        // start with index 0
        // We may use 'double' since we are memory-bound anyway.
        ReduceElemType aggregate = TensorOpReduce<ElemType, N, M, m - 1>::Compute(pointers, op, reducingOpDims, reducingStrides);
        // apply this index to the pointers
        C_size_t dim = reducingOpDims[m];
        for (C_size_t k = 1 /*done with k=0 already*/; k < dim; k++)
        {
            // bump the pointers
            for (C_size_t i = 0; i < N - 1; i++) // N-1 because output is not used here
                pointers[i] += reducingStrides(i, (C_size_t) m);
            ElemType val = TensorOpReduce<ElemType, N, M, m - 1>::Compute(pointers, op, reducingOpDims, reducingStrides);
            aggregate += val;
        }
        return (ElemType) aggregate;
    }
};

// this one terminates the template recursion over reduction dimensions
// The pointers are pointing to the input element.
template <class ElemType, C_size_t N, C_int M>
struct TensorOpReduce<ElemType, N, M, /*m=*/-1>
{
    // this version for m = -1
    // the pointers are pointing to the right location(s) to take the operation over
    static __device__ ElemType Compute(FixedArray<ElemType*, N> pointers, ElementWiseOperator op,
                                       const FixedArray<C_unsigned_int, M>& /*reducingOpDims*/, const FixedMatrix<C_int, N, M>& /*reducingStrides*/)
    {
        return TensorOps<ElemType>::Compute(pointers, op); // finally computing something!
    }
};

// -----------------------------------------------------------------------
// function to compute one constituent of the value for a given output location (this version has reduction done outside)
// -----------------------------------------------------------------------

template <class ElemType, C_size_t N, C_int M, C_int m>
struct TensorOpParallelReduce
{
    // this version for m >= 0
    static __device__ ElemType Compute(CUDA_LONG id, FixedArray<ElemType*, N> pointers, ElementWiseOperator op,
                                       const FixedArray<C_unsigned_int, M>& reducingOpDims, const FixedMatrix<C_int, N, M>& reducingStrides)
    {
        // map id (location on grid) to index[k]
        C_size_t stride = 1; // compute the stride. This seems expensive, but since we we only currently support M <= 2, this is just compile-time selection between 1 and reducingOpDims[0].
        for (int i = 0; i < m; i++)
            stride *= reducingOpDims[(C_size_t) i];
        C_size_t index = id / stride; // this dimension. For m=0, the stride is 1 and hence the division will be removed at compile time.
        id = id % stride;             // remaining dimensions inside this. For m=0 this value is ignored and hence not even computed.
        // apply this index to the pointers
        for (C_size_t i = 0; i < N - 1; i++)
            pointers[i] += index * reducingStrides(i, (C_size_t) m); // now this dimension is taken care of
        return TensorOpParallelReduce<ElemType, N, M, m - 1>::Compute(id, pointers, op, reducingOpDims, reducingStrides);
    }
};

// this one terminates the template recursion over reduction dimensions
// The pointers are pointing to the input element.
template <class ElemType, C_size_t N, C_int M>
struct TensorOpParallelReduce<ElemType, N, M, /*m=*/-1>
{
    // this version for m = -1
    // the pointers are pointing to the right location(s) to take the operation over
    static __device__ ElemType Compute(CUDA_LONG /*id*/, FixedArray<ElemType*, N> pointers, ElementWiseOperator op,
                                       const FixedArray<C_unsigned_int, M>& /*reducingOpDims*/, const FixedMatrix<C_int, N, M>& /*reducingStrides*/)
    {
        return TensorOps<ElemType>::Compute(pointers, op); // finally computing something!
    }
};

// -----------------------------------------------------------------------
// perform loop over regular index k for N-nary operations (N counting the output)
// -----------------------------------------------------------------------

// The canonical case, vector op without reduction, is this PTX function:
// _ZN9Microsoft3MSR4CNTK15_launchTensorOpIfLi3ELi0ELi1EEEvT_NS1_10FixedArrayIPS3_XT0_EEES3_NS1_19ElementWiseOperatorENS4_IiXT2_EEENS1_11FixedMatrixIiXT0_EXT2_EEENS4_IiXT1_EEENS9_IiXT0_EXT1_EEEi
//                                   float ^      ^ aggregate loop
//                                      args? ^       ^ input dims
// _ZN9Microsoft3MSR4CNTK15_launchTensorOpIfLi2ELi0ELi1EEEvT_NS1_10FixedArrayIPS3_XT0_EEES3_NS1_19ElementWiseOperatorENS4_IiXT2_EEENS1_11FixedMatrixIiXT0_EXT2_EEENS4_IiXT1_EEENS9_IiXT0_EXT1_EEEi

// The 'pointers' only refer to a single element, so we will bump them in-place to perform indexing.
template <class ElemType, C_size_t N, C_int M, C_int K, bool parallelReduce, C_int k>
struct TensorOpElement
{
    // template-recursive version loops over indices
    static __device__ void Compute(CUDA_LONG id, ElemType beta, FixedArray<ElemType*, N>& pointers, ElemType alpha, ElementWiseOperator op,
                                   const FixedArray<C_unsigned_int, K>& regularOpStrides, const FixedMatrix<C_int, N, K>& regularStrides,
                                   const FixedArray<C_unsigned_int, M>& reducingOpDims, const FixedMatrix<C_int, N, M>& reducingStrides,
                                   CUDA_LONG reductionBegin, CUDA_LONG reductionChunkSize)
    {
        // map id (location on grid) to index[k]
        C_size_t stride = regularOpStrides[(C_size_t) k];
        C_size_t index = id / stride; // this dimension
        id = id % stride;             // remaining dimensions inside this
        // apply this index to the pointers
        for (C_size_t i = 0; i < N; i++)
            pointers[i] += index * regularStrides(i, (C_size_t) k); // now this dimension is taken care of
        // process the previous index
        TensorOpElement<ElemType, N, M, K, parallelReduce, k - 1>::Compute(id, beta, pointers, alpha, op, regularOpStrides, regularStrides, reducingOpDims, reducingStrides, reductionBegin, reductionChunkSize);
    }
};

// specialization for k=0 where op stride is guaranteed to be 1
template <class ElemType, C_size_t N, C_int M, C_int K, bool parallelReduce>
struct TensorOpElement<ElemType, N, M, K, parallelReduce, /*k=*/0>
{
    // template-recursive version loops over indices
    static __device__ void Compute(CUDA_LONG id, ElemType beta, FixedArray<ElemType*, N>& pointers, ElemType alpha, ElementWiseOperator op,
                                   const FixedArray<C_unsigned_int, K>& regularOpStrides, const FixedMatrix<C_int, N, K>& regularStrides,
                                   const FixedArray<C_unsigned_int, M>& reducingOpDims, const FixedMatrix<C_int, N, M>& reducingStrides,
                                   CUDA_LONG reductionBegin, CUDA_LONG reductionChunkSize)
    {
        // map id (location on grid) to index[k]
        C_size_t index = id; // this dimension
        // apply this index to the pointers
        for (C_size_t i = 0; i < N; i++)
            pointers[i] += index * regularStrides(i, 0); // now this dimension is taken care of
        // process the previous index
        TensorOpElement<ElemType, N, M, K, parallelReduce, -1>::Compute(/*id*/ 0, beta, pointers, alpha, op, regularOpStrides, regularStrides, reducingOpDims, reducingStrides, reductionBegin, reductionChunkSize);
    }
};

//// apply beta and alpha and save
//template<class ElemType, class PointersType>
//static __device__ void SetFinalValue(ElemType val, ElemType beta, const PointersType & pointers, ElemType alpha)
//{
//    // scale
//    val *= alpha;
//    // combine with previous value in target matrix, then write it out
//    auto * pout = pointers[pointers.size() - 1];
//    if (beta != 0)
//        val += beta * *pout;
//    // save
//    *pout = val;
//}

// specialization for k = -1 terminates the template recursion, and computes reductions in a for loop
template <class ElemType, C_size_t N, C_int M, C_int K>
struct TensorOpElement<ElemType, N, M, K, /*parallelReduce=*/false, /*k=*/-1>
{
    // template-recursion-teminating version computes the actual value for this output location
    // now the output pointers point to the right element (input pointers may still iterate for reduction)
    static __device__ void Compute(CUDA_LONG /*id*/, ElemType beta, FixedArray<ElemType*, N>& pointers, ElemType alpha, ElementWiseOperator op,
                                   const FixedArray<C_unsigned_int, K>& /*regularOpStrides*/, const FixedMatrix<C_int, N, K>& /*regularStrides*/,
                                   const FixedArray<C_unsigned_int, M>& reducingOpDims, const FixedMatrix<C_int, N, M>& reducingStrides, CUDA_LONG /*reductionBegin*/, CUDA_LONG /*reductionChunkSize*/)
    {
        // compute the operation for this output coordinate
        // This may still involve a reduction over inverse-broadcasting dimensions.
        ElemType val = TensorOpReduce<ElemType, N, M, M - 1>::Compute(pointers, op, reducingOpDims, reducingStrides);
        // scale
        val *= alpha;
        // combine with previous value in target matrix, then write it out
        auto* pout = pointers[pointers.size() - 1];
        if (beta != 0)
            val += beta * *pout;
        // save
        *pout = val;
    }
};

// specialization for k = -1 terminates the template recursion, and computes reductions in parallel
template <class ElemType, C_size_t N, C_int M, C_int K>
struct TensorOpElement<ElemType, N, M, K, /*parallelReduce=*/true, /*k=*/-1>
{
    // template-recursion-teminating version computes the actual value for this output location
    // now the output pointers point to the right element (input pointers may still iterate for reduction)
    static __device__ void Compute(CUDA_LONG /*id*/, ElemType beta, FixedArray<ElemType*, N>& pointers, ElemType alpha, ElementWiseOperator op,
                                   const FixedArray<C_unsigned_int, K>& /*regularOpStrides*/, const FixedMatrix<C_int, N, K>& /*regularStrides*/,
                                   const FixedArray<C_unsigned_int, M>& reducingOpDims, const FixedMatrix<C_int, N, M>& reducingStrides, CUDA_LONG reductionBegin, CUDA_LONG reductionChunkSize)
    {
        CUDA_LONG reductionBlock = blockIdx.z; // block index  --larger reductions are split into blocks
        CUDA_LONG reductionBlocks = gridDim.z; // number of blocks
        CUDA_LONG tid = threadIdx.x;           // thread index
        CUDA_LONG tids = blockDim.x;           // out of how many threads  --note: last block is partial

        // determine our range  --this is a single int mul, we can stomach it (we could alternatively pass in yet another parameter)
        CUDA_LONG reductionDim = (CUDA_LONG) reducingOpDims[0];
        for (C_size_t i = 1; i < reducingOpDims.size(); i++)
            reductionDim *= reducingOpDims[i];

        // determine the redId range that we operate on
        // Each thread takes a stride tid + (multiples of tids) within this range.
        reductionBegin += reductionChunkSize * reductionBlock;
        CUDA_LONG reductionEnd = min(reductionBegin + reductionChunkSize, reductionDim);

        // compute the operation for this input coordinate
        ReduceElemType sum = 0;
        for (CUDA_LONG redId = reductionBegin + tid; redId < reductionEnd; redId += tids)
        {
            auto val = TensorOpParallelReduce<ElemType, N, M, M - 1>::Compute(redId, pointers, op, reducingOpDims, reducingStrides);
            sum += val;
        }

        // reduce    --cf https://docs.nvidia.com/cuda/samples/6_Advanced/reduction/doc/reduction.pdf
        __shared__ ReduceElemType accumulators[GridDim::maxThreadsPerBlock /*tids*/];
        accumulators[tid] = sum;
        __syncthreads();
        static_assert(GridDim::maxThreadsPerBlock <= 512, "GridDim::maxThreadsPerBlock too large, need to add manually unrolled steps");
        for (CUDA_LONG i = 256; i; i >>= 1)
        {
            if (tid < i && tid + i < tids)
                accumulators[tid] += accumulators[tid + i];
            if (0 + i < tids)
                __syncthreads(); // sync if condition true for at least one thread
            // TODO: use volatile* and then we can skip the __syncthreads() for the last 32 values. See Amit's allreduce() function implementation in MatrixQuantizer_kernel.cu.
        }

        // now set final value to output coordinate
        if (tid == 0)
        {
            ElemType val = (ElemType) accumulators[0];
            // scale
            val *= alpha;
            // combine with previous value in target matrix, then write it out
            auto* pout = pointers[pointers.size() - 1];
            if (reductionBlocks > 1) // multiple blocks: need to use atomicAdd()
            {
                // in this case, outer calling code must pass beta = 1
                val = atomicAdd(pout, val);
            }
            else
            {
                if (beta != 0)
                    val += beta * *pout;
                // save
                *pout = val;
            }
        }
    }
};

// -----------------------------------------------------------------------
// kernel and launch  --no reduction
// -----------------------------------------------------------------------

// launch tensor op with CUDA
template <class ElemType, C_size_t N, C_int M, C_int K>
__global__ void _launchTensorOp(ElemType beta, FixedArray<ElemType*, N> pointers, ElemType alpha, ElementWiseOperator op,
                                FixedArray<C_unsigned_int, K> regularOpStrides, FixedMatrix<C_int, N, K> regularStrides, CUDA_LONG numElements,
                                FixedArray<C_unsigned_int, M> reducingOpDims, FixedMatrix<C_int, N, M> reducingStrides)
{
    CUDA_LONG id = GridDim::GetLinearThreadId();
    if (id < numElements) // note: there are no __syncthread() calls inside
        TensorOpElement<ElemType, N, M, K, false, K - 1>::Compute(id, beta, pointers, alpha, op, regularOpStrides, regularStrides, reducingOpDims, reducingStrides, 0, 0);
}

template <class ElemType, C_size_t N, C_int K>
static void LaunchTensorOp(ElemType beta, array<ElemType*, N> pointerVector, ElemType alpha, ElementWiseOperator op,
                           const SmallVector<size_t>& regularOpDims, const array<SmallVector<ptrdiff_t>, N>& regularStrideVectors)
{
    // copy all parameters to CUDA-compatible data structures
    FixedArray<ElemType*, N> pointers(pointerVector);
    SmallVector<C_size_t> regularOpStrideVector; // kernel needs the strides for converting thread index back to multi-dimensional tensor index
    C_size_t numElements = 1;
    for (C_size_t k = 0; k < regularOpDims.size(); k++)
    {
        regularOpStrideVector.push_back(numElements);
        numElements *= (C_size_t) regularOpDims[k];
    }
    FixedArray<C_unsigned_int, K> regularOpStrides(regularOpStrideVector);
    FixedMatrix<C_int, N, K> regularStrides(regularStrideVectors);
    FixedArray<C_unsigned_int, /*M=*/0> reducingOpDims; // empty reduction dimensions
    FixedMatrix<C_int, N, /*M=*/0> reducingStrides;

    // launch the kernel
    CUDA_LONG NN = (CUDA_LONG) numElements; // linear space identifying each individual input element
    SyncGuard syncGuard;
    GridDim grid(NN);
    _launchTensorOp<ElemType, N, /*M=*/0, K><<<grid.m_blocksPerGrid, grid.m_threadsPerBlock, 0, t_stream>>>(beta, pointers, alpha, op, regularOpStrides, regularStrides, grid.m_N, reducingOpDims, reducingStrides);
}

// -----------------------------------------------------------------------
// kernel and launch  --with reduction
// -----------------------------------------------------------------------

template <class ElemType, C_size_t N, C_int M, C_int K>
__global__ void _launchTensorOpWithReduction(ElemType beta, FixedArray<ElemType*, N> pointers, ElemType alpha, ElementWiseOperator op,
                                             FixedArray<C_unsigned_int, K> regularOpStrides, FixedMatrix<C_int, N, K> regularStrides, CUDA_LONG numElements,
                                             FixedArray<C_unsigned_int, M> reducingOpDims, FixedMatrix<C_int, N, M> reducingStrides, CUDA_LONG reductionBegin, CUDA_LONG reductionChunkSize)
{
    CUDA_LONG id = gridDim.x * blockIdx.y + blockIdx.x; // input dimensions are Y dimension of blocks in this case, so we can use thread dim for shared-memory/parallelization
    if (id < numElements)                               // note: we have __syncthread() calls but only entire blocks in sync, so this is OK
        TensorOpElement<ElemType, N, M, K, true, K - 1>::Compute(id, beta, pointers, alpha, op, regularOpStrides, regularStrides, reducingOpDims, reducingStrides, reductionBegin, reductionChunkSize);
}

// All dimensions (N-ariness, number of input dimensions K and number of reduction dimensions M) are bound to template parameters now.
template <class ElemType, C_size_t N, C_int M, C_int K>
static void LaunchTensorOpWithReduction(ElemType beta, array<ElemType*, N> pointerVector, ElemType alpha, ElementWiseOperator op,
                                        const SmallVector<size_t>& regularOpDims, const array<SmallVector<ptrdiff_t>, N>& regularStrideVectors,
                                        const SmallVector<size_t>& reducingOpDimVector, const array<SmallVector<ptrdiff_t>, N>& reducingStrideVectors)
{
    // copy all parameters to CUDA-compatible data structures
    FixedArray<ElemType*, N> pointers(pointerVector);
    SmallVector<C_size_t> regularOpStrideVector; // kernel needs the strides for converting thread index back to multi-dimensional tensor index
    C_size_t numElements = 1;
    for (C_size_t k = 0; k < regularOpDims.size(); k++)
    {
        regularOpStrideVector.push_back(numElements);
        numElements *= (C_size_t) regularOpDims[k];
    }
    FixedArray<C_unsigned_int, K> regularOpStrides(regularOpStrideVector);
    FixedMatrix<C_int, N, K> regularStrides(regularStrideVectors);
    FixedArray<C_unsigned_int, M> reducingOpDims(reducingOpDimVector);
    FixedMatrix<C_int, N, M> reducingStrides(reducingStrideVectors);

    // launch the kernel
    CUDA_LONG NN = (CUDA_LONG) numElements; // linear space identifying each individual input element
    SyncGuard syncGuard;

    // do some optimization for reductions
    // Cases:
    //  - #output elements >= GPU procs  -->  use one proc per element, do reduction in inner loop
    //  - reduction dimension fits into a single kernel  -->  launch it that way
    //  - reduction dimension requires multiple kernels  -->  use atomic add, to avoid temp mem alloc
    //     - PlusNode: reducing to a bias for small matrices
    //     - ScaleNode: big elementwise product reduced to a scalar (dot product)
    //     - E.g. 3072 GPU procs:
    //       If >= 3072 reduced output values must be computed, just loop inside.
    //       If less, and reduction per value does not fit into a single proc,
    //       then we break it into procs, say, 24.
    //       This way we will need 24 atomicAdd()s of 3072/24 = 128 values.
    //       If reduction is along stride=1, then we'd have 24 atomicAdd()s of 32 coalesced writes.
    //       Does not sound scary at all.
    //       Precondition: matrix cannot at the same time participate in reduction and operation.
    C_size_t reductionDim = 1; // number of elements to reduce over
    for (C_size_t k = 0; k < reducingOpDimVector.size(); k++)
        reductionDim *= (C_size_t) reducingOpDimVector[k];
    let& props = GridDim::GetDeviceProps();
    GridDim grid(NN);
    if (reductionDim > 1 && grid.m_blocksPerGrid < props.multiProcessorCount /*    && NN == 10 && reductionDim <= GridDim::maxThreadsPerBlock*/)
    {
        // we are reducing and are underutilizing the multiprocs we have: get more parallelism by doing reduction in parallel
        // Change of strategy: All NN elements get their own block. Reduction gets split over blocks as well.

        // By how much do we underutilize?
        // We increase #blocks by that factor by breaking reduction into that many chunks.
        let numReductionChunks = CeilDiv(props.multiProcessorCount, NN);

        // NN may be too large for a single dimension
        let blockXOverBy = CeilDiv(NN, props.maxGridSize[0]);
        let numBlocksX = CeilDiv(NN, blockXOverBy);
        let numBlocksY = CeilDiv(NN, numBlocksX);
        let numBlocksZ = numReductionChunks;
        // Block dim is now:
        //  - X, Y: such that X*Y covers NN
        //  - Z: reduction chunks

        // reduction goes into thread dim X
        let reductionChunkSize = CeilDiv(reductionDim, numReductionChunks);
        let numThreadsX = min(reductionChunkSize, GridDim::maxThreadsPerBlock); // any that's over will be done by looping inside the kernel

        if (beta == 1 || numBlocksZ == 1)
        {
            _launchTensorOpWithReduction<ElemType, N, M, K><<<dim3(numBlocksX, numBlocksY, numBlocksZ), numThreadsX, numThreadsX * sizeof(ReduceElemType), t_stream>>>(/*beta=*/1, pointers, alpha, op, regularOpStrides, regularStrides, NN, reducingOpDims, reducingStrides, 0, reductionChunkSize);
        }
        else
        {
            // We need more than one chunk, we will use atomicAdd().
            // First reset/pre-multiply input; then do the remaining chunks using atomicAdd().
            _launchTensorOpWithReduction<ElemType, N, M, K><<<dim3(numBlocksX, numBlocksY, 1), numThreadsX, numThreadsX * sizeof(ReduceElemType), t_stream>>>(beta, pointers, alpha, op, regularOpStrides, regularStrides, NN, reducingOpDims, reducingStrides, 0, reductionChunkSize);
            _launchTensorOpWithReduction<ElemType, N, M, K><<<dim3(numBlocksX, numBlocksY, numBlocksZ - 1), numThreadsX, numThreadsX * sizeof(ReduceElemType), t_stream>>>(/*beta=*/1, pointers, alpha, op, regularOpStrides, regularStrides, NN, reducingOpDims, reducingStrides, reductionChunkSize, reductionChunkSize);
        }
    }
    else
    {
        // we got enough elements to generate: do one element per thread, and reduction inside
        _launchTensorOp<ElemType, N, M, K><<<grid.m_blocksPerGrid, grid.m_threadsPerBlock, 0, t_stream>>>(beta, pointers, alpha, op, regularOpStrides, regularStrides, grid.m_N, reducingOpDims, reducingStrides);
    }
}

// -----------------------------------------------------------------------
// kernel and launch  --linear unary
// -----------------------------------------------------------------------

// for linear unary ops, we need to define a functor for every function for use as a template parameter (lambda syntax doesn't work in CUDA 7)
#define DefineUnaryTensorFunctor(oper)           \
    struct Functor##oper                         \
    {                                            \
        template <class ElemType>                \
        static __device__ ElemType f(ElemType a) \
        {                                        \
            return Op##oper(a);                  \
        }                                        \
    };
ForAllUnaryOps(DefineUnaryTensorFunctor);

// the top-level kernel for linear unary ops
// Note: If we have a beta, we have 2 memory accesses, so this optimization may no longer be needed as we are memory-bound.
template <class ElemType, class FN>
__global__ void _launchUnaryTensorOp(ElemType beta, const ElemType* pa, ElemType* pb, ElemType alpha, CUDA_LONG numElements)
{
    CUDA_LONG id = GridDim::GetLinearThreadId();
    if (id >= numElements)
        return;
    ElemType a = pa[id];
    ElemType val = FN::f(a);
    val *= alpha;
    if (beta != 0)
        val += beta * pb[id];
    pb[id] = val;
}
// version without beta and alpha
template <class ElemType, class FN>
__global__ void _launchUnaryTensorOp(const ElemType* pa, ElemType* pb, CUDA_LONG numElements)
{
    CUDA_LONG id = GridDim::GetLinearThreadId();
    if (id >= numElements)
        return;
    ElemType a = pa[id];
    ElemType val = FN::f(a);
    pb[id] = val;
}

// special case of linear unary operation
template <class ElemType>
void LaunchUnaryTensorOp(ElemType beta, const ElemType* pa, ElemType* pb, ElemType alpha, ElementWiseOperator op, size_t regularOpDim)
{
    CUDA_LONG NN = (CUDA_LONG) regularOpDim;

#define CaseLaunchUnaryTensorOp(oper)                                                                                                        \
    case ElementWiseOperator::op##oper:                                                                                                      \
        if (beta == 0 && alpha == 1)                                                                                                         \
            _launchUnaryTensorOp<ElemType, Functor##oper><<<grid.m_blocksPerGrid, grid.m_threadsPerBlock, 0, t_stream>>>(pa, pb, NN); \
        else                                                                                                                                 \
            _launchUnaryTensorOp<ElemType, Functor##oper><<<grid.m_blocksPerGrid, grid.m_threadsPerBlock, 0, t_stream>>>(beta, pa, pb, alpha, NN);\
        break;

    SyncGuard syncGuard;
    GridDim grid(NN);
    switch (op)
    {
        ForAllUnaryOps(CaseLaunchUnaryTensorOp);
    default:
        LogicError("LaunchTensorOp1: Unknown op code %d.", (int) op);
    }
}

// -----------------------------------------------------------------------
// map runtime parameters N to template parameters
// -----------------------------------------------------------------------

// tensor operation with k+1 dimensions (-1 means scalar)
template <class ElemType, C_size_t N, C_int K>
static void TensorOpWithRegularLoop(ElemType beta, const array<ElemType*, N>& pointers, ElemType alpha, ElementWiseOperator op,
                                    const SmallVector<size_t>& regularOpDims, const array<SmallVector<ptrdiff_t>, N>& regularStrides,
                                    const SmallVector<size_t>& reducingOpDims, const array<SmallVector<ptrdiff_t>, N>& reducingStrides)
{
    size_t dims = reducingOpDims.size();
    switch (dims)
    {
    case 2:
        return LaunchTensorOpWithReduction<ElemType, N, 2, K>(beta, pointers, alpha, op, regularOpDims, regularStrides, reducingOpDims, reducingStrides);
    case 1:
        return LaunchTensorOpWithReduction<ElemType, N, 1, K>(beta, pointers, alpha, op, regularOpDims, regularStrides, reducingOpDims, reducingStrides);
    case 0:
        return LaunchTensorOp<ElemType, N, K>(beta, pointers, alpha, op, regularOpDims, regularStrides);
    default:
        LogicError("TensorOp: %d non-flattened reduction dimensions are not supported.", (C_int) dims);
    }
}

// tensor operation, generalized in number of arguments
// This function now expands into different k. It also eliminates the offsets by adding them to the pointers.
template <class ElemType, C_size_t N>
void TensorOpN(ElemType beta, array<ElemType*, N> pointers, ElemType alpha, ElementWiseOperator op,
               const array<size_t, N>& offsets,
               const SmallVector<size_t>& regularOpDims, const array<SmallVector<ptrdiff_t>, N>& regularStrides,
               const SmallVector<size_t>& reducingOpDims, const array<SmallVector<ptrdiff_t>, N>& reducingStrides)
{
    for (C_size_t i = 0; i < N; i++) // N = a small constant, this will be unrolled
        pointers[i] += offsets[i];
    size_t dims = regularOpDims.size();
    switch (dims)
    {
    case 4:
        return TensorOpWithRegularLoop<ElemType, N, 4>(beta, pointers, alpha, op, regularOpDims, regularStrides, reducingOpDims, reducingStrides);
    case 3:
        return TensorOpWithRegularLoop<ElemType, N, 3>(beta, pointers, alpha, op, regularOpDims, regularStrides, reducingOpDims, reducingStrides);
    case 2:
        return TensorOpWithRegularLoop<ElemType, N, 2>(beta, pointers, alpha, op, regularOpDims, regularStrides, reducingOpDims, reducingStrides);
    case 1:
        return TensorOpWithRegularLoop<ElemType, N, 1>(beta, pointers, alpha, op, regularOpDims, regularStrides, reducingOpDims, reducingStrides);
    case 0:
        return TensorOpWithRegularLoop<ElemType, N, 0>(beta, pointers, alpha, op, regularOpDims, regularStrides, reducingOpDims, reducingStrides);
    default:
        LogicError("TensorOp: %d non-flattened input dimensions are not supported.", (C_int) dims);
    }
}

//------------------------------------------------------------------------
// explicit instantiations--these are being called from GPUMatrix.cu
//------------------------------------------------------------------------

template void TensorOpN<float, 2>(float beta, array<float*, 2> pointers, float alpha, ElementWiseOperator op,
                                  const array<size_t, 2>& offsets,
                                  const SmallVector<size_t>& regularOpDims, const array<SmallVector<ptrdiff_t>, 2>& regularStrides,
                                  const SmallVector<size_t>& reducingOpDims, const array<SmallVector<ptrdiff_t>, 2>& reducingStrides);
template void TensorOpN<float, 3>(float beta, array<float*, 3> pointers, float alpha, ElementWiseOperator op,
                                  const array<size_t, 3>& offsets,
                                  const SmallVector<size_t>& regularOpDims, const array<SmallVector<ptrdiff_t>, 3>& regularStrides,
                                  const SmallVector<size_t>& reducingOpDims, const array<SmallVector<ptrdiff_t>, 3>& reducingStrides);
template void TensorOpN<float, 4>(float beta, array<float*, 4> pointers, float alpha, ElementWiseOperator op,
                                  const array<size_t, 4>& offsets,
                                  const SmallVector<size_t>& regularOpDims, const array<SmallVector<ptrdiff_t>, 4>& regularStrides,
                                  const SmallVector<size_t>& reducingOpDims, const array<SmallVector<ptrdiff_t>, 4>& reducingStrides);
template void TensorOpN<double, 2>(double beta, array<double*, 2> pointers, double alpha, ElementWiseOperator op,
                                   const array<size_t, 2>& offsets,
                                   const SmallVector<size_t>& regularOpDims, const array<SmallVector<ptrdiff_t>, 2>& regularStrides,
                                   const SmallVector<size_t>& reducingOpDims, const array<SmallVector<ptrdiff_t>, 2>& reducingStrides);
template void TensorOpN<double, 3>(double beta, array<double*, 3> pointers, double alpha, ElementWiseOperator op,
                                   const array<size_t, 3>& offsets,
                                   const SmallVector<size_t>& regularOpDims, const array<SmallVector<ptrdiff_t>, 3>& regularStrides,
                                   const SmallVector<size_t>& reducingOpDims, const array<SmallVector<ptrdiff_t>, 3>& reducingStrides);
template void TensorOpN<double, 4>(double beta, array<double*, 4> pointers, double alpha, ElementWiseOperator op,
                                   const array<size_t, 4>& offsets,
                                   const SmallVector<size_t>& regularOpDims, const array<SmallVector<ptrdiff_t>, 4>& regularStrides,
                                   const SmallVector<size_t>& reducingOpDims, const array<SmallVector<ptrdiff_t>, 4>& reducingStrides);

template void LaunchUnaryTensorOp(float beta, const float* pa, float* pb, float alpha, ElementWiseOperator op, size_t regularOpDim);
template void LaunchUnaryTensorOp(double beta, const double* pa, double* pb, double alpha, ElementWiseOperator op, size_t regularOpDim);

}}}

#endif // CPUONLY
