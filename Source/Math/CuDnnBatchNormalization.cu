#include "hip/hip_runtime.h"
//
// Copyright (c) Microsoft. All rights reserved.
// Licensed under the MIT license. See LICENSE.md file in the project root for full license information.
//

#include "stdafx.h"
#include "CuDnnFactories.h"
#include "BatchNormalizationEngine.h"
#include "CuDnnCommon.h"
#include "GPUMatrix.h"

namespace Microsoft { namespace MSR { namespace CNTK {

template <class ElemType>
class CuDnnBatchNormEngine : public BatchNormEngine<ElemType>
{
public:
    using Base = BatchNormEngine<ElemType>;
    using typename Base::Mat;

public:
    CuDnnBatchNormEngine(DEVICEID_TYPE deviceId, const TensorShape& inOutT,
                        bool spatial, ImageLayoutKind imageLayout)
                        : Base(deviceId, inOutT, spatial, imageLayout),
                        m_cudnn(CuDnn::Instance()),
                        m_inOutCuDnnT(GetInOutTensor(inOutT), CuDnnTensor::GetDataType<ElemType>()),
                        m_scaleBiasCuDnnT(GetScaleBiasTensor(inOutT, spatial), CuDnnTensor::GetDataType<ElemType>()),
                        m_cudnnEpsilon(HIPDNN_BN_MIN_EPSILON)
    {
    }

protected:
    using Base::m_deviceId;
    using Base::m_imageLayout;
    using Base::m_inOutT;
    using Base::m_spatial;

    void EnsureCompatible() override
    {
        if (m_spatial && m_imageLayout == ImageLayoutKind::HWC)
            InvalidArgument("cuDNN batch normalization supports only cudnn(CHW) layout.");
        if (m_inOutT.GetRank() > 4)
            InvalidArgument("cuDNN batch normalization supports tensors of max 4 dimensions.");
    }

    void ForwardCore(const Mat& in, const Mat& scale, const Mat& bias, bool inferenceOnly, double expAvgFactor, double blendFactor, Mat& runMean, Mat& runVariance,
                     Mat& out, double epsilon, Mat& savedMean, Mat& savedInvStdDev) override
    {
        // TODO batchSize == 1

        // REVIEW alexeyk: there might be a way to do this in cuDNN.
        if (blendFactor != 0 && (blendFactor != 1 || expAvgFactor > 0))
            InvalidArgument("cuDNN batch normalization engine currently supports blendTimeConstant of 0 or 1 only.");

        m_inOutCuDnnT.UpdateBatchSize(in.GetNumCols());
        hipdnnBatchNormMode_t mode = m_spatial ? HIPDNN_BATCHNORM_SPATIAL : HIPDNN_BATCHNORM_PER_ACTIVATION;
        // cuDNN will fail with BAD_PARAM if epsilon < HIPDNN_BN_MIN_EPSILON.
        m_cudnnEpsilon = max(epsilon, HIPDNN_BN_MIN_EPSILON);
        if (inferenceOnly)
        {
            assert(expAvgFactor == 0 && blendFactor == 1);
            savedMean.Resize(0, 0);      // (these are not produced in this case)
            savedInvStdDev.Resize(0, 0);
            CUDNN_CALL2(hipdnnBatchNormalizationForwardInference(*m_cudnn, mode, &C::One, &C::Zero, m_inOutCuDnnT, ptr(in), m_inOutCuDnnT, ptr(out),
                                                                  m_scaleBiasCuDnnT, ptr(scale), ptr(bias), ptr(runMean), ptr(runVariance), m_cudnnEpsilon),
                        "\nProbably hitting cuDNN limit on batch size, try reducing minibatch size");
        }
        else
        {
            savedMean.Resize(runMean);
            savedInvStdDev.Resize(runMean);
            CUDNN_CALL(hipdnnBatchNormalizationForwardTraining(*m_cudnn, mode, &C::One, &C::Zero, m_inOutCuDnnT, ptr(in),
                                                              m_inOutCuDnnT, ptr(out), m_scaleBiasCuDnnT, ptr(scale), ptr(bias), expAvgFactor, ptr(runMean), ptr(runVariance),
                                                              m_cudnnEpsilon, ptr(savedMean), ptr(savedInvStdDev)));
        }
    }

    void BackwardCore(const Mat& in, const Mat& srcGrad, Mat& grad, const Mat& scale, double blendFactor, const Mat& savedMean, const Mat& savedInvStdDev,
                      Mat& scaleGrad, Mat& biasGrad) override
    {
        UNUSED(blendFactor);  // BUGBUG: It should be used.
        m_inOutCuDnnT.UpdateBatchSize(srcGrad.GetNumCols());
        hipdnnBatchNormMode_t mode = m_spatial ? HIPDNN_BATCHNORM_SPATIAL : HIPDNN_BATCHNORM_PER_ACTIVATION;
        // REVIEW alexeyk: change betaParamDiff to 1 and update CNTK BN engine.
        CUDNN_CALL(hipdnnBatchNormalizationBackward(*m_cudnn, mode, &C::One, &C::One, &C::One, &C::Zero, m_inOutCuDnnT, ptr(in), m_inOutCuDnnT, ptr(srcGrad), m_inOutCuDnnT, ptr(grad),
                                                   m_scaleBiasCuDnnT, ptr(scale), ptr(scaleGrad), ptr(biasGrad), m_cudnnEpsilon, ptr(savedMean), ptr(savedInvStdDev)));
    }

private:
    static ElemType* ptr(Mat& src)
    {
        return src.Data();
    }
    static const ElemType* ptr(const Mat& src)
    {
        return src.Data();
    }

    static TensorShape GetInOutTensor(const TensorShape& inOutT)
    {
        // cuDNN supports only 3D and 4D tensors (in cuDNN docs it's 4D and 5D dues to N dimension)
        // even for non-spatial inputs so expand the tensor if needed.
        if (inOutT.GetRank() > 2)
            return inOutT;
        SmallVector<size_t> v(std::max(inOutT.GetRank(), (size_t)3), 1);
        for (size_t i = 0; i < inOutT.GetRank(); i++)
            v[i] = inOutT[i];
        return TensorShape(v);
    }

    static TensorShape GetScaleBiasTensor(const TensorShape& inOutT, bool spatial)
    {
        if (!spatial)
            return GetInOutTensor(inOutT);

        const auto& t = GetInOutTensor(inOutT);
        SmallVector<size_t> v(t.GetRank(), 1);
        v[v.size() - 1] = t[t.GetRank() - 1];
        return TensorShape(v);
    }

private:
    using C = Consts<ElemType>;

    CuDnn::ptr_t m_cudnn;
    CuDnnTensor m_inOutCuDnnT;
    CuDnnTensor m_scaleBiasCuDnnT;
    double m_cudnnEpsilon;
};

template class CuDnnBatchNormEngine<float>;
template class CuDnnBatchNormEngine<double>;

template <typename ElemType>
std::unique_ptr<BatchNormEngine<ElemType>> CuDnnBatchNormEngineFactory<ElemType>::Create(DEVICEID_TYPE deviceId, const TensorShape& inOutT,
                                                                                         bool spatial, ImageLayoutKind imageLayout)
{
    return std::make_unique<CuDnnBatchNormEngine<ElemType>>(deviceId, inOutT, spatial, imageLayout);
}

template class CuDnnBatchNormEngineFactory<float>;
template class CuDnnBatchNormEngineFactory<double>;

CudaTimer::~CudaTimer()
{
    // TODO: Should not throw if std::uncaught_exception()
    if (m_start != nullptr)
        CUDA_CALL(hipEventDestroy(reinterpret_cast<hipEvent_t>(m_start)));
    if (m_stop != nullptr)
        CUDA_CALL(hipEventDestroy(reinterpret_cast<hipEvent_t>(m_stop)));
}
void CudaTimer::Start()
{
    hipEvent_t start;
    hipEvent_t stop;
    if (m_start != nullptr)
        CUDA_CALL(hipEventDestroy(reinterpret_cast<hipEvent_t>(m_start)));
    if (m_stop != nullptr)
        CUDA_CALL(hipEventDestroy(reinterpret_cast<hipEvent_t>(m_stop)));
    CUDA_CALL(hipEventCreate(&start));
    CUDA_CALL(hipEventCreate(&stop));
    m_start = start;
    m_stop = stop;
    CUDA_CALL(hipEventRecord(start, GetStream()));
}
void CudaTimer::Stop()
{
    CUDA_CALL(hipEventRecord(reinterpret_cast<hipEvent_t>(m_stop), GetStream()));
    CUDA_CALL(hipEventSynchronize(reinterpret_cast<hipEvent_t>(m_stop)));
}
float CudaTimer::Elapsed()
{
    float ms;
    CUDA_CALL(hipEventElapsedTime(&ms, reinterpret_cast<hipEvent_t>(m_start), reinterpret_cast<hipEvent_t>(m_stop)));
    return ms;
}

} } }
