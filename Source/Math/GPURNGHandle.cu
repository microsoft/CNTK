//
// Copyright (c) Microsoft. All rights reserved.
// Licensed under the MIT license. See LICENSE.md file in the project root for full license information.
//
// CPUMatrix.cpp : full implementation of all matrix functions on the CPU side
//

#include "GPURNGHandle.h"
#include "GPUMatrix.h"

namespace Microsoft { namespace MSR { namespace CNTK {

GPURNGHandle::GPURNGHandle(int deviceId, unsigned long seed)
    : RNGHandle(deviceId)
{
    unsigned long long cudaSeed = seed;
    if (GetMathLibTraceLevel() > 0)
    {
        fprintf(stderr, "(GPU): creating hiprand object with seed %llu\n", cudaSeed);
    }

    CURAND_CALL(hiprandCreateGenerator(&m_generator, HIPRAND_RNG_PSEUDO_XORWOW));
    CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(m_generator, cudaSeed));
    CURAND_CALL(hiprandSetGeneratorOrdering(m_generator, HIPRAND_ORDERING_PSEUDO_SEEDED));
}

/*virtual*/ GPURNGHandle::~GPURNGHandle()
{
    if (std::uncaught_exception())
        hiprandDestroyGenerator(m_generator);
    else
        CURAND_CALL(hiprandDestroyGenerator(m_generator));
}

}}}
