#include "hip/hip_runtime.h"
//
// Copyright (c) Microsoft. All rights reserved.
// Licensed under the MIT license. See LICENSE.md file in the project root for full license information.
//

#include "Basics.h"
#include "BestGpu.h"

#ifndef CPUONLY

#include "GPUSparseMatrix.h"
#include "GPUMatrix.h"
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include "hipblas.h"
#include "GPUMatrixCUDAKernels.cuh"
#include <functional>
#include "CommonMatrix.h"
#include <iostream> // for cout/cerr
#include <assert.h>

typedef unsigned char byte;

#pragma warning(disable : 4267) // conversion from 'size_t' to 'unsigned int'; happens in CUDA <<<a,b>>> syntax if a and b are size_t
#pragma warning(disable : 4127) // conditional expression is constant; "if (sizeof(ElemType)==sizeof(float))" triggers this

#ifdef _WIN32
// thread local storage to access the current stream, initalize to default stream
extern __declspec(thread)
#else
static
#endif
    hipStream_t t_stream;

template <>
const char* CudaErrString<hipsparseStatus_t>(hipsparseStatus_t)
{
    hipDeviceSynchronize();
    return "(see hipsparse.h & look for hipsparseStatus_t or CUSPARSE_STATUS_xxx)";
}

namespace Microsoft { namespace MSR { namespace CNTK {

#pragma region Constructors and Destructor

template <class ElemType>
GPUSPARSE_INDEX_TYPE GPUSparseMatrix<ElemType>::SecondaryIndexValueAt(size_t idx) const
{
    if (idx + m_sliceViewOffset == 0) return 0;
    GPUSPARSE_INDEX_TYPE value;
    CUDA_CALL(hipMemcpy(&value, SecondaryIndexLocation() + idx, sizeof(GPUSPARSE_INDEX_TYPE), hipMemcpyDeviceToHost));

    return value;
}

//-------------------------------------------------------------------------
// construction and conversion
//-------------------------------------------------------------------------

template <class ElemType>
void GPUSparseMatrix<ElemType>::ZeroInit(const MatrixFormat matrixFormat, const DEVICEID_TYPE computeDevice)
{
    if (matrixFormat != MatrixFormat::matrixFormatSparseCSC && matrixFormat != MatrixFormat::matrixFormatSparseCSR &&
        matrixFormat != MatrixFormat::matrixFormatSparseBlockCol && matrixFormat != MatrixFormat::matrixFormatSparseBlockRow)
    {
        LogicError("GPUSparseMatrix:  unsupported sparse matrix format");
        // BUGBUG: Then why even define others?
    }
    Base::ZeroInit(matrixFormat, computeDevice);
}

template <class ElemType>
GPUSparseMatrix<ElemType>::GPUSparseMatrix(const size_t numRows, const size_t numCols, const size_t numNZ, DEVICEID_TYPE computeDevice, const MatrixFormat matrixFormat /*= MatrixFormat::matrixFormatSparseCSR*/)
{
    ZeroInit(matrixFormat, computeDevice);
    RequireSizeAndAllocate(numRows, numCols, numNZ, true, false);
}

template <class ElemType>
GPUSparseMatrix<ElemType>::GPUSparseMatrix(DEVICEID_TYPE computeDevice, const MatrixFormat matrixFormat /*= MatrixFormat::matrixFormatSparseCSR*/)
{
    ZeroInit(matrixFormat, computeDevice);
}

template <class ElemType>
GPUSparseMatrix<ElemType>::GPUSparseMatrix(const GPUMatrix<ElemType>& deepCopy, const MatrixFormat matrixFormat /*= MatrixFormat::matrixFormatSparseCSR*/)
{
    ZeroInit(matrixFormat, deepCopy.GetComputeDeviceId());
    if (!deepCopy.IsEmpty())
        SetValue(deepCopy, matrixFormat);
}

template <class ElemType>
GPUSparseMatrix<ElemType>::GPUSparseMatrix(const GPUSparseMatrix<ElemType>& deepCopy)
{
    ZeroInit(deepCopy.GetFormat(), deepCopy.GetComputeDeviceId());
    DeepCopy(deepCopy);
}

// PrepareDevice - Setup the correct cuda context for an operation
// deviceId - the device on which the operation will take place
//            defaults to -1, which means use matrices current device
template <class ElemType>
DEVICEID_TYPE GPUSparseMatrix<ElemType>::PrepareDevice(DEVICEID_TYPE deviceId /*=-1*/) const
{
    // if default value use current compute device
    DEVICEID_TYPE newId = deviceId >= 0 ? deviceId : GetComputeDeviceId();

    Microsoft::MSR::CNTK::PrepareDevice(newId);
    return newId;
}

template <class ElemType>
/*private*/ void GPUSparseMatrix<ElemType>::DeepCopy(const GPUSparseMatrix<ElemType>& deepCopy)
{
    ChangeDeviceTo(deepCopy.GetComputeDeviceId());
    deepCopy.PrepareDevice();

    RequireSizeAndAllocate(deepCopy.GetNumRows(), deepCopy.GetNumCols(), deepCopy.GetNumNZElements(), deepCopy.GetFormat(), true, false);
    m_sliceViewOffset = 0; // reset to zero as we only start copying the indices starting from the offset in the source matrix

    CUDA_CALL(hipMemcpy(Data(), deepCopy.NzValues(), deepCopy.NzSize(), hipMemcpyDeviceToDevice));
    CUDA_CALL(hipMemcpy(MajorIndexLocation(), deepCopy.MajorIndexLocationWithSliceViewOffset(), deepCopy.MajorIndexSize(), hipMemcpyDeviceToDevice));
    CUDA_CALL(hipMemcpy(SecondaryIndexLocation(), deepCopy.SecondaryIndexLocation(), deepCopy.SecondaryIndexSize(), hipMemcpyDeviceToDevice));

    if (deepCopy.m_sliceViewOffset > 0)
    {
        int blocksPerGrid = (int) ceil(1.0 * SecondaryIndexCount() / GridDim::maxThreadsPerBlock);
        SyncGuard syncGuard;
        _shiftColCSCIndexFromSliceViewToAbsolute<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(
            SecondaryIndexLocation(),
            SecondaryIndexCount(),
            GetNumNZElements());
    }

    // TODO: to copy other variables used only for class based LM
}

template <class ElemType>
void GPUSparseMatrix<ElemType>::SetValue(const GPUSparseMatrix<ElemType>& deepCopy)
{
    VerifyWritable(__FUNCTION__);

    DeepCopy(deepCopy);
}

// from CPU
template <class ElemType>
void GPUSparseMatrix<ElemType>::SetValue(const CPUSparseMatrix<ElemType>& deepCopy)
{
    VerifyWritable(__FUNCTION__);

    SetFormat(deepCopy.GetFormat());
    if (deepCopy.IsEmpty())
    {
        Reset();
        return;
    }

    if (deepCopy.GetFormat() == matrixFormatSparseCSR)
    {
        SetMatrixFromCSRFormat(deepCopy.RowLocation(), deepCopy.ColLocation(), deepCopy.Data(), deepCopy.GetNumElemAllocated(), deepCopy.GetNumRows(), deepCopy.GetNumCols());
    }
    else if (deepCopy.GetFormat() == matrixFormatSparseCSC)
    {
        SetMatrixFromCSCFormat(deepCopy.ColLocation(), deepCopy.RowLocation(), deepCopy.Data(), deepCopy.GetNumElemAllocated(), deepCopy.GetNumRows(), deepCopy.GetNumCols());
    }
    else
        NOT_IMPLEMENTED;
}

template <class ElemType>
void GPUSparseMatrix<ElemType>::CopyToCPUSparseMatrix(CPUSparseMatrix<ElemType>& cpuSparseMatrix) const
{
    cpuSparseMatrix.VerifyWritable(__FUNCTION__);

    cpuSparseMatrix.SetFormat(GetFormat());
    if (IsEmpty())
    {
        cpuSparseMatrix.Reset();
        return;
    }

    if (this->GetFormat() == matrixFormatSparseCSR)
    {
        // we need to do conversion because CPUSparseMatrix uses size_t for indexes while GPUSparseMatrix uses int
        cpuSparseMatrix.RequireSizeAndAllocate(GetNumRows(), GetNumCols(), GetNumElemAllocated(), true, false);

        PrepareDevice();

        if (sizeof(GPUSPARSE_INDEX_TYPE) == sizeof(CPUSPARSE_INDEX_TYPE))
        {
            CUDA_CALL(hipMemcpy(cpuSparseMatrix.RowLocation(), RowLocation(), RowSize(), hipMemcpyDeviceToHost));
            CUDA_CALL(hipMemcpy(cpuSparseMatrix.ColLocation(), ColLocation(), ColSize(), hipMemcpyDeviceToHost));
        }
        else
        {
            GPUSPARSE_INDEX_TYPE* h_CSRRow = (GPUSPARSE_INDEX_TYPE*) ReserveTempHostBuffer(RowSize());
            CUDA_CALL(hipMemcpy(h_CSRRow, RowLocation(), RowSize(), hipMemcpyDeviceToHost));
            ConvertBuffer(cpuSparseMatrix.RowLocation(), h_CSRRow, SecondaryIndexCount());

            GPUSPARSE_INDEX_TYPE* h_Col = (GPUSPARSE_INDEX_TYPE*) ReserveTempHostBuffer(ColSize());
            CUDA_CALL(hipMemcpy(h_Col, ColLocation(), ColSize(), hipMemcpyDeviceToHost));
            ConvertBuffer(cpuSparseMatrix.ColLocation(), h_Col, MajorIndexCount());
        }

        CUDA_CALL(hipMemcpy(cpuSparseMatrix.Data(), Data(), GetSizeElemAllocated(), hipMemcpyDeviceToHost));
    }
    else if (this->GetFormat() == matrixFormatSparseCSC)
    {
        // we need to do conversion because CPUSparseMatrix uses size_t for indexes while GPUSparseMatrix uses int
        cpuSparseMatrix.RequireSizeAndAllocate(GetNumRows(), GetNumCols(), GetNumElemAllocated(), true, false);

        PrepareDevice();
        if (sizeof(GPUSPARSE_INDEX_TYPE) == sizeof(CPUSPARSE_INDEX_TYPE))
        {
            CUDA_CALL(hipMemcpy(cpuSparseMatrix.RowLocation(), RowLocation(), RowSize(), hipMemcpyDeviceToHost));
            CUDA_CALL(hipMemcpy(cpuSparseMatrix.ColLocation(), ColLocation(), ColSize(), hipMemcpyDeviceToHost));
        }
        else
        {
            GPUSPARSE_INDEX_TYPE* h_CSCCol = (GPUSPARSE_INDEX_TYPE*) ReserveTempHostBuffer(ColSize());
            CUDA_CALL(hipMemcpy(h_CSCCol, ColLocation(), ColSize(), hipMemcpyDeviceToHost));
            ConvertBuffer(cpuSparseMatrix.ColLocation(), h_CSCCol, SecondaryIndexCount());

            GPUSPARSE_INDEX_TYPE* h_Row = (GPUSPARSE_INDEX_TYPE*) ReserveTempHostBuffer(RowSize());
            CUDA_CALL(hipMemcpy(h_Row, RowLocation(), RowSize(), hipMemcpyDeviceToHost));
            ConvertBuffer(cpuSparseMatrix.RowLocation(), h_Row, MajorIndexCount());
        }

        CUDA_CALL(hipMemcpy(cpuSparseMatrix.Data(), Data(), GetSizeElemAllocated(), hipMemcpyDeviceToHost));
    }
    else if (this->GetFormat() == matrixFormatSparseBlockCol)
    {
        cpuSparseMatrix.RequireSizeAndAllocate(GetNumRows(), GetNumCols(), GetNumNZElements(), true, false);

        PrepareDevice();
        std::vector<GPUSPARSE_INDEX_TYPE> temp(GetBlockSize());
        CUDA_CALL(hipMemcpy(temp.data(), BlockId2ColOrRow(), GetBlockSize() * sizeof(GPUSPARSE_INDEX_TYPE), hipMemcpyDeviceToHost));
        for (size_t i = 0; i < temp.size(); ++i)
            cpuSparseMatrix.BlockIdsLocation()[i] = temp[i];

        cpuSparseMatrix.SetBlockSize(GetBlockSize());

        CUDA_CALL(hipMemcpy(cpuSparseMatrix.Data(), Data(), NzSize(), hipMemcpyDeviceToHost));
    }
    else
        NOT_IMPLEMENTED;
}

template <class ElemType>
void GPUSparseMatrix<ElemType>::CopyToDenseMatrix(GPUMatrix<ElemType>& denseMatrix) const
{
    if (IsEmpty())
    {
        denseMatrix.RequireSize(0, 0);
        return;
    }

    PrepareDevice();
    hipsparseHandle_t cusparseHandle = 0;
    CUSPARSE_CALL(hipsparseCreate(&cusparseHandle));
    hipsparseMatDescr_t descr = 0;
    CUSPARSE_CALL(hipsparseCreateMatDescr(&descr));
    hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

    denseMatrix.RequireSize(GetNumRows(), GetNumCols());

    SyncGuard syncGuard;
    CUSPARSE_CALL(hipsparseSetStream(cusparseHandle, t_stream));
    if (GetFormat() == MatrixFormat::matrixFormatSparseCSR)
    {
        if (sizeof(ElemType) == sizeof(float))
        {
            CUSPARSE_CALL(hipsparseScsr2dense(cusparseHandle, int(GetNumRows()), int(GetNumCols()), descr, (float*) Buffer(), RowLocation(), ColLocation(), (float*) denseMatrix.Data(), int(GetNumRows())));
        }
        else
        {
            CUSPARSE_CALL(hipsparseDcsr2dense(cusparseHandle, int(GetNumRows()), int(GetNumCols()), descr, (double*) Buffer(), RowLocation(), ColLocation(), (double*) denseMatrix.Data(), int(GetNumRows())));
        }
    }
    else if (GetFormat() == MatrixFormat::matrixFormatSparseCSC)
    {
        if (sizeof(ElemType) == sizeof(float))
        {
            CUSPARSE_CALL(hipsparseScsc2dense(cusparseHandle, int(GetNumRows()), int(GetNumCols()), descr, (float*) Buffer(), RowLocation(), ColLocation(), (float*) denseMatrix.Data(), int(GetNumRows())));
        }
        else
        {
            CUSPARSE_CALL(hipsparseDcsc2dense(cusparseHandle, int(GetNumRows()), int(GetNumCols()), descr, (double*) Buffer(), RowLocation(), ColLocation(), (double*) denseMatrix.Data(), int(GetNumRows())));
        }
    }
    else if (GetFormat() == MatrixFormat::matrixFormatSparseBlockCol || GetFormat() == MatrixFormat::matrixFormatSparseBlockRow)
    {
        denseMatrix.SetValue((ElemType)0);
        ScaleAndAdd(1, *this, denseMatrix);
    }
    else
    {
        NOT_IMPLEMENTED;
    }
    CUSPARSE_CALL(hipsparseDestroy(cusparseHandle));

}

template <class ElemType>
void GPUSparseMatrix<ElemType>::ConvertToSparseFormat(MatrixFormat newFormat, GPUSparseMatrix<ElemType>& outMatrix) const
{
    outMatrix.VerifyWritable(__FUNCTION__);

    if (IsEmpty())
    {
        outMatrix.ZeroInit(newFormat, GetComputeDeviceId());
        return;
    }

    MatrixFormat oldFormat = GetFormat();
    if (oldFormat == newFormat)
    {
        outMatrix.SetValue(*this);
        return;
    }

    PrepareDevice();
    hipsparseHandle_t cusparseHandle = 0;
    CUSPARSE_CALL(hipsparseCreate(&cusparseHandle));

    SyncGuard syncGuard;
    CUSPARSE_CALL(hipsparseSetStream(cusparseHandle, t_stream));

    outMatrix.ChangeDeviceTo(GetComputeDeviceId());
    outMatrix.RequireSizeAndAllocate(GetNumRows(), GetNumCols(), NzCount(), newFormat, true, false);

    if ((oldFormat == matrixFormatSparseCSR && newFormat == matrixFormatSparseCSC) || (oldFormat == matrixFormatSparseCSC && newFormat == matrixFormatSparseCSR))
    {
        if (sizeof(ElemType) == sizeof(float))
        {
            CUSPARSE_CALL(hipsparseScsr2csc(cusparseHandle, int(GetNumRows()), int(GetNumCols()), int(GetSizeAllocated()),
                                           (float*) Data(), RowLocation(), ColLocation(), (float*) outMatrix.Data(),
                                           outMatrix.RowLocation(), outMatrix.ColLocation(), HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO));
        }
        else
        {
            CUSPARSE_CALL(hipsparseDcsr2csc(cusparseHandle, int(GetNumRows()), int(GetNumCols()), int(GetSizeAllocated()),
                                           (double*) Data(), RowLocation(), ColLocation(), (double*) outMatrix.Data(),
                                           outMatrix.RowLocation(), outMatrix.ColLocation(), HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO));
        }
    }
    else
    {
        NOT_IMPLEMENTED;
    }

    CUSPARSE_CALL(hipsparseDestroy(cusparseHandle));
}

template <class ElemType>
void GPUSparseMatrix<ElemType>::ConvertToSparseFormat(MatrixFormat newFormat)
{
    if (IsEmpty())
    {
        SetFormat(newFormat);
        return;
    }

    MatrixFormat oldFormat = GetFormat();
    if (oldFormat == newFormat)
        return;

    GPUSparseMatrix<ElemType> tempMatrix(GetComputeDeviceId(), newFormat);
    ConvertToSparseFormat(newFormat, tempMatrix);

    *this = std::move(tempMatrix);
}

template <class ElemType>
GPUMatrix<ElemType> GPUSparseMatrix<ElemType>::CopyToDenseMatrix() const
{
    GPUMatrix<ElemType> res(GetComputeDeviceId());
    if (!IsEmpty())
        CopyToDenseMatrix(res);
    return res;
}

template <class ElemType>
void GPUSparseMatrix<ElemType>::ChangeDeviceTo(DEVICEID_TYPE to_id)
{
    VerifyWritable(__FUNCTION__);
    if (to_id == CPUDEVICE)
        LogicError("to_id must be valid GPU");
    if (GetComputeDeviceId()== to_id)
        return;

    if (BufferSizeAllocated() == 0) // nothing to move
    {
        assert(Buffer() == nullptr);
    }
    else
    {
        ElemType* d_dst = reinterpret_cast<ElemType*>(TracingGPUMemoryAllocator::Allocate<char>(to_id, BufferSizeAllocated()));

        // first try peer access
        int canAccessPeer = false;
        CUDA_CALL(hipDeviceCanAccessPeer(&canAccessPeer, to_id, GetComputeDeviceId()));
        if (canAccessPeer)
        {
            hipError_t cudaStatus = hipDeviceEnablePeerAccess(GetComputeDeviceId(), 0);
            if (cudaStatus != hipErrorPeerAccessAlreadyEnabled)
            {
                CUDA_CALL(cudaStatus);
            }
            CUDA_CALL(hipMemcpyPeer(d_dst, to_id, Buffer(), GetComputeDeviceId(), BufferSizeAllocated()));
        }
        else
        {
            // peer access didn't work, just copy normal
            // make this more efficient by keeping some buffers available for each copy
            ElemType* h_dst = NULL;
            PrepareDevice();
            CUDA_CALL(hipHostMalloc((void**) &h_dst, BufferSizeAllocated()));
            CUDA_CALL(hipMemcpy(h_dst, Buffer(), BufferSizeAllocated(), hipMemcpyDeviceToHost));
            PrepareDevice((DEVICEID_TYPE) to_id);
            CUDA_CALL(hipMemcpy(d_dst, h_dst, BufferSizeAllocated(), hipMemcpyHostToDevice));
            CUDA_CALL(hipHostFree(h_dst));
        }

        TracingGPUMemoryAllocator::Free<ElemType>(GetComputeDeviceId(), Buffer());
        SetBuffer(d_dst, BufferSizeAllocated());
    }

    SetComputeDeviceId(PrepareDevice(to_id));
}

#if 0
template <class ElemType>
void GPUSparseMatrix<ElemType>::SetValue(const CPUMatrix<ElemType>& /*denseMatrix*/)
{
    NOT_IMPLEMENTED;
}
#endif

template <class ElemType>
void GPUSparseMatrix<ElemType>::SetValue(const GPUMatrix<ElemType>& denseMatrix)
{
    VerifyWritable(__FUNCTION__);

    SetValue(denseMatrix, GetFormat());
}

template <class ElemType>
void GPUSparseMatrix<ElemType>::SetValue(const GPUMatrix<ElemType>& denseMatrix, const MatrixFormat matrixFormat)
{
    VerifyWritable(__FUNCTION__);

    if (matrixFormat != matrixFormatSparseCSR && matrixFormat != matrixFormatSparseCSC)
    {
        NOT_IMPLEMENTED;
    }

    PrepareDevice();
    hipsparseHandle_t cusparseHandle = 0;
    CUSPARSE_CALL(hipsparseCreate(&cusparseHandle));
    hipsparseMatDescr_t descr = 0;
    CUSPARSE_CALL(hipsparseCreateMatDescr(&descr));
    hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

    int numRows = (int) denseMatrix.GetNumRows(); // m
    int numCols = (int) denseMatrix.GetNumCols(); // n

    int* nnzPerRowOrCol = TracingGPUMemoryAllocator::Allocate<GPUSPARSE_INDEX_TYPE>(GetComputeDeviceId(), ((matrixFormat & matrixFormatRowMajor) ? numRows : numCols));
    int nnzTotalDevHostPtr = -1;

    {
        SyncGuard syncGuard;
        if (sizeof(ElemType) == sizeof(float))
        {
            CUSPARSE_CALL(hipsparseSnnz(cusparseHandle, (matrixFormat & matrixFormatRowMajor) ? HIPSPARSE_DIRECTION_ROW : HIPSPARSE_DIRECTION_COLUMN, (int) numRows, (int) numCols, descr,
                                       reinterpret_cast<float*>(denseMatrix.Data()), (int) numRows, nnzPerRowOrCol, &nnzTotalDevHostPtr));
        }
        else
        {
            CUSPARSE_CALL(hipsparseDnnz(cusparseHandle, (matrixFormat & matrixFormatRowMajor) ? HIPSPARSE_DIRECTION_ROW : HIPSPARSE_DIRECTION_COLUMN, (int) numRows, (int) numCols, descr,
                                       reinterpret_cast<double*>(denseMatrix.Data()), (int) numRows, nnzPerRowOrCol, &nnzTotalDevHostPtr));
        }
        // ~SyncGuard
    }

    RequireSizeAndAllocate(numRows, numCols, nnzTotalDevHostPtr, matrixFormat, true, false);

    SyncGuard syncGuard;
    if (GetFormat() == MatrixFormat::matrixFormatSparseCSR)
    {
        if (sizeof(ElemType) == sizeof(float))
        {
            CUSPARSE_CALL(hipsparseSdense2csr(cusparseHandle, (int) GetNumRows(), (int) GetNumCols(), descr, reinterpret_cast<float*>(denseMatrix.Data()),
                                             (int) GetNumRows(), nnzPerRowOrCol, reinterpret_cast<float*>(Data()), RowLocation(), ColLocation()));
        }
        else
        {
            CUSPARSE_CALL(hipsparseDdense2csr(cusparseHandle, (int) GetNumRows(), (int) GetNumCols(), descr, reinterpret_cast<double*>(denseMatrix.Data()),
                                             (int) GetNumRows(), nnzPerRowOrCol, reinterpret_cast<double*>(Data()), RowLocation(), ColLocation()));
        }
    }
    else if (GetFormat() == MatrixFormat::matrixFormatSparseCSC)
    {
        if (sizeof(ElemType) == sizeof(float))
        {
            CUSPARSE_CALL(hipsparseSdense2csc(cusparseHandle, (int) GetNumRows(), (int) GetNumCols(), descr, reinterpret_cast<float*>(denseMatrix.Data()),
                                             (int) GetNumRows(), nnzPerRowOrCol, reinterpret_cast<float*>(Data()), RowLocation(), ColLocation()));
        }
        else
        {
            CUSPARSE_CALL(hipsparseDdense2csc(cusparseHandle, (int) GetNumRows(), (int) GetNumCols(), descr, reinterpret_cast<double*>(denseMatrix.Data()),
                                             (int) GetNumRows(), nnzPerRowOrCol, reinterpret_cast<double*>(Data()), RowLocation(), ColLocation()));
        }
    }
}

template <class ElemType>
GPUSPARSE_INDEX_TYPE* GPUSparseMatrix<ElemType>::GetCondensedVector() const
{
    if (GetFormat() == MatrixFormat::matrixFormatSparseCSC || GetFormat() == MatrixFormat::matrixFormatSparseCSR)
    {
        PrepareDevice();
        GPUSPARSE_INDEX_TYPE* pArray = new GPUSPARSE_INDEX_TYPE[SecondaryIndexCount()];
        CUDA_CALL(hipMemcpy(pArray, SecondaryIndexLocation(), sizeof(GPUSPARSE_INDEX_TYPE) * SecondaryIndexCount(), hipMemcpyDeviceToHost));
        return pArray;
    }
    else
    {
        return NULL;
    }
}

template <class ElemType>
void GPUSparseMatrix<ElemType>::MaskColumnsValue(const GPUMatrix<char>& columnsMask, ElemType val)
{
    VerifyWritable(__FUNCTION__);

    size_t n = GetNumCols();
    if (n != columnsMask.GetNumCols())
        RuntimeError("Matrix and column mask must have equal number of columns");

    if (val != 0)
        LogicError("MaskColumnsValue is not implmented for a non-zero mask for sparse matrices.");

#ifdef _DEBUG
    if (GetFormat() == MatrixFormat::matrixFormatSparseCSC)
    {
        // TODO: We could do this on the GPU, but for now C++ is easier.
        // Download the binary columns mask
        char* maskedCols = columnsMask.CopyToArray();

        // If we're CSC, we only need to verify that the columns to be zeroed are empty, since val == 0.
        // So just download the condensed column vector.
        GPUSPARSE_INDEX_TYPE* colVector = GetCondensedVector();

        // Verify that if the column is to be masked, there are no elements in it.
        #pragma omp parallel for
        for (long j = 0; j < n; j++)
            if (maskedCols[j] == 0 && colVector[j + 1] != colVector[j])
                RuntimeError("GPUSparseMatrix attempted to mask column %d, but it has %d elements in it.", (int)j, (int)(colVector[j + 1] - colVector[j]));
    }
    else
        NOT_IMPLEMENTED;
#endif
}


template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::operator=(const GPUSparseMatrix<ElemType>& deepCopy)
{
    if (this != &deepCopy)
        SetValue(deepCopy);

    return *this;
}

template <class ElemType>
GPUSparseMatrix<ElemType>::GPUSparseMatrix(GPUSparseMatrix<ElemType>&& moveFrom)
{
    Base::ShallowCopyFrom(moveFrom);
    moveFrom.ZeroValues(); // so that memory in moveFrom is not freed
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::operator=(GPUSparseMatrix<ElemType>&& moveFrom)
{
    if (this != &moveFrom)
    {
        Base::ShallowCopyFrom(moveFrom);
        moveFrom.ZeroValues();
    }

    return *this;
}

template <class ElemType>
GPUSparseMatrix<ElemType>::~GPUSparseMatrix()
{
    ZeroValues();
}

//ResizeAsAndCopyIndexFrom - Resize this sparse matrix to have the same element structure as the passed matrix
// a - sparse matrix whose structure we want to clone
// remark: this was done for element wise operations where the structure will be identical after an operation
template <class ElemType>
void GPUSparseMatrix<ElemType>::ResizeAsAndCopyIndexFrom(const GPUSparseMatrix<ElemType>& a, const bool growOnly /*= true*/)
{
    RequireSizeAndAllocate(a.GetNumRows(), a.GetNumCols(), a.NzCount(), a.GetFormat(), growOnly, false);

    CUDA_CALL(hipMemcpy(MajorIndexLocation(), a.MajorIndexLocation(), MajorIndexSize(), hipMemcpyDeviceToDevice));
    CUDA_CALL(hipMemcpy(SecondaryIndexLocation(), a.SecondaryIndexLocation(), SecondaryIndexSize(), hipMemcpyDeviceToDevice));
}

//-------------------------------------------------------------------------
// main operations
//-------------------------------------------------------------------------

template <class ElemType>
void GPUSparseMatrix<ElemType>::Reshape(const size_t numRows, const size_t numCols)
{
    if (GetNumRows() == numRows && GetNumCols() == numCols)
        return;

    VerifyWritable(__FUNCTION__);

    if (GetFormat() != MatrixFormat::matrixFormatSparseCSC)
        NOT_IMPLEMENTED;

    if (GetNumRows() * GetNumCols() != numRows * numCols)
        LogicError("GPUSparseMatrix::Reshape: new matrix size does not match current size, can't be reshaped. Did you mean to resize?");

    size_t bufferSizeNeeded = BufferSizeNeeded(numRows, numCols, GetSizeAllocated(), GetFormat());

    ElemType* pArray = reinterpret_cast<ElemType*>(TracingGPUMemoryAllocator::Allocate<char>(GetComputeDeviceId(), bufferSizeNeeded));

    if (Buffer() != nullptr)
    {
        CUDA_CALL(hipMemcpy(pArray, Data(), GetSizeElemAllocated(), hipMemcpyDeviceToDevice));

        GPUSPARSE_INDEX_TYPE* majorIndexInNewBuffer = (GPUSPARSE_INDEX_TYPE*) (pArray + GetSizeAllocated());
        GPUSPARSE_INDEX_TYPE* secondaryIndexInNewBuffer = majorIndexInNewBuffer + MajorIndexCount(numRows, numCols, GetSizeAllocated(), GetFormat());

        int blocksPerGrid = (int) ceil(1.0 * numCols / GridDim::maxThreadsPerBlock);
        SyncGuard syncGuard;
        _reshape<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(
            GetNumRows(),                // old row count
            GetNumCols(),                // old col count
            numRows,                  // new row count
            numCols,                  // new col count
            MajorIndexLocation(),     // old row index array
            SecondaryIndexLocation(), // old column index array
            majorIndexInNewBuffer,    // new row index array
            secondaryIndexInNewBuffer // new column index array
            );
        TracingGPUMemoryAllocator::Free<ElemType>(GetComputeDeviceId(), Buffer());
    }

    SetBuffer(pArray, bufferSizeNeeded);
    SetNumRows(numRows);
    SetNumCols(numCols);
}

// WARNING: When memory is reallocated, existing information will be lost.
// TODO: add keepExistingValues (default to true) argument so that the existing values are kept even after reallocation
template <class ElemType>
void GPUSparseMatrix<ElemType>::Allocate(const size_t numRows, const size_t numCols, const size_t numNZElemToReserve, const bool growOnly /*= true*/, bool keepExistingValues /*= true*/)
{
    // BugBug: This doesn't work because allocate is called from Resize sometimes and resize expects allocate to know the old values not the new values, so this won't work.
    if (GetNumRows() != numRows || GetNumCols() != numCols)
        LogicError("Error, calling allocate with dimensions (%d, %d), but the matrix has dimension (%d, %d).", (int)numRows, (int)numCols, (int)GetNumRows(), (int)GetNumCols());

    size_t bufferSizeNeeded = BufferSizeNeeded(numRows, numCols, numNZElemToReserve, GetFormat());
    bool reallocate = (BufferSizeAllocated() < bufferSizeNeeded || (!growOnly && BufferSizeAllocated() > bufferSizeNeeded));

    if (reallocate)
    {
        // Note that we are allocating one buffer for all of our data structures. Thus the ElemType* nzValues array lives directly next to
        // the GPUSPARSE_INDEX_TYPE* rowIndices/colIndices in sparseCSC/CSR formats. Thus we allocate the number of bytes, and then set the
        // start pointer to an ElemType*.
        char* buf = TracingGPUMemoryAllocator::Allocate<char>(GetComputeDeviceId(), bufferSizeNeeded);
        ElemType* pArray = (ElemType*)(buf);

        // Note this is required due to m_nz 
        CUDA_CALL(hipMemset(pArray, 0, bufferSizeNeeded));
        if (Buffer() != nullptr)
        {
            if (keepExistingValues)
            {
                if (NzCount() > numNZElemToReserve || BufferSizeAllocated() > bufferSizeNeeded)
                    LogicError("Resize: To keep values m_nz should <= numNZElemToReserve.");

                CUDA_CALL(hipMemcpy(pArray, Data(), GetSizeElemAllocated(), hipMemcpyDeviceToDevice));

                GPUSPARSE_INDEX_TYPE* majorIndexInNewBuffer = (GPUSPARSE_INDEX_TYPE*) (pArray + numNZElemToReserve);

                CUDA_CALL(hipMemcpy(majorIndexInNewBuffer, MajorIndexLocation(), MajorIndexSize(), hipMemcpyDeviceToDevice));

                GPUSPARSE_INDEX_TYPE* secondaryIndexInNewBuffer = majorIndexInNewBuffer + MajorIndexCount(numRows, numCols, numNZElemToReserve, GetFormat());
                CUDA_CALL(hipMemcpy(secondaryIndexInNewBuffer, SecondaryIndexLocation(), SecondaryIndexSize(), hipMemcpyDeviceToDevice));
            }
            TracingGPUMemoryAllocator::Free<ElemType>(GetComputeDeviceId(), Buffer());
        }

        SetBuffer(pArray, bufferSizeNeeded);
        SetSizeAllocated(numNZElemToReserve);
    }
    else // if requested size is smaller, keeping original values does not make sense
    {
        SetSizeAllocated(ElemCountFromBufferSize(numRows, numCols, GetFormat(), BufferSizeAllocated()));
        CUDA_CALL(hipMemset(Buffer(), 0, BufferSizeAllocated()));
    }
}

template <class ElemType>
void GPUSparseMatrix<ElemType>::RequireSizeAndAllocate(const size_t numRows, const size_t numCols, const size_t numNZElemToReserve /*= 10000*/, const bool growOnly /*= true*/, bool keepExistingValues /*= false*/)
{
    RequireSizeAndAllocate(numRows, numCols, numNZElemToReserve, GetFormat(), growOnly, keepExistingValues);
}

template <class ElemType>
void GPUSparseMatrix<ElemType>::RequireSizeAndAllocate(const size_t numRows, const size_t numCols, const size_t numNZElemToReserve, const MatrixFormat matrixFormat, const bool growOnly /*= true*/, bool keepExistingValues /*= true*/)
{
    RequireSize(numRows, numCols, matrixFormat, growOnly);
    
    size_t bufferSizeNeeded = BufferSizeNeeded(numRows, numCols, numNZElemToReserve, matrixFormat);
    bool reallocate = (BufferSizeAllocated() < bufferSizeNeeded || (!growOnly && BufferSizeAllocated() > bufferSizeNeeded));

    if (reallocate)
        Allocate(numRows, numCols, numNZElemToReserve, growOnly, keepExistingValues);

}

template <class ElemType>
void GPUSparseMatrix<ElemType>::RequireSize(const size_t numRows, const size_t numCols, const bool growOnly /*= true*/)
{
    RequireSize(numRows, numCols, GetFormat(), growOnly);
}

template <class ElemType>
void GPUSparseMatrix<ElemType>::RequireSize(const size_t numRows, const size_t numCols, const MatrixFormat matrixFormat, const bool growOnly /*= true*/)
{
    if (GetFormat() != matrixFormat || GetNumRows() != numRows || GetNumCols() != numCols)
        Resize(numRows, numCols, 0, matrixFormat, growOnly);
}

template <class ElemType>
void GPUSparseMatrix<ElemType>::Resize(const size_t numRows, const size_t numCols, const size_t numNZElemToReserve /*= 10000*/, const bool growOnly /*= true*/)
{
    Resize(numRows, numCols, numNZElemToReserve, GetFormat(), growOnly);
}

template <class ElemType>
void GPUSparseMatrix<ElemType>::Resize(const size_t numRows, const size_t numCols, const size_t numNZElemToReserve, const MatrixFormat matrixFormat, const bool growOnly /*= true*/)
{
    VerifyResizable(__FUNCTION__);

    m_sliceViewOffset = 0;
    SetNumRows(numRows);
    SetNumCols(numCols);
    SetNumStorageRows(numRows);
    SetNumStorageCols(numCols);
    SetFormat(matrixFormat);

    // If we really did resize the number of rows/columns, then we changed the number of nz elements allocated. That is, if we used to have a buffer capable of
    // stroring 100 nz elements and 10 columns in CSC format, but we resized to 20 columns, we can no longer store 100 elements, we can only store 95. 
    // Thus we must reset the number of nz elements which can be stored. So let's compute it now.
    size_t newNzElem = ComputeMaxNZElemFromBufferSize(numRows, numCols, BufferSizeAllocated(), matrixFormat);
    SetSizeAllocated(newNzElem);

    size_t bufferSizeNeeded = BufferSizeNeeded(numRows, numCols, numNZElemToReserve, matrixFormat);
    bool reallocate = (BufferSizeAllocated() < bufferSizeNeeded || (!growOnly && BufferSizeAllocated() > bufferSizeNeeded));

    if (reallocate)
        Allocate(numRows, numCols, numNZElemToReserve, growOnly, false);
    else
        ClearNzCount();
}

// Reset matrix to 0.
template <class ElemType>
void GPUSparseMatrix<ElemType>::Reset()
{
    VerifyWritable(__FUNCTION__);

    ClearNzCount();
}

template <class ElemType>
void GPUSparseMatrix<ElemType>::ClearNzCount()
{
    // We are now going to reset m_nz to 0. 
    // To reset m_nz to 0, we must do 2 things.
    //    1. We must clear the secondary column index.
    //    2. Set the block size to 0.
    // These requirements can be deduced by the NzCount method.
    CUDA_CALL(hipMemset(Buffer(), 0, BufferSizeAllocated()));
    SetBlockSize(0);
}


// copy features to GPU
template <class ElemType>
void GPUSparseMatrix<ElemType>::SetMatrixFromCSRFormat(const GPUSPARSE_INDEX_TYPE* h_CSRRow, const GPUSPARSE_INDEX_TYPE* h_Col, const ElemType* h_Val,
                                                       const size_t nz, const size_t numRows, const size_t numCols, const bool IsOnDevice /*= false*/, const DEVICEID_TYPE devId /*= -1*/)
{
    VerifyWritable(__FUNCTION__);

    if (h_CSRRow == nullptr || h_Col == nullptr || h_Val == nullptr)
        LogicError("SetMatrixFromCSRFormat: nullptr passed in.");

    SetComputeDeviceId(PrepareDevice(devId));

    SetFormat(matrixFormatSparseCSR);
    RequireSizeAndAllocate(numRows, numCols, nz, true, false);

    hipMemcpyKind kind = IsOnDevice ? hipMemcpyDeviceToDevice : hipMemcpyHostToDevice;
    CUDA_CALL(hipMemcpy(Data(), h_Val, nz * sizeof(ElemType), kind));

    if (sizeof(CPUSPARSE_INDEX_TYPE) == sizeof(GPUSPARSE_INDEX_TYPE))
    {
        // ColSize doesn't work since it requires NzCount() to be usable (RowSize doesn't, since it's the fixed, compressed,
        // dimension. Since NzCount is not available (because the sparse indices which is where the NzCount is copmuted from
        // haven't been copied in yet), we just tell it how many bytes to copy. That is, nz * sizeof(GPUSPARSE_INDEX_TYPE);
        CUDA_CALL(hipMemcpy(RowLocation(), h_CSRRow, RowSize(), kind));
        CUDA_CALL(hipMemcpy(ColLocation(), h_Col, nz * sizeof(GPUSPARSE_INDEX_TYPE), kind));
        assert(nz == NzCount());
    }
    else
    {
        GPUSPARSE_INDEX_TYPE* pCol = (GPUSPARSE_INDEX_TYPE*) ReserveTempHostBuffer(RowSize() + nz);
        ConvertBuffer(pCol, h_Col, MajorIndexCount());

        GPUSPARSE_INDEX_TYPE* pRow = pCol + MajorIndexCount();
        ConvertBuffer(pRow, h_CSRRow, nz);

        CUDA_CALL(hipMemcpy(RowLocation(), pRow, RowSize(), kind));
        CUDA_CALL(hipMemcpy(ColLocation(), pCol, nz * sizeof(GPUSPARSE_INDEX_TYPE), kind));
    }
}

// this function will allocate memory while the caller needs to release it
template <class ElemType>
void GPUSparseMatrix<ElemType>::GetMatrixFromCSRFormat(CPUSPARSE_INDEX_TYPE*& h_CSRRow, CPUSPARSE_INDEX_TYPE*& h_Col, ElemType*& h_Val, size_t& numElemAllocated, size_t& nz, size_t& numRows, size_t& numCols) const
{
    VerifyWritable(__FUNCTION__);

    if (h_CSRRow != nullptr || h_Col != nullptr || h_Val != nullptr)
        LogicError("GetMatrixFromCSRFormat: Passed pointers must be nullptr");

    numElemAllocated = GetNumElemAllocated();
    nz = GetNumNZElements();
    numRows = GetNumRows();
    numCols = GetNumCols();

    if (IsEmpty() || nz == 0)
        return;
    else
    {
        h_Val = new ElemType[numElemAllocated];
        h_CSRRow = new CPUSPARSE_INDEX_TYPE[GetNumRows() + 1];
        h_Col = new CPUSPARSE_INDEX_TYPE[nz];

        PrepareDevice();
        CUDA_CALL(hipMemcpy(h_Val, Data(), GetSizeElemAllocated(), hipMemcpyDeviceToHost));

        if (sizeof(CPUSPARSE_INDEX_TYPE) == sizeof(GPUSPARSE_INDEX_TYPE))
        {
            CUDA_CALL(hipMemcpy(h_CSRRow, RowLocation(), RowSize(), hipMemcpyDeviceToHost));
            CUDA_CALL(hipMemcpy(h_Col, ColLocation(), ColSize(), hipMemcpyDeviceToHost));
        }
        else
        {
            GPUSPARSE_INDEX_TYPE* pCol = (GPUSPARSE_INDEX_TYPE*) ReserveTempHostBuffer(RowSize() + ColSize());
            GPUSPARSE_INDEX_TYPE* pRow = pCol + MajorIndexCount();

            CUDA_CALL(hipMemcpy(pRow, RowLocation(), RowSize(), hipMemcpyDeviceToHost));
            CUDA_CALL(hipMemcpy(pCol, ColLocation(), ColSize(), hipMemcpyDeviceToHost));

            ConvertBuffer(h_Col, pCol, MajorIndexCount());
            ConvertBuffer(h_CSRRow, pRow, SecondaryIndexCount());
        }
    }
}

template <class ElemType>
void GPUSparseMatrix<ElemType>::SetMatrixFromCSCFormat(const CPUSPARSE_INDEX_TYPE* h_CSCCol, const CPUSPARSE_INDEX_TYPE* h_Row, const ElemType* h_Val,
    const size_t nz, const size_t numRows, const size_t numCols, const bool IsOnDevice /*= false*/, const DEVICEID_TYPE devId /*= -1*/, DataTransferer* transferer /*= nullptr*/)
{
    VerifyWritable(__FUNCTION__);

    if (h_CSCCol == nullptr || h_Row == nullptr || h_Val == nullptr)
        LogicError("SetMatrixFromCSCFormat: nullptr passed in.");

    SetComputeDeviceId(PrepareDevice(devId));
    SetFormat(matrixFormatSparseCSC);
    RequireSizeAndAllocate(numRows, numCols, nz, true, false);

    if (transferer && IsOnDevice)
        RuntimeError("Currently it is prohibited to copy data asynchronous from device to device.");

    // m_nz doesn't exist anymore. How are we going to deal with the NzSize, RowSize, and ColSize? Do it ourselves of course.

    hipMemcpyKind kind = IsOnDevice ? hipMemcpyDeviceToDevice : hipMemcpyHostToDevice;
    if (transferer)
    {
        // TODO: All RequireSizeAndAllocate should be async and use a transferer.
        // Currently there are some memset operations that can be still executing on the default stream,
        // Here we have to wait for them to finish.
        transferer->RecordComputeStreamSyncPoint();
        transferer->WaitForSyncPointOnAssignStreamAsync();
        transferer->CopyCPUToGPUAsync(h_Val, nz, sizeof(ElemType), Data());
    }
    else
        CUDA_CALL(hipMemcpy(Data(), h_Val, nz * sizeof(ElemType), kind));

    if (sizeof(CPUSPARSE_INDEX_TYPE) == sizeof(GPUSPARSE_INDEX_TYPE))
    {
        if (transferer)
        {
            transferer->CopyCPUToGPUAsync(h_Row, nz, sizeof(GPUSPARSE_INDEX_TYPE), RowLocation());
            transferer->CopyCPUToGPUAsync(h_CSCCol, numCols + 1, sizeof(GPUSPARSE_INDEX_TYPE), ColLocation());
        }
        else
        {
            CUDA_CALL(hipMemcpy(RowLocation(), h_Row, sizeof(GPUSPARSE_INDEX_TYPE) * nz, kind));
            CUDA_CALL(hipMemcpy(ColLocation(), h_CSCCol, sizeof(GPUSPARSE_INDEX_TYPE) * (numCols + 1), kind));
        }
    }
    else
    {
        size_t allocSize = sizeof(GPUSPARSE_INDEX_TYPE) * nz + sizeof(GPUSPARSE_INDEX_TYPE) * (numCols + 1);
        GPUSPARSE_INDEX_TYPE* pCol = (GPUSPARSE_INDEX_TYPE*) ReserveTempHostBuffer(allocSize);
        GPUSPARSE_INDEX_TYPE* pRow = pCol + nz;

        ConvertBuffer(pCol, h_CSCCol, (numCols+1));
        ConvertBuffer(pRow, h_Row, nz);

        if (transferer)
        {
            transferer->CopyCPUToGPUAsync(pRow, nz, sizeof(GPUSPARSE_INDEX_TYPE), RowLocation());
            transferer->CopyCPUToGPUAsync(pCol, numCols + 1, sizeof(GPUSPARSE_INDEX_TYPE), ColLocation());
        }
        else
        {
            CUDA_CALL(hipMemcpy(RowLocation(), pRow, sizeof(GPUSPARSE_INDEX_TYPE) * nz, kind));
            CUDA_CALL(hipMemcpy(ColLocation(), pCol, sizeof(GPUSPARSE_INDEX_TYPE) * (numCols + 1), kind));
        }
    }
}

#if 0 // add it back with test
template <class ElemType>
void GPUSparseMatrix<ElemType>::SetMatrixFromSBCFormat(const size_t* blockIds, const ElemType* val, const size_t numBlocks, const size_t numRows, const size_t numCols)
{
    VerifyWritable(__FUNCTION__);

    if (blockIds == nullptr || val == nullptr)
        LogicError("SetMatrixFromSBCFormat: nullptr passed in.");

    SetFormat(matrixFormatSparseBlockCol);
    SetBlockSize(numBlocks);

    if (numBlocks == 0) return; // ====>

    size_t nz = numBlocks * numRows;
    RequireSizeAndAllocate(numRows, numCols, nz, true, false);

    static std::vector<GPUSPARSE_INDEX_TYPE> gpuBlockId2Col(numBlocks);
    static std::vector<GPUSPARSE_INDEX_TYPE> gpuCol2BlockId(numCols);

    std::fill(gpuBlockId2Col.begin(), gpuBlockId2Col.end(), Id_NotAssigned);
    std::fill(gpuCol2BlockId.begin(), gpuCol2BlockId.end(), Id_NotAssigned);

    #pragma omp parallel for
    for (int i = 0; i < numBlocks; ++i)
    {
        gpuBlockId2Col[i] = (GPUSPARSE_INDEX_TYPE)blockIds[i];
        gpuCol2BlockId[blockIds[i]] = i;
    }

    CUDA_CALL(hipMemcpy(Data(), val, nz * sizeof(ElemType), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(BlockId2ColOrRow(), &gpuBlockId2Col[0], numBlocks * sizeof(GPUSPARSE_INDEX_TYPE), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(ColOrRow2BlockId(), &gpuCol2BlockId[0], numBlocks * sizeof(GPUSPARSE_INDEX_TYPE), hipMemcpyHostToDevice));
}
#endif

// this function will allocate memory while the caller needs to release it
template <class ElemType>
void GPUSparseMatrix<ElemType>::GetMatrixFromCSCFormat(GPUSPARSE_INDEX_TYPE*& h_CSCCol, GPUSPARSE_INDEX_TYPE*& h_Row, ElemType*& h_Val, size_t& numElemAllocated, size_t& nz, size_t& numRows, size_t& numCols) const
{
    if (h_CSCCol != nullptr || h_Row != nullptr || h_Val != nullptr)
        LogicError("GetMatrixFromCSCFormat: Passed pointers must be nullptr");

    numElemAllocated = GetNumElemAllocated();
    nz = GetNumNZElements();
    numRows = GetNumRows();
    numCols = GetNumCols();

    if (IsEmpty())
        return;
    else
    {
        h_Val = new ElemType[numElemAllocated];
        h_CSCCol = new GPUSPARSE_INDEX_TYPE[GetNumRows() + 1];
        h_Row = new GPUSPARSE_INDEX_TYPE[nz];

        PrepareDevice();
        CUDA_CALL(hipMemcpy(h_Val, Data(), GetSizeElemAllocated(), hipMemcpyDeviceToHost));

        if (sizeof(CPUSPARSE_INDEX_TYPE) == sizeof(GPUSPARSE_INDEX_TYPE))
        {
            CUDA_CALL(hipMemcpy(h_Row, RowLocation(), RowSize(), hipMemcpyDeviceToHost));
            CUDA_CALL(hipMemcpy(h_CSCCol, ColLocation(), ColSize(), hipMemcpyDeviceToHost));
        }
        else
        {
            GPUSPARSE_INDEX_TYPE* pCol = (GPUSPARSE_INDEX_TYPE*) ReserveTempHostBuffer(RowSize() + ColSize());
            GPUSPARSE_INDEX_TYPE* pRow = pCol + SecondaryIndexCount();

            CUDA_CALL(hipMemcpy(pRow, RowLocation(), RowSize(), hipMemcpyDeviceToHost));
            CUDA_CALL(hipMemcpy(pCol, ColLocation(), ColSize(), hipMemcpyDeviceToHost));

            ConvertBuffer(h_CSCCol, pCol, SecondaryIndexCount());
            ConvertBuffer(h_Row, pRow, MajorIndexCount());
        }
    }
}

#pragma endregion Constructors and Destructor

#pragma region Static BLAS Functions

// dense X sparse = dense
template <class ElemType>
void GPUSparseMatrix<ElemType>::MultiplyAndWeightedAdd(ElemType alpha, const GPUMatrix<ElemType>& lhs, const bool transposeA,
                                                       const GPUSparseMatrix<ElemType>& rhs, const bool transposeB, ElemType beta, GPUMatrix<ElemType>& c)
{
    if (lhs.GetComputeDeviceId() != rhs.GetComputeDeviceId() || (lhs.GetComputeDeviceId() != c.GetComputeDeviceId()))
        RuntimeError("GPUSparseMatrix::MultiplyAndWeightedAdd: All matrices must be on the same GPU");

    // BUGBUG: Below we fail if one of the factors is empty.That is wrong. We should be able to handle empty factors (e.g. worker of a minibatch got 0 samples).
    // Probably one should test further down and exit early, but we need to make sure that c is correct for beta != 0.
    if (lhs.IsEmpty() || rhs.IsEmpty())
        LogicError("GPUSparseMatrix::MultiplyAndWeightedAdd:  one of the input matrix is empty.");

    int m = transposeA ? (int) lhs.GetNumCols() : (int) lhs.GetNumRows();
    int k = transposeA ? (int) lhs.GetNumRows() : (int) lhs.GetNumCols();
    int l = transposeB ? (int) rhs.GetNumCols() : (int) rhs.GetNumRows();
    int n = transposeB ? (int) rhs.GetNumRows() : (int) rhs.GetNumCols();

    assert(m > 0 && k > 0 && l > 0 && n > 0); // converting from size_t to int may cause overflow
    assert(k == l);
    if (k != l)
    {
        InvalidArgument("GPUSparseMatrix::MultiplyAndWeightedAdd: The inner dimensions of a (= %d) and b (= %d) don't match.", k, l);
    }

    if (beta == 0)
        c.RequireSize(m, n);
    else
        c.VerifySize(m, n); // Can't resize if beta != 0

    c.PrepareDevice();
    if (rhs.GetFormat() == MatrixFormat::matrixFormatSparseCSC)
    {
        ConvolveAndWeightedAdd(alpha, lhs, transposeA, rhs, transposeB, beta, c, 1, 1, false, false);
    }
    else if (rhs.GetFormat() == matrixFormatSparseCSR)
    {
        GPUSparseMatrix<ElemType> tempMatrix(rhs.GetComputeDeviceId(), matrixFormatSparseCSC);
        rhs.ConvertToSparseFormat(matrixFormatSparseCSC, tempMatrix);
        MultiplyAndWeightedAdd(alpha, lhs, transposeA, tempMatrix, transposeB, beta, c);
    }
    else
    {
        NOT_IMPLEMENTED;
    }
}

// dense X sparse = dense
template <class ElemType>
void GPUSparseMatrix<ElemType>::ConvolveAndWeightedAdd(ElemType alpha, const GPUMatrix<ElemType>& lhs, const bool transposeA,
                                                       const GPUSparseMatrix<ElemType>& rhs, const bool transposeB, ElemType beta,
                                                       GPUMatrix<ElemType>& c, size_t numChannels, size_t horizontalSubsample, bool padding, bool channelwise)
{
    if (lhs.GetComputeDeviceId() != rhs.GetComputeDeviceId() || (lhs.GetComputeDeviceId() != c.GetComputeDeviceId()))
        RuntimeError("GPUSparseMatrix<ElemType>::ConvolveAndWeightedAdd: All matrices must be on the same GPU");

    if (lhs.IsEmpty() || rhs.IsEmpty())
        LogicError("GPUSparseMatrix<ElemType>::ConvolveAndWeightedAdd:  one of the input matrix is empty.");

    int m = transposeA ? (int) lhs.GetNumCols() : (int) lhs.GetNumRows();
    int k = transposeA ? (int) lhs.GetNumRows() : (int) lhs.GetNumCols();
    int l = transposeB ? (int) rhs.GetNumCols() : (int) rhs.GetNumRows();
    int n = transposeB ? (int) rhs.GetNumRows() : (int) rhs.GetNumCols();

    assert(m > 0 && k > 0 && l > 0 && n > 0); // converting from size_t to int may cause overflow

    int numSteps = 0;
    if (padding)
        numSteps = (int) ceil(1.0 * l / (horizontalSubsample * numChannels));
    else if (l >= k)
        numSteps = 1 + (l - k) / (horizontalSubsample * numChannels);

    if (numSteps == 0)
        LogicError("ConvolveAndWeightedAdd: number of steps is zero. Matrix dimensions are incorrect or set padding to true.");

    int cRows = m * numSteps;
    int cCols = n;

    if (beta == 0)
        c.RequireSize(cRows, cCols);
    else
        c.VerifySize(cRows, cCols); // Can't resize if beta != 0

    c.PrepareDevice();
    if (rhs.GetFormat() == MatrixFormat::matrixFormatSparseCSC)
    {
        if (!transposeB)
        {
            int blocksPerGrid = (int) ceil(1.0 * cRows * cCols / GridDim::maxThreadsPerBlock);
            SyncGuard syncGuard;
            _dense1DConvMultSparseCSCAndWeightedAddToDense<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(
                m,                   // rowDense
                k,                   // colDense
                n,                   // colSparse
                numChannels,         // number of input channels
                numSteps,            // convolution num steps
                horizontalSubsample, // convolution step size
                channelwise,         // channelwise or pixelwise multiplication
                alpha,
                reinterpret_cast<const ElemType*>(lhs.Data()), // dense
                transposeA,
                reinterpret_cast<const ElemType*>(rhs.Buffer()), // sparse nz values. Note that because of the offsets we use the array
                rhs.RowLocation(),
                rhs.ColLocation(),
                beta,
                reinterpret_cast<ElemType*>(c.Data()) // dense target
                );
        }
        else
        {
            if (beta != 1.0)
            {
                RuntimeError("Only support c += alpha * a operation");
            }

            int blocksPerGrid = (int) ceil(1.0 * cRows / GridDim::maxThreadsPerBlock);
            SyncGuard syncGuard;
            for (int rowInB = 0; rowInB < l; rowInB++)
            {
                _dense1DConvMultSparseCSCTransposeAndAddToDense<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(
                    m,                   // rowDense
                    k,                   // colDense
                    n,                   // colSparse
                    numChannels,         // number of input channels
                    numSteps,            // convolution num steps
                    horizontalSubsample, // convolution step size
                    channelwise,         // channelwise or pixelwise multiplication
                    rowInB,
                    alpha,
                    reinterpret_cast<const ElemType*>(lhs.Data()), // dense
                    transposeA,
                    reinterpret_cast<const ElemType*>(rhs.Buffer()), // sparse nz values
                    rhs.RowLocation(),
                    rhs.ColLocation(),
                    reinterpret_cast<ElemType*>(c.Data()) // dense target
                    );
            }
        }
    }
    else
    {
        NOT_IMPLEMENTED;
    }
}

template <class ElemType>
void GPUSparseMatrix<ElemType>::TensorShuffleScaleAndAdd(ElemType keepWeight, const GPUSparseMatrix<ElemType>& a, size_t D, size_t S, size_t M, size_t K, size_t T, 
    ElemType scaleFactor, const GPUSparseMatrix<ElemType>& b, GPUSparseMatrix<ElemType>& c)
{
    c.VerifyWritable(__FUNCTION__);

    if (a.GetComputeDeviceId() != c.GetComputeDeviceId() || b.GetComputeDeviceId() != c.GetComputeDeviceId())
        RuntimeError("GPUSparseMatrix<ElemType>::TensorShuffleScaleAndAdd: All matrices must be on the same GPU");

    if (a.GetFormat() != MatrixFormat::matrixFormatSparseCSC || b.GetFormat() != MatrixFormat::matrixFormatSparseCSC || c.GetFormat() != MatrixFormat::matrixFormatSparseCSC)
        NOT_IMPLEMENTED;

    // Can't distribute the operations if we need to move values across columns
    if (a.GetNumCols() != T || keepWeight != 0 || scaleFactor != 1)
        NOT_IMPLEMENTED;

    if (a.GetNumRows() != D * S * M * K)
        LogicError("GPUSparseMatrix<ElemType>::TensorShuffleScaleAndAdd: tensor dimensions and underlying matrix dimensions don't match");

    c.RequireSizeAndAllocate(a.GetNumRows(), a.GetNumCols(), a.NzCount(), true, false);

    if (a.NzCount() > 0)
    {
        c.PrepareDevice();
        SyncGuard syncGuard;
        CUDA_LONG N = (CUDA_LONG) a.NzCount();
        int blocksPerGrid = (int) ceil(1.0 * N / GridDim::maxThreadsPerBlock);
        _tensorShuffleScaleAndAddRowSparse<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(
            reinterpret_cast<const ElemType*>(a.Buffer()), // source nz values
            a.RowLocation(),
            a.ColLocation(),
            reinterpret_cast<ElemType*>(c.Buffer()), // target nz values
            c.RowLocation(),
            c.ColLocation(),
            D, S, M, K, T,
            a.NzCount());
    }
    else
    {
        CUDA_CALL(hipMemset(c.Buffer(), 0, c.BufferSizeAllocated()));
    }
}

// backward pass from hidden layer to feature weight
// dense X sparse = sparse
template <class ElemType>
void GPUSparseMatrix<ElemType>::MultiplyAndAdd(ElemType alpha, const GPUMatrix<ElemType>& lhs, const bool transposeA,
                                               const GPUSparseMatrix<ElemType>& rhs, const bool transposeB, GPUSparseMatrix<ElemType>& c)
{
    c.VerifyWritable(__FUNCTION__);

    if (lhs.GetComputeDeviceId() != rhs.GetComputeDeviceId())
        RuntimeError("GPUSparseMatrix::MultiplyAndAdd: All matrices must be on the same GPU");

    int m = transposeA ? (int) lhs.GetNumCols() : (int) lhs.GetNumRows();
    int k = transposeA ? (int) lhs.GetNumRows() : (int) lhs.GetNumCols();
    int l = transposeB ? (int) rhs.GetNumCols() : (int) rhs.GetNumRows();
    int n = transposeB ? (int) rhs.GetNumRows() : (int) rhs.GetNumCols();

    assert(m > 0 && k > 0 && l > 0 && n > 0);
    (void) m;
    (void) n; // converting from size_t to int may cause overflow
    assert(k == l);
    if (k != l)
    {
        InvalidArgument("GPUSparseMatrix::MultiplyAndAdd: The inner dimensions of a (= %d) and b (= %d) don't match.", k, l);
    }

    if (!transposeA && !transposeB)
    {
        NOT_IMPLEMENTED;
    }
    else if (!transposeA && transposeB)
    {
        if (rhs.GetFormat() != matrixFormatSparseCSC)
            NOT_IMPLEMENTED;

        c.SetFormat(matrixFormatSparseBlockCol);

        lhs.PrepareDevice();

        int blocksPerGrid = 0;
        SyncGuard syncGuard;

        // based on the size of m_nz in rhs and numCols in the resulted matrix we use different approaches
        size_t rhs_nz = rhs.NzCount();

        size_t blockSizePrev = c.GetBlockSize();
        if (blockSizePrev == 0)
        {
            c.Resize(m, n, 0);
            CUDA_CALL(hipMemset(c.ColOrRow2BlockId(), Id_NotAssigned, sizeof(GPUSPARSE_INDEX_TYPE) * (n)));
            CUDA_CALL(hipMemset(c.BlockId2ColOrRow(), Id_NotAssigned, sizeof(GPUSPARSE_INDEX_TYPE) * (n)));
        }

        size_t* blockSize = TracingGPUMemoryAllocator::Allocate<size_t>(lhs.GetComputeDeviceId(), 1);
        CUDA_CALL(hipMemcpy(blockSize, &blockSizePrev, sizeof(size_t), hipMemcpyHostToDevice));

        blocksPerGrid = (int) ceil(((double) rhs_nz) / GridDim::maxThreadsPerBlock);
        _findColsWithValues<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(
            rhs.RowLocation(), c.ColOrRow2BlockId(), rhs_nz);
                
        blocksPerGrid = (int) ceil(((double) n) / GridDim::maxThreadsPerBlock);
        _determineBlockIds<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(
            c.BlockId2ColOrRow(), c.ColOrRow2BlockId(), n, blockSize);

        size_t blockSizeCurr;
        CUDA_CALL(hipMemcpy(&blockSizeCurr, blockSize, sizeof(size_t), hipMemcpyDeviceToHost));
        TracingGPUMemoryAllocator::Free<size_t>(lhs.GetComputeDeviceId(), blockSize);
        c.SetBlockSize(blockSizeCurr);

        if (blockSizeCurr > blockSizePrev)
        {
            // zero initialize new blocks
            size_t nnz = m * blockSizeCurr;
            c.RequireSizeAndAllocate(m, n, nnz, true, true); // we need to keep the col2blockid and blockid2col info when resizing.
            CUDA_CALL(hipMemset(c.Data() + m * blockSizePrev, 0, sizeof(ElemType) * m * (blockSizeCurr - blockSizePrev)));
        }

        LONG64 N = (LONG64) lhs.GetNumElements(); // here we process for each row in lhs and each column in rhs (==columns in lhs)
        blocksPerGrid = (int) ceil(((double) N) / GridDim::maxThreadsPerBlock);
        _denseMulSparseCSCTransposeToSparseBlockCol2<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(
            alpha,
            lhs.Data(),
            m,
            l,
            rhs.Data(),
            rhs.RowLocation(),
            rhs.ColLocation(),
            c.ColOrRow2BlockId(),
            c.Data());
    }
    else if (transposeA && !transposeB)
    {
        NOT_IMPLEMENTED;
    }
    else
    {
        NOT_IMPLEMENTED;
    }
}

// find the rows of rhs with values
template <class ElemType>
size_t GPUSparseMatrix<ElemType>::IdentifyRowsWithValues() const
{
    if (GetFormat() != matrixFormatSparseCSC)
        NOT_IMPLEMENTED;

    let nnz = NzCount();
    this->ReserveTempDeviceBuffer(nnz);
    map<size_t, GPUSPARSE_INDEX_TYPE> indexer;
    GPUSPARSE_INDEX_TYPE* rowToId = (GPUSPARSE_INDEX_TYPE*) ReserveTempHostBuffer(sizeof(GPUSPARSE_INDEX_TYPE) * nnz * 2);

    // In the first nnz values of the 'rowToId' we will store the block ids of the nonzero-values (to be computed below).
    // In the next nnz values of 'rowToId' we store the row-ids of the non-zero values (copied from GPU).
    GPUSPARSE_INDEX_TYPE* h_Row = rowToId + nnz;
    CUDA_CALL(hipMemcpy(h_Row, RowLocation(), sizeof(GPUSPARSE_INDEX_TYPE) * nnz, hipMemcpyDeviceToHost));

    for (size_t i = 0; i < nnz; i++)
    {
        size_t row = h_Row[i];
        if (indexer.find(row) == indexer.end())
        {
            size_t id = indexer.size(); // We need to assign size to a temp variable due to difference in Linux and Windows
            indexer[row] = id;
        }
        rowToId[i] = indexer[row];
    }
    CUDA_CALL(hipMemcpy(GetTempDeviceBuffer(), rowToId, sizeof(GPUSPARSE_INDEX_TYPE) * nnz, hipMemcpyHostToDevice));
    return indexer.size();
}

// used for gradients udpate
template <class ElemType>
void GPUSparseMatrix<ElemType>::ScaleAndAdd(const ElemType alpha, const GPUSparseMatrix<ElemType>& lhs, GPUMatrix<ElemType>& rhs)
{
    if (lhs.GetNumRows() != rhs.GetNumRows() || lhs.GetNumCols() != rhs.GetNumCols())
        LogicError("ScaleAndAdd: dimension mismatch");

    if (lhs.GetComputeDeviceId() != rhs.GetComputeDeviceId())
        RuntimeError("GPUSparseMatrix::ScaleAndAdd: All matrices must be on the same GPU");

    if (lhs.GetFormat() == matrixFormatSparseBlockCol || lhs.GetFormat() == matrixFormatSparseBlockRow)
    {
        bool blockCol = (lhs.GetFormat() == matrixFormatSparseBlockCol);

        SyncGuard syncGuard;
        LONG64 N = (LONG64) lhs.GetNumNZElements();
        int blocksPerGrid = (int) ceil(((double) N) / GridDim::maxThreadsPerBlock);
        _scaleSparseBlockAndAddToDense<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(
            alpha,
            blockCol,
            lhs.GetNumRows(),
            lhs.GetNumCols(),
            lhs.GetBlockSize(),
            lhs.Data(),
            lhs.BlockId2ColOrRow(),
            rhs.Data());

    }
    else
    {
        ScaleAndAdd(alpha, lhs, 1, rhs, rhs);
    }
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::InplaceTruncate(const ElemType threshold)
{
    VerifyWritable(__FUNCTION__);

    CUDA_LONG N = (CUDA_LONG) GetNumNZElements();

    CUDA_LONG blocksPerGrid = (CUDA_LONG) ceil(N * 1.0 / GridDim::maxThreadsPerBlock);
    SyncGuard syncGuard;
    ElemType* values = NzValues();
    _inplaceTruncate<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(values, threshold, N);
    return *this;
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::InplaceSoftThreshold(const ElemType threshold)
{
    VerifyWritable(__FUNCTION__);

    CUDA_LONG N = (CUDA_LONG) GetNumNZElements();

    CUDA_LONG blocksPerGrid = (CUDA_LONG) ceil(N * 1.0 / GridDim::maxThreadsPerBlock);
    SyncGuard syncGuard;
    ElemType* values = NzValues();
    _inplaceSoftThreshold<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(values, threshold, N);
    return *this;
}

// A helper method used in MomentumSGDUpdate and NesterovAcceleratedMomentumSGDUpdate.
// Modifies the smoothed gradients "c", as well as the current gradients "this" on which this method is invoked. 
// Classic momentum (unitGainFactor == 1.0):
// 1) c = momentum * c + this
// Unit-gain momentum (unitGainFactor == 1.0 - momentum):
// 1) c = momentum * c + (1.0 - momentum) * this
// 2) this = c
// TODO: NormalGrad is a misnomer here. Come up with a better name.
template <class ElemType>
void GPUSparseMatrix<ElemType>::NormalGrad(GPUMatrix<ElemType>& c, const ElemType momentum, bool unitGainMomentum)
{
    VerifyWritable(__FUNCTION__);

    if (c.IsEmpty())
    {
        c.RequireSize(GetNumRows(), GetNumCols());
        c.SetValue(0.0);
    }

    if (GetFormat() == matrixFormatSparseBlockCol || GetFormat() == matrixFormatSparseBlockRow)
    {
        bool isBlockCol = (GetFormat() == MatrixFormat::matrixFormatSparseBlockCol);
        SyncGuard syncGuard;
        LONG64 N = (LONG64) GetNumNZElements();
        int blocksPerGrid = (int) ceil(((double) N) / GridDim::maxThreadsPerBlock);

        _normalGradForSparseBlock<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(
            momentum,
            isBlockCol,
            GetNumRows(),
            GetNumCols(),
            GetBlockSize(),
            Data(),
            BlockId2ColOrRow(),
            c.Data(),
            unitGainMomentum);
    }
    else
    {
        NOT_IMPLEMENTED;
    }
}

template <class ElemType>
ElemType GPUSparseMatrix<ElemType>::Adagrad(GPUMatrix<ElemType>& c, const bool needAveMultiplier)
{
    VerifyWritable(__FUNCTION__);

    size_t numColsNeeded = GetNumCols();
    if (needAveMultiplier)
        numColsNeeded += GetNumCols();

    if (c.IsEmpty() || c.GetNumCols() < numColsNeeded)
    {
        c.RequireSize(GetNumRows(), numColsNeeded);
        c.SetValue(0.0);
    }

    assert(c.GetNumRows() == GetNumRows() && c.GetNumCols() == numColsNeeded);

    size_t n = this->GetNumElements();

    ElemType* multipliers = nullptr;
    if (needAveMultiplier)
        multipliers = c.Buffer() + n; // temp memory used to store multipliers,

    if (GetFormat() == MatrixFormat::matrixFormatSparseCSC || GetFormat() == MatrixFormat::matrixFormatSparseCSR)
    {
        NOT_IMPLEMENTED;
    }
    else if (GetFormat() == MatrixFormat::matrixFormatSparseBlockCol || GetFormat() == MatrixFormat::matrixFormatSparseBlockRow)
    {
        let nz = NzCount();
        int blocksPerGrid = (nz + GridDim::maxThreadsPerBlock - 1) / GridDim::maxThreadsPerBlock;
        bool colMajor = GetFormat() == MatrixFormat::matrixFormatSparseBlockCol;
        size_t len = colMajor ? GetNumRows() : GetNumCols();
        _adagrad4BlockSparse<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(c.Buffer(), c.GetNumRows(), Data(), BlockId2ColOrRow(), multipliers, colMajor, len, nz);
    }
    else
        NOT_IMPLEMENTED;

    if (!needAveMultiplier)
        return 1;

    let nz = NzCount();
    hipblasHandle_t cuHandle = GPUMatrix<ElemType>::GetCublasHandle(GetComputeDeviceId());
    if (sizeof(ElemType) == sizeof(float))
    {
        float aveMultiplier = 0;
        CUBLAS_CALL(hipblasSasum(cuHandle, (LONG64) nz, reinterpret_cast<float*>(multipliers), 1, &aveMultiplier));
        return (ElemType) aveMultiplier / nz;
    }
    else
    {
        double aveMultiplier = 0;
        CUBLAS_CALL(hipblasDasum(cuHandle, (LONG64) nz, reinterpret_cast<double*>(multipliers), 1, &aveMultiplier));
        return (ElemType) aveMultiplier / nz;
    }
}

template <class ElemType>
void GPUSparseMatrix<ElemType>::FSAdagrad(
    GPUMatrix<ElemType>& c,
    GPUMatrix<ElemType>& functionValues,
    ElemType learnRatePerSample,
    ElemType momentum,
    ElemType adaWeight,
    ElemType adaMul,
    bool unitGainMomentum)
{
    if (GetFormat() != MatrixFormat::matrixFormatSparseBlockCol)
    {
        NOT_IMPLEMENTED;
    }

    size_t numColsNeeded = 2 * GetNumCols();

    if (c.IsEmpty() || (c.GetNumCols() < numColsNeeded))
    {
        c.RequireSize(GetNumRows(), numColsNeeded);
        c.SetValue(0.0);
    }

    assert((c.GetNumRows() == GetNumRows()) && (c.GetNumCols() == numColsNeeded));

    size_t n = GetNumElements();
    int blocksPerGrid = (n + GridDim::maxThreadsPerBlock - 1) / GridDim::maxThreadsPerBlock;
    _fsadagrad4BlockSparseCol<ElemType> << <blocksPerGrid, GridDim::maxThreadsPerBlock >> >(
        n, Data(), ColOrRow2BlockId(), GetNumRows(),
        c.Data(), c.Data() + n, functionValues.Data(),
        learnRatePerSample, momentum, adaWeight, adaMul, unitGainMomentum);
}

template <class ElemType>
void GPUSparseMatrix<ElemType>::Adam(
    GPUMatrix<ElemType>& c,
    GPUMatrix<ElemType>& functionValues,
    ElemType learnRatePerSample,
    ElemType momentum,
    ElemType adaWeight,
    ElemType adaMul,
    bool unitGainMomentum)
{
    if (GetFormat() != MatrixFormat::matrixFormatSparseBlockCol)
    {
        NOT_IMPLEMENTED;
    }

    size_t numColsNeeded = 2 * GetNumCols();

    if (c.IsEmpty() || (c.GetNumCols() < numColsNeeded))
    {
        c.RequireSize(GetNumRows(), numColsNeeded);
        c.SetValue(0.0);
    }

    assert((c.GetNumRows() == GetNumRows()) && (c.GetNumCols() == numColsNeeded));

    size_t n = GetNumElements();
    int blocksPerGrid = (n + GridDim::maxThreadsPerBlock - 1) / GridDim::maxThreadsPerBlock;
    _adam4BlockSparseCol<ElemType> << <blocksPerGrid, GridDim::maxThreadsPerBlock >> >(
        n, Data(), ColOrRow2BlockId(), GetNumRows(),
        c.Data(), c.Data() + n, functionValues.Data(),
        learnRatePerSample, momentum, adaWeight, adaMul, unitGainMomentum);
}

template <class ElemType>
ElemType GPUSparseMatrix<ElemType>::RmsProp(GPUMatrix<ElemType>& c,
    ElemType RMS_GAMMA,
    ElemType RMS_WGT_INC,
    ElemType RMS_WGT_MAX,
    ElemType RMS_WGT_DEC,
    ElemType RMS_WGT_MIN,
    const bool needAveMultiplier)
{
    if (GetFormat() != MatrixFormat::matrixFormatSparseBlockCol)
    {
        NOT_IMPLEMENTED;
    }

    const ElemType floor = 1e-6f;
    static ElemType* upd_gpu = (ElemType*)0;

    size_t n = GetNumElements();
    int blocksPerGrid = (c.GetNumElements() + GridDim::maxThreadsPerBlock - 1) / GridDim::maxThreadsPerBlock;

    size_t numColsNeeded = GetNumCols() * 3;
    if (needAveMultiplier)
        numColsNeeded += GetNumCols();

    if (c.IsEmpty() || c.GetNumCols() < numColsNeeded)
    {
        c.RequireSize(GetNumRows(), numColsNeeded);
        c.SetValue(0.0);

        ElemType* avars = c.Data();         // accumulated variances for RMS scaling
        ElemType* signs = c.Data() + n;     // sign of previous gradient
        ElemType* steps = c.Data() + 2 * n; // current step size
                                            // Data()+3*n is temp memory used to store multipliers, no need to initialize

        _rmsprop_init4BlockSparseCol<ElemType> << <blocksPerGrid, GridDim::maxThreadsPerBlock >> >(
            avars, signs, steps, 
            Data(), ColOrRow2BlockId(), GetNumRows(),
            n);
    }
    assert(c.GetNumRows() == GetNumRows() && c.GetNumCols() == numColsNeeded);

    ElemType* avars = c.Data();         // accumulated variances for RMS scaling
    ElemType* signs = c.Data() + n;     // sign of previous gradient
    ElemType* steps = c.Data() + 2 * n; // current step size

    ElemType* multipliers = nullptr;
    if (needAveMultiplier)
        multipliers = c.Data() + 3 * n; // temp memory used to store multipliers,

    if (!upd_gpu)
    {
        const ElemType upd[] = {
            2, 2, 0,
            2, 2, 0,
            1, 1, 1,
            2, 2, 0,
            1, 2, 1,
            0, 2, 2,
            1, 1, 1,
            0, 2, 2,
            0, 2, 2,
        };

        upd_gpu = TracingGPUMemoryAllocator::Allocate<ElemType>(GetComputeDeviceId(), 27);
        CUDA_CALL(hipMemcpy(upd_gpu, upd, sizeof(ElemType) * _countof(upd), hipMemcpyHostToDevice));
    }

    _rmsprop4BlockSparseCol<ElemType> << <blocksPerGrid, GridDim::maxThreadsPerBlock >> >(
        avars, signs, steps,
        Data(), ColOrRow2BlockId(), GetNumRows(),
        n,
        RMS_GAMMA, RMS_WGT_INC, RMS_WGT_MAX, RMS_WGT_DEC, RMS_WGT_MIN,
        floor, upd_gpu, multipliers);

    if (!needAveMultiplier)
        return 1;

    hipblasHandle_t cuHandle = GPUMatrix<ElemType>::GetCublasHandle(GetComputeDeviceId());
    if (sizeof(ElemType) == sizeof(float))
    {
        float aveMultiplier = 0;
        CUBLAS_CALL(hipblasSasum(cuHandle, (CUDA_LONG)n, reinterpret_cast<float*>(multipliers), 1, &aveMultiplier));
        return aveMultiplier / n;
    }
    else
    {
        double aveMultiplier = 0;
        CUBLAS_CALL(hipblasDasum(cuHandle, (CUDA_LONG)n, reinterpret_cast<double*>(multipliers), 1, &aveMultiplier));
        return (ElemType)aveMultiplier / n;
    }
}

// sparse X dense = dense
template <class ElemType>
void GPUSparseMatrix<ElemType>::MultiplyAndWeightedAdd(ElemType alpha, const GPUSparseMatrix<ElemType>& a, const bool transposeA,
                                                       const GPUMatrix<ElemType>& b, const bool transposeB, ElemType beta, GPUMatrix<ElemType>& c)
{
    if (transposeB)
        NOT_IMPLEMENTED;

    // Note: This function is written for 'a' being in CSR format. If 'a' is CSC, we reinterpret it as CSR by transposing it.
    if (a.GetFormat() != matrixFormatSparseCSR && a.GetFormat() != matrixFormatSparseCSC)
        NOT_IMPLEMENTED;
    const bool reinterpretAsCSR = a.GetFormat() == matrixFormatSparseCSC;

    if (a.GetComputeDeviceId() != b.GetComputeDeviceId() || (b.GetComputeDeviceId() != a.GetComputeDeviceId()))
        RuntimeError("MultiplyAndWeightedAdd: All matrices must be on the same GPU");

    a.PrepareDevice();
    hipsparseHandle_t cusparseHandle = 0;
    CUSPARSE_CALL(hipsparseCreate(&cusparseHandle));
    hipsparseMatDescr_t descr = 0;
    CUSPARSE_CALL(hipsparseCreateMatDescr(&descr));
    hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

    hipsparseOperation_t oper = (transposeA != reinterpretAsCSR) ? HIPSPARSE_OPERATION_TRANSPOSE : HIPSPARSE_OPERATION_NON_TRANSPOSE;

    int n = (int)b.GetNumCols();
    int m = (int)(reinterpretAsCSR ? a.GetNumCols() : a.GetNumRows());
    int k = (int)(reinterpretAsCSR ? a.GetNumRows() : a.GetNumCols());
    assert(n == (int) c.GetNumCols());

    const auto& aRowLocation = reinterpretAsCSR ? a.ColLocation() : a.RowLocation();
    const auto& aColLocation = reinterpretAsCSR ? a.RowLocation() : a.ColLocation();

    SyncGuard syncGuard;
    if (sizeof(ElemType) == sizeof(float))
    {
        CUSPARSE_CALL(hipsparseScsrmm(cusparseHandle, oper, m, n, k, (int) a.GetNumNZElements(), reinterpret_cast<float*>(&alpha), descr, reinterpret_cast<const float*>(a.Buffer()),
                                     aRowLocation, aColLocation, reinterpret_cast<float*>(b.Data()),
                                     (int) b.GetNumRows(), reinterpret_cast<float*>(&beta), reinterpret_cast<float*>(c.Data()), (int) c.GetNumRows()));
    }
    else
    {
        CUSPARSE_CALL(hipsparseDcsrmm(cusparseHandle, oper, m, n, k, (int) a.GetNumNZElements(), reinterpret_cast<double*>(&alpha), descr, reinterpret_cast<const double*>(a.Buffer()),
                                     aRowLocation, aColLocation, reinterpret_cast<double*>(b.Data()),
                                     (int) b.GetNumRows(), reinterpret_cast<double*>(&beta), reinterpret_cast<double*>(c.Data()), (int) c.GetNumRows()));
    }
    CUSPARSE_CALL(hipsparseDestroy(cusparseHandle));
}

template <class ElemType>
void GPUSparseMatrix<ElemType>::Multiply(const GPUSparseMatrix<ElemType>& S, const GPUMatrix<ElemType>& D, GPUMatrix<ElemType>& C)
{
    C.RequireSize(S.GetNumRows(), D.GetNumCols());

    MultiplyAndWeightedAdd(1, S, false, D, false, 0, C);
}

template <class ElemType>
void GPUSparseMatrix<ElemType>::Multiply(const GPUMatrix<ElemType>& D, const GPUSparseMatrix<ElemType>& S, GPUMatrix<ElemType>& C)
{
    C.RequireSize(S.GetNumCols(), D.GetNumRows());

    MultiplyAndWeightedAdd(1, D, false, S, false, 0, C);
}

// ElemCountFromBufferSize - Return the elemCountAllocated for a particular buffersize
// totalBufferSize - total buffer we have to use
// return: size of allocated elements/index slots available
template <class ElemType>
size_t GPUSparseMatrix<ElemType>::ElemCountFromBufferSize(const size_t numRows, const size_t numCols, const MatrixFormat format, const size_t totalBufferSize) const
{
    size_t elemSizeAllocated;
    if (format == matrixFormatSparseCSC)
    {
        elemSizeAllocated = (totalBufferSize - sizeof(GPUSPARSE_INDEX_TYPE) * (numCols + 1)) / (sizeof(GPUSPARSE_INDEX_TYPE) + sizeof(ElemType));
    }
    else if (format == matrixFormatSparseCSR)
    {
        elemSizeAllocated = (totalBufferSize - sizeof(GPUSPARSE_INDEX_TYPE) * (numRows + 1)) / (sizeof(GPUSPARSE_INDEX_TYPE) + sizeof(ElemType));
    }
    else if (format == matrixFormatSparseBlockCol)
    {
        elemSizeAllocated = (totalBufferSize - sizeof(GPUSPARSE_INDEX_TYPE) * 2 * numCols) / sizeof(ElemType);
    }
    else if (format == matrixFormatSparseBlockCol || format == matrixFormatSparseBlockRow)
    {
        elemSizeAllocated = (totalBufferSize - sizeof(GPUSPARSE_INDEX_TYPE) * 2 * numRows) / sizeof(ElemType);
    }
    else // uncompressed COO format
    {
        elemSizeAllocated = totalBufferSize / (2 * sizeof(GPUSPARSE_INDEX_TYPE) + sizeof(ElemType));
    }
    return elemSizeAllocated;
}

template <class ElemType>
size_t GPUSparseMatrix<ElemType>::ElemCountFromBufferSize() const
{
    return ElemCountFromBufferSize(GetNumRows(), GetNumCols(), GetFormat(), BufferSizeAllocated());
}

// PrepareBuffer - Get the dimensions start buffer, computes the starting row/column of each value
// m - rows in the source
// n - cols in the source
// canReuseBuffer - target matrix can be reused for temporary space
// func - function to call to count elements in the result (returns count, and fills csrRowPtr array)
template <class ElemType>
void GPUSparseMatrix<ElemType>::PrepareBuffer(size_t m, size_t n, bool canReuseBuffer, std::function<size_t(GPUSPARSE_INDEX_TYPE* csrRowPtrC)> func)
{
    VerifyWritable(__FUNCTION__);

    if (this->GetFormat() != matrixFormatSparseCSR)
        NOT_IMPLEMENTED;

    PrepareDevice();

    GPUSPARSE_INDEX_TYPE* csrRowPtrC = nullptr;
    GPUSparseMatrix<ElemType>& c = *this;
    size_t cSize = c.BufferSizeAllocated();
    size_t rowBufferRequired = (m + 1) * sizeof(GPUSPARSE_INDEX_TYPE);
    bool allocatedBuffer = false;

    // do we have enough memory to store just the row buffer?
    if (cSize >= rowBufferRequired && c.Data() != nullptr && canReuseBuffer)
    {
        csrRowPtrC = (GPUSPARSE_INDEX_TYPE*) c.Data();
    }
    else
    {
        csrRowPtrC = TracingGPUMemoryAllocator::Allocate<GPUSPARSE_INDEX_TYPE>(GetComputeDeviceId(), rowBufferRequired / sizeof(GPUSPARSE_INDEX_TYPE));
        allocatedBuffer = true;
    }

    // get the non-zero count from the function (and
    size_t nnzC = func(csrRowPtrC);

    // now we know the number of Non-zeros in the result set, set the output size
    c.RequireSizeAndAllocate(m, n, nnzC, true, false);

    CUDA_CALL(hipMemcpy(c.SecondaryIndexLocation(), csrRowPtrC, c.SecondaryIndexSize(), hipMemcpyDeviceToDevice));

    // if we allocated the buffer, free it here
    if (allocatedBuffer)
        TracingGPUMemoryAllocator::Free<GPUSPARSE_INDEX_TYPE>(GetComputeDeviceId(), csrRowPtrC);
}

// Multiply - multiply one spares matrix by another sparse matrix
// S1 - first sparse matrix
// transposeS1 - transpose first matrix?
// S2 - second sparse matrix
// transposeS2 - tanspose second matrix?
// c - result matrix
// NOTE: if c has enough space allocated, it will be reused, otherwise it will be freed and a new memory block used
template <class ElemType>
void GPUSparseMatrix<ElemType>::Multiply(const GPUSparseMatrix<ElemType>& S1, bool transposeS1, const GPUSparseMatrix<ElemType>& S2, bool transposeS2, GPUSparseMatrix<ElemType>& c)
{
    c.VerifyWritable(__FUNCTION__);

    if (S1.GetFormat() != matrixFormatSparseCSR || S2.GetFormat() != matrixFormatSparseCSR || c.GetFormat() != matrixFormatSparseCSR)
        NOT_IMPLEMENTED;

    if (S1.GetComputeDeviceId() != S2.GetComputeDeviceId())
        RuntimeError("Sparse matrix multiply: both matrices must be on the same device");

    S1.PrepareDevice();
    hipsparseHandle_t cusparseHandle = 0;
    CUSPARSE_CALL(hipsparseCreate(&cusparseHandle));
    hipsparseMatDescr_t descrA = 0, descrB = 0, descrC = 0;
    CUSPARSE_CALL(hipsparseCreateMatDescr(&descrA));
    CUSPARSE_CALL(hipsparseCreateMatDescr(&descrB));
    CUSPARSE_CALL(hipsparseCreateMatDescr(&descrC));
    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatType(descrB, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatType(descrC, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseSetMatIndexBase(descrB, HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseSetMatIndexBase(descrC, HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseOperation_t operA = transposeS1 ? HIPSPARSE_OPERATION_TRANSPOSE : HIPSPARSE_OPERATION_NON_TRANSPOSE;
    hipsparseOperation_t operB = transposeS2 ? HIPSPARSE_OPERATION_TRANSPOSE : HIPSPARSE_OPERATION_NON_TRANSPOSE;

    int m = int(transposeS1 ? S1.GetNumCols() : S1.GetNumRows());
    int n = int(transposeS2 ? S2.GetNumRows() : S2.GetNumCols());
    int k = int(transposeS1 ? S1.GetNumRows() : S1.GetNumCols());
    int l = int(transposeS2 ? S2.GetNumCols() : S2.GetNumRows());
    if (k != l)
        RuntimeError("Sparse matrix multiply: dimensionality mismatch");

    int nnzA = (int) S1.GetNumNZElements();
    int nnzB = (int) S2.GetNumNZElements();

    SyncGuard syncGuard;
    // Step 1
    c.PrepareBuffer(m, n, false, // false means we cannot reuse the "c" buffer if it exists for temporaries
                    [&](GPUSPARSE_INDEX_TYPE* csrRowPtrC) -> size_t
                    {
                        int nnzTotal = -1;
                        CUSPARSE_CALL(hipsparseXcsrgemmNnz(cusparseHandle, operA, operB, m, n, k, descrA, nnzA, S1.RowLocation(), S1.ColLocation(), descrB, nnzB,
                                                          S2.RowLocation(), S2.ColLocation(), descrC, csrRowPtrC, &nnzTotal));
                        return nnzTotal;
                    });

    // Step 2
    if (sizeof(float) == sizeof(ElemType))
    {
        CUSPARSE_CALL(hipsparseScsrgemm(cusparseHandle, operA, operB, m, n, k, descrA, nnzA, (const float*) S1.Buffer(), S1.RowLocation(), S1.ColLocation(),
                                       descrB, nnzB, (const float*) S2.Buffer(), S2.RowLocation(), S2.ColLocation(),
                                       descrC, (float*) c.Data(), c.RowLocation(), c.ColLocation()));
    }
    else
    {
        CUSPARSE_CALL(hipsparseDcsrgemm(cusparseHandle, operA, operB, m, n, k, descrA, nnzA, (const double*) S1.Buffer(), S1.RowLocation(), S1.ColLocation(),
                                       descrB, nnzB, (const double*) S2.Buffer(), S2.RowLocation(), S2.ColLocation(),
                                       descrC, (double*) c.Data(), c.RowLocation(), c.ColLocation()));
    }
    hipsparseDestroy(cusparseHandle);
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignProductOf(const GPUSparseMatrix<ElemType>& a, const bool transposeA, const GPUSparseMatrix<ElemType>& b, const bool transposeB)
{
    Multiply(a, transposeA, b, transposeB, *this);
    return *this;
}

template <class ElemType>
void GPUSparseMatrix<ElemType>::ScaleAndAdd(ElemType alpha, const GPUSparseMatrix<ElemType>& a, ElemType beta, const GPUSparseMatrix<ElemType>& b, GPUSparseMatrix<ElemType>& c)
{
    if (a.GetFormat() != matrixFormatSparseCSR || b.GetFormat() != matrixFormatSparseCSR )
    {
        NOT_IMPLEMENTED;
    }
    if (c.m_sob == nullptr)
        c.ZeroInit(a.GetFormat(), a.GetComputeDeviceId());

    if (a.GetNumCols() != b.GetNumCols() || a.GetNumRows() != b.GetNumRows())
        RuntimeError("Dimensions mismatch in ScaleAndAdd");
    if (a.GetComputeDeviceId() != b.GetComputeDeviceId())
        RuntimeError("ScaleAndAdd: matrices must be on the same device");

    c.SetFormat(a.GetFormat());
    c.SetComputeDeviceId(a.GetComputeDeviceId());
    int m = (int) a.GetNumRows();
    int n = (int) a.GetNumCols();
    int nnzA = (int) a.GetNumNZElements();
    int nnzB = (int) b.GetNumNZElements();

    a.PrepareDevice();
    hipsparseHandle_t cusparseHandle = 0;
    CUSPARSE_CALL(hipsparseCreate(&cusparseHandle));
    hipsparseMatDescr_t descrA = 0, descrB = 0, descrC = 0;
    CUSPARSE_CALL(hipsparseCreateMatDescr(&descrA));
    CUSPARSE_CALL(hipsparseCreateMatDescr(&descrB));
    CUSPARSE_CALL(hipsparseCreateMatDescr(&descrC));
    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatType(descrB, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatType(descrB, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseSetMatIndexBase(descrB, HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseSetMatIndexBase(descrC, HIPSPARSE_INDEX_BASE_ZERO);

    SyncGuard syncGuard;
    // Step 1
    bool inOutParameter = (&b == &c);
    c.PrepareBuffer(m, n, !inOutParameter, 
                    [&](GPUSPARSE_INDEX_TYPE* csrRowPtrC) -> size_t
                    {
                        int nnzTotal = -1;
                        CUSPARSE_CALL(hipsparseXcsrgeamNnz(cusparseHandle, m, n, descrA, nnzA, a.RowLocation(), a.ColLocation(), descrB, nnzB, b.RowLocation(), b.ColLocation(), descrC, csrRowPtrC, &nnzTotal));
                        return nnzTotal;
                    });

    // Step 2
    if (sizeof(ElemType) == sizeof(float))
    {
        CUSPARSE_CALL(hipsparseScsrgeam(cusparseHandle, m, n, reinterpret_cast<const float*>(&alpha), descrA, nnzA, reinterpret_cast<const float*>(a.Data()), a.RowLocation(), a.ColLocation(),
                                       reinterpret_cast<const float*>(&beta), descrB, nnzB, reinterpret_cast<const float*>(b.Data()), b.RowLocation(), b.ColLocation(), descrC, reinterpret_cast<float*>(c.Data()), c.RowLocation(), c.ColLocation()));
    }
    else
    {
        CUSPARSE_CALL(hipsparseDcsrgeam(cusparseHandle, m, n, reinterpret_cast<const double*>(&alpha), descrA, nnzA, reinterpret_cast<const double*>(a.Data()), a.RowLocation(), a.ColLocation(),
                                       reinterpret_cast<const double*>(&beta), descrB, nnzB, reinterpret_cast<const double*>(b.Data()), b.RowLocation(), b.ColLocation(), descrC, reinterpret_cast<double*>(c.Data()), c.RowLocation(), c.ColLocation()));
    }
    hipsparseDestroy(cusparseHandle);
}

template <class ElemType>
void GPUSparseMatrix<ElemType>::ScaleAndAdd(ElemType alpha, const GPUSparseMatrix<ElemType>& a, ElemType beta, const GPUMatrix<ElemType>& b, GPUMatrix<ElemType>& c)
{
    if (a.GetFormat() != matrixFormatSparseCSR)
        NOT_IMPLEMENTED;

    if (a.GetNumRows() != b.GetNumRows() || a.GetNumRows() != c.GetNumRows() || a.GetNumCols() != b.GetNumCols() || a.GetNumCols() != c.GetNumCols())
        LogicError("ScaleAndAdd: dimension mismatch");
    if (a.GetComputeDeviceId() != b.GetComputeDeviceId() || a.GetComputeDeviceId() != c.GetComputeDeviceId())
        RuntimeError("ScaleAndAdd: matrices must be on the same device");
    b.PrepareDevice();
    // copy b to c
    CUDA_CALL(hipMemcpy(c.Data(), b.Data(), sizeof(ElemType) * b.GetNumElements(), hipMemcpyDeviceToDevice));
    if (beta != 1)
    {
        c *= beta;
    }
    SyncGuard syncGuard;
    CUDA_LONG M = (CUDA_LONG) a.GetNumRows();
    int blocksPerGrid = (int) ceil(1.0 * M / GridDim::maxThreadsPerBlock);
    _sparseCSRPlusDense<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(alpha, a.Data(), a.RowLocation(), a.ColLocation(), c.Data(), M);
}

template <class ElemType>
void GPUSparseMatrix<ElemType>::ScaleAndAdd(ElemType alpha, const GPUMatrix<ElemType>& a, ElemType beta, const GPUSparseMatrix<ElemType>& b, GPUMatrix<ElemType>& c)
{
    ScaleAndAdd(beta, b, alpha, a, c);
}

template <class ElemType>
void GPUSparseMatrix<ElemType>::Scale(ElemType alpha, GPUSparseMatrix<ElemType>& a)
{
    a.VerifyWritable(__FUNCTION__);

    if (a.IsEmpty())
        return;

    CUDA_LONG N = (CUDA_LONG) a.GetNumNZElements();
    int blocksPerGrid = (int) ceil(1.0 * N / GridDim::maxThreadsPerBlock);
    SyncGuard syncGuard;
    _scaleArray<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(alpha, a.NzValues(), N);
}

template <class ElemType>
void GPUSparseMatrix<ElemType>::ElementWisePower(ElemType alpha, const GPUSparseMatrix<ElemType>& a, GPUSparseMatrix<ElemType>& c)
{
    c.VerifyWritable(__FUNCTION__);

    if (a.GetComputeDeviceId() != c.GetComputeDeviceId())
    {
        InvalidArgument("All matrices must be on the same GPU");
    }
    else
    {
        if (a.IsEmpty())
            LogicError("ElementWisePower:  The input matrix a is empty.");

        c.ResizeAsAndCopyIndexFrom(a);

        SyncGuard syncGuard;
        a.PrepareDevice();
        CUDA_LONG N = (CUDA_LONG) a.GetNumNZElements();
        int blocksPerGrid = (int) ceil(1.0 * N / GridDim::maxThreadsPerBlock);
        _elementWisePowerOnCuda<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(alpha, a.NzValues(), c.NzValues(), N);
    }
}

// sparse x dense = scalar
template <class ElemType>
ElemType GPUSparseMatrix<ElemType>::InnerProductOfMatrices(const GPUSparseMatrix<ElemType>& a, const GPUMatrix<ElemType>& b)
{
    if (a.GetFormat() != matrixFormatSparseCSR && a.GetFormat() != matrixFormatSparseCSC)
        NOT_IMPLEMENTED;

    if (a.GetComputeDeviceId() != b.GetComputeDeviceId())
        RuntimeError("a and b must be on the same device");

    int m = (int) a.GetNumRows();
    int n = (int) a.GetNumCols();
    int nnz = (int) a.GetNumNZElements();

    ElemType* cscValA = nullptr;
    GPUSPARSE_INDEX_TYPE* cscRowIndA = nullptr;
    GPUSPARSE_INDEX_TYPE* cscColPtrA = nullptr;

    hipsparseAction_t cpVals = HIPSPARSE_ACTION_NUMERIC;
    hipsparseIndexBase_t idxBase = HIPSPARSE_INDEX_BASE_ZERO;
    hipsparseHandle_t cusparseHandle = 0;
    CUSPARSE_CALL(hipsparseCreate(&cusparseHandle));

    bool allocTemp = (a.GetFormat() == matrixFormatSparseCSR);

    if (allocTemp) // need to put a in ColumnMajor format
    {
        cscValA = TracingGPUMemoryAllocator::Allocate<ElemType>(a.GetComputeDeviceId(), nnz);
        cscRowIndA = TracingGPUMemoryAllocator::Allocate<GPUSPARSE_INDEX_TYPE>(a.GetComputeDeviceId(), nnz);
        cscColPtrA = TracingGPUMemoryAllocator::Allocate<GPUSPARSE_INDEX_TYPE>(a.GetComputeDeviceId(), (n + 1));

        SyncGuard syncGuard;
        if (sizeof(ElemType) == sizeof(float))
        {
            CUSPARSE_CALL(hipsparseScsr2csc(cusparseHandle, m, n, nnz, reinterpret_cast<const float*>(a.Data()), a.RowLocation(), a.ColLocation(), reinterpret_cast<float*>(cscValA), cscRowIndA, cscColPtrA, cpVals, idxBase));
        }
        else
        {
            CUSPARSE_CALL(hipsparseDcsr2csc(cusparseHandle, m, n, nnz, reinterpret_cast<const double*>(a.Data()), a.RowLocation(), a.ColLocation(), reinterpret_cast<double*>(cscValA), cscRowIndA, cscColPtrA, cpVals, idxBase));
        }
    }
    else if (a.GetFormat() == matrixFormatSparseCSC)
    {
        cscValA = (ElemType*) a.Data();
        cscRowIndA = a.RowLocation();
        cscColPtrA = a.ColLocation();
    }
    else
    {
        NOT_IMPLEMENTED;
    }
    let a_nz = a.NzCount();
    // Given sparse matrix in column major format, calculate indices for corresponding sparse vector
    GPUSPARSE_INDEX_TYPE* vectArray = TracingGPUMemoryAllocator::Allocate<GPUSPARSE_INDEX_TYPE>(a.GetComputeDeviceId(), a_nz);
    CUDA_LONG M = n;
    CUDA_LONG N = m;
    // GPUSPARSE_INDEX_TYPE* h_vectArray= new int[a.m_nz];
    int blocksPerGrid = (int) ceil(1.0 * M / GridDim::maxThreadsPerBlock);
    SyncGuard syncGuard;
    _getSparseVectorRepresntationForCSCMatrix<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(cscColPtrA, cscRowIndA, vectArray, M, N);
    if (allocTemp)
    {
        TracingGPUMemoryAllocator::Free<GPUSPARSE_INDEX_TYPE>(a.GetComputeDeviceId(), cscRowIndA);
        TracingGPUMemoryAllocator::Free<GPUSPARSE_INDEX_TYPE>(a.GetComputeDeviceId(), cscColPtrA);
    }
    // CUDA_CALL(hipMemcpy(h_vectArray,vectArray,sizeof(GPUSPARSE_INDEX_TYPE)*a.m_nz,hipMemcpyDeviceToHost));

    // Actual dot product
    ElemType res = 0;
    if (sizeof(ElemType) == sizeof(float))
    {
        CUSPARSE_CALL(hipsparseSdoti(cusparseHandle, (int) a_nz, reinterpret_cast<float*>(cscValA), vectArray,
                                    reinterpret_cast<float*>(b.Data()),
                                    reinterpret_cast<float*>(&res), idxBase));
    }
    else
    {
        CUSPARSE_CALL(hipsparseDdoti(cusparseHandle, (int) a_nz, reinterpret_cast<double*>(cscValA), vectArray,
                                    reinterpret_cast<double*>(b.Data()),
                                    reinterpret_cast<double*>(&res), idxBase));
    }
    TracingGPUMemoryAllocator::Free<GPUSPARSE_INDEX_TYPE>(a.GetComputeDeviceId(), vectArray);
    if (allocTemp)
    {
        TracingGPUMemoryAllocator::Free<ElemType>(a.GetComputeDeviceId(), cscValA);
    }
    CUSPARSE_CALL(hipsparseDestroy(cusparseHandle));
    return res;
}

template <class ElemType>
ElemType GPUSparseMatrix<ElemType>::InnerProductOfMatrices(const GPUMatrix<ElemType>& a, const GPUSparseMatrix<ElemType>& b)
{
    return GPUSparseMatrix<ElemType>::InnerProductOfMatrices(b, a);
}

template <class ElemType>
void GPUSparseMatrix<ElemType>::InnerProduct(const GPUSparseMatrix<ElemType>& a, const GPUMatrix<ElemType>& b, GPUMatrix<ElemType>& c, const bool isColWise)
{
    if (a.GetComputeDeviceId() != b.GetComputeDeviceId() || b.GetComputeDeviceId() != c.GetComputeDeviceId()) // different GPUs
        InvalidArgument("All matrices must be on the same GPU");

    if (a.IsEmpty() || b.IsEmpty())
        LogicError("Scale:  one of the input matrices is empty.");

    if (a.GetFormat() != MatrixFormat::matrixFormatSparseCSC)
    {
        NOT_IMPLEMENTED;
    }

    const int m = (int)a.GetNumRows();
    const int n = (int)a.GetNumCols();
    const int k = (int)b.GetNumRows();
    const int l = (int)b.GetNumCols();

    assert(m > 0 && n > 0 && k > 0 && l > 0); // converting from size_t to int may cause overflow
    assert(m == k && n == l);                 // converting from size_t to int may cause overflow
    if (m != k || n != l)
        InvalidArgument("Matrices a and b should have same dimension.");

    if (isColWise)
        c.RequireSize(1, n);
    else
        c.RequireSize(m, 1);

    c.PrepareDevice();

    int blocksPerGrid = 0;
    if (isColWise) // col-wise
    {
        blocksPerGrid = (int)ceil(1.0 * n / GridDim::maxThreadsPerBlock);
    }
    else
    {
        blocksPerGrid = (int)ceil(1.0 * m / GridDim::maxThreadsPerBlock);
    }

    SyncGuard syncGuard;
    _innerProduct4SparseCSC<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(
        c.Data(),
        a.Data(), a.RowLocation(), a.ColLocation(),
        b.Data(),
        m, n, isColWise);
}

// This is an utility function useful for debugging issues with sparse matrices.
// It just checks that the CSC format indices are not corrupted / pointing to invalid memory.
template <class ElemType>
bool GPUSparseMatrix<ElemType>::IsValid() const
{
    if (GetFormat() != MatrixFormat::matrixFormatSparseCSC)
        NOT_IMPLEMENTED;

    long* res = new long[4];
    res[0] = 1;
    res[1] = 0;
    res[2] = 0;
    res[3] = 0;
    long* d_res = TracingGPUMemoryAllocator::Allocate<long>(GetComputeDeviceId(), 4);
    CUDA_CALL(hipMemcpy(d_res, res, sizeof(long) * 4, hipMemcpyHostToDevice));

    SyncGuard syncGuard;
    int blocksPerGrid = (int) ceil((1.0 * SecondaryIndexCount()) / GridDim::maxThreadsPerBlock);
    _isValid<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(MajorIndexLocation(), SecondaryIndexLocation(), GetNumRows(), GetNumCols(), GetNumNZElements(), d_res);

    CUDA_CALL(hipMemcpy(res, d_res, sizeof(long) * 4, hipMemcpyDeviceToHost));

    if (res[0] == 1)
    {
        return true;
    }
    else
    {
        fprintf(stderr, "GPUSparseMatrix::IsValid returned false (additional info: %ld %ld %ld %ld)\n", res[0], res[1], res[2], res[3]);
        return false;
    }
}

template <class ElemType>
/*static*/ bool GPUSparseMatrix<ElemType>::AreEqual(const GPUSparseMatrix<ElemType>& a, const GPUSparseMatrix<ElemType>& b,
                                                    const ElemType threshold)
{
    if (a.GetNumNZElements() != b.GetNumNZElements() || a.GetNumRows() != b.GetNumRows() || a.GetNumCols() != b.GetNumCols())
        return false;

    if (a.GetFormat() != b.GetFormat())
        NOT_IMPLEMENTED;

    long* res = new long[3];
    res[0] = 1;
    res[1] = 1;
    res[2] = 1;
    long* d_res = TracingGPUMemoryAllocator::Allocate<long>(a.GetComputeDeviceId(), 3);
    CUDA_CALL(hipMemcpy(d_res, res, sizeof(long) * 3, hipMemcpyHostToDevice));

    int blocksPerGrid = (int) ceil(1.0 * a.GetNumNZElements() / GridDim::maxThreadsPerBlock);
    _areEqual<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(a.NzValues(), b.NzValues(), (CUDA_LONG) a.GetNumNZElements(), threshold, d_res);
    _areEqual<int><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(a.MajorIndexLocation(), b.MajorIndexLocation(), (CUDA_LONG) a.MajorIndexCount(), (int) threshold, d_res + 1);
    blocksPerGrid = (int) ceil((1.0 * a.SecondaryIndexCount()) / GridDim::maxThreadsPerBlock);
    _areEqual<int><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(a.SecondaryIndexLocation(), b.SecondaryIndexLocation(), (CUDA_LONG) a.SecondaryIndexCount(), (int) threshold, d_res + 2);

    CUDA_CALL(hipMemcpy(res, d_res, sizeof(long) * 3, hipMemcpyDeviceToHost));
    if (res[0] * res[1] * res[2] == 1)
        return true;
    else
        return false;
}

template <class ElemType>
/*static*/ bool GPUSparseMatrix<ElemType>::AreEqual(const GPUMatrix<ElemType>& a, const GPUSparseMatrix<ElemType>& b,
                                                    const ElemType threshold)
{
    if (a.GetNumRows() != b.GetNumRows() || a.GetNumCols() != b.GetNumCols())
        return false;
    GPUSparseMatrix<ElemType> c(b.GetComputeDeviceId(), b.GetFormat());
    c.SetValue(a);
    return AreEqual(c, b, threshold);
}

template <class ElemType>
/*static*/ bool GPUSparseMatrix<ElemType>::AreEqual(const GPUSparseMatrix<ElemType>& a, const GPUMatrix<ElemType>& b,
                                                    const ElemType threshold)
{
    if (a.GetNumRows() != b.GetNumRows() || a.GetNumCols() != b.GetNumCols())
        return false;
    GPUSparseMatrix<ElemType> c(a.GetComputeDeviceId(), a.GetFormat());
    c.SetValue(b);
    return AreEqual(a, c, threshold);
}

template <class ElemType>
bool GPUSparseMatrix<ElemType>::IsEqualTo(const GPUSparseMatrix<ElemType>& a, const ElemType threshold) const
{
    return AreEqual(*this, a, threshold);
}

template <class ElemType>
bool GPUSparseMatrix<ElemType>::IsEqualTo(const GPUMatrix<ElemType>& a, const ElemType threshold) const
{
    return AreEqual(*this, a, threshold);
}

#pragma endregion Static BLAS Functions

#pragma region Member BLAS Functions

// sparse x dense = dense
template <class ElemType>
GPUMatrix<ElemType> GPUSparseMatrix<ElemType>::ElementProductOf(const GPUSparseMatrix<ElemType>& a, const GPUMatrix<ElemType>& b)
{
    if (a.GetFormat() != matrixFormatSparseCSR)
        NOT_IMPLEMENTED;

    if (a.GetNumRows() != b.GetNumRows() || a.GetNumCols() != b.GetNumCols())
        LogicError("ElementProductOf: matrix dimensions mismatch");

    b.PrepareDevice();
    GPUMatrix<ElemType> c(b.GetNumRows(), b.GetNumCols(), b.GetComputeDeviceId());

    SyncGuard syncGuard;
    CUDA_LONG M = (CUDA_LONG) a.GetNumRows();
    int blocksPerGrid = (int) ceil(1.0 * M / GridDim::maxThreadsPerBlock);
    _sparseCSRElemMulDense<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(a.Data(), a.RowLocation(), a.ColLocation(), b.Data(), c.Data(), M);
    return c;
}

// sparse x dense = dense
template <class ElemType>
GPUMatrix<ElemType> GPUSparseMatrix<ElemType>::ElementProductOf(const GPUMatrix<ElemType>& a, const GPUSparseMatrix<ElemType>& b)
{
    return GPUSparseMatrix<ElemType>::ElementProductOf(b, a);
}

template <class ElemType>
GPUSparseMatrix<ElemType> GPUSparseMatrix<ElemType>::operator+(const GPUSparseMatrix<ElemType>& a) const
{
    GPUSparseMatrix<ElemType> res(GetComputeDeviceId(), GetFormat());
    GPUSparseMatrix<ElemType>::ScaleAndAdd(1, *this, 1, a, res);
    return res;
}

template <class ElemType>
GPUSparseMatrix<ElemType> GPUSparseMatrix<ElemType>::operator-(const GPUSparseMatrix<ElemType>& a) const
{
    GPUSparseMatrix<ElemType> res(GetComputeDeviceId(), GetFormat());
    GPUSparseMatrix<ElemType>::ScaleAndAdd(1, *this, -1, a, res);
    return res;
}

// TODO: This is an unusual use of this operator. Remove this.
template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::operator^=(ElemType alpha)
{
    GPUSparseMatrix<ElemType>& us = *this;
    ElementWisePower(alpha, us, us);
    return us;
}

// TODO: This is an unusual use of this operator. Remove this.
template <class ElemType>
GPUSparseMatrix<ElemType> GPUSparseMatrix<ElemType>::operator^(ElemType alpha) const
{
    GPUSparseMatrix<ElemType> c(GetComputeDeviceId(), GetFormat());
    ElementWisePower(alpha, *this, c);
    return c;
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::operator*=(ElemType alpha)
{
    GPUSparseMatrix<ElemType>& us = *this;
    if (alpha != 1)
        Scale(alpha, us);
    return us;
}

template <class ElemType>
GPUSparseMatrix<ElemType> GPUSparseMatrix<ElemType>::operator*(ElemType alpha) const
{
    GPUSparseMatrix<ElemType> c(*this);
    if (alpha != 1)
        Scale(alpha, c);
    return c;
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignElementPowerOf(const GPUSparseMatrix<ElemType>& a, const ElemType power)
{
    ElementWisePower(power, a, *this);
    return *this;
}

template <class ElemType>
GPUSparseMatrix<ElemType> GPUSparseMatrix<ElemType>::Transpose() const
{
    int m = (int) GetNumRows();
    int n = (int) GetNumCols();
    int nnz = (int) GetNumNZElements();
    hipsparseAction_t cpVals = HIPSPARSE_ACTION_NUMERIC;
    hipsparseIndexBase_t idxBase = HIPSPARSE_INDEX_BASE_ZERO;

    assert(GetFormat() & matrixFormatCompressed); // for now this only supports compressed formats
    PrepareDevice();
    GPUSparseMatrix c(GetComputeDeviceId(), GetFormat());
    c.RequireSizeAndAllocate(n, m, nnz, GetFormat(), true, false);

    hipsparseHandle_t cusparseHandle = 0;
    CUSPARSE_CALL(hipsparseCreate(&cusparseHandle));

    SyncGuard syncGuard;
    if (GetFormat() == MatrixFormat::matrixFormatSparseCSR)
    {
        if (nnz > 0)
        {
            if (sizeof(ElemType) == sizeof(float))
            {
                CUSPARSE_CALL(hipsparseScsr2csc(cusparseHandle, m, n, nnz, reinterpret_cast<const float*>(Data()), RowLocation(), ColLocation(),
                                               reinterpret_cast<float*>(c.Data()), c.ColLocation(), c.RowLocation(), cpVals, idxBase));
            }
            else
            {
                CUSPARSE_CALL(hipsparseDcsr2csc(cusparseHandle, m, n, nnz, reinterpret_cast<const double*>(Data()), RowLocation(), ColLocation(),
                                               reinterpret_cast<double*>(c.Data()), c.ColLocation(), c.RowLocation(), cpVals, idxBase));
            }
        }
        else
        {
            CUDA_CALL(hipMemset(c.Buffer(), 0, c.BufferSizeAllocated()));
        }
    }
    else if (GetFormat() == matrixFormatSparseCSC)
    {
        if (nnz > 0)
        {
            if (sizeof(ElemType) == sizeof(float))
            {
                CUSPARSE_CALL(hipsparseScsr2csc(cusparseHandle, n, m, nnz, reinterpret_cast<const float*>(this->Data()), this->ColLocation(), this->RowLocation(),
                                               reinterpret_cast<float*>(c.Data()), c.RowLocation(), c.ColLocation(), cpVals, idxBase));
            }
            else
            {
                CUSPARSE_CALL(hipsparseDcsr2csc(cusparseHandle, n, m, nnz, reinterpret_cast<const double*>(this->Data()), this->ColLocation(), this->RowLocation(),
                                               reinterpret_cast<double*>(c.Data()), c.RowLocation(), c.ColLocation(), cpVals, idxBase));
            }
        }
        else
        {
            CUDA_CALL(hipMemset(c.Buffer(), 0, c.BufferSizeAllocated()));
        }
    }
    else
    {
        NOT_IMPLEMENTED;
    }
    CUSPARSE_CALL(hipsparseDestroy(cusparseHandle));
    return c;
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignTransposeOf(const GPUSparseMatrix<ElemType>& a)
{
    VerifyWritable(__FUNCTION__);

    if (this == &a)
        LogicError("AssignTransposeOf: a is the same as [this]. Does not support inplace transpose.");

    if (a.IsEmpty())
        LogicError("AssignTransposeOf: Matrix a is empty.");

    *this = a.Transpose();
    return *this;
}

template <class ElemType>
void GPUSparseMatrix<ElemType>::InplaceTranspose()
{
    if (IsEmpty())
        return;
    // transfer converted block over to this pointer
    *this = std::move(Transpose());
}

template <class ElemType>
GPUSparseMatrix<ElemType> GPUSparseMatrix<ElemType>::ColumnSlice(size_t startColumn, size_t numCols) const
{
    if (startColumn + numCols > GetNumCols())
        InvalidArgument("The slice (%d+%d) is out of range of the source matrix (%d).", (int) startColumn, (int) numCols, (int) GetNumCols());

    if (GetFormat() != MatrixFormat::matrixFormatSparseCSC && (startColumn != 0 || numCols != GetNumCols()))
        NOT_IMPLEMENTED;

    GPUSparseMatrix<ElemType> slice(GetComputeDeviceId());
    slice.ShallowCopyFrom(*this);
    slice.SetNumCols(numCols);
    slice.m_sliceViewOffset          = m_sliceViewOffset + startColumn; // Just shift the compressed index location to the new startColumn - that's it!
    // Note: m_nz is missing from here because it does not exist. We must compute it every time.

    return slice;
}
    
template <class ElemType>
void GPUSparseMatrix<ElemType>::AssignColumnSliceToDense(GPUMatrix<ElemType>& slice, size_t startColumn, size_t numCols) const
{
    int m = (int) GetNumRows();
    int n = (int) GetNumCols();

    // We can either error out or RequireSize. Because RequireSize will error out if it's not allowed, I think this makes more sense.
    slice.RequireSize(m, numCols);

    if (startColumn + numCols > n)
        InvalidArgument("The slice (%d+%d) is out of range of the source matrix (%d).", (int) startColumn, (int) numCols, (int) n);

    if (GetFormat() != MatrixFormat::matrixFormatSparseCSC)
        NOT_IMPLEMENTED;

    PrepareDevice();
    hipsparseHandle_t cusparseHandle = 0;
    CUSPARSE_CALL(hipsparseCreate(&cusparseHandle));
    hipsparseMatDescr_t descr = 0;
    CUSPARSE_CALL(hipsparseCreateMatDescr(&descr));
    hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

    SyncGuard syncGuard;
    CUSPARSE_CALL(hipsparseSetStream(cusparseHandle, t_stream));
    if (sizeof(ElemType) == sizeof(float))
    {
        CUSPARSE_CALL(hipsparseScsc2dense(cusparseHandle, m, numCols, descr, (float*) Buffer(), RowLocation(), ColLocation() + startColumn, (float*) slice.Data(), m));
    }
    else
    {
        CUSPARSE_CALL(hipsparseDcsc2dense(cusparseHandle, m, numCols, descr, (double*) Buffer(), RowLocation(), ColLocation() + startColumn, (double*) slice.Data(), m));
    }

    CUSPARSE_CALL(hipsparseDestroy(cusparseHandle));

}
template <class ElemType>
GPUMatrix<ElemType> GPUSparseMatrix<ElemType>::CopyColumnSliceToDense(size_t startColumn, size_t numCols) const
{
    GPUMatrix<ElemType> slice(GetNumRows(), numCols, GetComputeDeviceId());

    AssignColumnSliceToDense(slice, startColumn, numCols);

    return slice;
}

template <class ElemType>
GPUMatrix<ElemType> GPUSparseMatrix<ElemType>::DiagonalToDense() const
{
    int m = (int) GetNumRows();
    int n = (int) GetNumCols();

    if (m != n)
        LogicError("Diagonal can be called only for square matrix. (rows=%d, cols=%d)", m, n);

    if (GetFormat() != MatrixFormat::matrixFormatSparseCSC)
        NOT_IMPLEMENTED;

    GPUMatrix<ElemType> tmp(m, n, GetComputeDeviceId());

    // TODO: Implement optimized diagonal functions for sparse matrices. For now copy to dense first.
    CopyToDenseMatrix(tmp);

    return tmp.Diagonal();
}

template <class ElemType>
ElemType GPUSparseMatrix<ElemType>::SumOfAbsElements() const
{
    if (IsEmpty())
        return 0;

    hipblasHandle_t cuHandle = GPUMatrix<ElemType>::GetCublasHandle(GetComputeDeviceId());
    if (sizeof(ElemType) == sizeof(float))
    {
        float res = 0;
        hipblasSasum(cuHandle, (int) GetNumNZElements(), reinterpret_cast<const float*>(NzValues()), 1, &res);
        return res;
    }
    else
    {
        double res = 0;
        hipblasDasum(cuHandle, (int) GetNumNZElements(), reinterpret_cast<const double*>(NzValues()), 1, &res);
        return ElemType(res);
    }
}

template <class ElemType>
ElemType GPUSparseMatrix<ElemType>::SumOfElements() const
{
    if (IsEmpty())
        LogicError("SumOfElements: Matrix is empty");

    ElemType* d_sum = TracingGPUMemoryAllocator::Allocate<ElemType>(GetComputeDeviceId(), 1);
    ElemType h_sum;
    // WARNING: THIS kernel is not the most efficient way!
    _reductionSum1024Threads<ElemType><<<1, 1024>>>(NzValues(), d_sum, (LONG64) GetNumNZElements());
    CUDA_CALL(hipMemcpy(&h_sum, d_sum, sizeof(ElemType), hipMemcpyDeviceToHost));
    TracingGPUMemoryAllocator::Free<ElemType>(GetComputeDeviceId(), d_sum);

    return h_sum;
}

// sqrt(sum all elements^2)
template <class ElemType>
ElemType GPUSparseMatrix<ElemType>::FrobeniusNorm() const
{
    if (IsEmpty())
        return 0;

    ElemType* d_sum = TracingGPUMemoryAllocator::Allocate<ElemType>(GetComputeDeviceId(), 1);
    ElemType h_sum = 0;
    // WARNING: THIS kernel is not the most efficient way!
    _reductionSum21024Threads<ElemType><<<1, 1024>>>(NzValues(), d_sum, (int) GetNumNZElements());
    CUDA_CALL(hipMemcpy(&h_sum, d_sum, sizeof(ElemType), hipMemcpyDeviceToHost));
    TracingGPUMemoryAllocator::Free<ElemType>(GetComputeDeviceId(), d_sum);

    if (sizeof(ElemType) == sizeof(float))
        return (ElemType) sqrtf((float) h_sum);
    else
        return (ElemType) sqrt((double) h_sum);
}

template <class ElemType>
ElemType GPUSparseMatrix<ElemType>::MatrixNormInf() const
{
    if (IsEmpty())
        return 0;

    ElemType* d_maxAbs = TracingGPUMemoryAllocator::Allocate<ElemType>(GetComputeDeviceId(), 1);
    ElemType h_maxAbs = 0;
    // WARNING: THIS kernel is not the most efficient way!
    _reductionMatrixNormInf1024Threads<ElemType><<<1, 1024>>>(NzValues(), d_maxAbs, (int) GetNumNZElements());
    CUDA_CALL(hipMemcpy(&h_maxAbs, d_maxAbs, sizeof(ElemType), hipMemcpyDeviceToHost));
    TracingGPUMemoryAllocator::Free<ElemType>(GetComputeDeviceId(), d_maxAbs);

    if (sizeof(ElemType) == sizeof(float))
        return h_maxAbs;
    else
        return h_maxAbs;
}

template <class ElemType>
ElemType GPUSparseMatrix<ElemType>::MatrixNorm1() const
{
    return SumOfAbsElements();
}

#pragma endregion Member BLAS Functions

#pragma region Other Functions

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::ElementInverse()
{
#if 1
    // Note: This makes no sense because sparse matrices are defined by having lots of zeroes.
    NOT_IMPLEMENTED;
#else
    if (!OwnBuffer())
        LogicError("Cannot modify since the buffer is managed externally.");

    if (IsEmpty())
        LogicError("ElementInverse: Matrix is empty.");

    CUDA_LONG N = (CUDA_LONG) GetNumNZElements();
    int blocksPerGrid = (int) ceil(1.0 * N / GridDim::maxThreadsPerBlock);
    SyncGuard syncGuard;
    _elemInverse<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(NzValues(), N);
    return *this;
#endif
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignElementInverseOf(const GPUSparseMatrix<ElemType>& a)
{
#if 1
    // Note: This makes no sense because sparse matrices are defined by having lots of zeroes.
    UNUSED(a); NOT_IMPLEMENTED;
#else
    SetValue(a);
    return ElementInverse();
#endif
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::InplaceSigmoid()
{
#if 1
    // Note: This makes no sense because sigmoid(0) != 0.
    NOT_IMPLEMENTED;
#else
    performElementWiseFunction(ElementWiseOperator::opSigmoid, *this);
    return *this;
#endif
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignSigmoidOf(const GPUSparseMatrix<ElemType>& a)
{
#if 1
    // Note: This makes no sense because sigmoid(0) != 0.
    UNUSED(a); NOT_IMPLEMENTED;
#else
    if (this != &a)
        RequireSize(a.GetNumRows(), a.GetNumCols());
    performElementWiseFunction(ElementWiseOperator::opSigmoid, a);
    return *this;
#endif
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::InplaceLinearRectifierDerivative()
{
    performElementWiseFunction(ElementWiseOperator::opLinearRectifierDerivative, *this);
    return *this;
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignLinearRectifierDerivativeOf(const GPUSparseMatrix<ElemType>& a)
{
    if (this != &a)
        RequireSize(a.GetNumRows(), a.GetNumCols());
    performElementWiseFunction(ElementWiseOperator::opLinearRectifierDerivative, a);
    return *this;
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::InplaceTanh()
{
    performElementWiseFunction(ElementWiseOperator::opTanh, *this);
    return *this;
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignTanhOf(const GPUSparseMatrix<ElemType>& a)
{
    if (this != &a)
        RequireSize(a.GetNumRows(), a.GetNumCols());
    performElementWiseFunction(ElementWiseOperator::opTanh, a);
    return *this;
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::InplaceSqrt()
{
    performElementWiseFunction(ElementWiseOperator::opSqrt, *this);
    return *this;
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignSqrtOf(const GPUSparseMatrix<ElemType>& a)
{
    if (this != &a)
        RequireSize(a.GetNumRows(), a.GetNumCols());
    performElementWiseFunction(ElementWiseOperator::opSqrt, a);
    return *this;
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::InplaceExp()
{
#if 1
    // Note: This makes no sense because exp(0) != 0.
    NOT_IMPLEMENTED;
#else
    performElementWiseFunction(ElementWiseOperator::opExp, *this);
    return *this;
#endif
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignExpOf(const GPUSparseMatrix<ElemType>& a)
{
#if 1
    // Note: This makes no sense because exp(0) != 0.
    UNUSED(a); NOT_IMPLEMENTED;
#else
    if (this != &a)
        RequireSize(a.GetNumRows(), a.GetNumCols());
    performElementWiseFunction(ElementWiseOperator::opExp, a);
    return *this;
#endif
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::InplaceLog()
{
#if 1
    // Note: This makes no sense because log(0) != 0.
    NOT_IMPLEMENTED;
#else
    performElementWiseFunction(ElementWiseOperator::opLog, *this);
    return *this;
#endif
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignLogOf(const GPUSparseMatrix<ElemType>& a)
{
#if 1
    // Note: This makes no sense because log(0) != 0.
    UNUSED(a); NOT_IMPLEMENTED;
#else
    if (this != &a)
        RequireSize(a.GetNumRows(), a.GetNumCols());
    performElementWiseFunction(ElementWiseOperator::opLog, a);
    return *this;
#endif
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::InplaceAbs()
{
    performElementWiseFunction(ElementWiseOperator::opAbs, *this);
    return *this;
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignAbsOf(const GPUSparseMatrix<ElemType>& a)
{
    if (this != &a)
        RequireSizeAndAllocate(a.GetNumRows(), a.GetNumCols(), a.NzCount());
    performElementWiseFunction(ElementWiseOperator::opAbs, a);
    return *this;
}

// TODO: Check whether these functions always map 0 to 0.
template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::InplaceTruncateBottom(const ElemType threshold)
{
    VerifyWritable(__FUNCTION__);

    if (IsEmpty())
        LogicError("InplaceTruncateBottom: Matrix is empty.");
    CUDA_LONG N = (CUDA_LONG) GetNumNZElements();
    int blocksPerGrid = (int) ceil(N * 1.0 / GridDim::maxThreadsPerBlock);
    SyncGuard syncGuard;
    _assignTruncateBottom<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(NzValues(), NzValues(), threshold, N);
    return *this;
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignTruncateBottomOf(const GPUSparseMatrix<ElemType>& a, const ElemType threshold)
{
    VerifyWritable(__FUNCTION__);

    if (a.IsEmpty())
        LogicError("AssignTruncateBottomOf: Matrix a is empty.");

    if (this != &a)
    {
        // RequireSize(a.GetNumRows(), a.GetNumCols());
        ResizeAsAndCopyIndexFrom(a);
    }
    CUDA_LONG N = (CUDA_LONG) GetNumNZElements();
    int blocksPerGrid = (int) ceil(N * 1.0 / GridDim::maxThreadsPerBlock);
    SyncGuard syncGuard;
    _assignTruncateBottom<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(NzValues(), a.NzValues(), threshold, N);
    return *this;
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::InplaceTruncateTop(const ElemType threshold)
{
    VerifyWritable(__FUNCTION__);

    if (IsEmpty())
        LogicError("InplaceTruncateTop: Matrix is empty.");
    CUDA_LONG N = (CUDA_LONG) GetNumNZElements();
    int blocksPerGrid = (int) ceil(N * 1.0 / GridDim::maxThreadsPerBlock);
    SyncGuard syncGuard;
    _assignTruncateTop<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(NzValues(), NzValues(), threshold, N);
    return *this;
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignTruncateTopOf(const GPUSparseMatrix<ElemType>& a, const ElemType threshold)
{
    VerifyWritable(__FUNCTION__);

    if (a.IsEmpty())
        LogicError("AssignTruncateTopOf: Matrix a is empty.");

    if (this != &a)
    {
        ResizeAsAndCopyIndexFrom(a);
    }

    CUDA_LONG N = (CUDA_LONG) GetNumNZElements();
    int blocksPerGrid = (int) ceil(N * 1.0 / GridDim::maxThreadsPerBlock);
    SyncGuard syncGuard;
    _assignTruncateTop<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(NzValues(), a.NzValues(), threshold, N);
    return *this;
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::SetToZeroIfAbsLessThan(const ElemType threshold)
{
    VerifyWritable(__FUNCTION__);

    if (IsEmpty())
        LogicError("SetToZeroIfAbsLessThan: Matrix is empty.");
    CUDA_LONG N = (CUDA_LONG) GetNumNZElements();
    int blocksPerGrid = (int) ceil(N * 1.0 / GridDim::maxThreadsPerBlock);
    SyncGuard syncGuard;
    _setToZeroIfAbsLessThan<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(NzValues(), threshold, N);
    return *this;
}

#pragma endregion

#pragma region Helper Functions

//outBuffer should be allocated to be >= size by the caller
template <class ElemType>
template <class OutType, class InType>
/*private*/ void GPUSparseMatrix<ElemType>::ConvertBuffer(OutType* outBuffer, const InType* inBuffer, const size_t size)
{
#pragma omp parallel for
    for (size_t i = 0; i < (size & ~3); i += 4)
    {
        outBuffer[i] = inBuffer[i];
        outBuffer[i + 1] = inBuffer[i + 1];
        outBuffer[i + 2] = inBuffer[i + 2];
        outBuffer[i + 3] = inBuffer[i + 3];
    }
    // handle remaining stuffs
    for (size_t i = size & ~3; i < size; i++)
    {
        outBuffer[i] = inBuffer[i];
    }
}

template <class ElemType>
void* GPUSparseMatrix<ElemType>::ReserveTempHostBuffer(const size_t sizeInByte) const
{
    if (GetTempHostBufferSize() < sizeInByte)
    {
        delete[](byte*) GetTempHostBuffer();
        SetTempHostBuffer(new byte[sizeInByte]);
        SetTempHostBufferSize(sizeInByte);
    }
    return (void*) GetTempHostBuffer();
}

template <class ElemType>
void GPUSparseMatrix<ElemType>::performElementWiseFunction(ElementWiseOperator kind, const GPUSparseMatrix<ElemType>& src)
{
    VerifyWritable(__FUNCTION__);

    CUDA_LONG N = (CUDA_LONG) GetNumNZElements();
    int blocksPerGrid = (int) ceil(1.0 * N / GridDim::maxThreadsPerBlock);
    SyncGuard syncGuard;
    switch (kind)
    {
    case ElementWiseOperator::opSigmoid:
        return _elementWiseSigmoidOnCuda<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(src.NzValues(), NzValues(), N);
    case ElementWiseOperator::opTanh:
        return _elementWiseTanhOnCuda<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(src.NzValues(), NzValues(), N);
    case ElementWiseOperator::opSqrt:
        return _elementWiseSqrtOnCuda<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(src.NzValues(), NzValues(), N);
    case ElementWiseOperator::opExp:
        return _elementWiseExpOnCuda<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(src.NzValues(), NzValues(), N);
    case ElementWiseOperator::opLog:
        return _elementWiseLogOnCuda<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(src.NzValues(), NzValues(), N);
    case ElementWiseOperator::opAbs:
        return _elementWiseAbsOnCuda<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(src.NzValues(), NzValues(), N);
    case ElementWiseOperator::opLinearRectifierDerivative:
        return _elementWiseLinRectDerivativeOnCuda<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(src.NzValues(), NzValues(), N);
    default:
        NOT_IMPLEMENTED;
    }
}

#pragma endregion Helper Functions

template class MATH_API GPUSparseMatrix<float>;
template class MATH_API GPUSparseMatrix<double>;

// We use Matrix<char> as the backing store for QuantizedMatrix
// Let's explicitly instantiate the methods we need for that purpose
template GPUSparseMatrix<char>::GPUSparseMatrix(DEVICEID_TYPE, const MatrixFormat);
template GPUSparseMatrix<char>::GPUSparseMatrix(const size_t, const size_t, const size_t, DEVICEID_TYPE, const MatrixFormat);
template GPUSparseMatrix<char>::GPUSparseMatrix(GPUSparseMatrix<char> const&);
template GPUSparseMatrix<char>::GPUSparseMatrix(GPUSparseMatrix<char>&&);
template void GPUSparseMatrix<char>::SetValue(CPUSparseMatrix<char> const&);
template void GPUSparseMatrix<char>::SetValue(GPUSparseMatrix<char> const&);
template void GPUSparseMatrix<char>::SetValue(GPUMatrix<char> const&);
//template void GPUSparseMatrix<char>::SetValue(CPUMatrix<char> const&);
template GPUMatrix<char> GPUSparseMatrix<char>::CopyToDenseMatrix() const;
template void GPUSparseMatrix<char>::CopyToDenseMatrix(GPUMatrix<char>&) const;
template void GPUSparseMatrix<char>::CopyToCPUSparseMatrix(CPUSparseMatrix<char>&) const;
template void GPUSparseMatrix<char>::ChangeDeviceTo(int);
template void GPUSparseMatrix<char>::Resize(const size_t, const size_t, const size_t, const bool);
template void GPUSparseMatrix<char>::RequireSizeAndAllocate(const size_t, const size_t, const size_t, const bool, const bool);
template void GPUSparseMatrix<char>::Reset();
template GPUSPARSE_INDEX_TYPE GPUSparseMatrix<char>::SecondaryIndexValueAt(size_t) const;
template GPUSparseMatrix<char>::~GPUSparseMatrix();
template GPUSparseMatrix<char> GPUSparseMatrix<char>::ColumnSlice(size_t, size_t) const;
template GPUMatrix<char> GPUSparseMatrix<char>::CopyColumnSliceToDense(size_t, size_t) const;
template GPUSparseMatrix<char>& GPUSparseMatrix<char>::operator=(GPUSparseMatrix<char>&&);
template void GPUSparseMatrix<char>::Reshape(const size_t, const size_t);
template void GPUSparseMatrix<char>::ScaleAndAdd(char, GPUSparseMatrix<char> const &, GPUMatrix<char> &);

// Support <short>
template GPUSparseMatrix<short>::GPUSparseMatrix(DEVICEID_TYPE, const MatrixFormat);
template GPUSparseMatrix<short>::GPUSparseMatrix(const size_t, const size_t, const size_t, DEVICEID_TYPE, const MatrixFormat);
template GPUSparseMatrix<short>::GPUSparseMatrix(GPUSparseMatrix<short> const&);
template GPUSparseMatrix<short>::GPUSparseMatrix(GPUSparseMatrix<short>&&);
template void GPUSparseMatrix<short>::SetValue(CPUSparseMatrix<short> const&);
template void GPUSparseMatrix<short>::SetValue(GPUSparseMatrix<short> const&);
template void GPUSparseMatrix<short>::SetValue(GPUMatrix<short> const&);
//template void GPUSparseMatrix<short>::SetValue(CPUMatrix<short> const&);
template GPUMatrix<short> GPUSparseMatrix<short>::CopyToDenseMatrix() const;
template void GPUSparseMatrix<short>::CopyToDenseMatrix(GPUMatrix<short>&) const;
template void GPUSparseMatrix<short>::CopyToCPUSparseMatrix(CPUSparseMatrix<short>&) const;
template void GPUSparseMatrix<short>::ChangeDeviceTo(int);
template void GPUSparseMatrix<short>::Resize(const size_t, const size_t, const size_t, const bool);
template void GPUSparseMatrix<short>::RequireSizeAndAllocate(const size_t, const size_t, const size_t, const bool, const bool);
template void GPUSparseMatrix<short>::Reset();
template GPUSPARSE_INDEX_TYPE GPUSparseMatrix<short>::SecondaryIndexValueAt(size_t) const;
template GPUSparseMatrix<short>::~GPUSparseMatrix();
template GPUSparseMatrix<short> GPUSparseMatrix<short>::ColumnSlice(size_t, size_t) const;
template GPUMatrix<short> GPUSparseMatrix<short>::CopyColumnSliceToDense(size_t, size_t) const;
template GPUSparseMatrix<short>& GPUSparseMatrix<short>::operator=(GPUSparseMatrix<short>&&);
template void GPUSparseMatrix<short>::Reshape(const size_t, const size_t);
template void GPUSparseMatrix<short>::ScaleAndAdd(short, GPUSparseMatrix<short> const &, GPUMatrix<short> &);

template GPUSparseMatrix<int>::GPUSparseMatrix(DEVICEID_TYPE, const MatrixFormat);
template GPUSparseMatrix<int>::~GPUSparseMatrix();
template void GPUSparseMatrix<int>::RequireSizeAndAllocate(const size_t, const size_t, const size_t, const bool, const bool);

template <class ElemType>
MATH_API File& operator>>(File& stream, GPUSparseMatrix<ElemType>& us)
{
    us.VerifyWritable(__FUNCTION__);

    stream.GetMarker(fileMarkerBeginSection, std::wstring(L"BMAT"));
    size_t elsize;
    stream >> elsize;
    if (sizeof(ElemType) != elsize)
        RuntimeError("Template argument size doesn't match those in file");
    std::wstring matrixName;

    // now prepare this header to receive the data being read
    size_t nz, colnum, rownum;
    int format;

    // read in the header information
    stream >> matrixName >> format >> nz >> colnum >> rownum;

    us.SetFormat((MatrixFormat) format);
    if (us.GetFormat() != matrixFormatSparseCSC && us.GetFormat() != matrixFormatSparseCSR)
        NOT_IMPLEMENTED;

    us.RequireSizeAndAllocate(rownum, colnum, nz, true, false);

    if (nz > 0)
    {
        size_t compressedSize = (us.GetFormat() == matrixFormatSparseCSC) ? colnum + 1 : rownum + 1;
        ElemType* dataBuffer = new ElemType[nz];
        CPUSPARSE_INDEX_TYPE* unCompressedIndex = new CPUSPARSE_INDEX_TYPE[nz];
        CPUSPARSE_INDEX_TYPE* compressedIndex = new CPUSPARSE_INDEX_TYPE[compressedSize];

        // read in the sparse matrix info
        for (size_t i = 0; i < nz; ++i)
        {
            stream >> dataBuffer[i];
        }
        for (size_t i = 0; i < nz; ++i)
        {
            size_t val;
            stream >> val;
            unCompressedIndex[i] = val;
        }
        for (size_t i = 0; i < compressedSize; ++i)
        {
            size_t val;
            stream >> val;
            compressedIndex[i] = val;
        }

        if (us.GetFormat() == matrixFormatSparseCSC)
            us.SetMatrixFromCSCFormat(compressedIndex, unCompressedIndex, dataBuffer, nz, rownum, colnum);
        else if (us.GetFormat() == matrixFormatSparseCSR)
            us.SetMatrixFromCSRFormat(compressedIndex, unCompressedIndex, dataBuffer, nz, rownum, colnum);

        delete[] dataBuffer;
        delete[] unCompressedIndex;
        delete[] compressedIndex;
    }

    stream.GetMarker(fileMarkerEndSection, std::wstring(L"EMAT"));

    return stream;
}

template MATH_API File& operator>>(File& stream, GPUSparseMatrix<float>& us);
template MATH_API File& operator>>(File& stream, GPUSparseMatrix<double>& us);

template <class ElemType>
MATH_API File& operator<<(File& stream, const GPUSparseMatrix<ElemType>& us)
{
    if (us.GetFormat() != matrixFormatSparseCSC && us.GetFormat() != matrixFormatSparseCSR)
        NOT_IMPLEMENTED;

    stream.PutMarker(fileMarkerBeginSection, std::wstring(L"BMAT"));
    stream << sizeof(ElemType);
    std::wstring s(L"nnmatrix");
    stream << s;

    size_t nz = us.GetNumNZElements(), numElemAllocated = us.GetNumElemAllocated(), numRows = us.GetNumRows(), numCols = us.GetNumCols();
    size_t compressedSize = us.SecondaryIndexCount();
    int format = us.GetFormat();

    stream << format << nz << numCols << numRows;

    if (nz > 0)
    {
        ElemType* dataBuffer = nullptr;
        CPUSPARSE_INDEX_TYPE* compressedIndex = nullptr;
        CPUSPARSE_INDEX_TYPE* unCompressedIndex = nullptr;

        if (us.GetFormat() == matrixFormatSparseCSC)
            us.GetMatrixFromCSCFormat(compressedIndex, unCompressedIndex, dataBuffer, numElemAllocated, nz, numRows, numCols);
        else if (us.GetFormat() == matrixFormatSparseCSR)
            us.GetMatrixFromCSRFormat(compressedIndex, unCompressedIndex, dataBuffer, numElemAllocated, nz, numRows, numCols);
        else
            NOT_IMPLEMENTED;

        for (size_t i = 0; i < nz; ++i)
        {
            stream << dataBuffer[i];
        }
        for (size_t i = 0; i < nz; ++i)
        {
            size_t val = unCompressedIndex[i];
            stream << val;
        }
        for (size_t i = 0; i < compressedSize; ++i)
        {
            size_t val = compressedIndex[i];
            stream << val;
        }

        delete[] dataBuffer;
        delete[] unCompressedIndex;
        delete[] compressedIndex;
    }

    stream.PutMarker(fileMarkerEndSection, std::wstring(L"EMAT"));

    return stream;
}

template MATH_API File& operator<<(File& stream, const GPUSparseMatrix<float>& us);
template MATH_API File& operator<<(File& stream, const GPUSparseMatrix<double>& us);

}}}

#endif // CPUONLY
