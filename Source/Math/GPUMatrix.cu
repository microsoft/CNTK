#include "hip/hip_runtime.h"
//
// Copyright (c) Microsoft. All rights reserved.
// Licensed under the MIT license. See LICENSE.md file in the project root for full license information.
//

#include "stdafx.h"
#include "Basics.h"
#include "BestGpu.h"

#ifndef CPUONLY

#include "GPUMatrix.h"
#include "GPUMatrixCUDAKernels.cuh"
//#include "GPUSparseMatrix.h"
#include "GPUTensor.h"
#include "CommonMatrix.h"
#define TENSOR_OPS_DECL __device__ __host__
#include "TensorOps.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "hipblas.h"
#include <assert.h>
#include <memory>
#include "CntkBatchNormalization.cuh"
#include "Convolution.cuh"

#pragma comment(lib, "cudart.lib") // instruct linker to reference these libs
#pragma comment(lib, "cublas.lib")
#pragma comment(lib, "cusparse.lib")
#pragma comment(lib, "hiprand.lib")

#pragma warning(disable : 4267) // conversion from 'size_t' to 'unsigned int'; happens in CUDA <<<a,b>>> syntax if a and b are size_t
#pragma warning(disable : 4127) // conditional expression is constant; "if (sizeof(ElemType)==sizeof(float))" triggers this
#pragma warning(disable : 4702) // unreachable code; triggered for unknown reasons

#define DEFAULT_THREAD_PER_DIM 16

#define UNCONST(t, c, uc) GPUMatrix<t>& uc = const_cast<GPUMatrix<t>&>(c);

#ifdef _WIN32
// thread local storage to access the current stream, initalize to default stream
__declspec(thread)
#endif
    hipStream_t t_stream = hipStreamDefault;

#define DEFAULT_THREAD_PER_DIM 16

extern int _ConvertSMVer2Cores(int major, int minor); // forward declaration

// SetStream - set the stream that will be used by the GPU routines
void MATH_API SetStream(hipStream_t stream)
{
    t_stream = stream;
}

// GetStream - get the stream that will be used by the GPU routines
hipStream_t MATH_API GetStream()
{
    return t_stream;
}

// Helper macro patterns for elemtwise methods
#define DEF_ELEMWISE_INPLACE_FUNC(f)                                      \
    template <class ElemType>                                             \
    GPUMatrix<ElemType>& GPUMatrix<ElemType>::Inplace##f()                \
    {                                                                     \
        performElementWiseFunction(ElementWiseOperator::op##f, Data()); \
        return *this;                                                     \
    }
#define DEF_ELEMWISE_ASSIGN_FUNC(f)                                                       \
    template <class ElemType>                                                             \
    GPUMatrix<ElemType>& GPUMatrix<ElemType>::Assign##f##Of(const GPUMatrix<ElemType>& a) \
    {                                                                                     \
        if (a.IsEmpty())                                                                  \
            LogicError("Assign##f##Of: Matrix a is empty.");                              \
        if (this != &a)                                                                   \
            RequireSize(a.GetNumRows(), a.GetNumCols());                                       \
        performElementWiseFunction(ElementWiseOperator::op##f, a.Data());               \
        return *this;                                                                     \
    }

template <>
const char* CudaErrString<hipError_t>(hipError_t x)
{
    hipDeviceSynchronize();
    return hipGetErrorString(x);
}
template <>
const char* CudaErrString<hipblasStatus_t>(hipblasStatus_t e)
{
    hipDeviceSynchronize();
    switch (e)
    {
    case HIPBLAS_STATUS_SUCCESS:          return "HIPBLAS_STATUS_SUCCESS";
    case HIPBLAS_STATUS_NOT_INITIALIZED:  return "HIPBLAS_STATUS_NOT_INITIALIZED";
    case HIPBLAS_STATUS_ALLOC_FAILED:     return "HIPBLAS_STATUS_ALLOC_FAILED";
    case HIPBLAS_STATUS_INVALID_VALUE:    return "HIPBLAS_STATUS_INVALID_VALUE";
    case HIPBLAS_STATUS_ARCH_MISMATCH:    return "HIPBLAS_STATUS_ARCH_MISMATCH";
    case HIPBLAS_STATUS_MAPPING_ERROR:    return "HIPBLAS_STATUS_MAPPING_ERROR";
    case HIPBLAS_STATUS_EXECUTION_FAILED: return "HIPBLAS_STATUS_EXECUTION_FAILED";
    case HIPBLAS_STATUS_INTERNAL_ERROR:   return "HIPBLAS_STATUS_INTERNAL_ERROR";
    case HIPBLAS_STATUS_NOT_SUPPORTED:    return "HIPBLAS_STATUS_NOT_SUPPORTED";
    case HIPBLAS_STATUS_UNKNOWN:    return "HIPBLAS_STATUS_UNKNOWN";
    default:                             return "(look for CUBLAS_STATUS_xxx in hipblas.h)";
    }
}
template <>
const char* CudaErrString<hiprandStatus>(hiprandStatus)
{
    hipDeviceSynchronize();
    return "(see hiprand.h & look for hiprandStatus or CURAND_STATUS_xxx)";
}

namespace Microsoft { namespace MSR { namespace CNTK {

template <typename AllocatedElemType>
AllocatedElemType* TracingGPUMemoryAllocator::Allocate(int deviceId, size_t numRows, size_t numCols)
{
    if (IsTraceEnabled())
    {
        auto freeAndTotalMemory = GetFreeAndTotalMemoryInMBs(deviceId);
        fprintf(stderr, "Allocating Matrix<%s> (Rows = %d, Cols = %d) buffer on DeviceId = %d; GPU Memory Free = %d MB of %d MB\n", typeid(AllocatedElemType).name(), (int)numRows, (int)numCols, (int)deviceId, (int)freeAndTotalMemory.first, (int)freeAndTotalMemory.second);
        Microsoft::MSR::CNTK::DebugUtil::PrintCallStack();
    }

    AllocatedElemType* deviceBufferPtr = AllocateNoTrace<AllocatedElemType>(deviceId, numRows * numCols);

    if (IsTraceEnabled())
    {
        fprintf(stderr, "Allocated DeviceData = %p\n", (void*) deviceBufferPtr);
    }

    return deviceBufferPtr;
}

template <typename AllocatedElemType>
AllocatedElemType* TracingGPUMemoryAllocator::Allocate(int deviceId, size_t numElements)
{
    if (IsTraceEnabled())
    {
        auto freeAndTotalMemory = GetFreeAndTotalMemoryInMBs(deviceId);
        fprintf(stderr, "Allocating array<%s> (NumElements = %d) on DeviceId = %d; GPU Memory Free = %d MB of %d MB\n", typeid(AllocatedElemType).name(), (int)numElements, (int)deviceId, (int)freeAndTotalMemory.first, (int)freeAndTotalMemory.second);
        Microsoft::MSR::CNTK::DebugUtil::PrintCallStack();
    }

    AllocatedElemType* deviceBufferPtr = AllocateNoTrace<AllocatedElemType>(deviceId, numElements);
    
    if (IsTraceEnabled())
    {
        fprintf(stderr, "Allocated DeviceData = %p\n", (void*)deviceBufferPtr);
    }

    return deviceBufferPtr;
}

template <typename AllocatedElemType>
void TracingGPUMemoryAllocator::Free(int deviceId, AllocatedElemType* bufferPtr, bool ignoreCUDARetCode /*= false*/)
{
    PrepareDevice(deviceId);
    if (ignoreCUDARetCode)
        hipFree((void*) bufferPtr);
    else
        CUDA_CALL(hipFree((void*) bufferPtr));

    if (IsTraceEnabled())
    {
        auto freeAndTotalMemory = GetFreeAndTotalMemoryInMBs(deviceId);
        fprintf(stderr, "Freed buffer<%s> DeviceData = %p on DeviceId = %d; GPU Memory Free = %d MB of %d MB\n", typeid(AllocatedElemType).name(), (void*) bufferPtr, (int) deviceId, (int) freeAndTotalMemory.first, (int) freeAndTotalMemory.second);
        Microsoft::MSR::CNTK::DebugUtil::PrintCallStack();
    }
}

template <typename AllocatedElemType>
AllocatedElemType* TracingGPUMemoryAllocator::AllocateNoTrace(int deviceId, size_t numElements)
{
    AllocatedElemType* deviceBufferPtr;

    PrepareDevice(deviceId);
    CUDA_CALL(hipMalloc((void**) &deviceBufferPtr, sizeof(AllocatedElemType) * numElements));

    return deviceBufferPtr;
}

std::pair<size_t, size_t> TracingGPUMemoryAllocator::GetFreeAndTotalMemoryInMBs(int deviceId)
{
    PrepareDevice(deviceId);

    size_t free, total;
    CUDA_CALL(hipMemGetInfo(&free, &total));

    size_t numBytesPerMB = 1 << 20;
    return {free / numBytesPerMB, total / numBytesPerMB};
}

// PrepareDevice - Setup the correct cuda context for an operation
// deviceId - the device on which the operation will take place
void PrepareDevice(DEVICEID_TYPE deviceId)
{
    static DEVICEID_TYPE currentDevice = DEVICEID_NOTYETDETERMINED;
    // and if we last set the device to be this device we are good
    if (deviceId == currentDevice)
        return;
    CUDA_CALL(hipSetDevice(deviceId));
    currentDevice = deviceId;
}

#pragma region DeviceBoundNumber class

template <class ElemType>
DeviceBoundNumber<ElemType>::DeviceBoundNumber(const DeviceBoundNumber<ElemType>& /*deepCopy*/)
{
    NOT_IMPLEMENTED;
}

template <class ElemType>
DeviceBoundNumber<ElemType>::DeviceBoundNumber(DeviceBoundNumber<ElemType>&& shallowCopy)
{
    ShallowCopyFrom(shallowCopy.m_data, shallowCopy.m_computeDevice);
    shallowCopy.m_data = NULL;
}

template <class ElemType>
void DeviceBoundNumber<ElemType>::ShallowCopyFrom(ElemType* newVal, int newValsDevceId)
{
    m_computeDevice = newValsDevceId;
    m_data = newVal;
}

template <class ElemType>
DeviceBoundNumber<ElemType>::~DeviceBoundNumber()
{
    if (m_data != NULL)
    {
        if (m_computeDevice < 0)
        {
            delete m_data;
            m_data = NULL;
        }
        else
        {
            TracingGPUMemoryAllocator::Free<ElemType>(m_computeDevice, m_data);
        }
    }
}

#pragma endregion DeviceBoundNumber class

#pragma region Helper functions
template <class ElemType>
hipblasHandle_t _initCUBLAS(int devId)
{
    PrepareDevice((DEVICEID_TYPE) devId);
    hipblasHandle_t cuHandle;
    CUBLAS_CALL(hipblasCreate(&cuHandle));
    return cuHandle;
}

template <class ElemType>
void GPUMatrix<ElemType>::SetDevice(DEVICEID_TYPE deviceId)
{
    assert(deviceId >= 0);
    CUDA_CALL(hipSetDevice(deviceId));
}

// PrepareDevice - Setup the correct cuda context for an operation
// deviceId - the device on which the operation will take place
//            defaults to -1, which means use matrices current device
template <class ElemType>
DEVICEID_TYPE GPUMatrix<ElemType>::PrepareDevice(DEVICEID_TYPE deviceId /*=-1*/) const
{
    // if default value use current compute device
    DEVICEID_TYPE newId = deviceId >= 0 ? deviceId : GetComputeDeviceId();

    Microsoft::MSR::CNTK::PrepareDevice(newId);
    return newId;
}

template <class ElemType>
ElemType* GPUMatrix<ElemType>::CopyToArray() const
{
    size_t numElements = GetNumElements();
    if (numElements != 0)
    {
        PrepareDevice();
        ElemType* pArray = new ElemType[numElements];
        CUDA_CALL(hipMemcpy(pArray, Data(), sizeof(ElemType) * m_numRows * m_numCols, hipMemcpyDeviceToHost));
        return pArray;
    }
    else
    {
        return NULL;
    }
}

//memory will be allocated by the callee if not enough but need to be deleted by the caller after it's done
//return number of elements copied
template <class ElemType>
size_t GPUMatrix<ElemType>::CopyToArray(ElemType*& arrayCopyTo, size_t& currentArraySize) const
{
    size_t numElements = GetNumElements();

    if (numElements > currentArraySize)
    {
        delete arrayCopyTo;
        arrayCopyTo = new ElemType[numElements];
        currentArraySize = numElements;
    }

    if (numElements != 0)
    {
        PrepareDevice();
        CUDA_CALL(hipMemcpy(arrayCopyTo, Data(), sizeof(ElemType) * numElements, hipMemcpyDeviceToHost));
    }

    return numElements;
}

template <typename ElemType>
void GPUMatrix<ElemType>::CopySection(size_t numRows, size_t numCols, ElemType* dst, size_t colStride) const
{
    CUBLAS_CALL(hipblasGetMatrix((int) numRows, (int) numCols, sizeof(ElemType),
                                Data(), (int) GetNumRows(), dst, (int) colStride));
}
template <class ElemType>
void GPUMatrix<ElemType>::ChangeDeviceTo(DEVICEID_TYPE to_id)
{
    if (to_id == CPUDEVICE)
        LogicError("to_id must be valid GPU");
    if (GetComputeDeviceId() == to_id)
        return;

    ElemType* d_dst = TracingGPUMemoryAllocator::Allocate<ElemType>(to_id, m_numRows, m_numCols);

    SetSizeAllocated(m_numRows * m_numCols);

    // check to make sure we have something to copy (on init we often have zero sized allocations)
    if (GetSizeAllocated() > 0)
    {
        // first try peer access
        int canAccessPeer = false;
        CUDA_CALL(hipDeviceCanAccessPeer(&canAccessPeer, to_id, GetComputeDeviceId()));
        if (canAccessPeer)
        {
            hipError_t cudaStatus = hipDeviceEnablePeerAccess(GetComputeDeviceId(), 0);
            if (cudaStatus != hipErrorPeerAccessAlreadyEnabled)
            {
                CUDA_CALL(cudaStatus);
            }
            CUDA_CALL(hipMemcpyPeer(d_dst, to_id, Data(), GetComputeDeviceId(), sizeof(ElemType) * m_numRows * m_numCols));
        }
        else
        {
            // peer access didn't work, just copy normal
            // make this more efficient by keeping some buffers available for each copy
            ElemType* h_dst = NULL;
            PrepareDevice();
            CUDA_CALL(hipHostMalloc((void**) &h_dst, sizeof(ElemType) * m_numRows * m_numCols));
            CUDA_CALL(hipMemcpy(h_dst, Data(), sizeof(ElemType) * m_numRows * m_numCols, hipMemcpyDeviceToHost));
            PrepareDevice((DEVICEID_TYPE) to_id);
            CUDA_CALL(hipMemcpy(d_dst, h_dst, sizeof(ElemType) * m_numRows * m_numCols, hipMemcpyHostToDevice));
            CUDA_CALL(hipHostFree(h_dst));
        }
    }

    TracingGPUMemoryAllocator::Free<ElemType>(GetComputeDeviceId(), Buffer());
    SetBuffer(d_dst, m_numRows * m_numCols * sizeof(ElemType));

    PrepareDevice((DEVICEID_TYPE) to_id);
    SetComputeDeviceId(to_id);
}

template <class ElemType>
void GPUMatrix<ElemType>::performElementWiseFunction(ElementWiseOperator kind, const ElemType* src)
{
    PrepareDevice();
    CUDA_LONG N = (CUDA_LONG) GetNumElements();
    int blocksPerGrid = (int) ceil(1.0 * N / GridDim::maxThreadsPerBlock);
    SyncGuard syncGuard;
    switch (kind)
    {
    case ElementWiseOperator::opSigmoid:
        return _elementWiseSigmoidOnCuda<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(src, Data(), N);
    case ElementWiseOperator::opTanh:
        return _elementWiseTanhOnCuda<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(src, Data(), N);
    case ElementWiseOperator::opSqrt:
        return _elementWiseSqrtOnCuda<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(src, Data(), N);
    case ElementWiseOperator::opExp:
        return _elementWiseExpOnCuda<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(src, Data(), N);
    case ElementWiseOperator::opLog:
        return _elementWiseLogOnCuda<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(src, Data(), N);
    case ElementWiseOperator::opAbs:
        return _elementWiseAbsOnCuda<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(src, Data(), N);
    case ElementWiseOperator::opLinearRectifierDerivative:
        return _elementWiseLinRectDerivativeOnCuda<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(src, Data(), N);
    case ElementWiseOperator::opCosine:
        return _elementWiseCosineOnCuda<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(src, Data(), N);
    case ElementWiseOperator::opNegativeSine:
        return _elementWiseNegativeSineOnCuda<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(src, Data(), N);
    case ElementWiseOperator::opSigmoidDerivative:
        return _elementWiseSigmoidDerivativeOnCuda<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(src, Data(), N);
    default: LogicError("performElementWiseFunction: unexpected op code %d", (int)kind);
    }
}

#pragma endregion Helper functions

#pragma region Constructors and Destructor

// should only be used by constructors
template <class ElemType>
void GPUMatrix<ElemType>::ZeroInit(int deviceId)
{
    BaseMatrix<ElemType>::ZeroInit();
    SetComputeDeviceId(deviceId);
}

template <class ElemType>
GPUMatrix<ElemType>::GPUMatrix(int deviceId)
{
    ZeroInit(deviceId);
};

template <class ElemType>
GPUMatrix<ElemType>::GPUMatrix(const size_t numRows, const size_t numCols, int deviceId)
{
    ZeroInit(deviceId);
    m_numRows = numRows;
    m_numCols = numCols;
    SetSizeAllocated(GetNumElements());

    if (GetNumElements() != 0)
    {
        SetBuffer(TracingGPUMemoryAllocator::Allocate<ElemType>(GetComputeDeviceId(), m_numRows, m_numCols), GetNumElements() * sizeof(ElemType));
        CUDA_CALL(hipMemset(Buffer(), 0, sizeof(ElemType) * GetSizeAllocated()));
    }
};

template <class ElemType>
GPUMatrix<ElemType>::GPUMatrix(const size_t numRows, const size_t numCols, int deviceId, ElemType* pArray, const size_t matrixFlags)
{
    ZeroInit(deviceId);
    SetValue(numRows, numCols, deviceId, pArray, matrixFlags);
};

template <class ElemType>
GPUMatrix<ElemType>::GPUMatrix(const GPUMatrix<ElemType>& deepCopyFrom)
{
    ZeroInit();
    SetValue(deepCopyFrom);
}

template <class ElemType>
GPUMatrix<ElemType>::GPUMatrix(GPUMatrix<ElemType>&& moveFrom)
{
    ShallowCopyFrom(moveFrom);
    moveFrom.ZeroValues();
}

//assignment operator, deep copy
template <class ElemType>
GPUMatrix<ElemType>& GPUMatrix<ElemType>::operator=(const GPUMatrix<ElemType>& deepCopyFrom)
{
    if (this != &deepCopyFrom)
    {
        SetValue(deepCopyFrom);
    }
    return *this;
}

//move assignment operator, shallow copy
template <class ElemType>
GPUMatrix<ElemType>& GPUMatrix<ElemType>::operator=(GPUMatrix<ElemType>&& moveFrom)
{
    if (this != &moveFrom)
    {
        ShallowCopyFrom(moveFrom);
        moveFrom.ZeroValues();
    }
    return *this;
}

template <class ElemType>
GPUMatrix<ElemType>::~GPUMatrix(void)
{
}

// TODO: This should be in the storage object.
// Clear will clear your storage, zeroinit just drops it on the ground.
template <class ElemType>
void GPUMatrix<ElemType>::Clear()
{
    VerifyWritable(__func__);
    //if (OwnBuffer() && m_pArray != NULL)
    if (m_sob != nullptr)
    {
        if (GetComputeDeviceId()>= 0)
        {
            // BUG: We do not check the CUDA return code for hipFree here since this may get called
            // during processExit when hipFree will fail. The destruction of CUDA objects during
            // process exit must be avoided
            ReleaseStorageMemory();
        }
    }

    ZeroInit(GetComputeDeviceId());
}
#pragma endregion Constructors and Destructor

template <class ElemType>
std::unique_ptr<GPUMatrix<ElemType>> GPUMatrix<ElemType>::GetOrCreateWorkspace() const
{
    // REVIEW alexeyk: not thread-safe, fine for now.
    if (m_workspace == nullptr)
        m_workspace = std::make_unique<conc_stack<std::unique_ptr<GPUMatrix<ElemType>>>>();
    assert(m_workspace != nullptr);
    auto deviceId = GetComputeDeviceId();
    return m_workspace->pop_or_create([deviceId]()
                                      {
                                          return std::make_unique<GPUMatrix<ElemType>>(deviceId);
                                      });
}

template <class ElemType>
void GPUMatrix<ElemType>::ReleaseWorkspace(std::unique_ptr<GPUMatrix<ElemType>> src) const
{
    assert(m_workspace != nullptr);
    m_workspace->push(std::move(src));
}

#pragma region Basic Operators
template <class ElemType>
GPUMatrix<ElemType> GPUMatrix<ElemType>::ColumnSlice(size_t startColumn, size_t numCols) const
{
    if (startColumn + numCols > GetNumCols())
        InvalidArgument("The slice (%d+%d) is out of range of the source matrix (%d).", (int) startColumn, (int) numCols, (int) GetNumCols());

    GPUMatrix<ElemType> slice(GetComputeDeviceId());

    slice.ShallowCopyFrom(*this);
    slice.m_numCols = numCols;
    slice.m_sliceViewOffset = m_sliceViewOffset + startColumn * GetNumRows();

    return slice;
}

template <class ElemType>
GPUMatrix<ElemType>& GPUMatrix<ElemType>::AssignColumnSlice(const GPUMatrix<ElemType>& fromMatrix, size_t startColumn, size_t numCols)
{
    if (numCols == 0)
        LogicError("The slice cannot have 0 columns.");

    if (startColumn + numCols > fromMatrix.GetNumCols())
        InvalidArgument("The slice (%d+%d) is out of range of the source matrix (%d).", (int) startColumn, (int) numCols, (int) fromMatrix.GetNumCols());

    Clear();

    ShallowCopyFrom(fromMatrix);
    m_numCols = numCols;
    m_sliceViewOffset = fromMatrix.m_sliceViewOffset + startColumn * GetNumRows();

    return *this;
}

template <class ElemType>
GPUMatrix<ElemType>& GPUMatrix<ElemType>::SetColumnSlice(const GPUMatrix<ElemType>& fromMatrix, size_t startColumn, size_t numCols)
{
    if (startColumn + numCols > GetNumCols())
        LogicError("The slice is out of range of the destination matrix.");
    if (numCols > fromMatrix.GetNumCols())
        InvalidArgument("The slice (%d) is out of range of the source matrix (%d).", (int) numCols, (int) fromMatrix.GetNumCols());
    if (m_numRows != fromMatrix.m_numRows)
        LogicError("The number of rows in source and destination matrices do not match");

    if (m_numRows * numCols > 0) // TODO: remove if unnecessary
        CUDA_CALL(hipMemcpy(Data() + m_sliceViewOffset + LocateColumn(startColumn), fromMatrix.Data(), sizeof(ElemType) * m_numRows * numCols, hipMemcpyDeviceToDevice));
    return *this;
}

template <class ElemType>
void GPUMatrix<ElemType>::CopyColumnsStrided(const GPUMatrix<ElemType>& fromMatrix, size_t numCols, size_t srcNumColsStride, size_t destNumColsStride)
{
    if ((((numCols - 1) * srcNumColsStride) + 1) > fromMatrix.m_numCols)
        LogicError("The numCols to copy and srcNumColsStride specified is out of range of the source matrix.");
    if ((((numCols - 1) * destNumColsStride) + 1) > m_numCols)
        LogicError("The numCols to copy and srcNumColsStride specified is out of range of the destination matrix.");
    if (m_numRows != fromMatrix.m_numRows)
        LogicError("The number of rows in source and destination matrices do not match");

    if ((m_numRows * numCols) > 0)
    {
        // Launch a kernel to do the strided copy
        CUDA_LONG N = (CUDA_LONG)(m_numRows * numCols);
        int blocksPerGrid = (int) ceil(1.0 * N / GridDim::maxThreadsPerBlock);
        PrepareDevice();
        SyncGuard syncGuard;
        _copyColumnsStrided<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(Data(), fromMatrix.Data(), N, (CUDA_LONG) m_numRows, (CUDA_LONG) destNumColsStride, (CUDA_LONG) srcNumColsStride);
    }
}

//for each column of a, we assign all rows of a to this starting from startIndex
template <class ElemType>
GPUMatrix<ElemType>& GPUMatrix<ElemType>::AssignToRowSliceValuesOf(const GPUMatrix<ElemType>& a, const size_t startIndex, const size_t numRows)
{
    if (a.IsEmpty())
        LogicError("AddToRowSliceValuesOf: input matrix a is empty.");

    if (a.GetNumRows() != numRows)
        LogicError("AddToRowSliceValuesOf: a.GetNumRows() != numRows.");

    if (startIndex + numRows > GetNumRows())
        LogicError("AddToRowSliceValuesOf: startIndex + numRows exceeds GetNumRows().");

    if (a.GetNumCols() != GetNumCols())
        LogicError("AddToRowSliceValuesOf: columns does not match.");

    CUDA_LONG N = (CUDA_LONG) a.GetNumElements();
    int blocksPerGrid = (int) ceil(1.0 * N / GridDim::maxThreadsPerBlock);
    PrepareDevice();
    SyncGuard syncGuard;
    _assignToRowSliceValuesOf<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(Data(), a.Data(), N, (CUDA_LONG) startIndex, (CUDA_LONG) GetNumRows(), (CUDA_LONG) a.GetNumRows());
    return *this;
}

//for each column of a, we assign numRows starting from startIndex to this
template <class ElemType>
GPUMatrix<ElemType>& GPUMatrix<ElemType>::AssignRowSliceValuesOf(const GPUMatrix<ElemType>& a, const size_t startIndex, const size_t numRows)
{
    if (a.IsEmpty())
        LogicError("AssignRowSliceValuesOf: input matrix a is empty.");

    if (startIndex + numRows > a.GetNumRows())
        LogicError("AssignRowSliceValuesOf: startIndex + numRows exceeds a.GetNumRows().");

    RequireSize(numRows, a.GetNumCols());

    CUDA_LONG N = (CUDA_LONG) GetNumElements();
    int blocksPerGrid = (int) ceil(1.0 * N / GridDim::maxThreadsPerBlock);
    PrepareDevice();
    SyncGuard syncGuard;
    _assignRowSliceValuesOf<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(Data(), a.Data(), N, (CUDA_LONG) startIndex, (CUDA_LONG) numRows, (CUDA_LONG) a.GetNumRows());
    return *this;
}

//for the row slice of this starting from startIndex we add a to it.
template <class ElemType>
GPUMatrix<ElemType>& GPUMatrix<ElemType>::AddToRowSliceValuesOf(const GPUMatrix<ElemType>& a, const size_t startIndex, const size_t numRows)
{
    if (a.IsEmpty())
        LogicError("AddToRowSliceValuesOf: input matrix a is empty.");

    if (a.GetNumRows() != numRows)
        LogicError("AddToRowSliceValuesOf: a.GetNumRows() != numRows.");

    if (startIndex + numRows > GetNumRows())
        LogicError("AddToRowSliceValuesOf: startIndex + numRows exceeds GetNumRows().");

    if (a.GetNumCols() != GetNumCols())
        LogicError("AddToRowSliceValuesOf: columns does not match.");

    CUDA_LONG N = (CUDA_LONG) a.GetNumElements();
    int blocksPerGrid = (int) ceil(1.0 * N / GridDim::maxThreadsPerBlock);
    PrepareDevice();
    SyncGuard syncGuard;
    _addToRowSliceValuesOf<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(Data(), a.Data(), N, (CUDA_LONG) startIndex, (CUDA_LONG) GetNumRows(), (CUDA_LONG) a.GetNumRows());
    return *this;
}

//for each column of this, we add row slice of a starting from startIndex
template <class ElemType>
GPUMatrix<ElemType>& GPUMatrix<ElemType>::AddWithRowSliceValuesOf(const GPUMatrix<ElemType>& a, const size_t startIndex, const size_t numRows)
{
    if (a.IsEmpty())
        LogicError("AddWithRowSliceValuesOf: input matrix a is empty.");

    if (GetNumRows() != numRows)
        LogicError("AddWithRowSliceValuesOf: GetNumRows() != numRows.");

    if (startIndex + numRows > a.GetNumRows())
        LogicError("AddWithRowSliceValuesOf: startIndex + numRows exceeds a.GetNumRows().");

    if (a.GetNumCols() != GetNumCols())
        LogicError("AddWithRowSliceValuesOf: columns does not match.");

    CUDA_LONG N = (CUDA_LONG) GetNumElements();
    int blocksPerGrid = (int) ceil(1.0 * N / GridDim::maxThreadsPerBlock);
    PrepareDevice();
    SyncGuard syncGuard;
    _addWithRowSliceValuesOf<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(Data(), a.Data(), N, (CUDA_LONG) startIndex, (CUDA_LONG) GetNumRows(), (CUDA_LONG) a.GetNumRows());
    return *this;
}

template <class ElemType>
GPUMatrix<ElemType> GPUMatrix<ElemType>::Diagonal() const
{
    size_t m = GetNumRows();
    size_t n = GetNumCols();
    if (m != n)
        LogicError("Diagonal can be called only for square matrix. (rows=%d, cols=%d)", (int) m, (int) n);

    GPUMatrix<ElemType> diag(1, n, GetComputeDeviceId());

    CUDA_LONG N = (CUDA_LONG) GetNumElements();
    int blocksPerGrid = (int) ceil(1.0 * N / GridDim::maxThreadsPerBlock);
    PrepareDevice();
    SyncGuard syncGuard;
    _assignToDiagonalValuesOf<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(diag.Data(), Data(), N, (CUDA_LONG) n);
    return diag;
}

// c = c - 1.0 for a specific position
template <class ElemType>
void GPUMatrix<ElemType>::MinusOneAt(GPUMatrix<ElemType>& c, const size_t position)
{
    assert(position < c.GetNumElements());

    CUDA_LONG n = (CUDA_LONG) c.GetNumElements();
    CUDA_LONG p = (CUDA_LONG) position;

    int blocksPerGrid = (int) ceil(1.0 * n / GridDim::maxThreadsPerBlock);
    // BUGBUG: PrepareDevice() missing?
    SyncGuard syncGuard;
    _minusOneAt<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(c.Data(), p, n);
}

template <class ElemType>
GPUMatrix<ElemType>& GPUMatrix<ElemType>::AssignRepeatOf(const GPUMatrix<ElemType>& a, const size_t numRowRepeats, const size_t numColRepeats)
{
    if (this == &a)
        LogicError("AssignRepeatOf: a is the same as [this]. Does not support inplace repeat.");

    if (a.IsEmpty())
        LogicError("AssignRepeatOf: Matrix a is empty.");

    RequireSize(a.GetNumRows() * numRowRepeats, a.GetNumCols() * numColRepeats);

    CUDA_LONG N = (CUDA_LONG) GetNumElements();
    CUDA_LONG n = (CUDA_LONG) a.GetNumCols(), m = (CUDA_LONG) a.GetNumRows();
    int blocksPerGrid = (int) ceil(1.0 * N / GridDim::maxThreadsPerBlock);
    PrepareDevice();
    SyncGuard syncGuard;
    _assignRepeatOf<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(Data(), a.Data(), N, m, n, (CUDA_LONG) GetNumRows());
    return *this;
}

template <class ElemType>
GPUMatrix<ElemType>& GPUMatrix<ElemType>::AddToRowRepeatValuesOf(const GPUMatrix<ElemType>& a, const size_t numRepeats)
{
    if (a.IsEmpty())
        LogicError("AddToRowRepeatValuesOf: input matrix a is empty.");

    if (a.GetNumRows() != GetNumRows() * numRepeats)
        LogicError("AddToRowSliceValuesOf: a.GetNumRows() != GetNumRows() * numRepeats.");

    RequireSize(a.GetNumRows() / numRepeats, a.GetNumCols());

    CUDA_LONG N = (CUDA_LONG) a.GetNumElements();
    int blocksPerGrid = (int) ceil(1.0 * N / GridDim::maxThreadsPerBlock);
    PrepareDevice();
    SyncGuard syncGuard;
    _addToRowRepeatValuesOf<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(Data(), a.Data(), N, (CUDA_LONG) a.GetNumRows(), (CUDA_LONG) a.GetNumCols(), (CUDA_LONG) GetNumRows());
    return *this;
}

template <class ElemType>
GPUMatrix<ElemType>& GPUMatrix<ElemType>::AssignPositiveAndShiftedNegSample(const GPUMatrix<ElemType>& a, const size_t posNumber, const size_t negNumber, const size_t shiftNumber)
{
    if (this == &a)
        LogicError("AssignPositiveAndShiftedNegSample: a is the same as [this]. Does not support inplace assignment.");

    if (a.IsEmpty())
        LogicError("AssignPositiveAndShiftedNegSample: Matrix a is empty.");

    RequireSize(a.GetNumRows() * (posNumber + negNumber), a.GetNumCols());

    CUDA_LONG N = (CUDA_LONG) GetNumElements();
    CUDA_LONG n = (CUDA_LONG) a.GetNumCols(), m = (CUDA_LONG) a.GetNumRows();
    int blocksPerGrid = (int) ceil(1.0 * N / GridDim::maxThreadsPerBlock);
    PrepareDevice();
    SyncGuard syncGuard;
    _assignPositiveAndShiftedNegSample<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(Data(), a.Data(), N, m, n, (CUDA_LONG) GetNumRows(), posNumber, shiftNumber);
    return *this;
}

template <class ElemType>
GPUMatrix<ElemType>& GPUMatrix<ElemType>::AddFoldedPositiveAndShiftedNegSample(const GPUMatrix<ElemType>& a, const size_t posNumber, const size_t negNumber, const size_t shiftNumber)
{
    if (this == &a)
        LogicError("AddFoldedPositiveAndShiftedNegSample: a is the same as [this]. Does not support inplace assignment.");

    if (a.IsEmpty())
        LogicError("AddFoldedPositiveAndShiftedNegSample: Matrix a is empty.");

    if (a.GetNumRows() != GetNumRows() * (posNumber + negNumber) || a.GetNumCols() != GetNumCols())
        LogicError("AddFoldedPositiveAndShiftedNegSample: dimensions mismatch.");

    CUDA_LONG N = (CUDA_LONG) a.GetNumElements();
    CUDA_LONG n = (CUDA_LONG) a.GetNumCols(), m = (CUDA_LONG) a.GetNumRows();
    int blocksPerGrid = (int) ceil(1.0 * N / GridDim::maxThreadsPerBlock);
    PrepareDevice();
    SyncGuard syncGuard;
    _addFoldedPositiveAndShiftedNegSample<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(Data(), a.Data(), N, m, n, (CUDA_LONG) GetNumRows(), posNumber, shiftNumber);
    return *this;
}

template <class ElemType>
GPUMatrix<ElemType> GPUMatrix<ElemType>::Transpose() const
{
    if (IsEmpty())
        LogicError("Transpose: Matrix is empty.");

    GPUMatrix<ElemType> c(GetComputeDeviceId());
    c.AssignTransposeOf(*this);
    return c;
}

// GetCublasHandle - get a cublas handle for the given GPU, should only need one per GPU
// computeDevice - The compute device for which the cublas handle is desired
// returns: cublas handle
// NOTE: we currently don't bother to ever free the CUBLAS handle, it will be freed automatically by CUDA when the process ends
template <class ElemType>
hipblasHandle_t GPUMatrix<ElemType>::GetCublasHandle(int computeDevice /*=-1*/)
{
    // if the compute device is not passed, get the current device from CUDA
    if (computeDevice < 0)
        hipGetDevice(&computeDevice);

    if (computeDevice < 0 || computeDevice >= MaxGpus)
        LogicError("GetCublasHandle: Maximum GPU exceeded");
    hipblasHandle_t cuHandle = s_cuHandle[computeDevice];
    if (cuHandle == NULL)
    {
        s_cuHandle[computeDevice] = cuHandle = _initCUBLAS<ElemType>(computeDevice);
    }
    CUBLAS_CALL(hipblasSetStream(cuHandle, t_stream));

    return cuHandle;
}

template <class ElemType>
GPUMatrix<ElemType>& GPUMatrix<ElemType>::AssignTransposeOf(const GPUMatrix<ElemType>& a)
{
    if (this == &a)
        LogicError("AssignTransposeOf: a is the same as [this]. Does not support inplace transpose.");

    if (a.IsEmpty())
        LogicError("AssignTransposeOf: Matrix a is empty.");

    if (GetNumRows() != a.GetNumCols() || GetNumCols() != a.GetNumRows())
        RequireSize(a.GetNumCols(), a.GetNumRows());

    hipblasHandle_t cuHandle = GetCublasHandle(a.GetComputeDeviceId());
    hipblasOperation_t transA = HIPBLAS_OP_T;
    hipblasOperation_t transB = HIPBLAS_OP_T;
    int m = (int) a.m_numCols;
    int n = (int) a.m_numRows;
    ElemType alpha = 1;
    ElemType beta = 0;
    hipblasStatus_t st;
    if (sizeof(ElemType) == sizeof(float))
        st = hipblasSgeam(cuHandle, transA, transB, m, n, reinterpret_cast<float*>(&alpha), reinterpret_cast<float*>(a.Data()), (int) a.m_numRows, reinterpret_cast<float*>(&beta), reinterpret_cast<float*>(a.Data()), (int) a.m_numRows, reinterpret_cast<float*>(Data()), (int) m_numRows);
    else if (sizeof(ElemType) == sizeof(double))
        st = hipblasDgeam(cuHandle, transA, transB, m, n, reinterpret_cast<double*>(&alpha), reinterpret_cast<double*>(a.Data()), (int) a.m_numRows, reinterpret_cast<double*>(&beta), reinterpret_cast<double*>(a.Data()), (int) a.m_numRows, reinterpret_cast<double*>(Data()), (int) m_numRows);
    else
        RuntimeError("Unsupported template argument in GPUMatrix");
    if (st != HIPBLAS_STATUS_SUCCESS)
        RuntimeError("AssignTransposeOf failed");
    m_numRows = a.m_numCols;
    m_numCols = a.m_numRows;
    return *this;
}

template <class ElemType>
__global__ void _doGatherColumnsOf(ElemType* us, size_t usStride, const ElemType beta, const ElemType* idx, size_t idxStride, const ElemType* a, size_t aStride, size_t aCols, const ElemType alpha, CUDA_LONG numElements)
{
    CUDA_LONG id = GridDim::GetLinearThreadId();
    if (id >= numElements) // note: there are no __syncthread() calls inside
        return;

    // id = i + jOut * usStride;
    // Each thread processes one element of the output matrix.
    CUDA_LONG i    = id % usStride; // row index into 'us' and 'a'
    CUDA_LONG jOut = id / usStride; // col index into 'us' and 'idx'

    auto jInF = idx[jOut * idxStride]; // this is the column we need to get
    if (::isnan(jInF) || jInF < 0)     // negative index means gap
        return;
    size_t jIn = (size_t)jInF;
    //if (jIn >= aCols)
    //    return; // actually a failure

    const ElemType&  ra = a[    i + jIn  *  aStride  ];
    ElemType&       rus = us[id/*i + jOut * usStride*/];

    ElemType res = ra * alpha;
    if (beta != 0)
        res += rus * beta;
    rus = res;
}

// *this[:,j] = a[:,idx[j]] * alpha + *this[:,j] * beta
template <class ElemType>
GPUMatrix<ElemType>& GPUMatrix<ElemType>::DoGatherColumnsOf(ElemType beta, const GPUMatrix<ElemType>& idx, const GPUMatrix<ElemType>& a, ElemType alpha)
{
    if (idx.GetNumRows() != 1) // index is 1-dimensional only
        InvalidArgument("DoGatherColumnsOf: Map must be a row vector.");

    if (beta == 0)
        RequireSize(a.GetNumRows(), idx.GetNumCols()); // output has same column format as a, but number of columns comes from idx
    else
        VerifySize(a.GetNumRows(), idx.GetNumCols());

    if (idx.GetComputeDeviceId() != a.GetComputeDeviceId() || GetComputeDeviceId() != a.GetComputeDeviceId())
        InvalidArgument("All matrices must be on the same GPU");
    a.PrepareDevice();

    // launch the kernel
    CUDA_LONG NN = (CUDA_LONG)GetNumElements(); // linear space identifying each individual input element
    SyncGuard syncGuard;
    GridDim grid(NN);
    _doGatherColumnsOf<ElemType><<<grid.m_blocksPerGrid, grid.m_threadsPerBlock, 0, t_stream>>>(Data(), GetNumRows(), beta, idx.Data(), idx.GetNumRows(), a.Data(), a.GetNumRows(), a.GetNumCols(), alpha, grid.m_N);

    // Note: The following fails silently (no error, immediate or delayed) for numcols = 10000 under CUDA 7.0.
    //_doGatherColumnsOf<ElemType><<<GetNumCols(), GetNumRows(), 0, t_stream>>>(Data(), GetNumRows(), beta, idx.Data(), idx.GetNumRows(), a.Data(), a.GetNumRows(), a.GetNumCols(), alpha);

    return *this;
}

// little helper for debugging
template <class ElemType>
static void Peek(const GPUMatrix<ElemType>& m, const char* which)
{
    size_t rows = m.GetNumRows();
    size_t cols = m.GetNumCols();
    ElemType buf[10000] = { 0 };
    size_t n = min(rows * cols, _countof(buf));
    CUDA_CALL(hipMemcpy(buf, m.Data(), sizeof(ElemType) * n, hipMemcpyDeviceToHost));
    UNUSED(which); UNUSED(rows); UNUSED(cols); sin(1.0f); // set breakpoint here
    //CUDA_CALL(hipMemcpy(const_cast<ElemType*>(m.Data()), buf, sizeof(ElemType) * n, hipMemcpyHostToDevice));
}

#define ALLOW_ATOMIC_SCATTER // allow to disable this, until we know atomicAdd() works properly here

template <class ElemType>
__global__ void _doScatterColumnsOf(ElemType* us, size_t usStride, size_t usCols, const ElemType* idx, size_t idxStride, const ElemType* a, size_t aStride, const ElemType alpha, CUDA_LONG numElements)
{
    CUDA_LONG id = GridDim::GetLinearThreadId();
    if (id >= numElements) // note: there are no __syncthread() calls inside
        return;

    // id = i + jIn  *  aStride
    // Each thread processes one element of a
    CUDA_LONG i   = id % aStride; // row index into 'a' and 'us'
    CUDA_LONG jIn = id / aStride; // col index into 'a' and 'idx'

    auto jOutF = idx[jIn * idxStride];  // this is the column we copy/add into
    if (::isnan(jOutF) || jOutF < 0)    // negative index means gap
        return;
    size_t jOut = (size_t)jOutF;
    //if (jOut >= usCols)
    //    return; // actually a failure  --TODO: This should not be necessary. Why is it?

    const ElemType&  ra =  a[id/*i + jIn  *  aStride*/];
    ElemType&       rus = us[    i + jOut * usStride  ];

    ElemType res = ra * alpha;
    if (res != 0)             // avoid memory conflict if e.g. an entire column has no gradient
#ifdef ALLOW_ATOMIC_SCATTER
        atomicAdd(&rus, res); // rus += res;
#else
        rus += res;
#endif
    // Note: atomicAdd() is supposed to be fast in case of no conflict (the simple case of Scatter())
}

// *this[:,idx[j]] = a[:,j] * alpha + *this[:,idx[j]] * beta
template <class ElemType>
GPUMatrix<ElemType>& GPUMatrix<ElemType>::DoScatterColumnsOf(ElemType beta, const GPUMatrix<ElemType>& idx, const GPUMatrix<ElemType>& a, ElemType alpha)
{
    if (idx.GetNumRows() != 1) // index is 1-dimensional only
        InvalidArgument("DoScatterColumnsOf: Map must be a row vector.");
    if (idx.GetNumCols() != a.GetNumCols())
        InvalidArgument("DoScatterColumnsOf: Map must have width of input vector.");
    if (a.GetNumRows() != GetNumRows())
        InvalidArgument("DoScatterColumnsOf: Output must have same height as input vector.");

    if (idx.GetComputeDeviceId() != a.GetComputeDeviceId() || GetComputeDeviceId() != a.GetComputeDeviceId())
        InvalidArgument("All matrices must be on the same GPU");
    a.PrepareDevice();

    auto& us = *this;

#ifndef ALLOW_ATOMIC_SCATTER // verify that atomicAdd is not needed  --this is not efficient
    {
        vector<ElemType> buf(idx.GetNumRows() * idx.GetNumCols()); // idx(,)are the column(s) we copy/add into
        CUDA_CALL(hipMemcpy(buf.data(), idx.Data(), sizeof(ElemType) * buf.size(), hipMemcpyDeviceToHost));
        vector<bool> writtenTo(GetNumCols(), false); // remember whether an output column is in fact a target
        for (size_t i = 0; i < buf.size(); i++)
        {
            auto colF = buf[i];
            if (std::isnan(colF) || colF < 0)
                continue;
            size_t col = (size_t)colF;
            if (col >= GetNumCols())
                LogicError("DoScatterColumnsOf: Index value out of bounds.");
            if (writtenTo[col])
                LogicError("DoScatterColumnsOf: #ifndef ALLOW_ATOMIC_SCATTER then columns must be unique. Column idx(%d,%d)=%d is used twice.", (int)(i % idx.GetNumCols()), (int)(i / idx.GetNumCols()), (int)col);
            else
                writtenTo[col] = true;
        }
    }
#endif

    // pre-scale with beta upfront
    // Scatter may add more than one source column to the same target, so we must pre-scale with beta, and then just keep adding.
    Scale(beta, us); // if beta is 0, then this will be a memset()

    // launch the kernel
    CUDA_LONG NN = (CUDA_LONG)(a.GetNumElements()); // linear space identifying each individual input element
    SyncGuard syncGuard;
    GridDim grid(NN);
    _doScatterColumnsOf<ElemType><<<grid.m_blocksPerGrid, grid.m_threadsPerBlock, 0, t_stream>>>(Data(), GetNumRows(), GetNumCols(), idx.Data(), idx.GetNumRows(), a.Data(), a.GetNumRows(), alpha, NN);

    //SyncGuard syncGuard;
    //_doScatterColumnsOf<ElemType><<<a.GetNumCols(), a.GetNumRows(), 0, t_stream>>>(Data(), GetNumRows(), GetNumCols(), idx.Data(), idx.GetNumRows(), a.Data(), a.GetNumRows(), alpha, NN);

    return *this;
}

template <class ElemType>
void GPUMatrix<ElemType>::SetValue(const ElemType v)
{
    if (IsEmpty())
        return;

    CUDA_LONG N = (CUDA_LONG) GetNumElements();

    // Check if value is zero, which can be set using hipMemset
    bool isZero = true;
    const char* valArray = reinterpret_cast<const char*>(&v);

    for (int i = 0; i < sizeof(ElemType); i++)
    {
        if (valArray[i] != 0)
        {
            isZero = false;
            break;
        }
    }

    if (isZero)
    {
        CUDA_CALL(hipMemset(Data(), 0, N * sizeof(ElemType)));
    }
    else
    {
        int blocksPerGrid = (int) ceil(1.0 * N / GridDim::maxThreadsPerBlock);
        PrepareDevice();
        SyncGuard syncGuard;
        _setValue<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(Data(), v, N);
    }
}

template <class ElemType>
void GPUMatrix<ElemType>::SetValue(const ElemType* d_v) // d_v is pointer to the the value in GPU memory
{
    if (IsEmpty())
        LogicError("SetValue: Matrix is empty.");

    CUDA_LONG N = (CUDA_LONG) GetNumElements();
    int blocksPerGrid = (int) ceil(1.0 * N / GridDim::maxThreadsPerBlock);
    PrepareDevice();
    SyncGuard syncGuard;
    _setValue<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(Data(), d_v, N);
}

template <class ElemType>
void GPUMatrix<ElemType>::MaskColumnsValue(const GPUMatrix<char>& columnsMask, ElemType val)
{
    if (GetNumCols() != columnsMask.GetNumCols())
        RuntimeError("Matrix and column mask must have equal number of columns");

    if (GetComputeDeviceId() != columnsMask.GetComputeDeviceId())
        RuntimeError("Matrix and column mask must be on the same device");

    int blocksPerGrid = (int) GetNumCols();
    PrepareDevice();
    SyncGuard syncGuard;
    _maskColumnsValue<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(Data(), columnsMask.Data(), (CUDA_LONG) GetNumCols(), (CUDA_LONG) GetNumRows(), val);
}

template <class ElemType>
void GPUMatrix<ElemType>::SetColumn(const ElemType* colPointer, size_t colInd)
{
    if (IsEmpty())
        LogicError("SetValue: Matrix is empty.");
    if (colPointer == NULL)
        return;
    CUDA_CALL(hipMemcpy(Data() + LocateColumn(colInd), colPointer, sizeof(ElemType) * m_numRows, hipMemcpyHostToDevice));
}

template <class ElemType>
void GPUMatrix<ElemType>::SetColumn(const GPUMatrix<ElemType>& valMat, size_t colInd)
{
    if (IsEmpty())
        LogicError("SetColumn: Matrix is empty.");
    if (valMat.GetNumCols() != 1)
        LogicError("SetColumn: only support one column matrix now.");
    CUDA_CALL(hipMemcpy(Data() + LocateColumn(colInd), valMat.Data(), sizeof(ElemType) * m_numRows, hipMemcpyDeviceToDevice));
}

template <class ElemType>
void GPUMatrix<ElemType>::SetValue(const GPUMatrix<ElemType>& deepCopyFrom)
{
    if (this == &deepCopyFrom)
        return;

    SetValue(deepCopyFrom.GetNumRows(), deepCopyFrom.GetNumCols(), deepCopyFrom.GetComputeDeviceId(), deepCopyFrom.Data(), matrixFlagSetValueOnDevice);
}

#if 0
template <class ElemType>
void GPUMatrix<ElemType>::SetValue(const CPUMatrix<ElemType>& /*deepCopyFrom*/)
{
    NOT_IMPLEMENTED;
}

template <class ElemType>
void GPUMatrix<ElemType>::SetValue(const CPUSparseMatrix<ElemType>& /*deepCopyFrom*/)
{
    NOT_IMPLEMENTED;
}

template <class ElemType>
void GPUMatrix<ElemType>::SetValue(const GPUSparseMatrix<ElemType>& deepCopyFrom)
{
    deepCopyFrom.CopyToDenseMatrix(*this);
}
#endif

template <class ElemType>
void GPUMatrix<ElemType>::SetValue(const size_t numRows, const size_t numCols, int deviceId, ElemType* pArray, size_t matrixFlags)
{
    // handle externally managed case
	// BUGBUG: This is super super ugly, and needs to be fixed, but if matrixFlags has the right value, then we can't free anything,
    // and everything gets wonky. This should be fixed, and would go away if it is made a shared_ptr.
    if (matrixFlags & matrixFlagDontOwnBuffer)
    {
        // free the existing array if it used to be an owned array
        if ( Buffer() != NULL)
        {
            TracingGPUMemoryAllocator::Free<ElemType>(GetComputeDeviceId(), Buffer());
        }
        m_numRows = numRows;
        m_numCols = numCols;
        SetBuffer(pArray, GetNumElements() * sizeof(ElemType), true);
        SetSizeAllocated(GetNumElements());
        SetFormat(matrixFormatDense);
        SetComputeDeviceId(deviceId);
    }
    else
    {
        // if the devices are different move it now
        if (GetComputeDeviceId() != deviceId && deviceId >= 0)
        {
            Clear();
            ZeroInit(deviceId);
        }

        // now RequireSize/allocate as necessary
        RequireSize(numRows, numCols);

        // copy over the content to the buffer
        PrepareDevice();
        if (pArray != NULL)
        {
            if (!(matrixFlags & matrixFormatRowMajor))
            {
                CUDA_CALL(hipMemcpy(Data(), pArray, sizeof(ElemType) * GetNumElements(), (matrixFlags & matrixFlagSetValueOnDevice) ? hipMemcpyDeviceToDevice : hipMemcpyHostToDevice));
            }
            else // row major: must transpose (this is not meant to be efficient, but very useful for defining inline matrices for test code)
            {
                vector<ElemType> transposed(GetNumElements());
                for (size_t i = 0; i < numRows; i++)
                    for (size_t j = 0; j < numCols; j++)
                        transposed[i + numRows * j] = pArray[j + numCols * i];
                CUDA_CALL(hipMemcpy(Data(), transposed.data(), sizeof(ElemType) * GetNumElements(), (matrixFlags & matrixFlagSetValueOnDevice) ? hipMemcpyDeviceToDevice : hipMemcpyHostToDevice));
            }
        }
    }
    SetFormat(matrixFormatDense);
}

template <class ElemType>
void GPUMatrix<ElemType>::SetDiagonalValue(const ElemType v)
{
    CUDA_LONG N = (CUDA_LONG) GetNumRows();
    int blocksPerGrid = (int) ceil(1.0 * N / GridDim::maxThreadsPerBlock);
    PrepareDevice();
    SyncGuard syncGuard;
    _setDiagonalValue<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(Data(), v, N, (CUDA_LONG) GetNumRows());
}

template <class ElemType>
void GPUMatrix<ElemType>::SetDiagonalValue(const GPUMatrix<ElemType>& vector)
{
    if (IsEmpty() || vector.IsEmpty())
        LogicError("SetDiagonalValue: Matrix is empty.");

    if (GetNumRows() != GetNumCols())
        LogicError("SetDiagonalValue: NumRows and NumCols do not agree.");

    if (vector.GetNumRows() != 1 && vector.GetNumCols() != 1)
        LogicError("SetDiagonalValue: input vector must be a vector.");

    if (vector.GetNumElements() == 1) // reduce to simple form
        SetDiagonalValue(vector.Data()[0]);

    else if (vector.GetNumRows() != GetNumRows())
        LogicError("SetDiagonalValue: input vector's dimension does not agree with [this].");
    else
    {
        CUDA_LONG N = (CUDA_LONG) GetNumRows();
        int blocksPerGrid = (int) ceil(1.0 * N / GridDim::maxThreadsPerBlock);
        PrepareDevice();
        SyncGuard syncGuard;
        _setDiagonalValueFromVector<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(Data(), vector.Data(), N);
    }
}

template <class ElemType>
void GPUMatrix<ElemType>::SetUniformRandomValue(const ElemType low, const ElemType high, unsigned long seed)
{
    PrepareDevice();
    CreateCurandObject(seed, __FUNCTION__); // TODO call ResetCurandObject() instead?

    hipEvent_t done = nullptr;
    CUDA_CALL(hipEventCreate(&done)); // TODO: why not condition on do_sync, so that we can use SyncGuard?
    if (sizeof(ElemType) == sizeof(float))
        CURAND_CALL(hiprandGenerateUniform(((hiprandGenerator_t*) s_curandGenerator)[0], reinterpret_cast<float*>(Data()), GetNumElements()));
    else
        CURAND_CALL(hiprandGenerateUniformDouble(((hiprandGenerator_t*) s_curandGenerator)[0], reinterpret_cast<double*>(Data()), GetNumElements()));
    CUDA_CALL(hipEventRecord(done));
    CUDA_CALL(hipEventSynchronize(done));
    // CURAND_CALL(hiprandDestroyGenerator(gen));
    CUDA_CALL(hipEventDestroy(done));

    size_t N = GetNumElements();
    size_t blocksPerGrid = (size_t) ceil(N / (double) GridDim::maxThreadsPerBlock);

    SyncGuard syncGuard;
    _rescaleToRange<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(Data(), N, low, high);
}

template <class ElemType>
void GPUMatrix<ElemType>::SetGaussianRandomValue(const ElemType mean, const ElemType sigma, unsigned long seed)
{
    PrepareDevice();
    CreateCurandObject(seed, __FUNCTION__); // TODO call ResetCurandObject() instead?

    // TODO: Why not use SyncGuard?
    if (sizeof(ElemType) == sizeof(float))
        CURAND_CALL(hiprandGenerateNormal(((hiprandGenerator_t*) s_curandGenerator)[0], reinterpret_cast<float*>(Data()), GetNumElements(), (float) mean, (float) sigma));
    else
        CURAND_CALL(hiprandGenerateNormalDouble(((hiprandGenerator_t*) s_curandGenerator)[0], reinterpret_cast<double*>(Data()), GetNumElements(), (double) mean, (double) sigma));
    // CURAND_CALL(hiprandDestroyGenerator(gen));
}

//maskRate: percentage of values masked out (similar to dropout rate)
//scaleValue: which scale value to set to the left ones (unmasked items).
template <class ElemType>
void GPUMatrix<ElemType>::SetUniformRandomMask(const ElemType maskRate, const ElemType scaleValue, RNGHandle& rngHandle)
{
    PrepareDevice();

    GPURNGHandle* gpuRNGHandle = dynamic_cast<GPURNGHandle*>(&rngHandle);
    assert(gpuRNGHandle != nullptr);

    hipEvent_t done = nullptr;
    CUDA_CALL(hipEventCreate(&done)); // TODO: why not condition on do_sync, so that we can use SyncGuard?
    if (sizeof(ElemType) == sizeof(float))
        CURAND_CALL(hiprandGenerateUniform(gpuRNGHandle->Generator(), reinterpret_cast<float*>(Data()), GetNumElements()));
    else
        CURAND_CALL(hiprandGenerateUniformDouble(gpuRNGHandle->Generator(), reinterpret_cast<double*>(Data()), GetNumElements()));
    CUDA_CALL(hipEventRecord(done));
    CUDA_CALL(hipEventSynchronize(done));
    CUDA_CALL(hipEventDestroy(done));

    size_t N = GetNumElements();
    size_t blocksPerGrid = (size_t) ceil(N / (double) GridDim::maxThreadsPerBlock);
    SyncGuard syncGuard;
    _setMaskAndScale<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(Data(), N, maskRate, scaleValue);
}

template <class ElemType>
ElemType GPUMatrix<ElemType>::Adagrad(GPUMatrix<ElemType>& gradients, const bool needAveMultiplier)
{
    size_t numColsNeeded = gradients.GetNumCols();
    if (needAveMultiplier)
        numColsNeeded += gradients.GetNumCols();

    if (IsEmpty() || GetNumCols() < numColsNeeded)
    {
        RequireSize(gradients.GetNumRows(), numColsNeeded);
        SetValue(0.0);
    }

    assert(GetNumRows() == gradients.GetNumRows() && GetNumCols() == numColsNeeded);

    size_t n = gradients.GetNumElements();

    ElemType* multipliers = nullptr;
    if (needAveMultiplier)
        multipliers = Data() + n; // temp memory used to store multipliers,

    int blocksPerGrid = (n + GridDim::maxThreadsPerBlock - 1) / GridDim::maxThreadsPerBlock;
    _adagrad<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(Data(), gradients.Data(), n, multipliers);

    if (!needAveMultiplier)
        return 1;

    hipblasHandle_t cuHandle = GetCublasHandle(GetComputeDeviceId());
    if (sizeof(ElemType) == sizeof(float))
    {
        float aveMultiplier = 0;
        CUBLAS_CALL(hipblasSasum(cuHandle, (CUDA_LONG) n, reinterpret_cast<float*>(multipliers), 1, &aveMultiplier));
        return (ElemType) aveMultiplier / n;
    }
    else
    {
        double aveMultiplier = 0;
        CUBLAS_CALL(hipblasDasum(cuHandle, (CUDA_LONG) n, reinterpret_cast<double*>(multipliers), 1, &aveMultiplier));
        return (ElemType) aveMultiplier / n;
    }
}

template <class ElemType>
void GPUMatrix<ElemType>::FSAdagrad(GPUMatrix<ElemType>& gradients,
                                    GPUMatrix<ElemType>& functionValues,
                                    ElemType learnRatePerSample,
                                    ElemType momentum,
                                    ElemType adaWeight,
                                    ElemType adaMul)
{
    size_t numColsNeeded = 2 * gradients.GetNumCols();

    if (IsEmpty() || (GetNumCols() < numColsNeeded))
    {
        RequireSize(gradients.GetNumRows(), numColsNeeded);
        SetValue(0.0);
    }

    assert((GetNumRows() == gradients.GetNumRows()) && (GetNumCols() == numColsNeeded));

    size_t n = gradients.GetNumElements();
    int blocksPerGrid = (n + GridDim::maxThreadsPerBlock - 1) / GridDim::maxThreadsPerBlock;
    _fsadagrad<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(n, gradients.Data(), Data(), Data()+ n, functionValues.Data(),
                                                                         learnRatePerSample, momentum, adaWeight, adaMul);
}

template <class ElemType>
ElemType GPUMatrix<ElemType>::RmsProp(GPUMatrix<ElemType>& gradients,
                                      ElemType RMS_GAMMA,
                                      ElemType RMS_WGT_INC,
                                      ElemType RMS_WGT_MAX,
                                      ElemType RMS_WGT_DEC,
                                      ElemType RMS_WGT_MIN,
                                      const bool needAveMultiplier)
{
    const ElemType floor = 1e-6f;
    static ElemType* upd_gpu = (ElemType*) 0;

    size_t n = gradients.GetNumElements();
    int blocksPerGrid = (GetNumElements() + GridDim::maxThreadsPerBlock - 1) / GridDim::maxThreadsPerBlock;

    size_t numColsNeeded = gradients.GetNumCols() * 3;
    if (needAveMultiplier)
        numColsNeeded += gradients.GetNumCols();

    if (IsEmpty() || GetNumCols() < numColsNeeded)
    {
        RequireSize(gradients.GetNumRows(), numColsNeeded);
        SetValue(0.0);

        ElemType* avars = Data();         // accumulated variances for RMS scaling
        ElemType* signs = Data() + n;     // sign of previous gradient
        ElemType* steps = Data() + 2 * n; // current step size
        // Data()+3*n is temp memory used to store multipliers, no need to initialize

        _rmsprop_init<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(avars, signs, steps, gradients.Data(), n);
    }
    assert(GetNumRows() == gradients.GetNumRows() && GetNumCols() == numColsNeeded);

    ElemType* avars = Data();         // accumulated variances for RMS scaling
    ElemType* signs = Data() + n;     // sign of previous gradient
    ElemType* steps = Data() + 2 * n; // current step size

    ElemType* multipliers = nullptr;
    if (needAveMultiplier)
        multipliers = Data() + 3 * n; // temp memory used to store multipliers,

    if (!upd_gpu)
    {
        const ElemType upd[] = {
            2, 2, 0,
            2, 2, 0,
            1, 1, 1,
            2, 2, 0,
            1, 2, 1,
            0, 2, 2,
            1, 1, 1,
            0, 2, 2,
            0, 2, 2,
        };

        upd_gpu = TracingGPUMemoryAllocator::Allocate<ElemType>(GetComputeDeviceId(), 27);
        CUDA_CALL(hipMemcpy(upd_gpu, upd, sizeof(ElemType) * _countof(upd), hipMemcpyHostToDevice));
    }

    _rmsprop<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(avars, signs, steps, gradients.Data(), n,
                                                                       RMS_GAMMA, RMS_WGT_INC, RMS_WGT_MAX, RMS_WGT_DEC, RMS_WGT_MIN,
                                                                       floor, upd_gpu, multipliers);

    if (!needAveMultiplier)
        return 1;

    hipblasHandle_t cuHandle = GetCublasHandle(GetComputeDeviceId());
    if (sizeof(ElemType) == sizeof(float))
    {
        float aveMultiplier = 0;
        CUBLAS_CALL(hipblasSasum(cuHandle, (CUDA_LONG) n, reinterpret_cast<float*>(multipliers), 1, &aveMultiplier));
        return aveMultiplier / n;
    }
    else
    {
        double aveMultiplier = 0;
        CUBLAS_CALL(hipblasDasum(cuHandle, (CUDA_LONG) n, reinterpret_cast<double*>(multipliers), 1, &aveMultiplier));
        return (ElemType) aveMultiplier / n;
    }
}

template <class ElemType>
void GPUMatrix<ElemType>::Reshape(const size_t numRows, const size_t numCols)
{
    assert(numRows * numCols == GetNumElements());
    if (numRows * numCols != GetNumElements())
        InvalidArgument("Reshape: total number of elements does not match.");

    m_numRows = numRows;
    m_numCols = numCols;
}

template <class ElemType>
void GPUMatrix<ElemType>::RequireSize(const size_t numRows, const size_t numCols, bool growOnly)
{
    if (GetNumRows() != numRows || GetNumCols() != numCols)
        Resize(numRows, numCols, growOnly);
}

template <class ElemType>
void GPUMatrix<ElemType>::Resize(const size_t numRows, const size_t numCols, bool growOnly)
{
    VerifyResizable(__func__);

    if (GetNumRows() == numRows && GetNumCols() == numCols)
        return;

    size_t numElements = numRows * numCols;
    if (numElements > GetSizeAllocated() ||                 // grow allocation
        (!growOnly && numElements != GetSizeAllocated()))   // shrink allocation if not growOnly
    {
        // reallocate buffer if numElements > 0
        ElemType* pArray = nullptr;
        if (numElements > 0)
            pArray = TracingGPUMemoryAllocator::Allocate<ElemType>(GetComputeDeviceId(), numRows, numCols);

        // If the buffer exists, free it
        if (Buffer())
            TracingGPUMemoryAllocator::Free<ElemType>(GetComputeDeviceId(), Buffer());

        SetBuffer(pArray, numElements * sizeof(ElemType));
        SetSizeAllocated(numElements);
    }
    
    // success
    m_sliceViewOffset = 0;
    m_numRows = numRows;
    m_numCols = numCols;
}

template <class ElemType>
size_t GPUMatrix<ElemType>::LocateElement(const size_t row, const size_t col) const
{
    assert(row < m_numRows && col < m_numCols);
    return LocateColumn(col) + row; // matrix in column-wise storage
}

template <class ElemType>
size_t GPUMatrix<ElemType>::LocateColumn(const size_t col) const
{
    assert(col < GetNumCols());
    return col * m_numRows; // matrix in column-wise storage
}

template <class ElemType>
ElemType GPUMatrix<ElemType>::Get00Element() const
{
    ElemType res = 0;
    CUDA_CALL(hipMemcpy(&res, Data(), sizeof(ElemType), hipMemcpyDeviceToHost));
    return res;
}
#pragma endregion Basic Operators

#pragma region Member BLAS Functions
template <class ElemType>
GPUMatrix<ElemType>& GPUMatrix<ElemType>::operator+=(ElemType alpha)
{
    if (IsEmpty())
        LogicError("operator+=: Matrix is empty.");
    CUDA_LONG N = (CUDA_LONG) GetNumElements();
    int blocksPerGrid = (int) ceil(1.0 * N / GridDim::maxThreadsPerBlock);
    SyncGuard syncGuard;
    _addValue<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(Data(), alpha, N);
    return *this;
}

template <class ElemType>
GPUMatrix<ElemType> GPUMatrix<ElemType>::operator+(ElemType alpha) const
{
    if (IsEmpty())
        LogicError("operator+: Matrix is empty.");

    GPUMatrix<ElemType> c(*this);
    c += alpha;
    return c;
}

template <class ElemType>
GPUMatrix<ElemType>& GPUMatrix<ElemType>::AssignSumOf(const ElemType alpha, const GPUMatrix<ElemType>& a)
{
    SetValue(a);
    (*this) += alpha;
    return (*this);
}

template <class ElemType>
GPUMatrix<ElemType>& GPUMatrix<ElemType>::operator+=(const GPUMatrix<ElemType>& a)
{
    ScaleAndAdd(1, a, *this);
    return *this;
}

template <class ElemType>
GPUMatrix<ElemType> GPUMatrix<ElemType>::operator+(const GPUMatrix<ElemType>& a) const
{
    if (GetNumElements() == 1)
    {
        GPUMatrix<ElemType> c(a);
        c += Get00Element();
        return c;
    }
    else if (a.GetNumElements() == 1)
    {
        GPUMatrix<ElemType> c(*this);
        c += a.Get00Element();
        return c;
    }
    else
    {
        GPUMatrix<ElemType> c(*this); // this implementation will introduce a copy overhead. but make resue of the code
        c += a;
        return c;
    }
}

template <class ElemType>
GPUMatrix<ElemType>& GPUMatrix<ElemType>::AssignSumOf(const GPUMatrix<ElemType>& a, const GPUMatrix<ElemType>& b)
{
    SetValue(a);
    (*this) += b;
    return (*this);
}

template <class ElemType>
GPUMatrix<ElemType>& GPUMatrix<ElemType>::operator-=(ElemType alpha)
{
    if (IsEmpty())
        LogicError("operato-=: Matrix is empty.");
    return operator+=(-1 * alpha);
}

template <class ElemType>
GPUMatrix<ElemType> GPUMatrix<ElemType>::operator-(ElemType alpha) const
{
    if (IsEmpty())
        LogicError("operator-: Matrix is empty.");
    return operator+(-1 * alpha);
}

template <class ElemType>
GPUMatrix<ElemType>& GPUMatrix<ElemType>::AssignDifferenceOf(const ElemType alpha, const GPUMatrix<ElemType>& a)
{
    RequireSize(a.m_numRows, a.m_numCols);
    CUDA_LONG N = (CUDA_LONG) GetNumElements();
    int blocksPerGrid = (int) ceil(1.0 * N / GridDim::maxThreadsPerBlock);
    a.PrepareDevice();
    SyncGuard syncGuard;
    _assignDifferenceOf1<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(Data(), alpha, a.Data(), N);
    return *this;
}

template <class ElemType>
GPUMatrix<ElemType>& GPUMatrix<ElemType>::AssignDifferenceOf(const GPUMatrix<ElemType>& a, const ElemType alpha)
{
    RequireSize(a.m_numRows, a.m_numCols);
    CUDA_LONG N = (CUDA_LONG) GetNumElements();
    int blocksPerGrid = (int) ceil(1.0 * N / GridDim::maxThreadsPerBlock);
    a.PrepareDevice();
    SyncGuard syncGuard;
    _assignDifferenceOf2<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(Data(), alpha, a.Data(), N);
    return *this;
}

template <class ElemType>
GPUMatrix<ElemType>& GPUMatrix<ElemType>::operator-=(const GPUMatrix<ElemType>& a)
{
    ScaleAndAdd(-1, a, *this);

    return *this;
}

template <class ElemType>
GPUMatrix<ElemType> GPUMatrix<ElemType>::operator-(const GPUMatrix<ElemType>& a) const
{
    GPUMatrix<ElemType> c(*this); // this implementation will introduce a copy overhead. but make resue of the code
    c -= a;
    return c;
}

template <class ElemType>
GPUMatrix<ElemType>& GPUMatrix<ElemType>::AssignDifferenceOf(const GPUMatrix<ElemType>& a, const GPUMatrix<ElemType>& b)
{
    if (this != &a)
    {
        RequireSize(a.GetNumRows(), a.GetNumCols());
        SetValue(a);
    }
    (*this) -= b;
    return *this;
}

template <class ElemType>
GPUMatrix<ElemType>& GPUMatrix<ElemType>::operator*=(ElemType alpha)
{
    Scale(alpha, *this);
    return *this;
}

template <class ElemType>
GPUMatrix<ElemType> GPUMatrix<ElemType>::operator*(ElemType alpha) const
{
    GPUMatrix<ElemType> c(GetNumRows(), GetNumCols(), GetComputeDeviceId());
    Scale(alpha, *this, c);
    return c;
}

template <class ElemType>
GPUMatrix<ElemType>& GPUMatrix<ElemType>::AssignProductOf(const ElemType alpha, const GPUMatrix<ElemType>& a)
{
    Scale(alpha, a, *this);
    return *this;
}

template <class ElemType>
GPUMatrix<ElemType>& GPUMatrix<ElemType>::AssignProductOf(const GPUMatrix<ElemType>& a, const bool transposeA, const GPUMatrix<ElemType>& b, const bool transposeB)
{
    if (a.GetNumElements() == 1)
    {
        if (transposeB)
            AssignTransposeOf(b);
        (*this) *= a.Get00Element();
    }
    else if (b.GetNumElements() == 1)
    {
        if (transposeA)
            AssignTransposeOf(a);
        (*this) *= b.Get00Element();
    }
    else
        Multiply(a, transposeA, b, transposeB, *this);
    return *this;
}

template <class ElemType>
GPUMatrix<ElemType> GPUMatrix<ElemType>::operator*(const GPUMatrix<ElemType>& a) const
{
    const GPUMatrix<ElemType>& us = *this;
    if (GetNumElements() == 1)
    {
        GPUMatrix<ElemType> c(GetComputeDeviceId());
        c.AssignProductOf(Get00Element(), a);
        return c;
    }
    else if (a.GetNumElements() == 1)
    {
        GPUMatrix<ElemType> c(GetComputeDeviceId());
        c.AssignProductOf(a.Get00Element(), us);
        return c;
    }
    else
    {
        GPUMatrix<ElemType> c(GetNumRows(), a.GetNumCols(), GetComputeDeviceId());
        Multiply(*this, a, c);
        return c;
    }
}

template <class ElemType>
GPUMatrix<ElemType>& GPUMatrix<ElemType>::operator/=(ElemType alpha)
{
    (*this) *= 1 / alpha;
    return (*this);
}

template <class ElemType>
GPUMatrix<ElemType> GPUMatrix<ElemType>::operator/(ElemType alpha) const
{
    return ((*this) * (1 / alpha));
}

//element-wise power
template <class ElemType>
GPUMatrix<ElemType>& GPUMatrix<ElemType>::operator^=(ElemType alpha)
{
    GPUMatrix<ElemType>& us = *this;
    ElementWisePower(alpha, us, us);
    return us;
}

template <class ElemType>
GPUMatrix<ElemType> GPUMatrix<ElemType>::operator^(ElemType alpha) const
{
    GPUMatrix<ElemType> c(GetNumRows(), GetNumCols(), GetComputeDeviceId());
    ElementWisePower(alpha, *this, c);
    return c;
}

template <class ElemType>
GPUMatrix<ElemType>& GPUMatrix<ElemType>::AssignElementPowerOf(const GPUMatrix<ElemType>& a, const ElemType power)
{
    ElementWisePower(power, a, *this);
    return *this;
}

template <class ElemType>
GPUMatrix<ElemType>& GPUMatrix<ElemType>::AddElementProductOf(const GPUMatrix<ElemType>& a, const GPUMatrix<ElemType>& b)
{
    if (a.IsEmpty() || b.IsEmpty())
        LogicError("AddElementProductOf: Matrix is empty.");

    assert(a.GetNumRows() == b.GetNumRows() && a.GetNumCols() == b.GetNumCols());
    if (!(a.GetNumRows() == b.GetNumRows() && a.GetNumCols() == b.GetNumCols()))
        InvalidArgument("The input matrix dimensions do not match.");

    if (!(a.GetNumRows() == GetNumRows() && a.GetNumCols() == GetNumCols()))
        InvalidArgument("The input matrix dimensions do not match [this].");

    CUDA_LONG N = (CUDA_LONG) GetNumElements();
    int blocksPerGrid = (int) ceil(1.0 * N / GridDim::maxThreadsPerBlock);
    a.PrepareDevice();
    SyncGuard syncGuard;
    _addElementProductOf<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(Data(), a.Data(), b.Data(), N);
    return *this;
}

template <class ElemType>
GPUMatrix<ElemType>& GPUMatrix<ElemType>::ColumnElementMultiplyWith(const GPUMatrix<ElemType>& a)
{
    if (a.IsEmpty() || IsEmpty())
        LogicError("ColumnElementMultiplyWith: Matrix is empty.");

    if (!(a.GetNumRows() == GetNumRows() && a.GetNumCols() == 1))
        InvalidArgument("ColumnElementMultiplyWith: The input matrix should be a col vector and match [this]'s rows.");

    CUDA_LONG N = (CUDA_LONG) a.GetNumRows();
    CUDA_LONG M = (CUDA_LONG) GetNumCols();
    int blocksPerGrid = (int) ceil(1.0 * N / GridDim::maxThreadsPerBlock);
    a.PrepareDevice();
    SyncGuard syncGuard;
    _columnElementMultiplyWith<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(Data(), a.Data(), N, M);
    return *this;
}

template <class ElemType>
GPUMatrix<ElemType>& GPUMatrix<ElemType>::RowElementMultiplyWith(const GPUMatrix<ElemType>& a)
{
    if (a.IsEmpty() || IsEmpty())
        LogicError("RowElementMultiplyWith: Matrix is empty.");

    if (!(a.GetNumRows() == 1 && a.GetNumCols() == GetNumCols()))
        InvalidArgument("RowElementMultiplyWith: The input matrix should be a row vector and match [this]'s columns.");

    CUDA_LONG N = (CUDA_LONG) GetNumRows();
    CUDA_LONG M = (CUDA_LONG) a.GetNumCols();
    int blocksPerGrid = (int) ceil(1.0 * M / GridDim::maxThreadsPerBlock);
    a.PrepareDevice();
    SyncGuard syncGuard;
    _rowElementMultiplyWith<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(Data(), a.Data(), N, M);
    return *this;
}

template <class ElemType>
GPUMatrix<ElemType>& GPUMatrix<ElemType>::RowElementDivideBy(const GPUMatrix<ElemType>& a)
{
    if (a.IsEmpty() || IsEmpty())
        LogicError("RowElementDivideBy: Matrix is empty.");

    if (!(a.GetNumRows() == 1 && a.GetNumCols() == GetNumCols()))
        InvalidArgument("RowElementDivideBy: The input matrix should be a row vector and match [this]'s columns.");

    CUDA_LONG N = (CUDA_LONG) GetNumRows();
    CUDA_LONG M = (CUDA_LONG) a.GetNumCols();
    int blocksPerGrid = (int) ceil(1.0 * M / GridDim::maxThreadsPerBlock);
    a.PrepareDevice();
    SyncGuard syncGuard;
    _rowElementDivideBy<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(Data(), a.Data(), N, M);
    return *this;
}

template <class ElemType>
GPUMatrix<ElemType>& GPUMatrix<ElemType>::ColumnElementDivideBy(const GPUMatrix<ElemType>& a)
{
    if (a.IsEmpty() || IsEmpty())
        LogicError("ColumnElementDivideBy: Matrix is empty.");

    if (!(a.GetNumRows() == GetNumRows() && a.GetNumCols() == 1))
        InvalidArgument("ColumnElementDivideBy: The input matrix should be a col vector and match [this]'s rows.");

    CUDA_LONG N = (CUDA_LONG) a.GetNumRows();
    CUDA_LONG M = (CUDA_LONG) GetNumCols();
    int blocksPerGrid = (int) ceil(1.0 * N / GridDim::maxThreadsPerBlock);
    a.PrepareDevice();
    SyncGuard syncGuard;
    _ColumnElementDivideBy<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(Data(), a.Data(), N, M);
    return *this;
}

template <class ElemType>
GPUMatrix<ElemType>& GPUMatrix<ElemType>::ElementInverse()
{
    if (IsEmpty())
        LogicError("ElementInverse: Matrix is empty.");

    CUDA_LONG N = (CUDA_LONG) GetNumElements();
    int blocksPerGrid = (int) ceil(1.0 * N / GridDim::maxThreadsPerBlock);
    PrepareDevice();
    SyncGuard syncGuard;
    _elemInverse<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(Data(), N);
    return *this;
}

template <class ElemType>
GPUMatrix<ElemType>& GPUMatrix<ElemType>::AssignElementInverseOf(const GPUMatrix<ElemType>& a)
{
    SetValue(a);
    return ElementInverse();
}

DEF_ELEMWISE_INPLACE_FUNC(Sigmoid)

template <class ElemType>
GPUMatrix<ElemType>& GPUMatrix<ElemType>::AssignSigmoidOf(const GPUMatrix<ElemType>& a)
{
    RequireSize(a.GetNumRows(), a.GetNumCols());
    CUDA_LONG N = (CUDA_LONG) GetNumElements();
    int blocksPerGrid = (int) ceil(1.0 * N / GridDim::maxThreadsPerBlock);
    PrepareDevice();
    SyncGuard syncGuard;
    // _elementWIseSigmoidOnCuda has an implementation that avoids possible overflow errors, but has a slight accuracy regression.
#if 0
    _elementWiseSigmoidOnCuda<<<blocksPerGrid, threadsPerBlock, 0, t_stream>>>(a.Data(), Data(), N);
#else
    _assignSigmoidOf<<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(a.Data(), Data(), N);
#endif
    return *this;
}

DEF_ELEMWISE_INPLACE_FUNC(SigmoidDerivative)
DEF_ELEMWISE_ASSIGN_FUNC(SigmoidDerivative)

template <class ElemType>
void GPUMatrix<ElemType>::AssignNoiseContrastiveEstimation(const GPUMatrix<ElemType>& a,
                                                           const GPUMatrix<ElemType>& b, const GPUMatrix<ElemType>& bias, size_t sampleCount, GPUMatrix<ElemType>& tmp, GPUMatrix<ElemType>& c)
//this:   samples+probs
// a  :   hidden
// b  :   embedding
// tmp:   softmax
// c  :   loglikelihood
{
    UNCONST(ElemType, a, my_a);
    UNCONST(ElemType, b, my_b);
    UNCONST(ElemType, bias, my_bias);
    SyncGuard syncGuard;
    // a: dim * minibatch
    // b: dim * |vocab|
    int p = 512;
    int width = a.GetNumRows(); // dimension of hidden vector

    while (p / 2 > width)
        p = p / 2;

    _computeNceOutput<ElemType><<<GetNumElements() / 2, p>>>(
        Data(),
        sampleCount,
        m_numRows / 2,
        my_a.Data(), // a
        a.GetNumRows(),
        my_b.Data(), // b
        my_bias.Data(),
        tmp.Data()); // tmp

    p = 512;
    while (p / 2 > GetNumElements() / 2)
        p = p / 2;
    // summing up objective must be done in one block
    _assignNoiseContrastiveEstimation<ElemType><<<1, p>>>(
        Data(),
        sampleCount,
        m_numRows / 2,
        my_a.Data(),
        a.GetNumCols(),
        my_b.Data(),
        tmp.Data(),
        c.Data());
}

template <class ElemType>
void GPUMatrix<ElemType>::AssignNCEDerivative(GPUMatrix<ElemType>& tmp, const GPUMatrix<ElemType>& a,
                                              const GPUMatrix<ElemType>& b, size_t inputIndex, GPUMatrix<ElemType>& c)
{
    UNCONST(ElemType, a, my_a);
    UNCONST(ElemType, b, my_b);
    SyncGuard syncGuard;
    int p = 512;
    int width = a.GetNumRows();
    while (p / 2 > width)
        p = p / 2;

    _assignNceDerivativeNew<ElemType><<<(tmp.GetNumElements() + p - 1) / p, p>>>(
        Data(),
        tmp.GetNumCols(),
        m_numRows / 2,
        my_a.Data(),
        a.GetNumRows(),
        my_b.Data(),
        tmp.Data(),
        c.Data(),
        inputIndex);
}

template <class ElemType>
void GPUMatrix<ElemType>::AssignSoftmaxSum(const GPUMatrix<ElemType>& a, GPUMatrix<ElemType>& c)
{
    UNCONST(ElemType, a, my_a);
    SyncGuard syncGuard;
    int p = 512;
    int width = a.GetNumRows();
    while (p / 2 > width)
        p = p / 2;

    _assignSoftmaxSum<ElemType><<<1, p>>>(
        my_a.Data(),
        width,
        Data(),
        c.Data());
}

template <class ElemType>
void GPUMatrix<ElemType>::AssignNCEUnnormalizedEval(const GPUMatrix<ElemType>& a, const GPUMatrix<ElemType>& b, GPUMatrix<ElemType>& c)
{
    assert(a.GetComputeDeviceId() == b.GetComputeDeviceId());
    assert(GetNumRows() == a.GetNumRows());
    assert(GetNumCols() == b.GetNumRows());
    assert(a.GetNumCols() == b.GetNumRows());
    UNUSED(a);
    UNUSED(b);
    UNUSED(c); // TODO: this function seems like a stub
    /*
        EnsureAuxMemory();
        int p = 512;
        int width = a.GetNumCols();
        while (p / 2 > width) p = p / 2;

        // this kernel need be launched in nnz blocks
        _sparseInnerProductDenseTimesDense<ElemType> << <m_nz, p >> >(
        m_dVal,
        m_buf,
        m_dCol,
        m_nz,
        GetNumRows(),
        a.Buffer(),
        b.Buffer(),
        b.GetNumRows(),
        m_res);

        // sum up the results
        _reductionSum32<ElemType> << <1, 32 >> >(m_res, c.Buffer(), m_nz);*/
}

DEF_ELEMWISE_INPLACE_FUNC(Tanh)
DEF_ELEMWISE_ASSIGN_FUNC(Tanh)

template <class ElemType>
GPUMatrix<ElemType>& GPUMatrix<ElemType>::InplaceLogSoftmax(const bool isColWise)
{
    if (IsEmpty())
        LogicError("InplaceLogSoftmax: Matrix is empty.");

    PrepareDevice();
    if (isColWise)
    {
        CUDA_LONG N = (CUDA_LONG) GetNumCols(); // one kernel per column
        int blocksPerGrid = (int) ceil(N * 1.0 / GridDim::maxThreadsPerBlock);
        SyncGuard syncGuard;
        _logSoftMaxColWise<<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(Data(), (CUDA_LONG) m_numCols, (CUDA_LONG) m_numRows);
    }
    else
    {
        CUDA_LONG N = (CUDA_LONG) GetNumRows(); // one kernel per column
        int blocksPerGrid = (int) ceil(N * 1.0 / GridDim::maxThreadsPerBlock);
        SyncGuard syncGuard;
        _logSoftMaxRowWise<<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(Data(), (CUDA_LONG) m_numCols, (CUDA_LONG) m_numRows);
    }
    return *this;
}

template <class ElemType>
GPUMatrix<ElemType>& GPUMatrix<ElemType>::AssignLogSoftmaxOf(const GPUMatrix<ElemType>& a, const bool isColWise)
{
    RequireSize(a.GetNumRows(), a.GetNumCols());
    if (isColWise)
    {
        PrepareDevice();
        CUDA_LONG N = (CUDA_LONG) GetNumCols();
        CUDA_LONG M = (CUDA_LONG) GetNumRows();
        SyncGuard syncGuard;
        _assignColumnwiseLogSoftmaxOf<<<N, 512, 0, t_stream>>>(a.Data(), Data(), N, M);
    }
    else
    {
        NOT_IMPLEMENTED;
    }

    return *this;
}

template <class ElemType>
GPUMatrix<ElemType>& GPUMatrix<ElemType>::InplaceHardmax(const bool isColWise)
{
    return AssignHardmaxOf(*this, isColWise);
}

template <class ElemType>
GPUMatrix<ElemType>& GPUMatrix<ElemType>::AssignHardmaxOf(const GPUMatrix<ElemType>& a, const bool isColWise)
{
    RequireSize(a.GetNumRows(), a.GetNumCols());
    if (isColWise)
    {
        PrepareDevice();
        CUDA_LONG N = (CUDA_LONG) GetNumCols();
        CUDA_LONG M = (CUDA_LONG) GetNumRows();
        SyncGuard syncGuard;
        _assignColumnwiseHardmaxOf<<<N, 512, 0, t_stream>>>(a.Data(), Data(), N, M);
    }
    else
    {
        NOT_IMPLEMENTED;
    }

    return *this;
}

DEF_ELEMWISE_INPLACE_FUNC(Sqrt)
DEF_ELEMWISE_ASSIGN_FUNC(Sqrt)

DEF_ELEMWISE_INPLACE_FUNC(Exp)
DEF_ELEMWISE_ASSIGN_FUNC(Exp)

DEF_ELEMWISE_INPLACE_FUNC(Log)
DEF_ELEMWISE_ASSIGN_FUNC(Log)

DEF_ELEMWISE_INPLACE_FUNC(Abs)
DEF_ELEMWISE_ASSIGN_FUNC(Abs)

DEF_ELEMWISE_INPLACE_FUNC(LinearRectifierDerivative)
DEF_ELEMWISE_ASSIGN_FUNC(LinearRectifierDerivative)

DEF_ELEMWISE_INPLACE_FUNC(Cosine)
DEF_ELEMWISE_ASSIGN_FUNC(Cosine)

DEF_ELEMWISE_INPLACE_FUNC(NegativeSine)
DEF_ELEMWISE_ASSIGN_FUNC(NegativeSine)

template <class ElemType>
GPUMatrix<ElemType>& GPUMatrix<ElemType>::InplaceTruncateBottom(const ElemType threshold)
{
    return AssignTruncateBottomOf(*this, threshold);
}

template <class ElemType>
GPUMatrix<ElemType>& GPUMatrix<ElemType>::AssignTruncateBottomOf(const GPUMatrix<ElemType>& a, const ElemType threshold)
{
    if (a.IsEmpty())
        LogicError("AssignTruncateBottomOf: Matrix a is empty.");

    if (this != &a)
    {
        RequireSize(a.GetNumRows(), a.GetNumCols());
    }

    CUDA_LONG N = (CUDA_LONG) GetNumElements();
    int blocksPerGrid = (int) ceil(N * 1.0 / GridDim::maxThreadsPerBlock);
    PrepareDevice();
    SyncGuard syncGuard;
    _assignTruncateBottom<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(Data(), a.Data(), threshold, N);
    return *this;
}

template <class ElemType>
GPUMatrix<ElemType>& GPUMatrix<ElemType>::InplaceTruncateTop(const ElemType threshold)
{
    return AssignTruncateTopOf(*this, threshold);
}

template <class ElemType>
GPUMatrix<ElemType>& GPUMatrix<ElemType>::AssignTruncateTopOf(const GPUMatrix<ElemType>& a, const ElemType threshold)
{
    if (a.IsEmpty())
        LogicError("AssignTruncateTopOf: Matrix a is empty.");

    if (this != &a)
    {
        RequireSize(a.GetNumRows(), a.GetNumCols());
    }

    CUDA_LONG N = (CUDA_LONG) GetNumElements();
    int blocksPerGrid = (int) ceil(N * 1.0 / GridDim::maxThreadsPerBlock);
    a.PrepareDevice();
    SyncGuard syncGuard;
    _assignTruncateTop<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(Data(), a.Data(), threshold, N);
    return *this;
}

template <class ElemType>
GPUMatrix<ElemType>& GPUMatrix<ElemType>::InplaceTruncate(const ElemType threshold)
{
    if (IsEmpty())
        LogicError("InplaceTruncate: Matrix is empty.");

    CUDA_LONG N = (CUDA_LONG) GetNumElements();
    int blocksPerGrid = (int) ceil(N * 1.0 / GridDim::maxThreadsPerBlock);
    PrepareDevice();
    SyncGuard syncGuard;
    _inplaceTruncate<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(Data(), threshold, N);
    return *this;
}

template <class ElemType>
GPUMatrix<ElemType>& GPUMatrix<ElemType>::InplaceSoftThreshold(const ElemType threshold)
{
    if (IsEmpty())
        LogicError("InplaceSoftThreshold: Matrix is empty.");

    CUDA_LONG N = (CUDA_LONG) GetNumElements();
    int blocksPerGrid = (int) ceil(N * 1.0 / GridDim::maxThreadsPerBlock);
    PrepareDevice();
    SyncGuard syncGuard;
    _inplaceSoftThreshold<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(Data(), threshold, N);
    return *this;
}
template <class ElemType>
GPUMatrix<ElemType>& GPUMatrix<ElemType>::SetToZeroIfAbsLessThan(const ElemType threshold)
{
    if (IsEmpty())
        LogicError("SetToZeroIfAbsLessThan: Matrix is empty.");
    CUDA_LONG N = (CUDA_LONG) GetNumElements();
    int blocksPerGrid = (int) ceil(N * 1.0 / GridDim::maxThreadsPerBlock);
    PrepareDevice();
    SyncGuard syncGuard;
    _setToZeroIfAbsLessThan<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(Data(), threshold, N);
    return *this;
}

template <class ElemType>
ElemType GPUMatrix<ElemType>::SumOfAbsElements() const
{
    if (IsEmpty())
        LogicError("SumOfAbsElements: Matrix is empty");

    hipblasHandle_t cuHandle = GetCublasHandle(GetComputeDeviceId());
    if (sizeof(ElemType) == sizeof(float))
    {
        float res = 0;
        CUBLAS_CALL(hipblasSasum(cuHandle, (CUDA_LONG) GetNumElements(), reinterpret_cast<float*>(Data()), 1, &res));
        return res;
    }
    else
    {
        double res = 0;
        CUBLAS_CALL(hipblasDasum(cuHandle, (CUDA_LONG) GetNumElements(), reinterpret_cast<double*>(Data()), 1, &res));
        return ElemType(res);
    }
}

template <class ElemType>
ElemType GPUMatrix<ElemType>::SumOfElements() const
{
    if (IsEmpty())
        LogicError("SumOfElements: Matrix is empty");

    ElemType* d_sum = TracingGPUMemoryAllocator::Allocate<ElemType>(GetComputeDeviceId(), 1);
    ElemType h_sum;

    // WARNING: THIS kernel is not the most efficient way!
    _reductionSum<ElemType><<<1, 1024, 0, t_stream>>>(Data(), d_sum, (CUDA_LONG) GetNumElements());
    CUDA_CALL(hipMemcpy(&h_sum, d_sum, sizeof(ElemType), hipMemcpyDeviceToHost));
    TracingGPUMemoryAllocator::Free<ElemType>(GetComputeDeviceId(), d_sum);
    return h_sum;
}

template <class ElemType>
GPUMatrix<ElemType>& GPUMatrix<ElemType>::AssignSumOfElements(const GPUMatrix<ElemType>& a)
{
    if (a.IsEmpty())
        LogicError("AssignSumOfElements: Matrix a is empty");

    RequireSize(1, 1);

    PrepareDevice();
    SyncGuard syncGuard;
    // WARNING: THIS kernel is not the most efficient way!
    _reductionSumAndAssign<ElemType><<<1, 1024>>>(Data(), a.Data(), (CUDA_LONG) a.GetNumElements(), (CUDA_LONG) GetNumElements());
    return (*this);
}

template <class ElemType>
DeviceBoundNumber<ElemType> GPUMatrix<ElemType>::Sum_AsDeviceBoundNum() const
{
    if (IsEmpty())
        LogicError("Matrix is empty");
    ElemType* d_sum = TracingGPUMemoryAllocator::Allocate<ElemType>(GetComputeDeviceId(), 1);

    // WARNING: THIS kernel is not the most efficient way!
    _reductionSum<ElemType><<<1, 1024, 0, t_stream>>>(Data(), d_sum, (CUDA_LONG) GetNumElements());
    DeviceBoundNumber<ElemType> result;
    result.ShallowCopyFrom(d_sum, GetComputeDeviceId());
    return result;
}

template <class ElemType>
ElemType GPUMatrix<ElemType>::Max() const
{
    hipblasHandle_t cuHandle = GetCublasHandle(GetComputeDeviceId());
    ElemType res;
    if (sizeof(ElemType) == sizeof(float))
    {
        int resInd = 0;
        hipblasIsamax(cuHandle, (CUDA_LONG) GetNumElements(), reinterpret_cast<float*>(Data()), 1, &resInd);
        resInd--;
        CUDA_CALL(hipMemcpy(reinterpret_cast<float*>(&res), reinterpret_cast<float*>(Data()+ resInd), sizeof(float), hipMemcpyDeviceToHost));
        return res;
    }
    else
    {
        int resInd = 0;
        hipblasIdamax(cuHandle, (CUDA_LONG) GetNumElements(), reinterpret_cast<double*>(Data()), 1, &resInd);
        resInd--;
        CUDA_CALL(hipMemcpy(reinterpret_cast<double*>(&res), Data()+ resInd, sizeof(float), hipMemcpyDeviceToHost));
        return res;
    }
}

template <class ElemType>
GPUMatrix<ElemType>& GPUMatrix<ElemType>::ElementMultiplyWith(const GPUMatrix<ElemType>& a)
{
    if (IsEmpty() || a.IsEmpty())
        LogicError("ElementMultiplyWith: Matrix is empty.");

    GPUMatrix<ElemType>& us = *this;
    assert(us.GetNumRows() == a.GetNumRows() && us.GetNumCols() == a.GetNumCols());
    if (us.GetNumRows() != a.GetNumRows() || us.GetNumCols() != a.GetNumCols())
        InvalidArgument("The matrix dimensions do not match.");

    CUDA_LONG N = (CUDA_LONG) GetNumElements();
    int blocksPerGrid = (int) ceil(((double) N) / GridDim::maxThreadsPerBlock);
    a.PrepareDevice();
    SyncGuard syncGuard;
    _elemMul<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(Data(), a.Data(), N);
    return *this;
}

template <class ElemType>
GPUMatrix<ElemType>& GPUMatrix<ElemType>::AssignElementProductOf(const GPUMatrix<ElemType>& a, const GPUMatrix<ElemType>& b)
{
    if (a.IsEmpty() || b.IsEmpty())
        LogicError("AssignElementProductOf: Matrix is empty.");

    assert(a.GetNumRows() == b.GetNumRows() && a.GetNumCols() == b.GetNumCols());
    if (!(a.GetNumRows() == b.GetNumRows() && a.GetNumCols() == b.GetNumCols()))
        InvalidArgument("The input matrix dimensions do not match.");

    RequireSize(a.GetNumRows(), a.GetNumCols());
    CUDA_LONG N = (CUDA_LONG) GetNumElements();
    int blocksPerGrid = (int) ceil(((double) N) / GridDim::maxThreadsPerBlock);
    a.PrepareDevice();
    SyncGuard syncGuard;
    _assignElementProductOf<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(Data(), a.Data(), b.Data(), N);
    return *this;
}

template <class ElemType>
GPUMatrix<ElemType>& GPUMatrix<ElemType>::ElementDivideBy(const GPUMatrix<ElemType>& a)
{
    return AssignElementDivisionOf(*this, a);
}

template <class ElemType>
GPUMatrix<ElemType>& GPUMatrix<ElemType>::AssignElementDivisionOf(const GPUMatrix<ElemType>& a, const GPUMatrix<ElemType>& b)
{
    if (a.IsEmpty() || b.IsEmpty())
        LogicError("AssignElementDivisionOf: Matrix is empty.");

    assert(a.GetNumRows() == b.GetNumRows() && a.GetNumCols() == b.GetNumCols());
    if (!(a.GetNumRows() == b.GetNumRows() && a.GetNumCols() == b.GetNumCols()))
        InvalidArgument("The input matrix dimensions do not match.");

    RequireSize(a.GetNumRows(), a.GetNumCols());
    CUDA_LONG N = (CUDA_LONG) GetNumElements();
    int blocksPerGrid = (int) ceil(((double) N) / GridDim::maxThreadsPerBlock);
    a.PrepareDevice();
    SyncGuard syncGuard;
    _assignElementDivisionOf<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(Data(), a.Data(), b.Data(), N);
    return *this;
}

template <class ElemType>
bool GPUMatrix<ElemType>::IsEqualTo(const GPUMatrix<ElemType>& a, const ElemType threshold /*= 1e-8*/) const
{
    return AreEqual(*this, a, threshold);
}

template <class ElemType>
void GPUMatrix<ElemType>::VectorSum(const GPUMatrix<ElemType>& a, GPUMatrix<ElemType>& c, const bool isColWise)
{
    if (a.GetComputeDeviceId() != c.GetComputeDeviceId())
    {
        InvalidArgument("All matrices must be on the same GPU");
    }

    a.PrepareDevice();

    if (a.IsEmpty())
        LogicError("VectorSum:  Input matrix is empty.");

    const CUDA_LONG n = (CUDA_LONG) a.GetNumRows();
    const CUDA_LONG m = (CUDA_LONG) a.GetNumCols();
    assert(m > 0 && n > 0); // converting from size_t to int may cause overflow

    int blocksPerGrid = 0;
    if (isColWise) // col-wise
    {
        c.RequireSize(1, m);
        blocksPerGrid = (int) ceil(1.0 * m / GridDim::maxThreadsPerBlock);
    }
    else
    {
        c.RequireSize(n, 1);
        blocksPerGrid = (int) ceil(1.0 * n / GridDim::maxThreadsPerBlock);
    }

    SyncGuard syncGuard;
    _vectorSum<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(c.Data(), a.Data(), n, m, isColWise);
}
template <class ElemType>
void GPUMatrix<ElemType>::VectorNorm1(GPUMatrix<ElemType>& c, const bool isColWise) const
{
    if (IsEmpty())
        LogicError("VectorNorm1: Matrix is empty.");

    const CUDA_LONG n = (CUDA_LONG) GetNumRows();
    const CUDA_LONG m = (CUDA_LONG) GetNumCols();
    assert(m > 0 && n > 0); // converting from size_t to int may cause overflow

    PrepareDevice();
    c.ChangeDeviceTo(GetComputeDeviceId());

    int blocksPerGrid = 0;
    if (isColWise) // col-wise
    {
        c.RequireSize(1, m);
        blocksPerGrid = (int) ceil(1.0 * m / GridDim::maxThreadsPerBlock);
    }
    else
    {
        c.RequireSize(n, 1);
        blocksPerGrid = (int) ceil(1.0 * n / GridDim::maxThreadsPerBlock);
    }

    SyncGuard syncGuard;
    _vectorNorm1<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(c.Data(), Data(), n, m, isColWise);
}

template <class ElemType>
GPUMatrix<ElemType>& GPUMatrix<ElemType>::AssignVectorNorm1Of(GPUMatrix<ElemType>& a, const bool isColWise)
{
    a.VectorNorm1(*this, isColWise);
    return *this;
}

template <class ElemType>
void GPUMatrix<ElemType>::VectorNorm2(GPUMatrix<ElemType>& c, const bool isColWise) const
{
    if (IsEmpty())
        LogicError("VectorNorm2: Matrix is empty.");

    const CUDA_LONG n = (CUDA_LONG) GetNumRows();
    const CUDA_LONG m = (CUDA_LONG) GetNumCols();
    assert(m > 0 && n > 0); // converting from size_t to int may cause overflow

    PrepareDevice();
    c.ChangeDeviceTo(GetComputeDeviceId());

    int blocksPerGrid = 0;
    if (isColWise) // col-wise
    {
        c.RequireSize(1, m);
        blocksPerGrid = (int) ceil(1.0 * m / GridDim::maxThreadsPerBlock);
    }
    else
    {
        c.RequireSize(n, 1);
        c.ChangeDeviceTo(GetComputeDeviceId());
        blocksPerGrid = (int) ceil(1.0 * n / GridDim::maxThreadsPerBlock);
    }

    SyncGuard syncGuard;
    _vectorNorm2<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(c.Data(), Data(), n, m, isColWise);
}

template <class ElemType>
GPUMatrix<ElemType>& GPUMatrix<ElemType>::AssignVectorNorm2Of(GPUMatrix<ElemType>& a, const bool isColWise)
{
    a.VectorNorm2(*this, isColWise);
    return *this;
}

template <class ElemType>
void GPUMatrix<ElemType>::VectorNormInf(GPUMatrix<ElemType>& c, const bool isColWise) const
{
    if (IsEmpty())
        LogicError("VectorMax: Matrix is empty.");

    // this implementation is not efficient
    GPUMatrix<ElemType> tmp(GetComputeDeviceId());
    GPUMatrix<ElemType> tmp1(GetComputeDeviceId());
    tmp.AssignAbsOf((*this));
    tmp.VectorMax(tmp1, c, isColWise);
}

template <class ElemType>
GPUMatrix<ElemType>& GPUMatrix<ElemType>::AssignVectorNormInfOf(GPUMatrix<ElemType>& a, const bool isColWise)
{
    a.VectorNormInf(*this, isColWise);
    return *this;
}

template <class ElemType>
GPUMatrix<ElemType>& GPUMatrix<ElemType>::AssignInnerProductOf(const GPUMatrix<ElemType>& a, const GPUMatrix<ElemType>& b, const bool isColWise)
{
    InnerProduct(a, b, *this, isColWise);
    return *this;
}

template <class ElemType>
GPUMatrix<ElemType>& GPUMatrix<ElemType>::AssignKhatriRaoProductOf(const GPUMatrix<ElemType>& a, const GPUMatrix<ElemType>& b)
{
    if (a.IsEmpty() || b.IsEmpty())
        LogicError("AssignKhatriRaoProductOf: Matrix is empty.");

    CUDA_LONG cols = a.GetNumCols();
    assert(cols == b.GetNumCols());
    if (!(cols == b.GetNumCols()))
        InvalidArgument("AssignKhatriRaoProductOf: The input matrix dimensions do not match.");

    CUDA_LONG rowsA = (CUDA_LONG) a.GetNumRows();
    CUDA_LONG rowsB = (CUDA_LONG) b.GetNumRows();
    RequireSize(rowsA * rowsB, cols);
    float N = (float) GetNumElements();
    int blocksPerGrid = (int) ceil(N / GridDim::maxThreadsPerBlock);
    a.PrepareDevice();
    SyncGuard syncGuard;
    _assignKhatriRaoProductOf<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(Data(), a.Data(), b.Data(), rowsA, rowsB, cols);
    return *this;
}

//column-wise reshaped product. Used to compute KhatriRaoProduct Gradient
//   this = reshape each column of a from (K1xK2,1) to (K1, K2)
//   if each column of a is not transposed, each (K1, K2) times each column of b (K2, frames).
//   the output is a (K1, frames) matrix
//   if each column of a is tranposed, each (K1, K2)^T times each column of b(K1, frames) and output is (K2, frames)
template <class ElemType>
GPUMatrix<ElemType>& GPUMatrix<ElemType>::AddColumnReshapeProductOf(const GPUMatrix<ElemType>& a, const GPUMatrix<ElemType>& b, const bool transposeAColumn)
{
    if (a.IsEmpty() || b.IsEmpty())
        LogicError("AddColumnReshapeProductOf: Matrix is empty.");

    CUDA_LONG cols = a.GetNumCols();
    assert(cols == b.GetNumCols());
    if (!(cols == b.GetNumCols()))
        InvalidArgument("AddColumnReshapeProductOf: The input matrix dimensions do not match.");

    CUDA_LONG rowsA = (CUDA_LONG) a.GetNumRows();
    CUDA_LONG rowsB = (CUDA_LONG) b.GetNumRows();
    if (rowsA % rowsB != 0)
        InvalidArgument("AddColumnReshapeProductOf: number of rows in a should be multiples of that in b.");

    CUDA_LONG rowsC = rowsA / rowsB;
    if (rowsC != GetNumRows() || cols != GetNumCols())
        InvalidArgument("AddColumnReshapeProductOf: This matrix does not have the right size.");

    float N = (float) GetNumElements();
    int blocksPerGrid = (int) ceil(N / GridDim::maxThreadsPerBlock);
    a.PrepareDevice();
    SyncGuard syncGuard;
    _addColumnReshapeProductOf<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(Data(), a.Data(), b.Data(), rowsB, rowsC, cols, transposeAColumn);
    return *this;
}

template <class ElemType>
GPUMatrix<ElemType>& GPUMatrix<ElemType>::AddWithScaleOf(ElemType alpha, const GPUMatrix<ElemType>& a)
{
    ScaleAndAdd(alpha, a, *this);
    return *this;
}

template <class ElemType>
ElemType GPUMatrix<ElemType>::FrobeniusNorm() const
{
    if (IsEmpty())
        LogicError("FrobeniusNorm: Matrix is empty.");

    ElemType* d_sum = TracingGPUMemoryAllocator::Allocate<ElemType>(GetComputeDeviceId(), 1);

    ElemType h_sum = 0;
    // WARNING: THIS kernel is not the most efficient way!
    _reductionSum2<ElemType><<<1, 1024, 0, t_stream>>>(Data(), d_sum, (CUDA_LONG) GetNumElements(), true);
    CUDA_CALL(hipMemcpy(&h_sum, d_sum, sizeof(ElemType), hipMemcpyDeviceToHost));
    TracingGPUMemoryAllocator::Free<ElemType>(GetComputeDeviceId(), d_sum);

    return (h_sum);
}

template <class ElemType>
GPUMatrix<ElemType>& GPUMatrix<ElemType>::AssignFrobeniusNormOf(const GPUMatrix<ElemType>& a)
{
    if (a.IsEmpty())
        LogicError("AssignFrobeniusNormOf: Matrix a is empty.");

    RequireSize(1, 1);

    PrepareDevice();
    // WARNING: THIS kernel is not the most efficient way!
    _reductionSum2<ElemType><<<1, 1024, 0, t_stream>>>(a.Data(), Data(), (CUDA_LONG) a.GetNumElements(), true);

    return *this;
}

template <class ElemType>
ElemType GPUMatrix<ElemType>::MatrixNormInf() const
{
    if (IsEmpty())
        LogicError("MatrixNorm1: Matrix is empty.");

    ElemType* d_maxAbs = TracingGPUMemoryAllocator::Allocate<ElemType>(GetComputeDeviceId(), 1);

    ElemType h_maxAbs = 0;
    // WARNING: THIS kernel is not the most efficient way!
    _reductionMatrixNormInf<ElemType><<<1, 1024, 0, t_stream>>>(Data(), d_maxAbs, (CUDA_LONG) GetNumElements());
    CUDA_CALL(hipMemcpy(&h_maxAbs, d_maxAbs, sizeof(ElemType), hipMemcpyDeviceToHost));
    TracingGPUMemoryAllocator::Free<ElemType>(GetComputeDeviceId(), d_maxAbs);
    return h_maxAbs;
}

template <class ElemType>
ElemType GPUMatrix<ElemType>::MatrixNorm1() const
{
    if (IsEmpty())
        LogicError("MatrixNorm1: Matrix is empty.");
    return SumOfAbsElements();
}

template <class ElemType>
ElemType GPUMatrix<ElemType>::MatrixNorm0() const
{
    if (IsEmpty())
        LogicError("MatrixNorm0: Matrix is empty.");

    ElemType* d_nz = TracingGPUMemoryAllocator::Allocate<ElemType>(GetComputeDeviceId(), 1);
    ElemType h_nz = 0;
    // WARNING: THIS kernel is not the most efficient way!
    _reductionMatrixNorm0<ElemType><<<1, 1024, 0, t_stream>>>(Data(), d_nz, (CUDA_LONG) GetNumElements());
    CUDA_CALL(hipMemcpy(&h_nz, d_nz, sizeof(ElemType), hipMemcpyDeviceToHost));
    TracingGPUMemoryAllocator::Free<ElemType>(GetComputeDeviceId(), d_nz);
    return h_nz;
}

template <class ElemType>
GPUMatrix<ElemType>& GPUMatrix<ElemType>::AssignSignOf(const GPUMatrix<ElemType>& a)
{
    if (a.IsEmpty())
        LogicError("AssignSignOf: Matrix a is empty.");

    if (this != &a)
        RequireSize(a.GetNumRows(), a.GetNumCols());

    PrepareDevice();
    int blocksPerGrid = (int) ceil(1.0 * GetNumElements() / GridDim::maxThreadsPerBlock);
    SyncGuard syncGuard;
    _assignSignOf<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(Data(), a.Data(), (CUDA_LONG) GetNumElements());
    return *this;
}

template <class ElemType>
GPUMatrix<ElemType>& GPUMatrix<ElemType>::AddSignOf(const GPUMatrix<ElemType>& a)
{
    if (a.IsEmpty())
        LogicError("AddSignOf: Matrix a is empty.");

    if (this != &a)
        RequireSize(a.GetNumRows(), a.GetNumCols());

    PrepareDevice();
    int blocksPerGrid = (int) ceil(1.0 * GetNumElements() / GridDim::maxThreadsPerBlock);
    SyncGuard syncGuard;
    _addSignOf<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(Data(), a.Data(), (CUDA_LONG) GetNumElements());
    return *this;
}

template <class ElemType>
void GPUMatrix<ElemType>::VectorMax(GPUMatrix<ElemType>& maxIndexes, GPUMatrix<ElemType>& maxValues, const bool isColWise) const
{
    if (IsEmpty())
        LogicError("VectorMax: Matrix is empty.");

    const GPUMatrix<ElemType>& us = *this;
    const CUDA_LONG m = (CUDA_LONG) GetNumRows();
    const CUDA_LONG n = (CUDA_LONG) GetNumCols();
    assert(m > 0 && n > 0); // converting from size_t to int may cause overflow

    PrepareDevice();
    SyncGuard syncGuard;
    if (isColWise)
    {
        maxValues.RequireSize(1, n);
        maxIndexes.RequireSize(1, n);

        int blocksPerGrid = n; // we'll have 1 block processing 1 column
        _vectorMaxMinReduce<ElemType, true><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(us.Data(), maxIndexes.Data(), maxValues.Data(), m, n);

        /*int blocksPerGrid=(int)ceil(1.0*n/GridDim::maxThreadsPerBlock);
            _vectorMax<ElemType><<<blocksPerGrid,GridDim::maxThreadsPerBlock,0,t_stream>>>(us.Data(),maxIndexes.Data(),maxValues.Data(),m,n,isColWise);*/
    }
    else
    {
        maxValues.RequireSize(m, 1);
        maxIndexes.RequireSize(m, 1);
        int blocksPerGrid = (int) ceil(1.0 * m / GridDim::maxThreadsPerBlock);
        _vectorMax<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(us.Data(), maxIndexes.Data(), maxValues.Data(), m, n, isColWise);
    }
}

__global__ void _initIndicesForSort(uint64_t* indexes, CUDA_LONG crow, CUDA_LONG ccol)
{
    CUDA_LONG id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id >= crow * ccol)
        return;
    uint32_t irow = id % crow;
    uint32_t icol = id / crow;
    indexes[id] = (static_cast<uint64_t>(irow) << 32) | icol;
}

template <class ElemType>
void GPUMatrix<ElemType>::VectorMax(GPUMatrix<ElemType>& maxIndexes, GPUMatrix<ElemType>& maxValues, const bool isColWise, int topK) const
{
    if (IsEmpty())
        LogicError("VectorMax: Matrix is empty.");

    if (topK == 1)
    {
        VectorMax(maxIndexes, maxValues, isColWise);
        return;
    }

    if (!isColWise)
        RuntimeError("Row-wise TopK max is not supported.");

    const GPUMatrix<ElemType>& us = *this;
    const CUDA_LONG m = (CUDA_LONG) GetNumRows();
    const CUDA_LONG n = (CUDA_LONG) GetNumCols();
    assert(topK <= m);
    assert(m > 0 && n > 0); // converting from size_t to int may cause overflow

    PrepareDevice();
    SyncGuard syncGuard;
    maxValues.RequireSize(topK, n);
    maxIndexes.RequireSize(topK, n);

    // To sort matrix columns we use 2-pass _stable_ sort algorithm:
    // 1. Sort by values (descending) with corresponding row/col indexes.
    // 2. Sort by col indices (ascending) with corresponding values/row indices.
    // Indices are stored as 64-bit ints where low 32 bits represent column and high 32 bits - row index.
    // On the second pass only first 32 bits of the index are used in sorting, so SortPairs has
    // begin_bit and end_bit set accordingly.

    CUDA_LONG celt = static_cast<CUDA_LONG>(GetNumElements());
    ElemType* inVal = us.Data();
    ElemType* outVal1 = nullptr;
    ElemType* outVal2 = nullptr;
    uint64_t* inIdx = nullptr;
    uint64_t* outIdx = nullptr;
    // Determine temp buffer size needed for SortPairsDescending to sort values on the first pass.
    size_t cbtemp = 0;
    // If first param is nullptr then no actual work is done except writing result to cbtemp.
    CUDA_CALL(hipcub::DeviceRadixSort::SortPairsDescending(nullptr, cbtemp, inVal, outVal1, inIdx, outIdx, celt, 0, sizeof(ElemType) * 8, t_stream));
    size_t ctemp1 = (cbtemp + sizeof(ElemType) - 1) / sizeof(ElemType);
    // Determine temp buffer size needed for SortPairs to sort indices on the second pass.
    cbtemp = 0;
    CUDA_CALL(hipcub::DeviceRadixSort::SortPairs(nullptr, cbtemp, outIdx, inIdx, outVal1, outVal2, celt, 0, 32, t_stream));
    size_t ctemp2 = (cbtemp + sizeof(ElemType) - 1) / sizeof(ElemType);
    size_t ctemp = std::max(ctemp1, ctemp2);
    cbtemp = ctemp * sizeof(ElemType);
    // ElemType count needed to store indices, accounting for natural alignment for uint64_t type.
    size_t cidx = ((celt + 1) * sizeof(uint64_t) - 1 + sizeof(ElemType) - 1) / sizeof(ElemType);
    // Get temp workspace.
    auto workspace = GetOrCreateWorkspace();
    // RequireSize to store: output values for the 1st and 2nd passes, input indices, output indices, and temp storage.
    workspace->RequireSize(m, 2 * n + (2 * cidx + ctemp + m - 1) / m);
    outVal1 = workspace->Data();
    outVal2 = outVal1 + celt;
    inIdx = reinterpret_cast<uint64_t*>(outVal2 + celt);
    // Align indices pointer if needed.
    size_t cbAlign = reinterpret_cast<size_t>(inIdx) % sizeof(uint64_t);
    if (cbAlign != 0)
        reinterpret_cast<uint8_t*&>(inIdx) += sizeof(uint64_t) - cbAlign;
    outIdx = inIdx + celt;
    void* ptmp = outIdx + celt;
    assert(reinterpret_cast<ElemType*>(reinterpret_cast<uint8_t*>(ptmp) + cbtemp) <= workspace->Data() + workspace->GetNumElements());

    // Initialize indices.
    const int ThreadsPerBlock = 128;
    int cblock = (celt + ThreadsPerBlock - 1) / ThreadsPerBlock;
    _initIndicesForSort<<<cblock, ThreadsPerBlock, 0, t_stream>>>(inIdx, m, n);
    // Sort by values.
    CUDA_CALL(hipcub::DeviceRadixSort::SortPairsDescending(ptmp, cbtemp, inVal, outVal1, inIdx, outIdx, celt, 0, sizeof(ElemType) * 8, t_stream));
    // Sort by column indices. outIdx contains indices after the first pass so it's used as an input.
    CUDA_CALL(hipcub::DeviceRadixSort::SortPairs(ptmp, cbtemp, outIdx, inIdx, outVal1, outVal2, celt, 0, 32, t_stream));
    // Copy results.
    cblock = (topK * n + ThreadsPerBlock - 1) / ThreadsPerBlock;
    _copyTopKResults<<<cblock, ThreadsPerBlock, 0, t_stream>>>(inIdx, outVal2, maxIndexes.Data(), maxValues.Data(), m, n, topK);

    ReleaseWorkspace(std::move(workspace));

}

template <class ElemType>
void GPUMatrix<ElemType>::VectorMin(GPUMatrix<ElemType>& minIndexes, GPUMatrix<ElemType>& minValues, const bool isColWise) const
{
    if (IsEmpty())
        LogicError("VectorMax: Matrix is empty.");

    const GPUMatrix<ElemType>& us = *this;
    const int m = (int) GetNumRows();
    const int n = (int) GetNumCols();

    assert(m > 0 && n > 0); // converting from size_t to int may cause overflow
    PrepareDevice();
    SyncGuard syncGuard;
    if (isColWise)
    {
        minValues.RequireSize(1, n);
        minIndexes.RequireSize(1, n);

        int blocksPerGrid = n; // we'll have 1 block processing 1 column
        _vectorMaxMinReduce<ElemType, false><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(us.Data(), minIndexes.Data(), minValues.Data(), m, n);

        /*
            int blocksPerGrid=(int)ceil(1.0*n/GridDim::maxThreadsPerBlock);
            _vectorMin<ElemType><<<blocksPerGrid,GridDim::maxThreadsPerBlock,0,t_stream>>>(us.Data(),minIndexes.Data(),minValues.Data(),m,n,isColWise);*/
    }
    else
    {
        minValues.RequireSize(m, 1);
        minIndexes.RequireSize(m, 1);
        int blocksPerGrid = (int) ceil(1.0 * m / GridDim::maxThreadsPerBlock);
        _vectorMin<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(us.Data(), minIndexes.Data(), minValues.Data(), m, n, isColWise);
    }
}

template <class ElemType>
GPUMatrix<ElemType>& GPUMatrix<ElemType>::AssignNumOfDiff(const GPUMatrix<ElemType>& a, const GPUMatrix<ElemType>& b, bool searchInCol)
{
    if (a.GetNumCols() != b.GetNumCols())
        InvalidArgument("AssignNumOfDiff: a and b must have the same number of columns.");
    if (!searchInCol && a.GetNumRows() != b.GetNumRows())
        InvalidArgument("AssignNumOfDiff: a and b must have the same number of rows.");

    RequireSize(1, 1); // result should be one element

    PrepareDevice();
    SyncGuard syncGuard;
    if (!searchInCol)
    {
        // int blocksPerGrid=(int)ceil(1.0*a.GetNumElements()/GridDim::maxThreadsPerBlock);
        // _assignNumOfDiff<ElemType><<<blocksPerGrid,GridDim::maxThreadsPerBlock,0,t_stream>>>(a.Data(), b.Data(), Data(), a.GetNumElements());
        _assignNumOfDiff<ElemType><<<1, 1024, 0, t_stream>>>(a.Data(), b.Data(), Data(), (CUDA_LONG) a.GetNumElements());
    }
    else
    {
        const int blockSize = 1024;
        _assignNumOfDiffCol<blockSize><<<1, blockSize, 0, t_stream>>>(a.Data(), b.Data(), Data(),
                                                                      static_cast<CUDA_LONG>(b.GetNumRows()), static_cast<CUDA_LONG>(a.GetNumCols()));
    }
    return *this;
}

#pragma endregion Member BLAS Functions

#pragma region Other helper functions
template <class ElemType>
void GPUMatrix<ElemType>::Print(const char* /*matrixName*/, size_t /*rowStart*/, size_t /*rowEnd*/, size_t /*colStart*/, size_t /*colEnd*/) const
{
    NOT_IMPLEMENTED;
}

template <class ElemType>
void GPUMatrix<ElemType>::Print(const char* matrixName /*=nullptr*/) const
{
    Print(matrixName, 0, GetNumRows() - 1, 0, GetNumCols() - 1);
}

//helpfer function used for convolution neural network
template <class ElemType>
GPUMatrix<ElemType>& GPUMatrix<ElemType>::AssignPackedConvolutionInput(const GPUMatrix<ElemType>& inputSubBatch,
                                                                       const size_t inputWidth, const size_t inputHeight, const size_t inputChannels,
                                                                       const size_t outputWidth, const size_t outputHeight, const size_t outputChannels,
                                                                       const size_t kernelWidth, const size_t kernelHeight, const size_t horizontalSubsample, const size_t verticalSubsample,
                                                                       const bool zeroPadding)
{
    assert(verticalSubsample <= kernelHeight && horizontalSubsample <= kernelWidth);

    size_t packedInputRows = kernelWidth * kernelHeight * inputChannels;
    size_t packedInputColsPerSample = outputWidth * outputHeight;
    size_t smallBatchSize = inputSubBatch.GetNumCols();
    RequireSize(packedInputRows, packedInputColsPerSample * smallBatchSize);
    if (zeroPadding)
        SetValue((ElemType) 0);

    PrepareDevice();
    int numThreadPerBlock = GridDim::maxThreadsPerBlock;
#if 1
    int blocksPerGrid = (smallBatchSize * inputWidth * inputHeight * inputChannels + numThreadPerBlock - 1) / numThreadPerBlock;
#else
    dim3 blocksPerGrid((inputWidth * inputHeight * inputChannels + numThreadPerBlock - 1) / numThreadPerBlock, smallBatchSize);
#endif
    SyncGuard syncGuard;
    _assignPackedConvolutionInput<<<blocksPerGrid, numThreadPerBlock, 0, t_stream>>>(Data(),
                                                                                     inputSubBatch.Data(),
                                                                                     smallBatchSize,
                                                                                     inputWidth, inputHeight, inputChannels,
                                                                                     outputWidth, outputHeight, outputChannels,
                                                                                     kernelWidth, kernelHeight, horizontalSubsample, verticalSubsample, zeroPadding);

    return *this;
}

//helpfer function used for convolution neural network
template <class ElemType>
GPUMatrix<ElemType>& GPUMatrix<ElemType>::UnpackConvolutionInput(GPUMatrix<ElemType>& inputSubBatch,
                                                                 const size_t inputWidth, const size_t inputHeight, const size_t inputChannels,
                                                                 const size_t outputWidth, const size_t outputHeight, const size_t outputChannels,
                                                                 const size_t kernelWidth, const size_t kernelHeight, const size_t horizontalSubsample, const size_t verticalSubsample,
                                                                 const bool zeroPadding) const
{
    assert(verticalSubsample <= kernelHeight && horizontalSubsample <= kernelWidth);

    size_t smallBatchSize = inputSubBatch.GetNumCols();

    PrepareDevice();
    int numThreadPerBlock = GridDim::maxThreadsPerBlock;
#if 1
    int blocksPerGrid = (smallBatchSize * inputWidth * inputHeight * inputChannels + numThreadPerBlock - 1) / numThreadPerBlock;
#else
    dim3 blocksPerGrid((inputWidth * inputHeight * inputChannels + numThreadPerBlock - 1) / numThreadPerBlock, smallBatchSize);
#endif
    SyncGuard syncGuard;
    _unpackConvolutionInput<<<blocksPerGrid, numThreadPerBlock, 0, t_stream>>>(Data(),
                                                                               inputSubBatch.Data(),
                                                                               smallBatchSize,
                                                                               inputWidth, inputHeight, inputChannels,
                                                                               outputWidth, outputHeight, outputChannels,
                                                                               kernelWidth, kernelHeight, horizontalSubsample, verticalSubsample, zeroPadding);

    return inputSubBatch;
}

template <class ElemType>
GPUMatrix<ElemType>& GPUMatrix<ElemType>::AssignMaxPoolingResult(const GPUMatrix<ElemType>& inputBatch, const size_t channels,
                                                                 const size_t inputWidth, const size_t inputHeight, const size_t inputSizePerSample,
                                                                 const size_t outputWidth, const size_t outputHeight, const size_t outputSizePerSample,
                                                                 const size_t windowWidth, const size_t windowHeight, const size_t horizontalSubsample, const size_t verticalSubsample)
{
    assert(verticalSubsample <= windowHeight && horizontalSubsample <= windowWidth);

    unsigned int batchSize = inputBatch.GetNumCols();
    RequireSize(outputSizePerSample, batchSize);

    int numThreadPerBlock = GridDim::maxThreadsPerBlock;
    int blocksPerGrid = (batchSize * outputSizePerSample + numThreadPerBlock - 1) / numThreadPerBlock;

    PrepareDevice();
    SyncGuard syncGuard;
    _assignMaxPoolingResult<<<blocksPerGrid, numThreadPerBlock, 0, t_stream>>>(Data(), inputBatch.Data(), batchSize, channels,
                                                                               inputWidth, inputHeight, inputSizePerSample,
                                                                               outputWidth, outputHeight, outputSizePerSample,
                                                                               windowWidth, windowHeight, horizontalSubsample, verticalSubsample);

    return *this;
}

template <class ElemType>
GPUMatrix<ElemType>& GPUMatrix<ElemType>::AddMaxPoolingGradient(const GPUMatrix<ElemType>& outputGradientBatch, const GPUMatrix<ElemType>& inputBatch, const GPUMatrix<ElemType>& outputBatch,
                                                                const size_t channels,
                                                                const size_t inputWidth, const size_t inputHeight, const size_t inputSizePerSample,
                                                                const size_t outputWidth, const size_t outputHeight, const size_t outputSizePerSample,
                                                                const size_t windowWidth, const size_t windowHeight, const size_t horizontalSubsample, const size_t verticalSubsample)
{
    assert(verticalSubsample <= windowHeight && horizontalSubsample <= windowWidth);

    unsigned int batchSize = outputGradientBatch.GetNumCols();
    int numThreadPerBlock = GridDim::maxThreadsPerBlock;

    PrepareDevice();
    SyncGuard syncGuard;

    int blocksPerGrid = (batchSize * inputSizePerSample + numThreadPerBlock - 1) / numThreadPerBlock;
    _addMaxPoolingGradient<<<blocksPerGrid, numThreadPerBlock, 0, t_stream>>>(Data(), outputGradientBatch.Data(), inputBatch.Data(), outputBatch.Data(), batchSize, channels,
                                                                              inputWidth, inputHeight, inputSizePerSample,
                                                                              outputWidth, outputHeight, outputSizePerSample,
                                                                              windowWidth, windowHeight, horizontalSubsample, verticalSubsample);

    return *this;
}

template <class ElemType>
GPUMatrix<ElemType>& GPUMatrix<ElemType>::AssignAveragePoolingResult(const GPUMatrix<ElemType>& inputBatch, const size_t channels,
                                                                     const size_t inputWidth, const size_t inputHeight, const size_t inputSizePerSample,
                                                                     const size_t outputWidth, const size_t outputHeight, const size_t outputSizePerSample,
                                                                     const size_t windowWidth, const size_t windowHeight, const size_t horizontalSubsample, const size_t verticalSubsample)
{
    assert(verticalSubsample <= windowHeight && horizontalSubsample <= windowWidth);

    unsigned int batchSize = inputBatch.GetNumCols();
    RequireSize(outputSizePerSample, batchSize);

    int numThreadPerBlock = GridDim::maxThreadsPerBlock;
    int blocksPerGrid = (batchSize * outputSizePerSample + numThreadPerBlock - 1) / numThreadPerBlock;

    PrepareDevice();
    SyncGuard syncGuard;
    _assignAveragePoolingResult<<<blocksPerGrid, numThreadPerBlock, 0, t_stream>>>(Data(), inputBatch.Data(), batchSize, channels,
                                                                                   inputWidth, inputHeight, inputSizePerSample,
                                                                                   outputWidth, outputHeight, outputSizePerSample,
                                                                                   windowWidth, windowHeight, horizontalSubsample, verticalSubsample);

    return *this;
}

template <class ElemType>
GPUMatrix<ElemType>& GPUMatrix<ElemType>::AddAveragePoolingGradient(const GPUMatrix<ElemType>& outputGradientBatch,
                                                                    const size_t channels,
                                                                    const size_t inputWidth, const size_t inputHeight, const size_t inputSizePerSample,
                                                                    const size_t outputWidth, const size_t outputHeight, const size_t outputSizePerSample,
                                                                    const size_t windowWidth, const size_t windowHeight, const size_t horizontalSubsample, const size_t verticalSubsample)
{
    assert(verticalSubsample <= windowHeight && horizontalSubsample <= windowWidth);

    size_t batchSize = outputGradientBatch.GetNumCols();
    int numThreadPerBlock = GridDim::maxThreadsPerBlock;

    PrepareDevice();
    SyncGuard syncGuard;
    size_t blocksPerGrid = (batchSize * inputSizePerSample + numThreadPerBlock - 1) / numThreadPerBlock;
    _addAveragePoolingGradient<<<blocksPerGrid, numThreadPerBlock, 0, t_stream>>>(Data(), outputGradientBatch.Data(), (CUDA_LONG) batchSize, channels,
                                                                                  inputWidth, inputHeight, inputSizePerSample,
                                                                                  outputWidth, outputHeight, outputSizePerSample,
                                                                                  windowWidth, windowHeight, horizontalSubsample, verticalSubsample);

    return *this;
}

#pragma endregion Other helper functions

template <class ElemType>
void GPUMatrix<ElemType>::ConvolutionForward(const GPUMatrix<ElemType>& kernel, const GPUMatrix<int>& mpRowCol, const GPUMatrix<int>& mpRowIwht,
                                             const GPUMatrix<int>& mpRowRun, const GPUMatrix<int>& runs, GPUMatrix<ElemType>& output) const
{
    const int BlockSize = 128;
    auto gdim = dim3((output.GetNumRows() + BlockSize - 1)/ BlockSize, std::min((int)GetNumCols(), 65535));
    PrepareDevice();
    SyncGuard syncGuard;
    kConvolutionForward<<<gdim, BlockSize, 0, t_stream>>>((int)GetNumCols(), kernel.Data(), mpRowCol.Data(), mpRowIwht.Data(), mpRowRun.Data(),
                                                            runs.Data(), Data(), (int)GetNumRows(), output.Data(), (int)output.GetNumRows());
}

template <class ElemType>
void GPUMatrix<ElemType>::ConvolutionBackwardData(const GPUMatrix<ElemType>& kernel, const GPUMatrix<int>& mpRowCol, const GPUMatrix<int>& mpRowIwht,
                                                  const GPUMatrix<int>& mpRowRun, const GPUMatrix<int>& runs, GPUMatrix<ElemType>& grad) const
{
    const int BlockSize = 128;
    auto gdim = dim3((GetNumRows() + BlockSize - 1)/ BlockSize, std::min((int)GetNumCols(), 65535));
    PrepareDevice();
    SyncGuard syncGuard;
    kConvolutionBackwardData<<<gdim, BlockSize, 0, t_stream>>>((int)GetNumCols(), kernel.Data(), mpRowCol.Data(), mpRowIwht.Data(), mpRowRun.Data(),
                                                                 runs.Data(), Data(), (int)GetNumRows(), grad.Data(), (int)grad.GetNumRows());
}

template <class ElemType>
void GPUMatrix<ElemType>::ConvolutionBackwardKernel(const GPUMatrix<ElemType>& in, const GPUMatrix<int>& mpRowCol, const GPUMatrix<int>& mpRowIwht,
                                                    const GPUMatrix<int>& mpRowRun, const GPUMatrix<int>& runs, GPUMatrix<ElemType>& kernelGrad) const
{
    const int BlockSize = 128;
    auto gdim = dim3((GetNumRows() + BlockSize - 1)/ BlockSize, std::min((int)GetNumCols(), 65535));
    PrepareDevice();
    SyncGuard syncGuard;
    kConvolutionBackwardKernel<<<gdim, BlockSize, 0, t_stream>>>((int)GetNumCols(), (int)in.GetNumRows(), (int)GetNumRows(),
                                                                   in.Data(), mpRowCol.Data(), mpRowIwht.Data(), mpRowRun.Data(),
                                                                   runs.Data(), Data(), kernelGrad.Data());
}

template <class ElemType>
void GPUMatrix<ElemType>::MaxPoolingForward(const GPUMatrix<int>& mpRowCol, const GPUMatrix<int>& mpRowIndices, const GPUMatrix<int>& indices, GPUMatrix<ElemType>& output) const
{
    const int BlockSize = 128;
    auto gdim = dim3((output.GetNumRows() + BlockSize - 1)/ BlockSize, std::min((int)GetNumCols(), 65535));
    PrepareDevice();
    SyncGuard syncGuard;
    kMaxPoolingForward<<<gdim, BlockSize, 0, t_stream>>>((int)GetNumCols(), mpRowCol.Data(), mpRowIndices.Data(), indices.Data(),
                                                           Data(), (int)GetNumRows(), output.Data(), (int)output.GetNumRows());
}

template <class ElemType>
void GPUMatrix<ElemType>::MaxPoolingBackward(const GPUMatrix<ElemType>& out, const GPUMatrix<ElemType>& in,
                                             const GPUMatrix<int>& mpRowCol, const GPUMatrix<int>& mpRowIndices, const GPUMatrix<int>& indices,
                                             GPUMatrix<ElemType>& grad) const
{
    const int BlockSize = 128;
    auto gdim = dim3((GetNumRows() + BlockSize - 1)/ BlockSize, std::min((int)GetNumCols(), 65535));
    PrepareDevice();
    SyncGuard syncGuard;
    kMaxPoolingBackward<<<gdim, BlockSize, 0, t_stream>>>((int)GetNumCols(), out.Data(), in.Data(),
                                                            mpRowCol.Data(), mpRowIndices.Data(), indices.Data(),
                                                            Data(), (int)GetNumRows(), grad.Data(), (int)grad.GetNumRows());
}

template <class ElemType>
void GPUMatrix<ElemType>::MaxUnpooling(const GPUMatrix<int>& mpRowCol, const GPUMatrix<int>& mpRowIndices, const GPUMatrix<int>& indices, const GPUMatrix<ElemType>& poolInput, GPUMatrix<ElemType>& input) const
{
    const int BlockSize = 128;
    auto gdim = dim3((GetNumRows() + BlockSize - 1)/ BlockSize, std::min((int)GetNumCols(), 65535));
    PrepareDevice();
    SyncGuard syncGuard;
    kMaxUnpooling<<<gdim, BlockSize, 0, t_stream>>>((int)GetNumCols(), mpRowCol.Data(), mpRowIndices.Data(), indices.Data(),
                                                     Data(), poolInput.Data(), (int)GetNumRows(), input.Data(), (int)input.GetNumRows());
}

template <class ElemType>
void GPUMatrix<ElemType>::AveragePoolingForward(const GPUMatrix<int>& mpRowCol, const GPUMatrix<int>& mpRowIndices, const GPUMatrix<int>& indices, GPUMatrix<ElemType>& output) const
{
    const int BlockSize = 128;
    auto gdim = dim3((output.GetNumRows() + BlockSize - 1)/ BlockSize, std::min((int)GetNumCols(), 65535));
    PrepareDevice();
    SyncGuard syncGuard;
    kAveragePoolingForward<<<gdim, BlockSize, 0, t_stream>>>((int)GetNumCols(), mpRowCol.Data(), mpRowIndices.Data(), indices.Data(),
                                                               Data(), (int)GetNumRows(), output.Data(), (int)output.GetNumRows());
}

template <class ElemType>
void GPUMatrix<ElemType>::AveragePoolingBackward(const GPUMatrix<int>& mpRowCol, const GPUMatrix<int>& mpRowIndices, const GPUMatrix<int>& indices, GPUMatrix<ElemType>& grad) const
{
    const int BlockSize = 128;
    auto gdim = dim3((GetNumRows() + BlockSize - 1)/ BlockSize, std::min((int)GetNumCols(), 65535));
    PrepareDevice();
    SyncGuard syncGuard;
    kAveragePoolingBackward<<<gdim, BlockSize, 0, t_stream>>>((int)GetNumCols(), mpRowCol.Data(), mpRowIndices.Data(), indices.Data(),
                                                                Data(), (int)GetNumRows(), grad.Data(), (int)grad.GetNumRows());
}

template <class ElemType>
void GPUMatrix<ElemType>::BatchNormalizationForward(const GPUMatrix<ElemType>& scale, const GPUMatrix<ElemType>& bias, double expAvgFactor, double blendFactor,
                                                    GPUMatrix<ElemType>& runMean, GPUMatrix<ElemType>& runInvStdDev, GPUMatrix<ElemType>& out, double epsilon,
                                                    GPUMatrix<ElemType>& saveMean, GPUMatrix<ElemType>& saveInvStdDev) const
{
    assert((GetNumRows() % scale.GetNumRows()) == 0);

    bool spatial = GetNumRows() != scale.GetNumRows();
    size_t vectorSize = GetNumRows();
    size_t spatialSize = spatial ? (GetNumRows() / scale.GetNumRows()) : 1;
    size_t batchSize = GetNumCols();

    assert(0 < vectorSize && vectorSize <= std::numeric_limits<int>::max());
    assert(0 < batchSize  && batchSize  <= std::numeric_limits<int>::max());

    SyncGuard syncGuard;
    // If expAvgFactor == 0 && blendFactor == 1 then we don't need to compute current minibatch statistics.
    if (expAvgFactor > 0 || blendFactor < 1)
    {
        if (spatial)
        {
            Call<ComputeSpatialBatchMeanAndInvStdDev, ElemType>(spatialSize, vectorSize, spatialSize, batchSize, Data(),
                                                                expAvgFactor, runMean.Data(), runInvStdDev.Data(), epsilon,
                                                                saveMean.Data(), saveInvStdDev.Data(), GetStream());
        }
        else
        {
            Call<ComputeBatchMeanAndInvStdDev, ElemType>(vectorSize, vectorSize, batchSize, Data(),
                                                         expAvgFactor, runMean.Data(), runInvStdDev.Data(), epsilon,
                                                         saveMean.Data(), saveInvStdDev.Data(), GetStream());
        }
    }
    // When:
    //     blendFactor == 1 - use running mean/var instead of the current minibatch mean/var.
    // 0 < blendFactor <  1 - blend running mean/var with mean/var of the current minibatch: saveMean = (1 - blendFactor) * saveMean + blendFactor * runMean
    //     blendFactor == 0 - use mean/var of the current minibatch.
    if (blendFactor < 1)
    {
        if (blendFactor > 0)
        {
            // REVIEW alexeyk: can be rolled into NormalizeBatchTraining to save bandwidth.
            // TODO: add a 'beta' parameter to ScaleAndAdd()
            Scale((ElemType)(1 - blendFactor), saveMean);
            ScaleAndAdd((ElemType)blendFactor, runMean, saveMean);
            Scale((ElemType)(1 - blendFactor), saveInvStdDev);
            ScaleAndAdd((ElemType)blendFactor, runInvStdDev, saveInvStdDev);
        }
        Call<NormalizeBatchTraining, ElemType>(spatial ? spatialSize : vectorSize, vectorSize, spatialSize, batchSize,
                                               spatial, Data(), out.Data(), scale.Data(), bias.Data(),
                                               saveMean.Data(), saveInvStdDev.Data(), GetStream());
    }
    else
    {
        Call<NormalizeBatchTraining, ElemType>(spatial ? spatialSize : vectorSize, vectorSize, spatialSize, batchSize,
                                               spatial, Data(), out.Data(), scale.Data(), bias.Data(),
                                               runMean.Data(), runInvStdDev.Data(), GetStream());
    }
}

template <class ElemType>
void GPUMatrix<ElemType>::BatchNormalizationBackward(const GPUMatrix<ElemType>& in, GPUMatrix<ElemType>& grad, const GPUMatrix<ElemType>& scale, 
                                                     const GPUMatrix<ElemType>& saveMean, const GPUMatrix<ElemType>& saveInvStdDev,
                                                     GPUMatrix<ElemType>& scaleGrad, GPUMatrix<ElemType>& biasGrad) const
{
    assert((GetNumRows() % scale.GetNumRows()) == 0);

    bool spatial = GetNumRows() != scale.GetNumRows();
    size_t vectorSize = GetNumRows();
    size_t spatialSize = spatial ? (GetNumRows() / scale.GetNumRows()) : 1;
    size_t batchSize = GetNumCols();

    assert(0 < vectorSize && vectorSize <= std::numeric_limits<int>::max());
    assert(0 < batchSize  && batchSize  <= std::numeric_limits<int>::max());

    SyncGuard syncGuard;
    if (spatial)
    {
        Call<ComputeSpatialScaleAndBiasGradients, ElemType>(spatialSize, vectorSize, spatialSize, batchSize, in.Data(), Data(), scaleGrad.Data(), biasGrad.Data(),
                                                            saveMean.Data(), saveInvStdDev.Data(), GetStream());
    }
    else
    {
        Call<ComputeScaleAndBiasGradients, ElemType>(vectorSize, vectorSize, batchSize, in.Data(), Data(), scaleGrad.Data(), biasGrad.Data(),
                                                     saveMean.Data(), saveInvStdDev.Data(), GetStream());
    }
    Call<BackpropagateBatchNormGradients, ElemType>(spatial ? spatialSize : vectorSize, vectorSize, spatialSize, batchSize, spatial,
                                                    in.Data(), Data(), grad.Data(), scale.Data(), scaleGrad.Data(), biasGrad.Data(), saveMean.Data(), saveInvStdDev.Data(), GetStream());
}

#pragma region Static BLAS Functions
// float/double overloads of hipblasSgemm()/hipblasDgemm()
static hipblasStatus_t cublas_gemm(hipblasHandle_t handle, hipblasOperation_t transa, hipblasOperation_t transb, int m, int n, int k, const float* alpha, const float* A, int lda, const float* B, int ldb, const float* beta, float* C, int ldc)
{
    return hipblasSgemm(handle, transa, transb, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
}
static hipblasStatus_t cublas_gemm(hipblasHandle_t handle, hipblasOperation_t transa, hipblasOperation_t transb, int m, int n, int k, const double* alpha, const double* A, int lda, const double* B, int ldb, const double* beta, double* C, int ldc)
{
    return hipblasDgemm(handle, transa, transb, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
}
static hipblasStatus_t cublas_axpy(hipblasHandle_t handle, int n, const float* alpha, const float* x, int incx, float* y, int incy)
{
    return hipblasSaxpy(handle, n, alpha, x, incx, y, incy);
}
static hipblasStatus_t cublas_axpy(hipblasHandle_t handle, int n, const double* alpha, const double* x, int incx, double* y, int incy)
{
    return hipblasDaxpy(handle, n, alpha, x, incx, y, incy);
}

template <class ElemType>
void GPUMatrix<ElemType>::MultiplyAndWeightedAdd(ElemType alpha, const GPUMatrix<ElemType>& a, const bool transposeA, const GPUMatrix<ElemType>& b, const bool transposeB,
                                                 ElemType beta, GPUMatrix<ElemType>& c)
{
    a.PrepareDevice();
    if ((a.GetComputeDeviceId() != b.GetComputeDeviceId()) || (b.GetComputeDeviceId() != c.GetComputeDeviceId())) // different GPUs
        InvalidArgument("All matrices must be on the same GPU");

    hipblasHandle_t cuHandle = GetCublasHandle(b.GetComputeDeviceId());
    hipblasOperation_t transA = transposeA ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    hipblasOperation_t transB = transposeB ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    int m = int(transposeA ? a.m_numCols : a.m_numRows);
    int n = int(transposeB ? b.m_numRows : b.m_numCols);
    int k = int(transposeA ? a.m_numRows : a.m_numCols);
    int l = int(transposeB ? b.m_numCols : b.m_numRows);

    c.RequireSize(m, n);

    if (!(m > 0 && k > 0 && l > 0 && n > 0))
        RuntimeError("!(m>0 && k>0 && l>0 && n>0)"); // converting from size_t to int may cause overflow
    if (k != l)
        RuntimeError("matrix dim mismatch in MultiplyAndWeightedAdd");
    CUBLAS_CALL(cublas_gemm(cuHandle, transA, transB, m, n, k, &alpha, a.Data(), (int) a.m_numRows, b.Data(), (int) b.m_numRows, &beta, c.Data(), (int) c.m_numRows));
    c.m_numRows = m;
    c.m_numCols = n;
}

template <class ElemType>
void GPUMatrix<ElemType>::Multiply1x1AndWeightedAdd(ElemType alpha, const GPUMatrix<ElemType>& a, const GPUMatrix<ElemType>& b, ElemType beta, GPUMatrix<ElemType>& c)
{
    a.PrepareDevice();
    if ((a.GetComputeDeviceId() != b.GetComputeDeviceId()) || (b.GetComputeDeviceId() != c.GetComputeDeviceId())) // different GPUs
        InvalidArgument("All matrices must be on the same GPU");
    CUDA_LONG N = (CUDA_LONG) c.GetNumElements();
    int blocksPerGrid = (int) ceil(1.0 * N / GridDim::maxThreadsPerBlock);
    SyncGuard syncGuard;
    _multiply1x1AndWeightedAdd<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(alpha, a.Data(), b.Data(), beta, c.Data(), N);
}

template <class ElemType>
void GPUMatrix<ElemType>::MultiplyAndAdd(const GPUMatrix<ElemType>& a, const bool transposeA, const GPUMatrix<ElemType>& b, const bool transposeB, GPUMatrix<ElemType>& c)
{
    return GPUMatrix<ElemType>::MultiplyAndWeightedAdd(1, a, transposeA, b, transposeB, 1, c);
}

template <class ElemType>
void GPUMatrix<ElemType>::Multiply(const GPUMatrix<ElemType>& a, const bool transposeA, const GPUMatrix<ElemType>& b, const bool transposeB, GPUMatrix<ElemType>& c)
{
    return GPUMatrix<ElemType>::MultiplyAndWeightedAdd(1, a, transposeA, b, transposeB, 0, c);
}

template <class ElemType>
void GPUMatrix<ElemType>::Multiply(const GPUMatrix<ElemType>& a, const GPUMatrix<ElemType>& b, GPUMatrix<ElemType>& c)
{
    return GPUMatrix<ElemType>::MultiplyAndWeightedAdd(1, a, false, b, false, 0, c);
}

/// <summary>Matrix-scalar multiply with col-major matrices: c = alpha * a + c</summary>
/// if a is a column vector, add to all columns of c
/// if a is a row vector, add to all rows of c
/// if a is a scalar, add to all elements of c
/// <param name="alpha">Scalar</param>
/// <param name="a">Input matrix</param>
/// <param name="c">Resulting matrix, user is responsible for allocating this</param>
template <class ElemType>
/*static*/ void GPUMatrix<ElemType>::ScaleAndAdd(ElemType alpha, const GPUMatrix<ElemType>& a, GPUMatrix<ElemType>& c)
{
    if (a.GetComputeDeviceId() != c.GetComputeDeviceId())
    {
        InvalidArgument("All matrices must be on the same GPU");
    }
    else
    {
        if (a.IsEmpty() && c.IsEmpty())
            return;
        a.PrepareDevice();
        if (a.IsEmpty() || c.IsEmpty())
            LogicError("ScaleAndAdd:  one of the input matrices is empty.");
        // if (a.GetNumRows() != 1 && a.GetNumCols() != 1) // a is not a col or row vector
        if (a.GetNumRows() == c.GetNumRows() && a.GetNumCols() == c.GetNumCols()) // dimensions match
        {
            const int m = (int) a.GetNumRows();
            const int n = (int) a.GetNumCols();
            const int len = m * n;
            const int incx = 1;
            const int incy = 1;

            assert(m > 0 && n > 0 && len > 0); // converting from size_t to int may cause overflow
            assert((int) c.GetNumRows() == m && (int) c.GetNumCols() == n);
            if ((int) c.GetNumRows() != m || (int) c.GetNumCols() != n)
                InvalidArgument("dimension of matrix c does not match dimension of matrix a.");

            hipblasHandle_t cuHandle = GetCublasHandle(a.GetComputeDeviceId());
            // TODO: Overload the call to cublas_axpy to remove these ugly if/else statements.
            if (sizeof(ElemType) == sizeof(float))
            {
                CUBLAS_CALL(hipblasSaxpy(cuHandle, len, reinterpret_cast<float*>(&alpha), reinterpret_cast<float*>(a.Data()), incx, reinterpret_cast<float*>(c.Data()), incy));
            }
            else if (sizeof(ElemType) == sizeof(double))
            {
                CUBLAS_CALL(hipblasDaxpy(cuHandle, len, reinterpret_cast<double*>(&alpha), reinterpret_cast<double*>(a.Data()), incx, reinterpret_cast<double*>(c.Data()), incy));
            }
            else
            {
                RuntimeError("Unsupported template argument in GPUMatrix");
            }
        }
        else if (a.GetNumElements() == 1)
        {
            CUDA_LONG N = (CUDA_LONG) c.GetNumElements();
            int blocksPerGrid = (int) ceil(1.0 * N / GridDim::maxThreadsPerBlock);
            c.PrepareDevice();
            SyncGuard syncGuard;
            _scaleAndAddScalar<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(c.Data(), N, alpha, a.Data(), c.Data());
                                }
        else if (a.GetNumCols() == 1) // col vector, add it to all columns
        {
            CUDA_LONG m = (CUDA_LONG) c.GetNumRows();
            CUDA_LONG n = (CUDA_LONG) c.GetNumCols();
            if (m != (CUDA_LONG) a.GetNumRows())
                InvalidArgument("To add column vector, rows should match.");

            int blocksPerGrid = (int) (ceil(1.0 * m * n / GridDim::maxThreadsPerBlock));
            SyncGuard syncGuard;
#ifdef VALIDATION
            printf(">>>> CUDA compute device is %d\n", a.GetComputeDeviceId());
            printf(">>>> a.Data()= %p, c.Data()= %p, alpha = %f, m = %ld, n = %ld\n", a.Data(), c.Data(), alpha, m, n);
            for (int i = 0; i < 2; i++)
            {
                ElemType buffer[10] = {-1.234f};
                hipError_t error = hipMemcpy(buffer, !i ? a.Data(): c.Data(), sizeof(buffer), hipMemcpyKind::hipMemcpyDeviceToHost);
                if (error == hipError_t::hipSuccess)
                    printf("buffer valid\n");
            }
#endif

            _matrixVectorColumnWiseAddWithThreadPerElem<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(a.Data(), c.Data(), c.Data(), alpha, m, n);

                                }
        else if (a.GetNumRows() == 1) // row vector, add it to all rows
        {
            hipblasHandle_t cuHandle = GetCublasHandle(a.GetComputeDeviceId());
            int m = (int) c.GetNumRows();
            int n = (int) c.GetNumCols();
            assert(n == (int) a.GetNumCols());
            if (n != (int) a.GetNumCols())
                InvalidArgument("To add row vector, cols should match.");

            // TODO: Overload the call to cublas_axpy to remove these ugly if/else statements.
            if (sizeof(ElemType) == sizeof(double))
            {
                foreach_row (i, c)
                {
                    CUBLAS_CALL(hipblasDaxpy(cuHandle, n, reinterpret_cast<double*>(&alpha), reinterpret_cast<double*>(a.Data()), 1, reinterpret_cast<double*>(c.Data()+ i), m));
                }
            }
            else
            {
                foreach_row (i, c)
                {
                    CUBLAS_CALL(hipblasSaxpy(cuHandle, n, reinterpret_cast<float*>(&alpha), reinterpret_cast<float*>(a.Data()), 1, reinterpret_cast<float*>(c.Data()+ i), m));
                }
            }
        }
        else
            InvalidArgument("dimension of matrix c does not match dimension of matrix a.");
    }
}

/// <summary>Matrix-scalar multiply with col-major matrices: c = alpha * a + b</summary>
/// if a is a column vector, add to all columns of b
/// if a is a row vector, add to all rows of b
/// if a is a scalar, add to all elements of b
/// <param name="alpha">Scalar</param>
/// <param name="a">Input matrix</param>
/// <param name="b">Input matrix</param>
/// <param name="c">Resulting matrix, user is responsible for allocating this</param>
template <class ElemType>
/*static*/ void GPUMatrix<ElemType>::ScaleAndAdd(ElemType alpha, const GPUMatrix<ElemType>& a, const GPUMatrix<ElemType>& b, GPUMatrix<ElemType>& c)
{
    if (a.GetComputeDeviceId() != c.GetComputeDeviceId() || a.GetComputeDeviceId() != b.GetComputeDeviceId())
    {
        InvalidArgument("All matrices must be on the same GPU");
    }
    else
    {
        if (a.IsEmpty() && b.IsEmpty())
            return;
        a.PrepareDevice();
        if (a.IsEmpty() || b.IsEmpty())
            LogicError("ScaleAndAdd: One of the input matrices is empty.");
        c.RequireSize(b.GetNumRows(), b.GetNumCols());
        // if (a.GetNumRows() != 1 && a.GetNumCols() != 1) // a is not a col or row vector
        if (a.GetNumRows() == b.GetNumRows() && a.GetNumCols() == b.GetNumCols()) // dimensions match
        {
            /*
                const int m = (int)a.GetNumRows();
                const int n = (int)a.GetNumCols();
                const int len = m * n;
                const int incx = 1;
                const int incy = 1;
                assert (m>0 && n>0 && len>0); // converting from size_t to int may cause overflow
                */
            CUDA_LONG N = (CUDA_LONG) c.GetNumElements();
            int blocksPerGrid = (int) ceil(1.0 * N / GridDim::maxThreadsPerBlock);
            c.PrepareDevice();
            SyncGuard syncGuard;
            _matrixMatrixAddOnCuda<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(alpha, a.Data(), b.Data(), c.Data(), N);
        }
        else if (a.GetNumElements() == 1)
        {
            CUDA_LONG N = (CUDA_LONG) c.GetNumElements();
            int blocksPerGrid = (int) ceil(1.0 * N / GridDim::maxThreadsPerBlock);
            c.PrepareDevice();
            SyncGuard syncGuard;
            _scaleAndAddScalar<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(c.Data(), N, alpha, a.Data(), b.Data());
        }
        else if (a.GetNumCols() == 1) // col vector, add it to all columns
        {
            CUDA_LONG m = (CUDA_LONG) c.GetNumRows();
            CUDA_LONG n = (CUDA_LONG) c.GetNumCols();
            if (m != (CUDA_LONG) a.GetNumRows())
                InvalidArgument("To add column vector, rows should match.");

            int blocksPerGrid = (int) (ceil(1.0 * m * n / GridDim::maxThreadsPerBlock));
            SyncGuard syncGuard;
            _matrixVectorColumnWiseAddWithThreadPerElem<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(a.Data(), b.Data(), c.Data(), alpha, m, n);

        }
        else if (a.GetNumRows() == 1) // row vector, add it to all rows
        {
            CUDA_LONG m = (CUDA_LONG) c.GetNumRows();
            CUDA_LONG n = (CUDA_LONG) c.GetNumCols();
            if (m != (CUDA_LONG) a.GetNumRows())
                InvalidArgument("To add column vector, rows should match.");

            int blocksPerGrid = (int) (ceil(1.0 * m * n / GridDim::maxThreadsPerBlock));
            SyncGuard syncGuard;
            _matrixVectorRowWiseAddWithThreadPerElem<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(a.Data(), b.Data(), c.Data(), alpha, m, n);
        }
        else
            InvalidArgument("Dimension of matrix c does not match dimension of matrix a.");
    }
}

/// <summary>c += alpha * (a-b)</summary>
/// if a, b, c  must have same dim
/// <param name="alpha">Scalar</param>
/// <param name="a">Input matrix</param>
/// <param name="b">Input matrix</param>
/// <param name="c">Resulting matrix, user is responsible for allocating this</param>
template <class ElemType>
void GPUMatrix<ElemType>::AddScaledDifference(const ElemType alpha, const GPUMatrix<ElemType>& a, const GPUMatrix<ElemType>& b, GPUMatrix<ElemType>& c)
{
    if (a.GetComputeDeviceId() != c.GetComputeDeviceId())
    {
        InvalidArgument("All matrices must be on the same GPU");
    }
    else
    {
        a.PrepareDevice();

        assert(a.GetNumRows() == b.GetNumRows() && a.GetNumRows() == c.GetNumRows() &&
               a.GetNumCols() == b.GetNumCols() && a.GetNumCols() == c.GetNumCols());

        if (!(a.GetNumRows() == b.GetNumRows() && a.GetNumRows() == c.GetNumRows() &&
              a.GetNumCols() == b.GetNumCols() && a.GetNumCols() == c.GetNumCols()))
        {
            InvalidArgument("AddScaledDifference: a, b, and c must have same dimension.");
        }

        if (a.IsEmpty())
            LogicError("AddScaledDifference: Input matrix a is empty.");

        CUDA_LONG n = (CUDA_LONG) a.GetNumElements();
        int blocksPerGrid = (int) ceil(1.0 * n / GridDim::maxThreadsPerBlock);
        SyncGuard syncGuard;
        _addScaledDifference<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(alpha, a.Data(), b.Data(), c.Data(), n);
    }
}

/// <summary> c = alpha * (a-b)</summary>
/// if a, b, c  must have same dim
/// <param name="alpha">Scalar</param>
/// <param name="a">Input matrix</param>
/// <param name="b">Input matrix</param>
/// <param name="c">Resulting matrix, user is responsible for allocating this</param>
template <class ElemType>
void GPUMatrix<ElemType>::AssignScaledDifference(const ElemType alpha, const GPUMatrix<ElemType>& a, const GPUMatrix<ElemType>& b, GPUMatrix<ElemType>& c)
{
    if (a.GetComputeDeviceId() != c.GetComputeDeviceId())
    {
        InvalidArgument("All matrices must be on the same GPU");
    }
    else
    {
        a.PrepareDevice();

        assert(a.GetNumRows() == b.GetNumRows() && a.GetNumCols() == b.GetNumCols());

        if (!(a.GetNumRows() == b.GetNumRows() && a.GetNumCols() == b.GetNumCols()))
            InvalidArgument("AssignScaledDifference: a, b must have same dimension.");

        if (a.IsEmpty())
            LogicError("AssignScaledDifference: Input matrix a is empty.");

        if (&c != &a && &c != &b)
            c.RequireSize(a.GetNumRows(), a.GetNumCols());

        CUDA_LONG n = (CUDA_LONG) a.GetNumElements();
        int blocksPerGrid = (int) ceil(1.0 * n / GridDim::maxThreadsPerBlock);
        SyncGuard syncGuard;
        _assignScaledDifference<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(alpha, a.Data(), b.Data(), c.Data(), n);
    }
}

/// <summary>c += alpha * (a-b)</summary>
/// if a, b, c  must have same dim
/// <param name="alpha">1X1 matrix</param>
/// <param name="a">Input matrix</param>
/// <param name="b">Input matrix</param>
/// <param name="c">Resulting matrix, user is responsible for allocating this</param>
template <class ElemType>
void GPUMatrix<ElemType>::AddScaledDifference(const GPUMatrix<ElemType>& alpha, const GPUMatrix<ElemType>& a, const GPUMatrix<ElemType>& b, GPUMatrix<ElemType>& c)
{
    assert(alpha.GetNumElements() == 1);
    if (!(alpha.GetNumElements() == 1))
        InvalidArgument("AddScaledDifference: alpha must be a 1X1 matrix.");

    if (a.GetComputeDeviceId() != c.GetComputeDeviceId())
    {
        InvalidArgument("All matrices must be on the same GPU");
    }
    else
    {
        a.PrepareDevice();

        assert(a.GetNumRows() == b.GetNumRows() && a.GetNumRows() == c.GetNumRows() &&
               a.GetNumCols() == b.GetNumCols() && a.GetNumCols() == c.GetNumCols());

        if (!(a.GetNumRows() == b.GetNumRows() && a.GetNumRows() == c.GetNumRows() &&
              a.GetNumCols() == b.GetNumCols() && a.GetNumCols() == c.GetNumCols()))
        {
            InvalidArgument("AddScaledDifference: a, b, and c must have same dimension.");
        }

        if (a.IsEmpty())
            LogicError("AddScaledDifference: Input matrix a is empty.");

        CUDA_LONG n = (CUDA_LONG) a.GetNumElements();
        int blocksPerGrid = (int) ceil(1.0 * n / GridDim::maxThreadsPerBlock);
        SyncGuard syncGuard;
        _addScaledDifference<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(alpha.Data(), a.Data(), b.Data(), c.Data(), n);
    }
}

/// <summary> c = alpha * (a-b)</summary>
/// if a, b, c  must have same dim
/// <param name="alpha">Scalar</param>
/// <param name="a">Input matrix</param>
/// <param name="b">Input matrix</param>
/// <param name="c">Resulting matrix, user is responsible for allocating this</param>
template <class ElemType>
void GPUMatrix<ElemType>::AssignScaledDifference(const GPUMatrix<ElemType>& alpha, const GPUMatrix<ElemType>& a, const GPUMatrix<ElemType>& b, GPUMatrix<ElemType>& c)
{
    assert(alpha.GetNumElements() == 1);
    if (!(alpha.GetNumElements() == 1))
        InvalidArgument("AddScaledDifference: alpha must be a 1X1 matrix.");

    if (a.GetComputeDeviceId() != c.GetComputeDeviceId())
    {
        InvalidArgument("All matrices must be on the same GPU");
    }
    else
    {
        a.PrepareDevice();

        assert(a.GetNumRows() == b.GetNumRows() && a.GetNumCols() == b.GetNumCols());

        if (!(a.GetNumRows() == b.GetNumRows() && a.GetNumCols() == b.GetNumCols()))
        {
            InvalidArgument("AssignScaledDifference: a, b must have same dimension.");
        }

        if (a.IsEmpty())
            LogicError("AssignScaledDifference: Input matrix a is empty.");

        c.RequireSize(a.GetNumRows(), a.GetNumCols());

        CUDA_LONG n = (CUDA_LONG) a.GetNumElements();
        int blocksPerGrid = (int) ceil(1.0 * n / GridDim::maxThreadsPerBlock);
        SyncGuard syncGuard;
        _assignScaledDifference<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(alpha.Data(), a.Data(), b.Data(), c.Data(), n);
    }
}

//c[ci,cj] += a[ai,aj]
template <class ElemType>
void GPUMatrix<ElemType>::AddElementToElement(ElemType beta, const GPUMatrix<ElemType>& a, const size_t ai, const size_t aj, GPUMatrix<ElemType>& c, const size_t ci, const size_t cj)
{
    if (ai >= a.GetNumRows() || aj >= a.GetNumCols() ||
        ci >= c.GetNumRows() || cj >= c.GetNumCols())
        InvalidArgument("AddElementToElement: Index out of range.");

    a.PrepareDevice();
    SyncGuard syncGuard;
    _addElementToElement<ElemType><<<1, 1, 0, t_stream>>>(beta, a.Data(), (CUDA_LONG) a.LocateElement(ai, aj), c.Data(), (CUDA_LONG) c.LocateElement(ci, cj));
}

template <class ElemType>
/*static*/ void GPUMatrix<ElemType>::Scale(ElemType alpha, GPUMatrix<ElemType>& a)
{
    if (alpha == 0) // if 0 then do not access the value, so that we can use this to multiply uninitialized matrices with beta=0
    {
        CUDA_CALL(hipMemset(a.Data(), 0, a.m_numRows * a.m_numCols * sizeof(ElemType)));
        return;
    }

    hipblasHandle_t cuHandle = GetCublasHandle(a.GetComputeDeviceId());
    // TODO: Overload the call to cublas_axpy to remove these ugly if/else statements.
    if (sizeof(ElemType) == sizeof(float))
    {
        float alph = (float) alpha;
        CUBLAS_CALL(hipblasSscal(cuHandle, int(a.m_numRows * a.m_numCols), &alph, (float*) a.Data(), 1));
    }
    else if (sizeof(ElemType) == sizeof(double))
    {
        double alph = alpha;
        CUBLAS_CALL(hipblasDscal(cuHandle, int(a.m_numRows * a.m_numCols), &alph, (double*) a.Data(), 1));
    }
    else
    {
        RuntimeError("Unsupported template argument in GPUMatrix");
    }
}

template <class ElemType>
/*static*/ void GPUMatrix<ElemType>::Scale(GPUMatrix<ElemType>& alpha, GPUMatrix<ElemType>& a)
{
    if (alpha.GetNumElements() != 1)
    {
        RuntimeError("Matrix alpha must be 1x1");
    }
    hipblasHandle_t cuHandle = GetCublasHandle(a.GetComputeDeviceId());
    hipblasSetPointerMode(cuHandle, HIPBLAS_POINTER_MODE_DEVICE);
    if (sizeof(ElemType) == sizeof(float))
    {
        CUBLAS_CALL(hipblasSscal(cuHandle, int(a.m_numRows * a.m_numCols), (float*) alpha.Data(), (float*) a.Data(), 1));
    }
    else if (sizeof(ElemType) == sizeof(double))
    {
        CUBLAS_CALL(hipblasDscal(cuHandle, int(a.m_numRows * a.m_numCols), (double*) alpha.Data(), (double*) a.Data(), 1));
    }
    else
    {
        hipblasSetPointerMode(cuHandle, HIPBLAS_POINTER_MODE_HOST);
        RuntimeError("Unsupported template argument in GPUMatrix");
    }
    hipblasSetPointerMode(cuHandle, HIPBLAS_POINTER_MODE_HOST);
}

template <class ElemType> // c = alpha * a
/*static*/ void GPUMatrix<ElemType>::Scale(ElemType alpha, const GPUMatrix<ElemType>& a, GPUMatrix<ElemType>& c)
{
    c = a;
    Scale(alpha, c);
}

template <class ElemType>
void GPUMatrix<ElemType>::InnerProduct(const GPUMatrix<ElemType>& a, const GPUMatrix<ElemType>& b, GPUMatrix<ElemType>& c, const bool isColWise)
{
    if (a.GetComputeDeviceId() != b.GetComputeDeviceId() || b.GetComputeDeviceId() != c.GetComputeDeviceId()) // different GPUs
        InvalidArgument("All matrices must be on the same GPU");

    if (a.IsEmpty() || b.IsEmpty())
        LogicError("Scale:  one of the input matrices is empty.");

    const int m = (int) a.GetNumRows();
    const int n = (int) a.GetNumCols();
    const int k = (int) b.GetNumRows();
    const int l = (int) b.GetNumCols();

    assert(m > 0 && n > 0 && k > 0 && l > 0); // converting from size_t to int may cause overflow
    assert(m == k && n == l);                 // converting from size_t to int may cause overflow
    if (m != k || n != l)
        InvalidArgument("Matrices a and b should have same dimension.");

    if (isColWise)
        c.RequireSize(1, n);
    else
        c.RequireSize(m, 1);

    if ((isColWise && m == 1) || !isColWise && n == 1) // in this case it's equivalent to element-wise product
    {
        c.AssignElementProductOf(a, b);
    }
    else
    {
        c.PrepareDevice();

        int blocksPerGrid = 0;
        if (isColWise) // col-wise
        {
            c.RequireSize(1, n);
            blocksPerGrid = (int) ceil(1.0 * n / GridDim::maxThreadsPerBlock);
        }
        else
        {
            c.RequireSize(m, 1);
            blocksPerGrid = (int) ceil(1.0 * m / GridDim::maxThreadsPerBlock);
        }

        SyncGuard syncGuard;
        _innerProduct<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(c.Data(), a.Data(), b.Data(), m, n, isColWise);
    }
}

template <class ElemType>
ElemType GPUMatrix<ElemType>::InnerProductOfMatrices(const GPUMatrix<ElemType>& a, const GPUMatrix<ElemType>& b)
{
    if (a.IsEmpty() || b.IsEmpty())
        LogicError("InnerProductOfMatrices:  one of the input matrices is empty.");

    const int m = (int) a.GetNumRows();
    const int n = (int) a.GetNumCols();
    const int k = (int) b.GetNumRows();
    const int l = (int) b.GetNumCols();

    assert(m > 0 && n > 0 && k > 0 && l > 0); // converting from size_t to int may cause overflow
    assert(m == k && n == l);                 // converting from size_t to int may cause overflow
    if (m != k || n != l)
        InvalidArgument("InnerProductOfMatrices: Matrices a and b should have same dimension.");

    hipblasHandle_t cuHandle = GetCublasHandle(a.GetComputeDeviceId());
    if (sizeof(ElemType) == sizeof(double))
    {
        double tmp = 0;
        CUBLAS_CALL(hipblasDdot(cuHandle, m * n, reinterpret_cast<double*>(a.Data()), 1, reinterpret_cast<double*>(b.Data()), 1, &tmp));
        return ElemType(tmp);
        // return (ElemType)ddot((int)a.GetNumElements(), reinterpret_cast <double*>(a.Data()), 1, reinterpret_cast <double*>(b.Data()), 1);
    }
    else
    {
        float tmp = 0;
        CUBLAS_CALL(hipblasSdot(cuHandle, m * n, reinterpret_cast<float*>(a.Data()), 1, reinterpret_cast<float*>(b.Data()), 1, &tmp));
        return tmp;
        // return (ElemType)sdot((int)a.GetNumElements(), reinterpret_cast <float*>(a.Data()), 1, reinterpret_cast <float*>(b.Data()), 1);
    }
}

template <class ElemType>
GPUMatrix<ElemType>& GPUMatrix<ElemType>::AssignInnerProductOfMatrices(const GPUMatrix<ElemType>& a, const GPUMatrix<ElemType>& b)
{
    if (a.IsEmpty() || b.IsEmpty())
        LogicError("InnerProductOfMatrices:  one of the input matrices is empty.");

    RequireSize(1, 1);

    const int m = (int) a.GetNumRows();
    const int n = (int) a.GetNumCols();
    const int k = (int) b.GetNumRows();
    const int l = (int) b.GetNumCols();

    assert(m > 0 && n > 0 && k > 0 && l > 0); // converting from size_t to int may cause overflow
    assert(m == k && n == l);                 // converting from size_t to int may cause overflow
    if (m != k || n != l)
        InvalidArgument("InnerProductOfMatrices: Matrices a and b should have same dimension.");

    hipblasHandle_t cuHandle = GetCublasHandle(a.GetComputeDeviceId());
    hipblasSetPointerMode(cuHandle, HIPBLAS_POINTER_MODE_DEVICE);
    if (sizeof(ElemType) == sizeof(double))
    {
        CUBLAS_CALL(hipblasDdot(cuHandle, m * n, reinterpret_cast<double*>(a.Data()), 1, reinterpret_cast<double*>(b.Data()), 1, reinterpret_cast<double*>(Data())));
    }
    else
    {
        CUBLAS_CALL(hipblasSdot(cuHandle, m * n, reinterpret_cast<float*>(a.Data()), 1, reinterpret_cast<float*>(b.Data()), 1, reinterpret_cast<float*>(Data())));
    }
    hipblasSetPointerMode(cuHandle, HIPBLAS_POINTER_MODE_HOST);
    return *this;
}

template <class ElemType>
void GPUMatrix<ElemType>::ElementWisePower(ElemType alpha, const GPUMatrix<ElemType>& a, GPUMatrix<ElemType>& c)
{
    if (a.GetComputeDeviceId() != c.GetComputeDeviceId())
    {
        InvalidArgument("All matrices must be on the same GPU");
    }
    else
    {
        if (a.IsEmpty())
            LogicError("ElementWisePower:  The input matrix a is empty.");

        c.RequireSize(a.GetNumRows(), a.GetNumCols());

        a.PrepareDevice();
        SyncGuard syncGuard;
        CUDA_LONG N = (CUDA_LONG) a.GetNumElements();
        int blocksPerGrid = (int) ceil(1.0 * N / GridDim::maxThreadsPerBlock);
        _elementWisePowerOnCuda<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(alpha, a.Data(), c.Data(), N);
    }
}

template <class ElemType>
bool GPUMatrix<ElemType>::AreEqual(const GPUMatrix<ElemType>& a, const GPUMatrix<ElemType>& b, const ElemType threshold /*= 1e-8*/)
{
    if (a.IsEmpty() || b.IsEmpty())
        LogicError("AreEqual: one of the input matrices is empty.");

    if (a.GetNumRows() != b.GetNumRows() || a.GetNumCols() != b.GetNumCols())
        return false;

    bool bResult = false;

    long* res = new long[1];
    res[0] = 1;
    long* d_res = TracingGPUMemoryAllocator::Allocate<long>(a.GetComputeDeviceId(), 1);
    CUDA_CALL(hipMemcpy(d_res, res, sizeof(long) * 1, hipMemcpyHostToDevice));
    CUDA_LONG N = (CUDA_LONG) a.GetNumElements();
    int blocksPerGrid = (int) ceil(1.0 * N / GridDim::maxThreadsPerBlock);
    _areEqual<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(a.Data(), b.Data(), N, threshold, d_res);
    CUDA_CALL(hipMemcpy(res, d_res, sizeof(long) * 1, hipMemcpyDeviceToHost));
    TracingGPUMemoryAllocator::Free<long>(a.GetComputeDeviceId(), d_res);
    if (res[0] != 0)
        bResult = true;
    delete[] res;
    return bResult;
}

// see Matrix<ElemType>::TensorShuffleScaleAndAdd() for comments
template <class ElemType>
void GPUMatrix<ElemType>::TensorShuffleScaleAndAdd(ElemType keepWeight, const GPUMatrix<ElemType>& a, size_t D, size_t S, size_t M, size_t K, size_t T, ElemType scaleFactor, const GPUMatrix<ElemType>& b, GPUMatrix<ElemType>& c)
{
    CUDA_LONG N = (CUDA_LONG) c.GetNumElements();
    assert(N == (CUDA_LONG) a.GetNumElements() && N == (CUDA_LONG) b.GetNumElements());
    assert(a.GetComputeDeviceId() == c.GetComputeDeviceId() && b.GetComputeDeviceId() == c.GetComputeDeviceId());
    a.PrepareDevice();
    SyncGuard syncGuard;
    int blocksPerGrid = (int) ceil(1.0 * N / GridDim::maxThreadsPerBlock);
    _tensorShuffleScaleAndAdd<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(keepWeight, a.Data(), D, S, M, K, T, scaleFactor, b.Data(), c.Data());
}

template <class ElemType>
bool GPUMatrix<ElemType>::HasElement(const GPUMatrix<ElemType>& a, const ElemType v)
{
    if (a.IsEmpty())
        LogicError("HasElement: the input matrix is empty.");

    bool bResult = false;
    ElemType* res = new ElemType[2];
    res[0] = v;
    res[1] = 0;
    ElemType* d_res = TracingGPUMemoryAllocator::Allocate<ElemType>(a.GetComputeDeviceId(), 2);
    CUDA_CALL(hipMemcpy(d_res, res, sizeof(ElemType) * 2, hipMemcpyHostToDevice));
    CUDA_LONG N = (CUDA_LONG) a.GetNumElements();
    int blocksPerGrid = (int) ceil(1.0 * N / GridDim::maxThreadsPerBlock);
    _hasElement<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(a.Data(), N, d_res);
    CUDA_CALL(hipMemcpy(res, d_res, sizeof(ElemType) * 2, hipMemcpyDeviceToHost));
    TracingGPUMemoryAllocator::Free<ElemType>(a.GetComputeDeviceId(), d_res);
    if (res[1] != 0)
        bResult = true;
    else
        bResult = false;

    delete[] res;
    return bResult;
}

template <class ElemType>
void GPUMatrix<ElemType>::CreateCurandObject(unsigned long seed, const char* caller)
{
    assert(caller != nullptr);

    if (s_curandGenerator == NULL)
    {
        unsigned long long cudaSeed = (seed == USE_TIME_BASED_SEED) ? time(NULL) : seed;
        fprintf(stderr, "%s (GPU): creating hiprand object with seed %llu, sizeof(ElemType)==%lu\n",
                caller, cudaSeed, (unsigned long)sizeof(ElemType));
        s_curandGenerator = new hiprandGenerator_t;
        // Create pseudo-random number generator
        CURAND_CALL(hiprandCreateGenerator(&(((hiprandGenerator_t*) s_curandGenerator)[0]), HIPRAND_RNG_PSEUDO_XORWOW));
        CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(((hiprandGenerator_t*) s_curandGenerator)[0], cudaSeed));
        CURAND_CALL(hiprandSetGeneratorOrdering(((hiprandGenerator_t*) s_curandGenerator)[0], HIPRAND_ORDERING_PSEUDO_SEEDED));
    }
}

template <class ElemType>
void GPUMatrix<ElemType>::ResetCurandObject(unsigned long seed, const char* caller)
{
    assert(caller != nullptr);

    if (s_curandGenerator && (seed != USE_TIME_BASED_SEED))
    {
        // Note: this might be slow.
        CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(((hiprandGenerator_t*) s_curandGenerator)[0], seed));
        CURAND_CALL(hiprandSetGeneratorOffset(((hiprandGenerator_t*) s_curandGenerator)[0], 0));
    }
    else
    {
        CreateCurandObject(seed, caller);
    }
}

template <class ElemType>
GPUMatrix<ElemType> GPUMatrix<ElemType>::Ones(const size_t rows, const size_t cols, int deviceId)
{
    GPUMatrix<ElemType> c(rows, cols, deviceId); // will initialize to 0
    c.SetValue(1);
    return c;
}

template <class ElemType>
GPUMatrix<ElemType> GPUMatrix<ElemType>::Zeros(const size_t rows, const size_t cols, int deviceId)
{
    GPUMatrix<ElemType> c(rows, cols, deviceId); // will initialize to 0
    // c.SetValue(0);
    return c;
}

template <class ElemType>
GPUMatrix<ElemType> GPUMatrix<ElemType>::Eye(const size_t rows, int deviceId)
{
    GPUMatrix<ElemType> c(rows, rows, deviceId); // will initialize to 0
    c.SetDiagonalValue(1);
    return c;
}

template <class ElemType>
GPUMatrix<ElemType> GPUMatrix<ElemType>::RandomUniform(const size_t rows, const size_t cols, int deviceId, const ElemType low, const ElemType high, unsigned long seed)
{
    GPUMatrix<ElemType> c(rows, cols, deviceId); // will initialize to 0
    c.SetUniformRandomValue(low, high, seed);
    return c;
}

template <class ElemType>
GPUMatrix<ElemType> GPUMatrix<ElemType>::RandomGaussian(const size_t rows, const size_t cols, int deviceId, const ElemType mean, const ElemType sigma, unsigned long seed)
{
    GPUMatrix<ElemType> c(rows, cols, deviceId); // will initialize to 0
    c.SetGaussianRandomValue(mean, sigma, seed);
    return c;
}

template <class ElemType>
ElemType GPUMatrix<ElemType>::GetLearnRateForBlock_Helper(const GPUMatrix<ElemType>& Gradients, const GPUMatrix<ElemType>& SmoothedGradients)
{
    ElemType* d_res = TracingGPUMemoryAllocator::Allocate<ElemType>(Gradients.GetComputeDeviceId(), 1);

    // Compute inner product of matrices and keep it on device
    const int m = (int) Gradients.GetNumRows();
    const int n = (int) Gradients.GetNumCols();
    const int k = (int) SmoothedGradients.GetNumRows();
    const int l = (int) SmoothedGradients.GetNumCols();
    assert(m > 0 && n > 0 && k > 0 && l > 0); // converting from size_t to int may cause overflow
    assert(m == k && n == l);                 // converting from size_t to int may cause overflow
    if (m != k || n != l)
        InvalidArgument("InnerProductOfMatrices: Matrices a and b should have same dimension.");

    if (sizeof(ElemType) == sizeof(double))
    {
        hipblasHandle_t cuHandle = GetCublasHandle(Gradients.GetComputeDeviceId());
        hipblasSetPointerMode(cuHandle, HIPBLAS_POINTER_MODE_DEVICE);
        CUBLAS_CALL(hipblasDdot(cuHandle, m * n, reinterpret_cast<double*>(Gradients.Data()), 1, reinterpret_cast<double*>(SmoothedGradients.Data()), 1, reinterpret_cast<double*>(d_res)));
        hipblasSetPointerMode(cuHandle, HIPBLAS_POINTER_MODE_HOST);
    }
    else
    {
        hipblasHandle_t cuHandle = GetCublasHandle(Gradients.GetComputeDeviceId());
        hipblasSetPointerMode(cuHandle, HIPBLAS_POINTER_MODE_DEVICE);
        CUBLAS_CALL(hipblasSdot(cuHandle, m * n, reinterpret_cast<float*>(Gradients.Data()), 1, reinterpret_cast<float*>(SmoothedGradients.Data()), 1, reinterpret_cast<float*>(d_res)));
        hipblasSetPointerMode(cuHandle, HIPBLAS_POINTER_MODE_HOST);
    }
    // d_res[0] should now contain inner product of matrices
    // Compute squared Frobenius norms (squared sums of elements)
    _lrHelper<ElemType><<<1, 512, 0, t_stream>>>(Gradients.Data(), SmoothedGradients.Data(), (CUDA_LONG) Gradients.GetNumElements(), d_res);
    ElemType res;
    CUDA_CALL(hipMemcpy(&res, d_res, sizeof(ElemType), hipMemcpyDeviceToHost));
    TracingGPUMemoryAllocator::Free<ElemType>(Gradients.GetComputeDeviceId(), d_res);
    return res;
}
// The inputs are two row vectors [a1 a2 a3 a4] [b1 b2 b3 b4]
// The outputs are one matrix of size (nt+1)*4
// The first row is just element multiplication
// The rest rows will be with shift
template <class ElemType>
GPUMatrix<ElemType>& GPUMatrix<ElemType>::AssignElementProductOfWithShiftNeg(const GPUMatrix<ElemType>& a, const GPUMatrix<ElemType>& b, const size_t shift, const size_t nt)
{
    if (a.IsEmpty() || b.IsEmpty())
        LogicError("AssignElementProductOf: Matrix is empty.");

    assert(a.GetNumRows() == b.GetNumRows() && a.GetNumCols() == b.GetNumCols());
    if (!(a.GetNumRows() == b.GetNumRows() && a.GetNumCols() == b.GetNumCols()))
        InvalidArgument("The input matrix dimensions do not match.");

    if (!(a.GetNumRows() == 1))
        InvalidArgument("The input matrix must be a row vector.");

    RequireSize(nt + 1, a.GetNumCols());
    int BS = a.GetNumCols();

    // the output matrix is of size (nt+1, BS)
    dim3 thread_tail(DEFAULT_THREAD_PER_DIM, DEFAULT_THREAD_PER_DIM);
    dim3 block_tail((nt + 1 + DEFAULT_THREAD_PER_DIM - 1) / DEFAULT_THREAD_PER_DIM, (BS + DEFAULT_THREAD_PER_DIM - 1) / DEFAULT_THREAD_PER_DIM);

    a.PrepareDevice();
    SyncGuard syncGuard;
    _assignElementProductOfWithShiftNeg<ElemType><<<block_tail, thread_tail, 0, t_stream>>>(Data(), a.Data(), b.Data(), shift, nt + 1, BS);
    //      _assignElementProductOf<ElemType> << <block_tail, thread_tail, 0, t_stream >> >(Data(), a.Data(), b.Data(), nt);

    return *this;
}

template <class ElemType>
void GPUMatrix<ElemType>::InnerProductWithShiftNeg(const GPUMatrix<ElemType>& a, const GPUMatrix<ElemType>& b, GPUMatrix<ElemType>& c, const size_t shift, const size_t nt)
{
    if (a.GetComputeDeviceId() != b.GetComputeDeviceId() || b.GetComputeDeviceId() != c.GetComputeDeviceId()) // different GPUs
        InvalidArgument("All matrices must be on the same GPU");

    if (a.IsEmpty() || b.IsEmpty())
        LogicError("Scale:  one of the input matrices is empty.");

    const int m = (int) a.GetNumRows();
    const int n = (int) a.GetNumCols();
    const int k = (int) b.GetNumRows();
    const int l = (int) b.GetNumCols();

    assert(m > 0 && n > 0 && k > 0 && l > 0); // converting from size_t to int may cause overflow
    assert(m == k && n == l);                 // converting from size_t to int may cause overflow
    if (m != k || n != l)
        InvalidArgument("Matrices a and b should have same dimension.");

    c.RequireSize(nt + 1, n);

    if (true)
    {
        c.PrepareDevice();

        dim3 thread_tail(DEFAULT_THREAD_PER_DIM, DEFAULT_THREAD_PER_DIM);
        dim3 block_tail((nt + 1 + DEFAULT_THREAD_PER_DIM - 1) / DEFAULT_THREAD_PER_DIM, (n + DEFAULT_THREAD_PER_DIM - 1) / DEFAULT_THREAD_PER_DIM);

        SyncGuard syncGuard;
        _innerProductWithShiftNeg<ElemType><<<block_tail, thread_tail, 0, t_stream>>>(c.Data(), a.Data(), b.Data(), m, n, shift, nt + 1);
    }
}

template <class ElemType>
GPUMatrix<ElemType>& GPUMatrix<ElemType>::GetARowByIndex(const GPUMatrix<ElemType>& a, const size_t m)
{
    if (a.IsEmpty())
        LogicError("GetARowByIndex: Matrix is empty.");

    RequireSize(1, a.GetNumCols());

    int n = a.GetNumRows();
    int P = a.GetNumCols();

    if (m >= n)
        LogicError("GetARowByIndex: m is out of range.");

    int blocksPerGrid = (int) ceil(((double) P) / GridDim::maxThreadsPerBlock);

    a.PrepareDevice();
    SyncGuard syncGuard;
    _getARowByIndex<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(Data(), a.Data(), n, P, m);
    //      _assignElementProductOf<ElemType> << <block_tail, thread_tail, 0, t_stream >> >(Data(), a.Data(), b.Data(), nt);
    return *this;
}

template <class ElemType>
void GPUMatrix<ElemType>::ConductRowElementMultiplyWithShift(const GPUMatrix<ElemType>& a, const GPUMatrix<ElemType>& b, GPUMatrix<ElemType>& c, const size_t shift, const bool isafixed)
{
    if (a.GetComputeDeviceId() != b.GetComputeDeviceId() || b.GetComputeDeviceId() != c.GetComputeDeviceId()) // different GPUs
        InvalidArgument("All matrices must be on the same GPU");

    if (a.IsEmpty() || b.IsEmpty())
        LogicError("Scale:  one of the input matrices is empty.");

    const int m = (int) a.GetNumRows();
    const int n = (int) a.GetNumCols();
    const int O = (int) b.GetNumRows();
    const int P = (int) b.GetNumCols();

    assert(m > 0 && n > 0 && O > 0 && P > 0); // converting from size_t to int may cause overflow
    if (m != 1 || n != P)
        InvalidArgument("Matrices a and b should have same dimension.");

    c.RequireSize(O, P);

    if (true)
    {
        c.PrepareDevice();

        dim3 thread_tail(DEFAULT_THREAD_PER_DIM, DEFAULT_THREAD_PER_DIM);
        dim3 block_tail((O + DEFAULT_THREAD_PER_DIM - 1) / DEFAULT_THREAD_PER_DIM, (P + DEFAULT_THREAD_PER_DIM - 1) / DEFAULT_THREAD_PER_DIM);

        SyncGuard syncGuard;
        _conductRowElementMultiplyWithShift<ElemType><<<block_tail, thread_tail, 0, t_stream>>>(c.Data(), a.Data(), b.Data(), O, P, shift, isafixed);
    }
}

template <class ElemType>
GPUMatrix<ElemType>& GPUMatrix<ElemType>::AssignElementProductOfWithShift(const GPUMatrix<ElemType>& a, const GPUMatrix<ElemType>& b, const size_t shift)
{
    if (a.IsEmpty() || b.IsEmpty())
        LogicError("AssignElementProductOfWithShift: Matrix is empty.");

    assert(a.GetNumRows() == b.GetNumRows() && a.GetNumCols() == b.GetNumCols());
    if (!(a.GetNumRows() == b.GetNumRows() && a.GetNumCols() == b.GetNumCols()))
        InvalidArgument("The input matrix dimensions do not match.");

    // int O = a.GetNumRows();
    int P = a.GetNumCols();

    RequireSize(1, P);
    CUDA_LONG N = (CUDA_LONG) GetNumElements();
    int blocksPerGrid = (int) ceil(((double) N) / GridDim::maxThreadsPerBlock);
    a.PrepareDevice();
    SyncGuard syncGuard;
    _assignElementProductOfWithShift<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(Data(), a.Data(), b.Data(), shift, N);
    return *this;
}

//sequence training
template <class ElemType>
GPUMatrix<ElemType>& GPUMatrix<ElemType>::DropFrame(const GPUMatrix<ElemType>& label, const GPUMatrix<ElemType>& gamma, const ElemType& threshhold)
{
    if (IsEmpty())
        LogicError("DropFrame: Matrix is empty.");

    PrepareDevice();

    long N = (long) GetNumCols(); // one kernel per column
    int blocksPerGrid = (int) ceil(N * 1.0 / GridDim::maxThreadsPerBlock);
    SyncGuard syncGuard;
    _DropFrame<<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(Data(), label.Data(), gamma.Data(), threshhold, (long) m_numCols, (long) m_numRows);
    return *this;
}

template <class ElemType>
GPUMatrix<ElemType>& GPUMatrix<ElemType>::AssignSequenceError(const ElemType hsmoothingWeight, const GPUMatrix<ElemType>& label,
                                                              const GPUMatrix<ElemType>& dnnoutput, const GPUMatrix<ElemType>& gamma, ElemType alpha)
{
    if (IsEmpty())
        LogicError("AssignSequenceError: Matrix is empty.");

    PrepareDevice();

    SyncGuard syncGuard;
    long N = (LONG64) label.GetNumElements();
    int blocksPerGrid = (int) ceil(1.0 * N / GridDim::maxThreadsPerBlock);
    _AssignSequenceError<<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(hsmoothingWeight, Data(), label.Data(), dnnoutput.Data(), gamma.Data(), alpha, N);
    return *this;
}

#pragma endregion Static BLAS Functions

/// f = logadd(f, vec) to get the logadd sum of vector elments
template <class ElemType>
ElemType GPUMatrix<ElemType>::LogSumOfElements() const
{
    if (IsEmpty())
        LogicError("SumOfElements: Matrix is empty");

    ElemType* d_sum = TracingGPUMemoryAllocator::Allocate<ElemType>(GetComputeDeviceId(), 1);

    ElemType h_sum;
    CUDA_LONG N = (CUDA_LONG) GetNumElements();
    int blocksPerGrid = (int) ceil(((double) N) / GridDim::maxThreadsPerBlock);

    _reductionLogAddSum<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(Data(),
                                                                                  d_sum, 1, N);
    CUDA_CALL(hipMemcpy(&h_sum, d_sum, sizeof(ElemType), hipMemcpyDeviceToHost));
    TracingGPUMemoryAllocator::Free<ElemType>(GetComputeDeviceId(), d_sum);

    return h_sum;
}

template <class ElemType>
void GPUMatrix<ElemType>::RCRFBackwardCompute(
    const GPUMatrix<ElemType>& alpha, GPUMatrix<ElemType>& beta,
    const GPUMatrix<ElemType>& /*lbls*/,
    const GPUMatrix<ElemType>& pos_scores, const GPUMatrix<ElemType>& pair_scores, const int shift)
{
    if (alpha.IsEmpty() || pos_scores.IsEmpty() || pair_scores.IsEmpty())
        LogicError("RCRFBackwardCompute: one of the input matrices is empty.");

    if (alpha.GetNumRows() != pos_scores.GetNumRows() || alpha.GetNumCols() != pos_scores.GetNumCols())
        LogicError("RCRFBackwardCompute: matrix dimensions mismatched.");

    size_t iNumLab = alpha.GetNumRows();
    size_t iNumPos = alpha.GetNumCols();

    alpha.PrepareDevice();
    beta.RequireSize(iNumLab, iNumPos);

    ElemType* d_zeta = TracingGPUMemoryAllocator::Allocate<ElemType>(alpha.GetComputeDeviceId(), iNumLab);

    CUDA_LONG N = iNumLab;
    int blocksPerGrid = (int) ceil(1.0 * N / GridDim::maxThreadsPerBlock);
    size_t szMemSize;
    for (int t = iNumPos - 1; t >= 0; t--)
    {
        szMemSize = sizeof(ElemType) * iNumLab;
        _rcrfBackwardComputeZeta<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, szMemSize>>>(t, iNumPos, alpha.Data(), d_zeta, pair_scores.Data(), iNumLab, shift);
        szMemSize = iNumLab * 3;
        szMemSize *= sizeof(ElemType);
        _rcrfBackwardCompute<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, szMemSize>>>(t, iNumPos, alpha.Data(), beta.Data(),
                                                                                                  d_zeta, pair_scores.Data(), iNumLab, shift);
    }
    /*
        error = hipGetErrorString(hipPeekAtLastError());
        printf("%s\n", error);
        error = hipGetErrorString(hipDeviceSynchronize());
        printf("%s\n", error);
        */

    TracingGPUMemoryAllocator::Free<ElemType>(alpha.GetComputeDeviceId(), d_zeta);
}

/**
    Compute the gradient for the first order Markov transition probabilities
    It uses equations derived in R. Collobert's paper "Natural language processing (almost) from scratch"
    */
template <class ElemType>
void GPUMatrix<ElemType>::RCRFTransGrdCompute(const GPUMatrix<ElemType>& lbls,
                                              const GPUMatrix<ElemType>& alpha,
                                              const GPUMatrix<ElemType>& beta,
                                              const GPUMatrix<ElemType>& pair_scores,
                                              GPUMatrix<ElemType>& grd,
                                              const int startLbl,
                                              const int shift)
{
    assert(shift == 1);
    int iNumPos = alpha.GetNumCols();
    int iNumLab = alpha.GetNumRows();

    ElemType* d_zeta = TracingGPUMemoryAllocator::Allocate<ElemType>(alpha.GetComputeDeviceId(), iNumLab);

    CUDA_LONG N = iNumLab;
    int blocksPerGrid = (int) ceil(1.0 * N / GridDim::maxThreadsPerBlock);
    size_t szMemSize;
    for (int t = 0; t < iNumPos; t++)
    {
        szMemSize = sizeof(ElemType) * iNumLab;
        _rcrfTransGrdComputeZeta<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, szMemSize>>>(t - 1, iNumPos, alpha.Data(), d_zeta, pair_scores.Data(), iNumLab, startLbl, shift);
        szMemSize = iNumLab * 3;
        szMemSize *= sizeof(ElemType);
        _rcrfTransGrdCompute<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, szMemSize>>>(t, startLbl, alpha.Data(), beta.Data(),
                                                                                                  d_zeta, pair_scores.Data(), lbls.Data(), grd.Data(), iNumPos, iNumLab, shift);
    }
    TracingGPUMemoryAllocator::Free<ElemType>(alpha.GetComputeDeviceId(), d_zeta);
};

// -----------------------------------------------------------------------
// TensorView entry points from Matrix.cpp
// -----------------------------------------------------------------------

// helper to provide a vector of ones of at least the given number of elements
// TODO: Use this to implement ComputationNode::ConstOnes? Or do we even need that anymore?
template <class ElemType>
static shared_ptr<GPUMatrix<ElemType>> GetOnesVector(size_t N, DEVICEID_TYPE deviceId)
{
    // using an array of shared_ptrs because those are thread-safe. The objects themselves are immutable.
    // And using a plain array so this will never get freed, avoiding free-after-DLL-unload issues.
    static shared_ptr<GPUMatrix<ElemType>> onesCache[32]; // cache of objects
    if (deviceId >= _countof(onesCache))
        LogicError("GetOnesVector: onesCache[] too small (%d entries), increase (you need %d) and recompile.", (int) _countof(onesCache), (int) deviceId + 1);
    auto p = onesCache[deviceId];
    if (!p || p->GetNumRows() < N) // must (re-)allocate
    {
        p = make_shared<GPUMatrix<ElemType>>(GPUMatrix<ElemType>::Ones(N, 1, deviceId));
        onesCache[deviceId] = p; // this will replace the pointer thread-safely (although weird race conditions may happen where a larger entry is overwritten by a smaller one; will still run correctly)
    }
    return p;
}

// perform unary operation 'op' on a giving 'this', reinterpreting the matrices as tensors as specified by the dims and strides
// This binds the N-ariness to a template parameter N, and gets the data pointers out from the matrix objects.
template <class ElemType>
void GPUMatrix<ElemType>::TensorOp(ElemType beta, const GPUMatrix<ElemType>& a, ElemType alpha, ElementWiseOperator op, ElementWiseOperator reductionOp,
                                   const array<size_t, 2>& offsets,
                                   const SmallVector<size_t>& regularOpDims, const array<SmallVector<ptrdiff_t>, 2>& regularStrides,
                                   const SmallVector<size_t>& reducingOpDims, const array<SmallVector<ptrdiff_t>, 2>& reducingStrides)
{
    if (reductionOp != ElementWiseOperator::opSum) // TODO: enable the reduction ops
        InvalidArgument("TensorOp: Unary reduction operations other than opSum not yet implemented.");

    a.PrepareDevice();
    if (a.GetComputeDeviceId() != GetComputeDeviceId())
        InvalidArgument("All matrices must be on the same GPU");

    // special case: linear processing
    // The case statement has measurable impact for unary ops (but not for binary ops it seems, due to double mem access).
    // Linear gap-free unary ops happen so regularly that we will eliminate the case statement from the CUDA kernel, and instead expand all.
    if (regularOpDims.size() == 1 && regularStrides[0][0] == 1 && regularStrides[1][0] == 1 && reducingOpDims.size() == 0)
    {
        // special case: for copy, use hipMemcpy() instead, or cublas_axpy()
        // TODO: We should observe if these actually make a speed difference, and if not, remove these special cases.
        if (op == ElementWiseOperator::opCopy && beta == 0 && alpha == 1)
            return CUDA_CALL(hipMemcpy(Data()+ offsets[1], a.Data()+ offsets[0], sizeof(ElemType) * regularOpDims[0], hipMemcpyDeviceToDevice));
        else if (op == ElementWiseOperator::opCopy && beta == 1)
            return CUBLAS_CALL(cublas_axpy(GetCublasHandle(GetComputeDeviceId()), (int) regularOpDims[0], &alpha, a.Data()+ offsets[0], 1, Data()+ offsets[1], 1));
        else
            return LaunchUnaryTensorOp<ElemType>(beta, a.Data()+ offsets[0], Data()+ offsets[1], alpha, op, regularOpDims[0]);
    }

    // special case: reducing a matrix onto a column vector; can be done with SGEMM
    // Note: A minor risk is that with this, our own reduction function will rarely be used.
    // That function was tested to give the same results with 'double', and nearly the same with 'float' (different summation order matters).
    else if (op == ElementWiseOperator::opCopy && // we are just adding to target without any further operation
#ifdef _DEBUG
             sizeof(ElemType) == sizeof(float) && // in debug don't shortcut 'double' so we have some test of our own codepath
#endif
             regularOpDims.size() == 1 && regularStrides[0][0] == 1 && regularStrides[1][0] == 1 && // we are processing a column
             reducingOpDims.size() == 1 && reducingStrides[0][0] >= (ptrdiff_t) regularOpDims[0])   // reducing across columns and no overlap
    {
        assert(reducingStrides[1][0] == 0);
        auto ARows = regularOpDims[0];    // vertical steps
        auto ACols = reducingOpDims[0];   // horizontal steps (reduction)
        auto ALd = reducingStrides[0][0]; // horizontal step width through matrix
        hipblasHandle_t cuHandle = GetCublasHandle(a.GetComputeDeviceId());
        CUBLAS_CALL(cublas_gemm(cuHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, (int) /*CRows=*/ARows, /*CCols=*/1, (int) ACols, &alpha,
                                /*A00=*/a.Data()+ offsets[0], (int) ALd,
                                /*B00=*/GetOnesVector<ElemType>(ACols, a.GetComputeDeviceId())->Data(), (int) /*BRows=*/ACols, &beta,
                                /*C00=*/Data()+ offsets[1], (int) /*CRows=*/ARows));
        return;
    }

    // TODO: Add a special case for tensor bias reduction. cudnn is ~7% faster on Image/QuickE2E.

    // regular case
    else
        return TensorOpN<ElemType, 2>(beta, array<ElemType*, 2>{a.Data(), Data()}, alpha, op, offsets, regularOpDims, regularStrides, reducingOpDims, reducingStrides);
}

// perform binary operation 'op' on a and b giving 'this', reinterpreting the matrices as tensors as specified by the dims and strides
template <class ElemType>
void GPUMatrix<ElemType>::TensorOp(ElemType beta, const GPUMatrix<ElemType>& a, const GPUMatrix<ElemType>& b, ElemType alpha, ElementWiseOperator op, ElementWiseOperator reductionOp,
                                   const array<size_t, 3>& offsets,
                                   const SmallVector<size_t>& regularOpDims, const array<SmallVector<ptrdiff_t>, 3>& regularStrides,
                                   const SmallVector<size_t>& reducingOpDims, const array<SmallVector<ptrdiff_t>, 3>& reducingStrides)
{
    if (reductionOp != ElementWiseOperator::opSum)
        InvalidArgument("TensorOp: The only permitted binary reduction operation is opSum.");

    a.PrepareDevice();
    if (a.GetComputeDeviceId() != GetComputeDeviceId() || b.GetComputeDeviceId() != GetComputeDeviceId())
        InvalidArgument("All matrices must be on the same GPU");

    return TensorOpN<ElemType, 3>(beta, array<ElemType*, 3>{a.Data(), b.Data(), Data()}, alpha, op, offsets, regularOpDims, regularStrides, reducingOpDims, reducingStrides);
}

// perform ternary operation 'op' on a, and c giving 'this', reinterpreting the matrices as tensors as specified by the dims and strides
template <class ElemType>
void GPUMatrix<ElemType>::TensorOp(ElemType beta, const GPUMatrix<ElemType>& a, const GPUMatrix<ElemType>& b, const GPUMatrix<ElemType>& c, ElemType alpha, ElementWiseOperator op, ElementWiseOperator reductionOp,
                                   const array<size_t, 4>& offsets,
                                   const SmallVector<size_t>& regularOpDims, const array<SmallVector<ptrdiff_t>, 4>& regularStrides,
                                   const SmallVector<size_t>& reducingOpDims, const array<SmallVector<ptrdiff_t>, 4>& reducingStrides)
{
    if (reductionOp != ElementWiseOperator::opSum)
        InvalidArgument("TensorOp: The only permitted ternary reduction operation is opSum.");

    a.PrepareDevice();
    if (a.GetComputeDeviceId() != GetComputeDeviceId() || b.GetComputeDeviceId() != GetComputeDeviceId() || c.GetComputeDeviceId() != GetComputeDeviceId())
        InvalidArgument("All matrices must be on the same GPU");
    return TensorOpN<ElemType, 4>(beta, array<ElemType*, 4>{a.Data(), b.Data(), c.Data(), Data()}, alpha, op, offsets, regularOpDims, regularStrides, reducingOpDims, reducingStrides);
}

// =======================================================================
// explicit instantiations business
// =======================================================================

template class GPUMatrix<float>;
template class GPUMatrix<double>;
template class DeviceBoundNumber<float>;
template class DeviceBoundNumber<double>;

template <class ElemType>
hipblasHandle_t GPUMatrix<ElemType>::s_cuHandle[GPUMatrix<ElemType>::MaxGpus] = {0};

template <class ElemType>
void* GPUMatrix<ElemType>::s_curandGenerator = NULL;

// We use Matrix<char> as the backing store for QuantizedMatrix
// Let's explicitly instantiate the methods we need for that purpose
template GPUMatrix<char>::GPUMatrix(const size_t numRows, const size_t numCols, int deviceId);
template GPUMatrix<char>::GPUMatrix(const size_t numRows, const size_t numCols, int deviceId, char* pArray, const size_t matrixFlags);
template GPUMatrix<char>::GPUMatrix(const GPUMatrix<char>&);
template GPUMatrix<char>::GPUMatrix(GPUMatrix<char>&&);
template char* GPUMatrix<char>::CopyToArray() const;
template void GPUMatrix<char>::ChangeDeviceTo(int);
template void GPUMatrix<char>::Resize(size_t, size_t, bool);
template void GPUMatrix<char>::RequireSize(size_t, size_t, bool);

template GPUMatrix<char>::~GPUMatrix();
template GPUMatrix<char> GPUMatrix<char>::ColumnSlice(size_t startColumn, size_t numCols) const;
template GPUMatrix<char>& GPUMatrix<char>::operator=(GPUMatrix<char>&&);
template GPUMatrix<char>::GPUMatrix(int);
template void GPUMatrix<char>::SetValue(const char);
template void GPUMatrix<char>::SetValue(const size_t numRows, const size_t numCols, int deviceId, char* pArray, size_t matrixFlags);
//template void GPUMatrix<char>::SetValue(CPUMatrix<char> const&);
template void GPUMatrix<char>::SetValue(GPUMatrix<char> const&);
//template void GPUMatrix<char>::SetValue(CPUSparseMatrix<char> const&);
//template void GPUMatrix<char>::SetValue(GPUSparseMatrix<char> const&);

template void GPUMatrix<char>::CopySection(size_t numRows, size_t numCols, char* dst, size_t colStride) const;
template void GPUMatrix<char>::Reshape(const size_t, const size_t);
template GPUMatrix<char>& GPUMatrix<char>::operator*=(char);
template DEVICEID_TYPE GPUMatrix<char>::PrepareDevice(DEVICEID_TYPE deviceId) const;

template GPUMatrix<int>::GPUMatrix(const size_t, const size_t, int, int*, const size_t);
template GPUMatrix<int>::~GPUMatrix();

template int* TracingGPUMemoryAllocator::Allocate<int>(int, size_t);
template size_t* TracingGPUMemoryAllocator::Allocate<size_t>(int, size_t);
template long* TracingGPUMemoryAllocator::Allocate<long>(int, size_t);
template char* TracingGPUMemoryAllocator::Allocate<char>(int, size_t);
template float* TracingGPUMemoryAllocator::Allocate<float>(int, size_t);
template double* TracingGPUMemoryAllocator::Allocate<double>(int, size_t);

template void TracingGPUMemoryAllocator::Free<int>(int, int*, bool);
template void TracingGPUMemoryAllocator::Free<size_t>(int, size_t*, bool);
template void TracingGPUMemoryAllocator::Free<char>(int, char*, bool);
template void TracingGPUMemoryAllocator::Free<float>(int, float*, bool);
template void TracingGPUMemoryAllocator::Free<double>(int, double*, bool);

}}}

// !!!!This is from hip/hip_runtime_api.h which comes with CUDA samples!!!! Consider if it is beneficial to just include all hip/hip_runtime_api.h
// TODO: This is duplicated in BestGpu.cpp
// Beginning of GPU Architecture definitions
int _ConvertSMVer2Cores(int major, int minor)
{
    // Defines for GPU Architecture types (using the SM version to determine the # of cores per SM
    typedef struct
    {
        int SM; // 0xMm (hexidecimal notation), M = SM Major version, and m = SM minor version
        int Cores;
    } sSMtoCores;

    sSMtoCores nGpuArchCoresPerSM[] =
        {
            {0x10, 8},   // Tesla Generation (SM 1.0) G80 class
            {0x11, 8},   // Tesla Generation (SM 1.1) G8x class
            {0x12, 8},   // Tesla Generation (SM 1.2) G9x class
            {0x13, 8},   // Tesla Generation (SM 1.3) GT200 class
            {0x20, 32},  // Fermi Generation (SM 2.0) GF100 class
            {0x21, 48},  // Fermi Generation (SM 2.1) GF10x class
            {0x30, 192}, // Kepler Generation (SM 3.0) GK10x class
            {0x35, 192}, // Kepler Generation (SM 3.5) GK11x class
            {-1, -1}};

    int index = 0;

    while (nGpuArchCoresPerSM[index].SM != -1)
    {
        if (nGpuArchCoresPerSM[index].SM == ((major << 4) + minor))
        {
            return nGpuArchCoresPerSM[index].Cores;
        }

        index++;
    }
    return nGpuArchCoresPerSM[7].Cores;
};
// end of GPU Architecture definitions

//inline CUDA_LONG _GetFreeMemoryOnCUDADevice(int devId)
//{
//    hipDevice_t cudaDevice;
//    hipError_t result = hipDeviceGet(&cudaDevice, devId);
//    if(result!= hipSuccess)
//    {
//        return 0;
//    }
//
//    // create cuda context
//    hipCtx_t cudaContext;
//    result = hipCtxCreate(&cudaContext, hipDeviceScheduleAuto, cudaDevice);
//    if(result != hipSuccess)
//    {
//        return 0;
//    }
//
//    // get the amount of free memory on the graphics card
//    size_t free;
//    size_t total;
//    result = hipMemGetInfo(&free, &total);
//    if (result!=hipSuccess)
//    {
//        return 0;
//    }
//    else
//        return (CUDA_LONG)free;
//}

#endif // CPUONLY
