#include "hip/hip_runtime.h"
//
// Copyright (c) Microsoft. All rights reserved.
// Licensed under the MIT license. See LICENSE.md file in the project root for full license information.
//

#include "stdafx.h"
#include "CuDnnFactories.h"
#include "GPUMatrix.h"
#include <typeinfo>
#include <typeindex>
#include "CuDnnCommon.h"

template <>
const char* CudaErrString<hipdnnStatus_t>(hipdnnStatus_t x)
{
    return hipdnnGetErrorString(x);
}

// A note on the formats: CNTK originally used NHWC for input/output tensors and CHWN for kernels.
// Such formats have very limited support in cuDNN and not used in other frameworks.
// CNTK with cuDNN by default uses NCHW formats for both inputs/outputs and kernels.
#define TENSOR_FORMAT HIPDNN_TENSOR_NCHW
#define FILTER_FORMAT HIPDNN_TENSOR_NCHW

namespace Microsoft { namespace MSR { namespace CNTK {

static bool IsGpu(DEVICEID_TYPE deviceId)
{
    return deviceId >= 0;
}

class CuDnnKernel
{
public:
    CuDnnKernel(const ConvolveGeometry& geometry, hipdnnDataType_t dataType)
        : m_kernel(nullptr)
    {
        CUDNN_CALL(hipdnnCreateFilterDescriptor(&m_kernel));
        // Set cuDNN kernel dimensions. cuDNN uses row-major format while TensorShape - column-major
        // so conversion is required.
        const auto& filt = geometry.KernelShape();
        size_t mapCount = geometry.GetMapCount(geometry.InputShape().GetRank() - 1);
        if (mapCount != geometry.MapCount().GetNumElements())
            InvalidArgument("cuDNN does not support map tensor of this configuration.");
        SmallVector<int> dims(filt.GetRank() + 1);
        for (int i = 0; i < filt.GetRank(); i++)
            dims[dims.size() - 1 - i] = (int)filt[i];
        // Set map count(aka K) dimension.
        dims[0] = (int)mapCount;
        CUDNN_CALL(cudnnSetFilterNdDescriptor_v4(m_kernel, dataType, FILTER_FORMAT, (int)dims.size(), dims.data()));
    }

    ~CuDnnKernel()
    {
        if (m_kernel != nullptr)
        {
            hipdnnDestroyFilterDescriptor(m_kernel);
            m_kernel = nullptr;
        }
    }

    operator hipdnnFilterDescriptor_t() const
    {
        return m_kernel;
    }

    DISABLE_COPY_AND_MOVE(CuDnnKernel);

private:
    hipdnnFilterDescriptor_t m_kernel;
};

class CuDnnConv
{
public:
    CuDnnConv(const ConvolveGeometry& geometry, hipdnnDataType_t dataType)
        : m_conv(nullptr)
    {
        CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&m_conv));
        // Set cuDNN convolution parameters. cuDNN uses row-major format while TensorShape - column-major
        // so conversion is required. Also, for 2D convolutions (which have 3D tensor shapes)
        // cuDNN uses 2D descriptors while for 3D convolutions - 3D so we need to ignore
        // rightmost dimension in ConvolveGeometry tensors.
        SmallVector<int> stride(geometry.InputShape().GetRank() - 1);
        SmallVector<int> pad(stride.size());
        for (int i = 0; i < stride.size(); i++)
        {
            stride[stride.size() - 1 - i] = (int)geometry.GetStride(i);
            pad[stride.size() - 1 - i] = geometry.GetLowerPad(i);
        }
        SmallVector<int> upscale(stride.size(), 1);
        CUDNN_CALL(hipdnnSetConvolutionNdDescriptor(m_conv, (int)stride.size(), pad.data(),
                                                   stride.data(), upscale.data(),
                                                   HIPDNN_CROSS_CORRELATION, dataType));
    }

    ~CuDnnConv()
    {
        if (m_conv != nullptr)
        {
            hipdnnDestroyConvolutionDescriptor(m_conv);
            m_conv = nullptr;
        }
    }

    operator hipdnnConvolutionDescriptor_t() const
    {
        return m_conv;
    }

    DISABLE_COPY_AND_MOVE(CuDnnConv);

private:
    hipdnnConvolutionDescriptor_t m_conv;
};

class CuDnnPool
{
public:
    CuDnnPool(const ConvolveGeometry& geometry, PoolKind kind)
        : m_pool(nullptr)
    {
        assert(kind == PoolKind::Max || kind == PoolKind::Average);

        CUDNN_CALL(hipdnnCreatePoolingDescriptor(&m_pool));
        // Set cuDNN pooling parameters. cuDNN uses row-major format while TensorShape - column-major
        // so conversion is required. Same as in convolution descriptor, cuDNN uses 2D descriptors
        // for 3D inputs.
        SmallVector<int> dims(geometry.InputShape().GetRank() - 1);
        SmallVector<int> stride(dims.size());
        SmallVector<int> pad(stride.size());
        int j = (int)dims.size() - 1;
        for (int i = 0; i < stride.size(); i++, j--)
        {
            dims[j] = (int)geometry.KernelShape()[i];
            stride[j] = (int)geometry.GetStride(i);
            pad[j] = geometry.GetLowerPad(i);
        }

        // Must use HIPDNN_POOLING_AVERAGE_COUNT_EXCLUDE_PADDING to get the same results as in reference engine.
#if CUDNN_MAJOR >= 5
        CUDNN_CALL(hipdnnSetPoolingNdDescriptor(m_pool,
                                               kind == PoolKind::Max ? HIPDNN_POOLING_MAX : HIPDNN_POOLING_AVERAGE_COUNT_EXCLUDE_PADDING,
                                               HIPDNN_PROPAGATE_NAN,
                                               (int)dims.size(), dims.data(), pad.data(), stride.data()));
#else
        CUDNN_CALL(hipdnnSetPoolingNdDescriptor(m_pool,
                                               kind == PoolKind::Max ? HIPDNN_POOLING_MAX : HIPDNN_POOLING_AVERAGE_COUNT_EXCLUDE_PADDING,
                                               (int)dims.size(), dims.data(), pad.data(), stride.data()));
#endif
    }

    ~CuDnnPool()
    {
        if (m_pool != nullptr)
        {
            hipdnnDestroyPoolingDescriptor(m_pool);
            m_pool = nullptr;
        }
    }

    operator hipdnnPoolingDescriptor_t() const
    {
        return m_pool;
    }

    DISABLE_COPY_AND_MOVE(CuDnnPool);

private:
    hipdnnPoolingDescriptor_t m_pool;
};

template <class ElemType>
class CuDnnConvolutionEngine : public ConvolutionEngine<ElemType>
{
public:
    using Base = ConvolutionEngine<ElemType>;
    using typename Base::Mat;

public:
    CuDnnConvolutionEngine(ConvolveGeometryPtr geometry, DEVICEID_TYPE deviceId, ImageLayoutKind imageLayout,
                           size_t maxTempMemSizeInSamples, PoolKind poolKind)
                           : Base(geometry, deviceId, imageLayout, maxTempMemSizeInSamples, poolKind),
                           m_cudnn(CuDnn::Instance()),
                           m_dataType(CuDnnTensor::GetDataType<ElemType>()),
                           m_inT(geometry->InputShape(), m_dataType),
                           m_outT(geometry->OutputShape(), m_dataType)
    {
    }

protected:
    using Base::m_geometry;
    using Base::m_deviceId;
    using Base::m_imageLayout;
    using Base::m_maxTempMemSizeInSamples;
    using Base::m_poolKind;

    void EnsureCompatible() override
    {
        if (m_imageLayout != ImageLayoutKind::CHW)
            RuntimeError("cuDNN convolution engine supports only CHW/cudnn layout.");
        if (!IsGpu(m_deviceId))
            RuntimeError("cuDNN convolution engine supports GPU devices only.");
    }

    void EnsureConvolutionInitialized() override
    {
        if (m_kernelT == nullptr)
        {
            m_kernelT = std::make_unique<CuDnnKernel>(*m_geometry, m_dataType), 
            m_conv = std::make_unique<CuDnnConv>(*m_geometry, m_dataType);
        }
    }

    void ForwardCore(const Mat& in, const Mat& kernel, Mat& out, Mat& workspace) override
    {
        size_t batchSize = in.GetNumCols();
        // Find best algo and allocate temp buffer, if needed.
        auto finder = [this](int& calgo, hipdnnConvolutionFwdAlgoPerf_t algoPerf[MaxAlgoCount]) -> hipdnnStatus_t
        {
            return hipdnnFindConvolutionForwardAlgorithm(*m_cudnn, m_inT, *m_kernelT, *m_conv, m_outT, MaxAlgoCount, &calgo, algoPerf);
        };
        auto staticFinder = [this](hipdnnConvolutionFwdAlgo_t& algo) -> hipdnnStatus_t
        {
            return hipdnnGetConvolutionForwardAlgorithm(*m_cudnn, m_inT, *m_kernelT, *m_conv, m_outT, HIPDNN_CONVOLUTION_FWD_NO_WORKSPACE, 0, &algo);
        };
        FindBestAlgo(batchSize, m_fwdAlgo, finder, staticFinder);
        if (m_fwdAlgo.Algo.memory > 0)
            workspace.Resize((m_fwdAlgo.Algo.memory + sizeof(ElemType) - 1) / sizeof(ElemType), 1);
        // Perform forward convolution operation.
        auto err = hipdnnConvolutionForward(*m_cudnn, &C::One, m_inT, ptr(in), *m_kernelT, ptr(kernel), *m_conv,
                                           m_fwdAlgo.Algo.algo, ptr(workspace), m_fwdAlgo.Algo.memory, &C::Zero, m_outT, ptr(out));
        // There might be a case where cuDNN fails due to workspace being too small, try using no-workspace algo instead.
        // REVIEW alexeyk: NVIDIA is currently reviewing this issue.
        if (HIPDNN_STATUS_INVALID_VALUE == err && m_fwdAlgo.Algo.memory > 0)
        {
            auto err2 = hipdnnConvolutionForward(*m_cudnn, &C::One, m_inT, ptr(in), *m_kernelT, ptr(kernel), *m_conv,
                                                m_fwdAlgo.NoWorkspaceAlgo, nullptr, 0, &C::Zero, m_outT, ptr(out));
            // Update original error in case of success.
            if (HIPDNN_STATUS_SUCCESS == err2)
                err = HIPDNN_STATUS_SUCCESS;
        }
        CUDNN_CALL(err);
    }

    void BackwardDataCore(const Mat& srcGrad, const Mat& kernel, Mat& grad, Mat& workspace) override
    {
        size_t batchSize = srcGrad.GetNumCols();
        // Find best algo and allocate temp buffer, if needed.
        auto finder = [this](int& calgo, hipdnnConvolutionBwdDataAlgoPerf_t algoPerf[MaxAlgoCount]) -> hipdnnStatus_t
        {
            return hipdnnFindConvolutionBackwardDataAlgorithm(*m_cudnn, *m_kernelT, m_outT, *m_conv, m_inT, MaxAlgoCount, &calgo, algoPerf);
        };
        auto staticFinder = [this](hipdnnConvolutionBwdDataAlgo_t& algo) -> hipdnnStatus_t
        {
            return hipdnnGetConvolutionBackwardDataAlgorithm(*m_cudnn, *m_kernelT, m_outT, *m_conv, m_inT, HIPDNN_CONVOLUTION_BWD_DATA_NO_WORKSPACE, 0, &algo);
        };
        FindBestAlgo(batchSize, m_backDataAlgo, finder, staticFinder);
        if (m_backDataAlgo.Algo.memory > 0)
            workspace.Resize((m_backDataAlgo.Algo.memory + sizeof(ElemType) - 1) / sizeof(ElemType), 1);
        // Compute gradients with respect to the output tensor (data).
        CUDNN_CALL(hipdnnConvolutionBackwardData(*m_cudnn, &C::One, *m_kernelT, ptr(kernel), m_outT, ptr(srcGrad), *m_conv, m_backDataAlgo.Algo.algo,
                                                ptr(workspace), m_backDataAlgo.Algo.memory, &C::One, m_inT, ptr(grad)));
    }

    void BackwardKernelCore(const Mat& srcGrad, const Mat& in, Mat& kernelGrad, bool /*allowReuse*/, Mat& workspace) override
    {
        size_t batchSize = in.GetNumCols();
        // Find best algo and allocate temp buffer, if needed.
        auto finder = [this](int& calgo, hipdnnConvolutionBwdFilterAlgoPerf_t algoPerf[MaxAlgoCount]) -> hipdnnStatus_t
        {
            return hipdnnFindConvolutionBackwardFilterAlgorithm(*m_cudnn, m_inT, m_outT, *m_conv, *m_kernelT, MaxAlgoCount, &calgo, algoPerf);
        };
        auto staticFinder = [this](hipdnnConvolutionBwdFilterAlgo_t& algo) -> hipdnnStatus_t
        {
            return hipdnnGetConvolutionBackwardFilterAlgorithm(*m_cudnn, m_inT, m_outT, *m_conv, *m_kernelT, HIPDNN_CONVOLUTION_BWD_FILTER_NO_WORKSPACE, 0, &algo);
        };
        FindBestAlgo(batchSize, m_backFiltAlgo, finder, staticFinder);
        if (m_backFiltAlgo.Algo.memory > 0)
            workspace.Resize((m_backFiltAlgo.Algo.memory + sizeof(ElemType) - 1) / sizeof(ElemType), 1);
        // Compute gradients with respect to the output tensor (data).
        CUDNN_CALL(hipdnnConvolutionBackwardFilter(*m_cudnn, &C::One, m_inT, ptr(in), m_outT, ptr(srcGrad), *m_conv, m_backFiltAlgo.Algo.algo,
                                                  ptr(workspace), m_backFiltAlgo.Algo.memory, &C::One, *m_kernelT, ptr(kernelGrad)));
    }

    void EnsurePoolingInitialized() override
    {
        if (m_pool == nullptr)
            m_pool = std::make_unique<CuDnnPool>(*m_geometry, m_poolKind);
    }

    void ForwardPoolingCore(const Mat& in, Mat& out) override
    {
        size_t batchSize = in.GetNumCols();
        m_inT.UpdateBatchSize(batchSize);
        m_outT.UpdateBatchSize(batchSize);
        CUDNN_CALL(hipdnnPoolingForward(*m_cudnn, *(m_pool), &C::One, m_inT, ptr(in), &C::Zero, m_outT, ptr(out)));
    }

    void BackwardPoolingCore(const Mat& out, const Mat& srcGrad, const Mat& in, Mat& grad) override
    {
        size_t batchSize = in.GetNumCols();
        m_inT.UpdateBatchSize(batchSize);
        m_outT.UpdateBatchSize(batchSize);
        CUDNN_CALL(hipdnnPoolingBackward(*m_cudnn, *(m_pool), &C::One, m_outT, ptr(out), m_outT, ptr(srcGrad),
                                        m_inT, ptr(in), &C::One, m_inT, ptr(grad)));
    }

    void MaxUnpoolingCore(const Mat& out, const Mat& poolIn, Mat& in) override
    {
        UNUSED(out);
        UNUSED(poolIn);
        UNUSED(in);
        // Not implemented but potentially can make a fallback to reference engine.
        LogicError("MaxUnpooling is not implemented for cuDNN engine.");
    }

private:
    using C = Consts<ElemType>;

    static const int MaxAlgoCount = 10;

    template <typename TAlgo, typename TFinder, typename TStaticFinder>
    void FindBestAlgo(size_t batchSize, TAlgo& algo, TFinder finder, TStaticFinder staticFinder)
    {
        m_inT.UpdateBatchSize(batchSize);
        m_outT.UpdateBatchSize(batchSize);

        if (!algo.NeedAutotuning(batchSize))
            return;

        using CuDnnAlgoT = decltype(TAlgo::Algo);
        CuDnnAlgoT algoPerf[MaxAlgoCount];
        int calgo = 0;
        hipdnnStatus_t err = finder(calgo, algoPerf);
        // Alloc failed - usually means cuDNN runtime auto-tuner could not allocate workspace.
        // In such case, use static auto-tuner with no workspace.
        if (err == HIPDNN_STATUS_ALLOC_FAILED)
        {
            decltype(CuDnnAlgoT::algo) noMemAlgo;
            CUDNN_CALL(staticFinder(noMemAlgo));
            algo.MaxAllowedMBSizeForCurrentAlgo = batchSize;
            algo.Algo = algoPerf[0];
            algo.Algo.algo = noMemAlgo;
            algo.Algo.memory = 0;
            algo.Algo.status = HIPDNN_STATUS_SUCCESS;
            algo.NoWorkspaceAlgo = noMemAlgo;
            return;
        }
        CUDNN_CALL(err);
        assert(calgo > 0);
        size_t inputSampleSize = m_geometry->InputShape().GetNumElements();
        size_t maxMem = m_maxTempMemSizeInSamples == 0 ? (std::numeric_limits<size_t>::max)() : inputSampleSize * m_maxTempMemSizeInSamples * sizeof(ElemType);
        // Find best (fastest) algorithm which satisfies workspace requirements.
        auto res = std::find_if(algoPerf, algoPerf + calgo,
            [=](const CuDnnAlgoT& cur)
            {
                return cur.status == HIPDNN_STATUS_SUCCESS && cur.memory <= maxMem;
            });
        if (res == algoPerf + calgo)
            RuntimeError("cuDNN could not find suitable algorithm for the current convolution configuration.");
        algo.MaxAllowedMBSizeForCurrentAlgo = batchSize;
        algo.Algo = *res;
        // Find fastest algorithm that does NOT require workspace. It is used as a fallback algo in Forward function.
        res = std::find_if(algoPerf, algoPerf + calgo,
            [](const CuDnnAlgoT& cur)
            {
                return cur.status == HIPDNN_STATUS_SUCCESS && cur.memory == 0;
            });
        if (res == algoPerf + calgo)
        {
            // In theory, this should never happen.
            RuntimeError("cuDNN could not find no-workspace algorithm for the current convolution configuration.");
        }
        else
            algo.NoWorkspaceAlgo = (*res).algo;
    }

    static ElemType* ptr(Mat& src)
    {
        return src.Data();
    }
    static const ElemType* ptr(const Mat& src)
    {
        return src.Data();
    }

private:
    template <typename T>
    struct ConvAlgoInfo
    {
        using CuDnnAlgoT = decltype(T::algo);

        ConvAlgoInfo()
            : MaxAllowedMBSizeForCurrentAlgo(0)
        {
            Algo.status = HIPDNN_STATUS_NOT_INITIALIZED;
            NoWorkspaceAlgo = (CuDnnAlgoT)-1;
        }
        // Current mini-batch size, needed for re-computing statistics in auto-tuner.
        size_t MaxAllowedMBSizeForCurrentAlgo;

        T Algo;
        CuDnnAlgoT NoWorkspaceAlgo;

        bool NeedAutotuning(size_t batchSize)
        {
            // Need to re-run auto-tuner in case minibatch size is increased.
            // If minibatch size is decreased we assume that previously selected algorithm requires less or the same amount of workspace.
            // This is done to avoid re-running auto-tuner every time in case minibatch size changes frequently (e.g. when distributed reading is enabled).
            // REVIEW alexeyk: potentially, this might cause some perf issues if better (faster) algo can be selected for a smaller mininbatch.
            // We also need to reset auto-tuning status at the beginning of each epoch but ComputationNode currently does not provide such notification.
            // We assume no other dimensions of tensors can change so we don't check it.
            // REVIEW alexeyk: review once we get response from NVIDIA.
            return (Algo.status != HIPDNN_STATUS_SUCCESS || batchSize > MaxAllowedMBSizeForCurrentAlgo);
        }
    };

    CuDnn::ptr_t m_cudnn;
    hipdnnDataType_t m_dataType;
    CuDnnTensor m_inT;
    CuDnnTensor m_outT;
    // Convolution specific.
    std::unique_ptr<CuDnnKernel> m_kernelT;
    std::unique_ptr<CuDnnConv> m_conv;
    // Pooling specific.
    std::unique_ptr<CuDnnPool> m_pool;

    ConvAlgoInfo<hipdnnConvolutionFwdAlgoPerf_t> m_fwdAlgo;
    ConvAlgoInfo<hipdnnConvolutionBwdDataAlgoPerf_t> m_backDataAlgo;
    ConvAlgoInfo<hipdnnConvolutionBwdFilterAlgoPerf_t> m_backFiltAlgo;
};

template <class ElemType>
std::unique_ptr<ConvolutionEngine<ElemType>> CuDnnConvolutionEngineFactory<ElemType>::Create(ConvolveGeometryPtr geometry,
                                                                                             DEVICEID_TYPE deviceId, ImageLayoutKind imageLayout,
                                                                                             size_t maxTempMemSizeInSamples, PoolKind poolKind)
{
    return std::make_unique<CuDnnConvolutionEngine<ElemType>>(geometry, deviceId, imageLayout, maxTempMemSizeInSamples, poolKind);
}

template <class ElemType>
bool CuDnnConvolutionEngineFactory<ElemType>::IsSupported(DEVICEID_TYPE deviceId, ConvolveGeometryPtr geometry, PoolKind poolKind)
{
    // REVIEW alexeyk: IsSupported check should be performed by cuDNN itself. Is there a good way to do that?

    hipDeviceProp_t props = {0};
    // Note that hipGetDeviceProperties also sets CUDA last error so need to check/clear both.
    if (deviceId < 0 || (hipGetDeviceProperties(&props, deviceId) | hipGetLastError()) != hipSuccess || props.major < 3)
        return false;

    const auto& input = geometry->InputShape();
    const auto& kernel = geometry->KernelShape();
    const auto& sharing = geometry->Sharing();
    const auto& mapCount = geometry->MapCount();
    // cuDNN supports 2D and 3D convolutions at the moment with full sharing.
    // In case map count size > 1, then it should have all ones except last dimension.
    // If pooling is requested, then cuDNN supports only 2D/3D inputs and 2D pooling kernels.
    return (input.GetRank() <= 4 &&
            std::find(begin(sharing), end(sharing), false) == sharing.end() &&
            mapCount.GetNumElements() == mapCount[mapCount.GetRank() - 1] &&
            (poolKind == PoolKind::None || 
             input.GetRank() <= 3 && (kernel.GetRank() < 3 || kernel[2] == 1)));
}

template class CuDnnConvolutionEngineFactory<float>;
template class CuDnnConvolutionEngineFactory<double>;

} } }
