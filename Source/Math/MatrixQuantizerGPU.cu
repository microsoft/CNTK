#include "hip/hip_runtime.h"
#include "stdafx.h"
#include "MatrixQuantizerGPU.h"
#include "MatrixQuantizer_kernel.cu"
#include "GPUMatrix.h"
#include "GPUDataTransferer.h"

#pragma comment(lib, "cudart.lib") // instruct linker to reference these libs
#pragma comment(lib, "cublas.lib")
#pragma comment(lib, "cusparse.lib")
#pragma comment(lib, "hiprand.lib")

#pragma warning(disable : 4267) // conversion from 'size_t' to 'unsigned int'; happens in CUDA <<<a,b>>> syntax if a and b are size_t
#pragma warning(disable : 4127) // conditional expression is constant; "if (sizeof(ElemType)==sizeof(float))" triggers this
#pragma warning(disable : 4702) // unreachable code; triggered for unknown reasons

namespace Microsoft { namespace MSR { namespace CNTK {

// CUDA failed
// Since the outer code sometimes does not recover properly, as an option we log and die right away.
// This is needed for our GCD farm which has intermittent CUDA errors that sometimes cause the DBN tool, when running with MPI, to hang instead of terminating.
void cudafail(const char* msg)
{
    // TODO: get from an env variable
    bool dieoncudafailure = false;
    if (!dieoncudafailure)
    {
        RuntimeError("%s", msg);
    }
    fprintf(stderr, "%s\n", msg);
    fprintf(stderr, "cudafail: terminating\n"), fflush(stderr);
#ifdef WIN32
    TerminateProcess(GetCurrentProcess(), EXIT_FAILURE); // fail the hard way to ensure it won't hang elsewhere
#else
    exit(1);
#endif
}

// allows to write cudaFunction() || "error"   (CUDA runtime)
static
#ifdef WIN32
    __declspec(noinline)
#endif
        void
        operator||(hipError_t rc, const char* msg)
{
    if (rc != hipSuccess)
    {
        char buf[1000];
        sprintf_s(buf, 1000, "%s: %s (cuda error %d)", msg, hipGetErrorString(rc), rc);
        cudafail(buf);
    }
}

template <class ElemType>
void MatrixQuantizerGPU<ElemType>::Sync()
{
    hipDeviceSynchronize() || "hipDeviceSynchronize failed";
}

// wait until stream has completed all scheduled operations
template <class ElemType>
void MatrixQuantizerGPU<ElemType>::SyncStream(hipStream_t stream)
{
    hipStreamSynchronize(stream) || "hipStreamSynchronize failed";
}

// same but for event
template <class ElemType>
void MatrixQuantizerGPU<ElemType>::SyncEvent(hipEvent_t ev)
{
    auto rc = hipEventQuery(ev);
    if (rc != hipErrorNotReady)
    {
        // if Event is ready then no need to wait
        rc || "hipEventQuery failed";
        return;
    }
    // we must wait
    hipEventSynchronize(ev) || "hipEventSynchronize failed";
}

//streams
template <class ElemType>
hipStream_t MatrixQuantizerGPU<ElemType>::m_computeStream = NULL;

template <class ElemType>
hipStream_t MatrixQuantizerGPU<ElemType>::m_fetchStream = NULL;

template <class ElemType>
hipStream_t MatrixQuantizerGPU<ElemType>::m_assignStream = NULL;

template <class ElemType>
hipStream_t MatrixQuantizerGPU<ElemType>::GetComputeStream()
{
    return m_computeStream;
}

template <class ElemType>
hipStream_t MatrixQuantizerGPU<ElemType>::GetFetchStream()
{
    return m_fetchStream;
}

template <class ElemType>
hipStream_t MatrixQuantizerGPU<ElemType>::GetAssignStream()
{
    return m_assignStream;
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// computestream: the stream the caller issued the quant op on
template <class ElemType>
void MatrixQuantizerGPU<ElemType>::RecordQuantizeCompleteEvent(hipStream_t computestream) const
{
    // schedule to flag the quantize-complete event (on main stream)
    hipEventRecord(m_quantizeCompleteEvent, computestream) || "hipEventRecord failed";

    // when running synchronously (for time measurements), then we (CPU) wait right here
    if (m_forceSync)
    {
        SyncStream(computestream);
    }
}

template <class ElemType>
void MatrixQuantizerGPU<ElemType>::SyncQuantizeCompleEventAndFetchAndRecordFetchCompleteEvent(char* cpuBuffer, char* gpuBuffer, size_t size) const
{
    // schedule fetch stream to wait until the last quantize op is complete, i.e. the data in the buffer is now valid
    // wait until commencement
    hipStreamWaitEvent(GetFetchStream(), m_quantizeCompleteEvent, 0 /*flags 'must be 0'*/) || "hipStreamWaitEvent failed";

    // schedule to fetch that quantized data into CPU buffer (on a separate transfer stream)
    hipMemcpyAsync(cpuBuffer, gpuBuffer, size, hipMemcpyDeviceToHost, GetFetchStream()) || "hipMemcpyAsync failed";

    hipEventRecord(m_fetchCompleteEvent, GetFetchStream()) || "hipEventRecord failed"; // for next GPU operation

    // when running synchronously (for time measurements), then we (CPU) wait right here
    if (m_forceSync)
    {
        SyncStream(GetFetchStream());
    }
}

template <class ElemType>
void MatrixQuantizerGPU<ElemType>::SyncAssignCompleteEvent(hipStream_t computestream) const
{
    // schedule to wait for the assign-complete event (on main/compute stream)     --CPU buffer free once main stream does anything after this
    hipStreamWaitEvent(computestream, m_assignCompleteEvent, 0 /*flags 'must be 0'*/) || "hipStreamWaitEvent failed";

    // Note that the NVidia doc says somewhat confusingly:
    //  * If \p stream is NULL, any future work submitted in any stream will wait for
    //  * \p event to complete before beginning execution. This effectively creates a
    //  * barrier for all future work submitted to the device on this thread.
    // -> it says that this may bring the whole machinery to stall. Or does hipStreamWaitEvent() honor hipStreamNonBlocking?
    // According to NVidia (Jiri Kraus), this works as expected.
}

template <class ElemType>
QuantizedMatrix<ElemType>& MatrixQuantizerGPU<ElemType>::GetTempGPUQuantizedMatrix(size_t numRows, size_t numCols, size_t nBits, bool& newlyAllocated)
{
    newlyAllocated = false;

    // Check if the existing one is good for our needs
    if ((m_tempGPUQuantizedMatrix != nullptr) && (m_tempGPUQuantizedMatrix->GetNumBits() == nBits) && (m_tempGPUQuantizedMatrix->GetNumRows() >= numRows) && (m_tempGPUQuantizedMatrix->GetNumCols() >= numCols))
    {
        return *m_tempGPUQuantizedMatrix;
    }

    if (m_tempGPUQuantizedMatrix != nullptr)
    {
        delete m_tempGPUQuantizedMatrix;
        m_tempGPUQuantizedMatrix = nullptr;
    }

    m_tempGPUQuantizedMatrix = new QuantizedMatrix<ElemType>(numRows, numCols, nBits, (short) this->GetDeviceId());
    newlyAllocated = true;

    return *m_tempGPUQuantizedMatrix;
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
///cpubuffer should be page-locked memory allocated, otherwise CUDA will not be efficient (hence we don't use STL)
template <class ElemType>
MatrixQuantizerGPU<ElemType>::MatrixQuantizerGPU(int deviceId, bool useDedicatedComputeStream, bool forceSync /*= false*/)
    : MatrixQuantizerImpl<ElemType>(deviceId), m_quantizeCompleteEvent(NULL), m_fetchCompleteEvent(NULL), m_tempMatrixZeroingCompleteEvent(NULL), m_assignCompleteEvent(NULL), m_forceSync(forceSync), m_tempGPUQuantizedMatrix(nullptr), m_quantizeOpIncludedFetch(false)
{
    PrepareDevice(this->GetDeviceId());

    // events
    // Note: Do NOT use hipEventBlockingSync (which supposedly yields the process)--it will totally break hipEventSynchronize(), causing it to take 50 or 100 ms randomly.
    hipEventCreateWithFlags(&m_tempMatrixZeroingCompleteEvent, hipEventDisableTiming) || "hipEventCreateWithFlags failed";
    hipEventCreateWithFlags(&m_quantizeCompleteEvent, hipEventDisableTiming) || "hipEventCreateWithFlags failed";
    hipEventCreateWithFlags(&m_fetchCompleteEvent, hipEventDisableTiming) || "hipEventCreateWithFlags failed";
    hipEventCreateWithFlags(&m_assignCompleteEvent, hipEventDisableTiming) || "hipEventCreateWithFlags failed";

#pragma warning(disable : 4127)
    if (useDedicatedComputeStream && (m_computeStream == NULL))
    {
        hipStreamCreateWithFlags(&m_computeStream, hipStreamNonBlocking) || "hipStreamCreateWithFlags failed";
    }

    if (m_fetchStream == NULL)
    {
        hipStreamCreateWithFlags(&m_fetchStream, hipStreamNonBlocking) || "hipStreamCreateWithFlags failed";
        hipStreamCreateWithFlags(&m_assignStream, hipStreamNonBlocking) || "hipStreamCreateWithFlags failed";
    }
}

template <class ElemType>
MatrixQuantizerGPU<ElemType>::~MatrixQuantizerGPU()
{
    if (nullptr != m_tempGPUQuantizedMatrix)
    {
        delete m_tempGPUQuantizedMatrix;
        m_tempGPUQuantizedMatrix = nullptr;
    }

    // BUGBUG: we don't destroy our streams (they are static variables); we need a static destructor, I am too lazy now
    // TODO: Check for error code and throw if !std::uncaught_exception()
    hipEventDestroy(m_assignCompleteEvent);
    hipEventDestroy(m_fetchCompleteEvent);
    hipEventDestroy(m_quantizeCompleteEvent);
    hipEventDestroy(m_tempMatrixZeroingCompleteEvent);
}

template <class ElemType>
void MatrixQuantizerGPU<ElemType>::QuantizeAsync(const Matrix<ElemType>& inMatrix, const Matrix<ElemType>& inResidual, QuantizedMatrix<ElemType>& outQMatrix, Matrix<ElemType>& outResidual, bool zeroThresholdFor1Bit)
{
    // Verify various input matrix parameter's dimensions
    assert((inMatrix.GetNumRows() == outQMatrix.GetNumRows()) && (inMatrix.GetNumCols() == outQMatrix.GetNumCols()));
    assert((inMatrix.GetNumRows() == inResidual.GetNumRows()) && (inMatrix.GetNumCols() == inResidual.GetNumCols()));
    assert((inMatrix.GetNumRows() == outResidual.GetNumRows()) && (inMatrix.GetNumCols() == outResidual.GetNumCols()));

    size_t nBits = outQMatrix.GetNumBits();

    PrepareDevice(this->GetDeviceId());
    if (m_forceSync)
    {
        Sync();
    }

    bool GPUMatrixNewlyAllocated = false;
    QuantizedMatrix<ElemType>& outQMatrixGPU = (outQMatrix.GetDeviceId() == CPUDEVICE) ? GetTempGPUQuantizedMatrix(outQMatrix.GetNumRows(), outQMatrix.GetNumCols(), nBits, GPUMatrixNewlyAllocated) : outQMatrix;

    // If we newly allocated the target GPU matrix then the aysnc zeroing of the matrix is still in procgress on
    // the main compute stream. We must synchroniz with the mail compute stream in case the quantization
    // compute stream is different from the main compute stream
    if (GPUMatrixNewlyAllocated && (GetComputeStream() != GetStream()))
    {
        hipEventRecord(m_tempMatrixZeroingCompleteEvent, GetStream()) || "hipEventRecord failed";
        hipStreamWaitEvent(GetComputeStream(), m_tempMatrixZeroingCompleteEvent, 0 /*flags 'must be 0'*/) || "hipStreamWaitEvent failed";
    }

    // Do the quantization on compute sstream and insert event into stream
    _QuantizeMatrix<ElemType>(inMatrix.Data(), inResidual.Data(),
                              inMatrix.GetNumRows(), inMatrix.GetNumCols(),
                              outQMatrixGPU.Buffer(), nBits, GetComputeStream(),
                              outResidual.Data(), zeroThresholdFor1Bit);

    RecordQuantizeCompleteEvent(GetComputeStream());

    // copy from gpu to cpu if needed
    m_quantizeOpIncludedFetch = false;
    if (outQMatrix.GetDeviceId() == CPUDEVICE)
    {
        SyncQuantizeCompleEventAndFetchAndRecordFetchCompleteEvent(outQMatrix.Buffer(), outQMatrixGPU.Buffer(), outQMatrixGPU.GetSize());
        m_quantizeOpIncludedFetch = true;
    }
}

template <class ElemType>
void MatrixQuantizerGPU<ElemType>::WaitQuantizeAsyncDone()
{
    PrepareDevice(this->GetDeviceId());

    if (m_quantizeOpIncludedFetch)
    {
        SyncEvent(m_fetchCompleteEvent);
    }
    else
    {
        SyncEvent(m_quantizeCompleteEvent);
    }
}

template <class ElemType>
void MatrixQuantizerGPU<ElemType>::UnquantizeAsync(QuantizedMatrix<ElemType>& inQMatrix, Matrix<ElemType>& outMatrix, bool add /*= false*/)
{
    // The outMatrix should be on the same GPU as m_inMatrix
    assert(outMatrix.GetDeviceId() == this->GetDeviceId());

    PrepareDevice(this->GetDeviceId());

    size_t nBits = inQMatrix.GetNumBits();

    // Verify  input matrix parameter's dimensions
    assert((inQMatrix.GetNumRows() == outMatrix.GetNumRows()) && (inQMatrix.GetNumCols() == outMatrix.GetNumCols()));

    bool GPUMatrixNewlyAllocated = false;
    QuantizedMatrix<ElemType>& inQMatrixGPU = (inQMatrix.GetDeviceId() == CPUDEVICE) ? GetTempGPUQuantizedMatrix(inQMatrix.GetNumRows(), inQMatrix.GetNumCols(), nBits, GPUMatrixNewlyAllocated) : inQMatrix;

    if (inQMatrix.GetDeviceId() == CPUDEVICE)
    {
        // If the intermediate GPU Matrix was newly allocated, we need to wait for its zeroing to finish
        // before assigning the inQMatrix contents
        if (GPUMatrixNewlyAllocated)
        {
            hipEventRecord(m_tempMatrixZeroingCompleteEvent, GetStream()) || "hipEventRecord failed";
            hipStreamWaitEvent(GetAssignStream(), m_tempMatrixZeroingCompleteEvent, 0 /*flags 'must be 0'*/) || "hipStreamWaitEvent failed";
        }

        // schedule assign to GPU (on transfer stream)
        hipMemcpyAsync(inQMatrixGPU.Buffer(), inQMatrix.Buffer(), inQMatrix.GetSize(), hipMemcpyHostToDevice, GetAssignStream()) || "hipMemcpyAsync failed";

        // schedule to flag the assign-complete event
        hipEventRecord(m_assignCompleteEvent, GetAssignStream()) || "hipEventRecord failed"; // for subsequent GPU operation to consume this buffer

        if (m_forceSync)
        {
            SyncStream(GetAssignStream());
        }

        // let the computing stream wait for the assign complete
        SyncAssignCompleteEvent(GetComputeStream());
    }

    // do the actually unquantization
    _UnquantizeMatrix(inQMatrixGPU.Buffer(), inQMatrixGPU.GetSize(),
                      outMatrix.Data(), outMatrix.GetNumRows(), outMatrix.GetNumCols(),
                      nBits, add, GetComputeStream());

    // Record the event of unquantization
    RecordQuantizeCompleteEvent(GetComputeStream());
}

template <class ElemType>
void MatrixQuantizerGPU<ElemType>::WaitUnquantizeAsyncDone()
{
    PrepareDevice(this->GetDeviceId());
    SyncEvent(m_quantizeCompleteEvent);
}

//explicit
template class MatrixQuantizerGPU<float>;
template class MatrixQuantizerGPU<double>;

GPUMatrixComputeStreamEvent::GPUMatrixComputeStreamEvent(int deviceId)
    : MatrixComputeStreamEvent(deviceId)
{
    // Note: Do NOT use hipEventBlockingSync (which supposedly yields the process)--it will totally break hipEventSynchronize(), causing it to take 50 or 100 ms randomly.
    hipEventCreateWithFlags(&m_mainGPUComputeStreamCUDAEvent, hipEventDisableTiming) || "hipEventCreateWithFlags failed";

    // Record an event on the main GPU compute stream
    hipEventRecord(m_mainGPUComputeStreamCUDAEvent, GetStream()) || "hipEventRecord failed";
}

GPUMatrixComputeStreamEvent::~GPUMatrixComputeStreamEvent()
{
    // TODO: Check for error code and throw if !std::uncaught_exception()
    hipEventDestroy(m_mainGPUComputeStreamCUDAEvent) || "hipEventDestroy failed";
}

void GPUMatrixComputeStreamEvent::SynchronizeEvent()
{
    hipEventSynchronize(m_mainGPUComputeStreamCUDAEvent) || "hipEventSynchronize failed";
}

template <typename ElemType>
void GPUMatrixComputeStreamEvent::SynchronizeQuantizationComputeStreamWithEvent()
{
    hipStreamWaitEvent(MatrixQuantizerGPU<ElemType>::GetComputeStream(), m_mainGPUComputeStreamCUDAEvent, 0 /*flags 'must be 0'*/) || "hipStreamWaitEvent failed";
}

template <typename ElemType>
void GPUMatrixComputeStreamEvent::SynchronizeDataTransferFetchStreamWithEvent()
{
    hipStreamWaitEvent(GPUDataTransferer::GetFetchStream(), m_mainGPUComputeStreamCUDAEvent, 0 /*flags 'must be 0'*/) || "hipStreamWaitEvent failed";
}

// Explicit template instantiations
template void GPUMatrixComputeStreamEvent::SynchronizeQuantizationComputeStreamWithEvent<float>();
template void GPUMatrixComputeStreamEvent::SynchronizeQuantizationComputeStreamWithEvent<double>();
template void GPUMatrixComputeStreamEvent::SynchronizeDataTransferFetchStreamWithEvent<float>();
template void GPUMatrixComputeStreamEvent::SynchronizeDataTransferFetchStreamWithEvent<double>();
} } }
