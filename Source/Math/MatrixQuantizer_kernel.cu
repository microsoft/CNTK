#include "hip/hip_runtime.h"
#ifndef __MATRIX_QUANTIZER_KERNEL_CUH__
#define __MATRIX_QUANTIZER_KERNEL_CUH__
#include <float.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>

#include "ValueQuantizer.h"
#include "ColumnQuantizer.h"
#include "QuantizedMatrix.h"

namespace Microsoft { namespace MSR { namespace CNTK {

// =======================================================================
// thread layout helpers
// =======================================================================

// --- distribute array elements naively over threads
__host__ static void ParallelizeOverRangeDim(size_t size, dim3& griddim, dim3& blockdim, const size_t warpsize = 64)
{
    // <<< griddim, blockdim, sharedmemsize, stream >>>
    griddim = (unsigned int) ((size + warpsize - 1) / warpsize); // 'warpsize' threads on each block (-> threadIdx.x)
    blockdim = (unsigned int) warpsize;                          // -> blockIdx.x
}
// get the array index for the current thread
__device__ __inline__ static size_t ParallelizeOverRangeIndex()
{
    return threadIdx.x + (blockIdx.x * blockDim.x);
}

// =======================================================================
// quantization
// =======================================================================

// helper to reduce all T across all threads of a block
template <typename T, int BLOCKSIZE>
__device__ void allreduce(T& var)
{
    __shared__ T buf[BLOCKSIZE];
    volatile T* vBuf = buf;

    buf[threadIdx.x] = var;
    __syncthreads();

    // We assume BLOCKSIZE is a power of 2
    if (BLOCKSIZE >= 1024)
    {
        if (threadIdx.x < 512)
        {
            var = var + buf[threadIdx.x + 512];
            buf[threadIdx.x] = var;
        }
        __syncthreads();
    }

    if (BLOCKSIZE >= 512)
    {
        if (threadIdx.x < 256)
        {
            var = var + buf[threadIdx.x + 256];
            buf[threadIdx.x] = var;
        }
        __syncthreads();
    }

    if (BLOCKSIZE >= 256)
    {
        if (threadIdx.x < 128)
        {
            var = var + buf[threadIdx.x + 128];
            buf[threadIdx.x] = var;
        }
        __syncthreads();
    }

    if (BLOCKSIZE >= 128)
    {
        if (threadIdx.x < 64)
        {
            var = var + buf[threadIdx.x + 64];
            buf[threadIdx.x] = var;
        }
        __syncthreads();
    }

    // Intra warp reduce
    if ((BLOCKSIZE >= 64) && (threadIdx.x < 32))
    {
        var = var + vBuf[threadIdx.x + 32];
        vBuf[threadIdx.x] = var;
        __syncwarp(0xffffffff);
    }

    if ((BLOCKSIZE >= 32) && (threadIdx.x < 16))
    {
        var = var + vBuf[threadIdx.x + 16];
        vBuf[threadIdx.x] = var;
        __syncwarp(0xffff);
    }

    if ((BLOCKSIZE >= 16) && (threadIdx.x < 8))
    {
        var = var + vBuf[threadIdx.x + 8];
        vBuf[threadIdx.x] = var;
        __syncwarp(0xff);
    }

    if ((BLOCKSIZE >= 8) && (threadIdx.x < 4))
    {
        var = var + vBuf[threadIdx.x + 4];
        vBuf[threadIdx.x] = var;
        __syncwarp(0xf);
   }

    if ((BLOCKSIZE >= 4) && (threadIdx.x < 2))
    {
        var = var + vBuf[threadIdx.x + 2];
        vBuf[threadIdx.x] = var;
        __syncwarp(0x3);
    }

    if ((BLOCKSIZE >= 2) && (threadIdx.x == 0))
    {
        var = var + vBuf[1];
        vBuf[0] = var;
    }

    __syncthreads();

    var = buf[0];
}

#define REDUCTION_BLOCK_SIZE 128 // 256 is much worse; 64 is somewhat worse

// version optimized for collated memory access
template <class ElemType, bool ZeroThresholdFor1Bit>
__global__ void _ComputeQuantiStatParj(const ElemType* us, const ElemType* inResidual, long M, long N, size_t ldNbits, char* qpackage)
{
    size_t subset = threadIdx.x; // first thread computes 0, 64, 128; second thread 1, 65, 129 etc.
    size_t j = blockIdx.x;       // we process one column per *block*, j=column index; note: j is never out of range

    size_t rows = M; // we compute from 0..rows-1
    size_t bits = 1 << ldNbits;
    const size_t colSizeByte = Microsoft::MSR::CNTK::QuantizedColumn<ElemType>::QuantizedColumnSize(bits, rows);
    auto& qcol = *(Microsoft::MSR::CNTK::QuantizedColumn<ElemType>*) &qpackage[colSizeByte * j];

    Microsoft::MSR::CNTK::ColumnQuantizer<ElemType>::ComputeRangeStatColjSubset<ZeroThresholdFor1Bit>(us, inResidual, M, j, bits, qcol.lower, qcol.upper,
                                                                                                      subset, REDUCTION_BLOCK_SIZE, allreduce<ElemType, REDUCTION_BLOCK_SIZE>, allreduce<unsigned int, REDUCTION_BLOCK_SIZE>);
}

//caller: griddim and blockdim should be both 1d
//total thread number is: totalNumQWordsAlMatrix = numCols() * numQWordsPerCol
//called to quantize a GPU matrix
template <class ElemType, bool ZeroThresholdFor1Bit>
__global__ void _QuantizeStripjOneQWord(
    const ElemType* us,
    ElemType* curResidual,
    long M, long N,
    char* qMat,
    size_t qColSize,
    size_t numQWordsPerCol,
    size_t ldNbits,
    ElemType* newResidual)
{
    // map our thread index into a linear index
    const size_t linindex = ParallelizeOverRangeIndex();

    // map to (QWord index, column index)
    const size_t j = linindex / numQWordsPerCol;
    if (j >= N) // out of col range
        return;

    const size_t iQWord = linindex % numQWordsPerCol;

    // get data pointers to the quantized column
    auto& qCol = *(Microsoft::MSR::CNTK::QuantizedColumn<ElemType>*) &qMat[qColSize * j];

    // and quantizer
    const Microsoft::MSR::CNTK::ColumnQuantizer<ElemType> q(ldNbits, qCol.lower, qCol.upper);

    // quantize one QWord to qCol[iQWord]
    qCol.bits[iQWord] = q.QuantizeOneQWord<ZeroThresholdFor1Bit>(us, curResidual, M, iQWord, M, numQWordsPerCol, j, newResidual);
}

template <class ElemType>
__global__ void UnquantizeStripejOneQWord(ElemType* us, const long M, const long N, const char* qpackage, size_t colsize, size_t numQWordsPerCol, size_t ldNbits, bool add)
{
    // this follows the same as  quantizestripej()
    // map our thread index into a linear index
    const size_t linindex = ParallelizeOverRangeIndex();
    // map to (QWord index, column index)
    const size_t j = linindex / numQWordsPerCol;

    if (j >= N) // out of col range
        return;

    const size_t iQWord = linindex % numQWordsPerCol;

    // get data pointers and quantizer
    const auto& qcol = *(const Microsoft::MSR::CNTK::QuantizedColumn<ElemType>*) &qpackage[colsize * j];
    const ElemType lower = qcol.lower;
    const ElemType upper = qcol.upper;
    Microsoft::MSR::CNTK::ColumnQuantizer<ElemType> q(ldNbits, lower, upper);
    // unquantize from this one QWord
    q.UnquantizeOneQWord(us, M, iQWord, M, numQWordsPerCol, j, qcol.bits[iQWord], add);
}

//maybe should move out into another class?
template <class ElemType>
void _QuantizeMatrix(
    const ElemType* us,
    ElemType* curResidual,
    long M, long N,
    char* qPackage,
    size_t Nbits,
    hipStream_t stream,
    ElemType* newResidual,
    bool zeroThresholdFor1Bit)
{

    /* verify buffer allocation size
        if (msra::math::matrixquantizer::buffersize(bits, rows(), cols()) != gpubuffer.size())
        LogicError("quantizestripe: dimension of patch to be quantized does not match allocated buffer size for quantized data");
        if (rows() != curresidual.rows() || cols() != curresidual.cols()
        || rows() != newresidual.rows() || cols() != newresidual.cols())
        LogicError("quantizestripe: dimension of patch to be quantized does not match residual buffer");
        if (gpubuffer.size() == 0)      // empty buffer: empty matrix, we are done (explicit test needed since launch will fail with 0 threads)
        return;*/
    // determine mean and variance -> value range (stored in quant package)   --for 1 bit, refine it in a second pass
    const size_t ldNbits = ValueQuantizer<ElemType>::ld(Nbits);

    size_t nRow = M;
    size_t nCol = N;
    dim3 mvgriddim, mvblockdim;
    // using specialized CUDA code (not shared with CPU) for collated memory access
    // each thread column computes 'warpsize' elements
    mvgriddim = (unsigned int) nCol; // column number
    mvblockdim = REDUCTION_BLOCK_SIZE;

    if (zeroThresholdFor1Bit)
    {
        _ComputeQuantiStatParj<ElemType, true><<<mvgriddim, mvblockdim, 0, stream>>>(us, curResidual, M, N, ldNbits, qPackage);
    }
    else
    {
        _ComputeQuantiStatParj<ElemType, false><<<mvgriddim, mvblockdim, 0, stream>>>(us, curResidual, M, N, ldNbits, qPackage);
    }

    // quantize data (also computing the residual at once)
    // optimizing for collated memory access:
    //  - each 32-bit word represents an interleaved (not consecutive) set of floats -> parallel threads can do collated accesses
    // example:
    //  - total number of 32-bit words(1-bit quant): 1100 * 2048 / 32 = 70k
    //  - thread x dimension: index into 32-bit word (e.g. 1100/32 = 35 threads)
    //  - thread y dimension and thread position: column (e.g. 2048)
    //  - using 128 threads on one proc -> 70k/128 = 550 blocks
    //  - threads are indexed by a global index into quantized 32-bit words in increasing order; each thread must
    //     - re-linearize block index and thread index
    //     - map to (i,j) coordinate (start of the set of floats)

    const size_t numQWordsPerCol = Microsoft::MSR::CNTK::ColumnQuantizer<ElemType>::QWordsPerCol(nRow, Nbits);
    const size_t totalQWords = nCol * numQWordsPerCol;

    const size_t colsizebyte = Microsoft::MSR::CNTK::QuantizedColumn<ElemType>::QuantizedColumnSize(Nbits, nRow);

    dim3 griddim, blockdim;
    ParallelizeOverRangeDim(totalQWords, griddim, blockdim, 256);
    if (zeroThresholdFor1Bit)
    {
        _QuantizeStripjOneQWord<ElemType, true><<<griddim, blockdim, 0, stream>>>(us, curResidual, M, N, qPackage, colsizebyte, numQWordsPerCol, ldNbits, newResidual);
    }
    else
    {
        _QuantizeStripjOneQWord<ElemType, false><<<griddim, blockdim, 0, stream>>>(us, curResidual, M, N, qPackage, colsizebyte, numQWordsPerCol, ldNbits, newResidual);
    }
}

// unquantize
// Process the quantization package to recover (unquantize) the matrix patch.
template <class ElemType>
void _UnquantizeMatrix(const char* gpuBuffer, size_t gpuBufferSize,
                       ElemType* us, long M, long N,
                       size_t nBits, bool add, hipStream_t stream)
{
    // verify buffer allocation size
    /*if (msra::math::matrixquantizer::buffersize(bits, rows(), cols()) != gpubuffer.size())
            LogicError("unquantizestripe: dimension of patch to be unquantized does not match size of quantized data");
        if (gpubuffer.size() == 0)      // empty buffer: empty matrix, we are done (explicit test needed since launch will fail with 0 threads)
            return;
        */
    size_t qSize = QuantizedColumn<ElemType>::QuantizedColumnSize(nBits, M) * N;
    if (qSize != gpuBufferSize)
        LogicError("unquantizestripe: dimension of patch to be unquantized does not match size of quantized data");
    if (gpuBufferSize == 0) // empty buffer: empty matrix, we are done (explicit test needed since launch will fail with 0 threads)
        return;

    // #bits must be a power of two; we operate on shift values
    const size_t ldNbits = ValueQuantizer<ElemType>::ld(nBits);
    // unquantize in the same thread layout as quantize(), see there
    const size_t numQWordsPerCol = ColumnQuantizer<ElemType>::QWordsPerCol(M, nBits);
    const size_t totalQWords = N * numQWordsPerCol;

    const size_t colsize = QuantizedColumn<ElemType>::QuantizedColumnSize(nBits, M);

    dim3 griddim, blockdim;
    ParallelizeOverRangeDim(totalQWords, griddim, blockdim, 256);
    UnquantizeStripejOneQWord<<<griddim, blockdim, 0, stream>>>(us, M, N, gpuBuffer, colsize, numQWordsPerCol, ldNbits, add);
}
}
}
}

#endif
